#include "hip/hip_runtime.h"
/*
 !=====================================================================
 !
 !               S p e c f e m 3 D  V e r s i o n  2 . 0
 !               ---------------------------------------
 !
 !          Main authors: Dimitri Komatitsch and Jeroen Tromp
 !    Princeton University, USA and University of Pau / CNRS / INRIA
 ! (c) Princeton University / California Institute of Technology and University of Pau / CNRS / INRIA
 !                            August 2013
 !
 ! This program is free software; you can redistribute it and/or modify
 ! it under the terms of the GNU General Public License as published by
 ! the Free Software Foundation; either version 2 of the License, or
 ! (at your option) any later version.
 !
 ! This program is distributed in the hope that it will be useful,
 ! but WITHOUT ANY WARRANTY; without even the implied warranty of
 ! MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 ! GNU General Public License for more details.
 !
 ! You should have received a copy of the GNU General Public License along
 ! with this program; if not, write to the Free Software Foundation, Inc.,
 ! 51 Franklin Street, Fifth Floor, Boston, MA 02110-1301 USA.
 !
 !=====================================================================
 */

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

#include <sys/types.h>
#include <unistd.h>
#include <sys/time.h>
#include <sys/resource.h>

#include "config.h"
#include "mesh_constants_cuda.h"


/* ----------------------------------------------------------------------------------------------- */

// elastic domain sources

/* ----------------------------------------------------------------------------------------------- */

__global__ void compute_add_sources_kernel(realw* accel,
                                           int* ibool,
                                           realw* sourcearrays,
                                           double* stf_pre_compute,
                                           int myrank,
                                           int* islice_selected_source,
                                           int* ispec_selected_source,
                                           int NSOURCES) {
  int ispec,iglob;
  realw stf;

  int i = threadIdx.x;
  int j = threadIdx.y;
  int k = threadIdx.z;
  int isource  = blockIdx.x + gridDim.x*blockIdx.y; // bx

  // when NSOURCES > MAXIMUM_GRID_DIM, but mod(nspec_top,2) > 0, we end up with an extra block.
  if(isource < NSOURCES) {
    if(myrank == islice_selected_source[isource]) {

      ispec = ispec_selected_source[isource]-1;

      stf = (realw) stf_pre_compute[isource];
      iglob = ibool[INDEX4(NGLLX,NGLLX,NGLLX,i,j,k,ispec)]-1;

      // note: for global version, sourcearrays has dimensions
      //            sourcearrays(NDIM,NGLLX,NGLLY,NGLLZ,NSOURCES)
      atomicAdd(&accel[3*iglob], sourcearrays[INDEX5(NDIM,NGLLX,NGLLX,NGLLX,0,i,j,k,isource)]*stf);
      atomicAdd(&accel[3*iglob+1], sourcearrays[INDEX5(NDIM,NGLLX,NGLLX,NGLLX,1,i,j,k,isource)]*stf);
      atomicAdd(&accel[3*iglob+2], sourcearrays[INDEX5(NDIM,NGLLX,NGLLX,NGLLX,2,i,j,k,isource)]*stf);
    }
  }
}


/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(compute_add_sources_cuda,
              COMPUTE_ADD_SOURCES_CUDA)(long* Mesh_pointer_f,
                                        int* NSOURCESf,
                                        double* h_stf_pre_compute) {

  TRACE("compute_add_sources_cuda");

  Mesh* mp = (Mesh*)(*Mesh_pointer_f); //get mesh pointer out of fortran integer container

  // checks if anything to do
  if( mp->nsources_local == 0 ) return;

  int NSOURCES = *NSOURCESf;

  int num_blocks_x, num_blocks_y;
  get_blocks_xy(NSOURCES,&num_blocks_x,&num_blocks_y);

  dim3 grid(num_blocks_x,num_blocks_y);
  dim3 threads(NGLLX,NGLLX,NGLLX);

  // copies source time function buffer values to GPU
  print_CUDA_error_if_any(hipMemcpy(mp->d_stf_pre_compute,h_stf_pre_compute,
                                     NSOURCES*sizeof(double),hipMemcpyHostToDevice),71018);

  // adds source contributions
  compute_add_sources_kernel<<<grid,threads,0,mp->compute_stream>>>(mp->d_accel_crust_mantle,
                                               mp->d_ibool_crust_mantle,
                                               mp->d_sourcearrays,
                                               mp->d_stf_pre_compute,
                                               mp->myrank,
                                               mp->d_islice_selected_source,
                                               mp->d_ispec_selected_source,
                                               NSOURCES);

#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
  exit_on_cuda_error("compute_add_sources_cuda");
#endif
}

/* ----------------------------------------------------------------------------------------------- */

// backward sources

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(compute_add_sources_backward_cuda,
              COMPUTE_ADD_SOURCES_BACKWARD_CUDA)(long* Mesh_pointer_f,
                                                 int* NSOURCESf,
                                                 double* h_stf_pre_compute) {
  TRACE("compute_add_sources_backward_cuda");
  // debug
  DEBUG_BACKWARD_SOURCES();

  Mesh* mp = (Mesh*)(*Mesh_pointer_f); //get mesh pointer out of fortran integer container

  // checks if anything to do
  if( mp->nsources_local == 0 ) return;

  int NSOURCES = *NSOURCESf;

  int num_blocks_x, num_blocks_y;
  get_blocks_xy(NSOURCES,&num_blocks_x,&num_blocks_y);

  dim3 grid(num_blocks_x,num_blocks_y);
  dim3 threads(NGLLX,NGLLX,NGLLX);

  // copies source time function buffer values to GPU
  print_CUDA_error_if_any(hipMemcpy(mp->d_stf_pre_compute,h_stf_pre_compute,
                                     NSOURCES*sizeof(double),hipMemcpyHostToDevice),71019);

  compute_add_sources_kernel<<<grid,threads,0,mp->compute_stream>>>(mp->d_b_accel_crust_mantle,
                                               mp->d_ibool_crust_mantle,
                                               mp->d_sourcearrays,
                                               mp->d_stf_pre_compute,
                                               mp->myrank,
                                               mp->d_islice_selected_source,
                                               mp->d_ispec_selected_source,
                                               NSOURCES);

#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
  exit_on_cuda_error("compute_add_sources_backward_cuda");
#endif
}


/* ----------------------------------------------------------------------------------------------- */

// ADJOINT sources

/* ----------------------------------------------------------------------------------------------- */

__global__ void compute_add_sources_adjoint_cuda_kernel(realw* accel,
                                                        int nrec,
                                                        realw* adj_sourcearrays,
                                                        int* ibool,
                                                        int* ispec_selected_rec,
                                                        int* pre_computed_irec,
                                                        int nadj_rec_local) {

  int ispec,iglob;
  int irec,i,j,k;

  int irec_local = blockIdx.x + gridDim.x*blockIdx.y;

  // when nrec > MAXIMUM_GRID_DIM, but mod(nspec_top,2) > 0, we end up with an extra block.
  if(irec_local < nadj_rec_local) {
    irec = pre_computed_irec[irec_local];
    ispec = ispec_selected_rec[irec]-1;

    i = threadIdx.x;
    j = threadIdx.y;
    k = threadIdx.z;
    iglob = ibool[INDEX4(NGLLX,NGLLX,NGLLX,i,j,k,ispec)]-1;

    // atomic operations are absolutely necessary for correctness!
    atomicAdd(&accel[3*iglob], adj_sourcearrays[INDEX5(NDIM,NGLLX,NGLLX,NGLLX,0,i,j,k,irec_local)]);
    atomicAdd(&accel[3*iglob+1], adj_sourcearrays[INDEX5(NDIM,NGLLX,NGLLX,NGLLX,1,i,j,k,irec_local)]);
    atomicAdd(&accel[3*iglob+2], adj_sourcearrays[INDEX5(NDIM,NGLLX,NGLLX,NGLLX,2,i,j,k,irec_local)]);
  }
}

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(compute_add_sources_adjoint_cuda,
              COMPUTE_ADD_SOURCES_ADJOINT_CUDA)(long* Mesh_pointer,
                                                int* h_nrec) {

// adds adjoint sources
// note: call this routine after transfer_adj_to_device**() to have correct adjoint sourcearrays in array d_adj_sourcearrays

  TRACE("compute_add_sources_adjoint_cuda");

  Mesh* mp = (Mesh*)(*Mesh_pointer); //get mesh pointer out of fortran integer container

  // check if anything to do
  if( mp->nadj_rec_local == 0 ) return;

  // total number of receivers/adjoint sources
  int nrec = *h_nrec;

  // waits for previous transfer_** calls to be finished
  if( GPU_ASYNC_COPY ){
    // waits for asynchronous copy to finish
    hipStreamSynchronize(mp->copy_stream);
  }

  int num_blocks_x, num_blocks_y;
  get_blocks_xy(mp->nadj_rec_local,&num_blocks_x,&num_blocks_y);

  dim3 grid(num_blocks_x,num_blocks_y,1);
  dim3 threads(NGLLX,NGLLX,NGLLX);

  // the irec_local variable needs to be precomputed (as
  // h_pre_comp..), because normally it is in the loop updating accel,
  // and due to how it's incremented, it cannot be parallelized
  compute_add_sources_adjoint_cuda_kernel<<<grid,threads,0,mp->compute_stream>>>(mp->d_accel_crust_mantle,
                                                                                 nrec,
                                                                                 mp->d_adj_sourcearrays,
                                                                                 mp->d_ibool_crust_mantle,
                                                                                 mp->d_ispec_selected_rec,
                                                                                 mp->d_pre_computed_irec,
                                                                                 mp->nadj_rec_local);

#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
  exit_on_cuda_error("compute_add_sources_adjoint_cuda");
#endif
}

/* ----------------------------------------------------------------------------------------------- */

// adjoint memory transfers

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(transfer_adj_to_device,
              TRANSFER_ADJ_TO_DEVICE)(long* Mesh_pointer,
                                      int* h_nrec,
                                      realw* h_adj_sourcearrays,
                                      int* h_islice_selected_rec) {

// transfers adjoint source arrays synchronuously to GPU

  TRACE("transfer_adj_to_device");

  Mesh* mp = (Mesh*)(*Mesh_pointer); //get mesh pointer out of fortran integer container

  // check if anything to do
  if( mp->nadj_rec_local == 0 ) return;

  // total number of receivers/adjoint sources
  int nrec = *h_nrec;

  // build slice of adj_sourcearrays because full array is *very* large.
  //
  // note: this copies array values for local adjoint sources at given time step "iadj_vec(it)"
  //          from large adj_sourcearrays array into h_adj_sourcearrays_slice
  //
  // dimension of global array version
  //   adj_sourcearrays is (NDIM,NGLLX,NGLLY,NGLLZ,nadj_rec_local,NTSTEP_BETWEEN_READ_ADJSRC)
  // passed as function argument here is pointer to slice at time iadj_vec(it)
  //    which has dimension (NDIM,NGLLX,NGLLY,NGLLZ,nadj_rec_local)
  int i,j,k,irec_local;

  irec_local = 0;
  for(int irec = 0; irec < nrec; irec++) {
    if(mp->myrank == h_islice_selected_rec[irec]) {
      // takes only local sources
      for(k=0;k<NGLLX;k++) {
        for(j=0;j<NGLLX;j++) {
          for(i=0;i<NGLLX;i++) {
            mp->h_adj_sourcearrays_slice[INDEX5(NDIM,NGLLX,NGLLX,NGLLX,0,i,j,k,irec_local)]
              = h_adj_sourcearrays[INDEX5(NDIM,NGLLX,NGLLX,NGLLX,0,i,j,k,irec_local)];

            mp->h_adj_sourcearrays_slice[INDEX5(NDIM,NGLLX,NGLLX,NGLLX,1,i,j,k,irec_local)]
              = h_adj_sourcearrays[INDEX5(NDIM,NGLLX,NGLLX,NGLLX,1,i,j,k,irec_local)];

            mp->h_adj_sourcearrays_slice[INDEX5(NDIM,NGLLX,NGLLX,NGLLX,2,i,j,k,irec_local)]
              = h_adj_sourcearrays[INDEX5(NDIM,NGLLX,NGLLX,NGLLX,2,i,j,k,irec_local)];
          }
        }
      }
      // increases local receivers counter
      irec_local++;
    }
  }

  // check all local sources were added
  if( irec_local != mp->nadj_rec_local) exit_on_error("irec_local not equal to nadj_rec_local\n");

  // copies extracted array values onto GPU
  print_CUDA_error_if_any(hipMemcpy(mp->d_adj_sourcearrays, mp->h_adj_sourcearrays_slice,
                                     (mp->nadj_rec_local)*NDIM*NGLL3*sizeof(realw),hipMemcpyHostToDevice),71000);

#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
  exit_on_cuda_error("transfer_adj_to_device");
#endif
}

/* ----------------------------------------------------------------------------------------------- */


extern "C"
void FC_FUNC_(transfer_adj_to_device_async,
              TRANSFER_ADJ_TO_DEVICE_ASYNC)(long* Mesh_pointer,
                                            int* h_nrec,
                                            realw* h_adj_sourcearrays,
                                            int* h_islice_selected_rec) {

// asynchronous transfer for next adjoint source arrays from host to device

  TRACE("transfer_adj_to_device_async");

  Mesh* mp = (Mesh*)(*Mesh_pointer); //get mesh pointer out of fortran integer container

  // check if anything to do
  if( mp->nadj_rec_local == 0 ) return;

  // checks async-memcpy
  if( GPU_ASYNC_COPY == 0 ){
    exit_on_error("transfer_adj_to_device_async must be called with GPU_ASYNC_COPY == 1, please check mesh_constants_cuda.h");
  }

  // total number of receivers/adjoint sources
  int nrec = *h_nrec;

  // build slice of adj_sourcearrays because full array is *very* large.
  //
  // note: this copies array values for local adjoint sources at given time step "iadj_vec(it)"
  //          from large adj_sourcearrays array into h_adj_sourcearrays_slice
  //
  // dimension of global array version
  //   adj_sourcearrays is (NDIM,NGLLX,NGLLY,NGLLZ,nadj_rec_local,NTSTEP_BETWEEN_READ_ADJSRC)
  // passed as function argument here is pointer to slice at time iadj_vec(it)
  //    which has dimension (NDIM,NGLLX,NGLLY,NGLLZ,nadj_rec_local)
  int i,j,k,irec_local;

  // waits for previous copy_stream call to be finished
  hipStreamSynchronize(mp->copy_stream);

  irec_local = 0;
  for(int irec = 0; irec < nrec; irec++) {
    if(mp->myrank == h_islice_selected_rec[irec]) {
      // takes only local sources
      for(k=0;k<NGLLX;k++) {
        for(j=0;j<NGLLX;j++) {
          for(i=0;i<NGLLX;i++) {
            mp->h_adj_sourcearrays_slice[INDEX5(NDIM,NGLLX,NGLLX,NGLLX,0,i,j,k,irec_local)]
              = h_adj_sourcearrays[INDEX5(NDIM,NGLLX,NGLLX,NGLLX,0,i,j,k,irec_local)];

            mp->h_adj_sourcearrays_slice[INDEX5(NDIM,NGLLX,NGLLX,NGLLX,1,i,j,k,irec_local)]
              = h_adj_sourcearrays[INDEX5(NDIM,NGLLX,NGLLX,NGLLX,1,i,j,k,irec_local)];

            mp->h_adj_sourcearrays_slice[INDEX5(NDIM,NGLLX,NGLLX,NGLLX,2,i,j,k,irec_local)]
              = h_adj_sourcearrays[INDEX5(NDIM,NGLLX,NGLLX,NGLLX,2,i,j,k,irec_local)];
          }
        }
      }
      // increases local receivers counter
      irec_local++;
    }
  }

  // check all local sources were added
  if( irec_local != mp->nadj_rec_local) exit_on_error("irec_local not equal to nadj_rec_local\n");

  // waits for previous compute_add_sources_adjoint_cuda_kernel() call to be finished
  hipStreamSynchronize(mp->compute_stream);

  // copies extracted array values onto GPU
  // (asynchronous copy to GPU using copy_stream)
  hipMemcpyAsync(mp->d_adj_sourcearrays, mp->h_adj_sourcearrays_slice,(mp->nadj_rec_local)*NDIM*NGLL3*sizeof(realw),
                  hipMemcpyHostToDevice,mp->copy_stream);

}

