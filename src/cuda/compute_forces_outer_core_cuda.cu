#include "hip/hip_runtime.h"
/*
 !=====================================================================
 !
 !               S p e c f e m 3 D  V e r s i o n  2 . 0
 !               ---------------------------------------
 !
 !          Main authors: Dimitri Komatitsch and Jeroen Tromp
 !    Princeton University, USA and University of Pau / CNRS / INRIA
 ! (c) Princeton University / California Institute of Technology and University of Pau / CNRS / INRIA
 !                            April 2011
 !
 ! This program is free software; you can redistribute it and/or modify
 ! it under the terms of the GNU General Public License as published by
 ! the Free Software Foundation; either version 2 of the License, or
 ! (at your option) any later version.
 !
 ! This program is distributed in the hope that it will be useful,
 ! but WITHOUT ANY WARRANTY; without even the implied warranty of
 ! MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 ! GNU General Public License for more details.
 !
 ! You should have received a copy of the GNU General Public License along
 ! with this program; if not, write to the Free Software Foundation, Inc.,
 ! 51 Franklin Street, Fifth Floor, Boston, MA 02110-1301 USA.
 !
 !=====================================================================
 */

#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hipblas.h>

#include <sys/time.h>
#include <sys/resource.h>

#include "config.h"
#include "mesh_constants_cuda.h"

/* ----------------------------------------------------------------------------------------------- */

// elemental routines

/* ----------------------------------------------------------------------------------------------- */

// fluid rotation

__device__ void compute_element_oc_rotation(int tx,int working_element,
                                            realw time,
                                            realw two_omega_earth,
                                            realw deltat,
                                            realw* d_A_array_rotation,
                                            realw* d_B_array_rotation,
                                            reald dpotentialdxl, reald dpotentialdyl,
                                            reald* dpotentialdx_with_rot,
                                            reald* dpotentialdy_with_rot) {

  reald two_omega_deltat,cos_two_omega_t,sin_two_omega_t;
  reald A_rotation,B_rotation;
  reald ux_rotation,uy_rotation;
  reald source_euler_A,source_euler_B;

  // non-padded offset
  int offset_nonpadded = tx + working_element*NGLL3;

  // store the source for the Euler scheme for A_rotation and B_rotation
  two_omega_deltat = deltat * two_omega_earth;

  cos_two_omega_t = cos(two_omega_earth*time);
  sin_two_omega_t = sin(two_omega_earth*time);

  // time step deltat of Euler scheme is included in the source
  source_euler_A = two_omega_deltat * (cos_two_omega_t * dpotentialdyl + sin_two_omega_t * dpotentialdxl);
  source_euler_B = two_omega_deltat * (sin_two_omega_t * dpotentialdyl - cos_two_omega_t * dpotentialdxl);

  A_rotation = d_A_array_rotation[offset_nonpadded];
  B_rotation = d_B_array_rotation[offset_nonpadded];

  ux_rotation =   A_rotation*cos_two_omega_t + B_rotation*sin_two_omega_t;
  uy_rotation = - A_rotation*sin_two_omega_t + B_rotation*cos_two_omega_t;

  *dpotentialdx_with_rot = dpotentialdxl + ux_rotation;
  *dpotentialdy_with_rot = dpotentialdyl + uy_rotation;

  // updates rotation term with Euler scheme
  d_A_array_rotation[offset_nonpadded] += source_euler_A;
  d_B_array_rotation[offset_nonpadded] += source_euler_B;

  return;
}


/* ----------------------------------------------------------------------------------------------- */

// KERNEL 2
//
// for outer core ( acoustic domain )
/* ----------------------------------------------------------------------------------------------- */


__global__ void Kernel_2_outer_core_impl(int nb_blocks_to_compute,
                                       int NGLOB, int* d_ibool,
                                       int* d_phase_ispec_inner,
                                       int num_phase_ispec,
                                       int d_iphase,
                                       int use_mesh_coloring_gpu,
                                       realw* d_potential, realw* d_potential_dot_dot,
                                       realw* d_xix, realw* d_xiy, realw* d_xiz,
                                       realw* d_etax, realw* d_etay, realw* d_etaz,
                                       realw* d_gammax, realw* d_gammay, realw* d_gammaz,
                                       realw* hprime_xx, realw* hprime_yy, realw* hprime_zz,
                                       realw* hprimewgll_xx, realw* hprimewgll_yy, realw* hprimewgll_zz,
                                       realw* wgllwgll_xy,realw* wgllwgll_xz,realw* wgllwgll_yz,
                                       int GRAVITY,
                                       realw* d_xstore, realw* d_ystore, realw* d_zstore,
                                       realw* d_d_ln_density_dr_table,
                                       realw* d_minus_rho_g_over_kappa_fluid,
                                       realw* wgll_cube,
                                       int ROTATION,
                                       realw time,
                                       realw two_omega_earth,
                                       realw deltat,
                                       realw* d_A_array_rotation,realw* d_B_array_rotation){

  int bx = blockIdx.y*gridDim.x+blockIdx.x;
  int tx = threadIdx.x;

  //const int NGLL3 = NGLL3;
  const int NGLL3_ALIGN = NGLL3_PADDED;
  // R_EARTH_KM is the radius of the bottom of the oceans (radius of Earth in km)
  const reald R_EARTH_KM = 6371.0f;
  // uncomment line below for PREM with oceans
  //const reald R_EARTH_KM = 6368.0f;

  int K = (tx/NGLL2);
  int J = ((tx-K*NGLL2)/NGLLX);
  int I = (tx-K*NGLL2-J*NGLLX);

  int active,offset;
  int iglob = 0;
  int working_element;
  reald temp1l,temp2l,temp3l;
  reald xixl,xiyl,xizl,etaxl,etayl,etazl,gammaxl,gammayl,gammazl,jacobianl;
  reald dpotentialdxl,dpotentialdyl,dpotentialdzl;
  reald dpotentialdx_with_rot,dpotentialdy_with_rot;
  reald fac1,fac2,fac3;
  reald sum_terms;
  reald gravity_term;
  reald gxl,gyl,gzl;
  reald radius,theta,phi;
  reald cos_theta,sin_theta,cos_phi,sin_phi;
  reald grad_x_ln_rho,grad_y_ln_rho,grad_z_ln_rho;
  int int_radius;


#ifndef MANUALLY_UNROLLED_LOOPS
  int l;
  int offset1,offset2,offset3;
  realw hp1,hp2,hp3;
#endif

  __shared__ reald s_dummy_loc[NGLL3];

  __shared__ reald s_temp1[NGLL3];
  __shared__ reald s_temp2[NGLL3];
  __shared__ reald s_temp3[NGLL3];

// use only NGLL^3 = 125 active threads, plus 3 inactive/ghost threads,
// because we used memory padding from NGLL^3 = 125 to 128 to get coalescent memory accesses
  active = (tx < NGLL3 && bx < nb_blocks_to_compute) ? 1:0;

// copy from global memory to shared memory
// each thread writes one of the NGLL^3 = 125 data points
  if (active) {

#ifdef USE_MESH_COLORING_GPU
    working_element = bx;
#else
    //mesh coloring
    if( use_mesh_coloring_gpu ){
      working_element = bx;
    }else{
      // iphase-1 and working_element-1 for Fortran->C array conventions
      working_element = d_phase_ispec_inner[bx + num_phase_ispec*(d_iphase-1)]-1;
    }
#endif

    // iglob = d_ibool[working_element*NGLL3_ALIGN + tx]-1;
    iglob = d_ibool[working_element*NGLL3 + tx]-1;

#ifdef USE_TEXTURES
    s_dummy_loc[tx] = tex1Dfetch(tex_potential, iglob);
#else
    // changing iglob indexing to match fortran row changes fast style
    s_dummy_loc[tx] = d_potential[iglob];
#endif
  }

// synchronize all the threads (one thread for each of the NGLL grid points of the
// current spectral element) because we need the whole element to be ready in order
// to be able to compute the matrix products along cut planes of the 3D element below
  __syncthreads();

#ifndef MAKE_KERNEL2_BECOME_STUPID_FOR_TESTS

  if (active) {

#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
//      if(iglob == 0 )printf("kernel 2: iglob %i  hprime_xx %f %f %f \n",iglob,hprime_xx[0],hprime_xx[1],hprime_xx[2]);
#endif


#ifndef MANUALLY_UNROLLED_LOOPS

    temp1l = 0.f;
    temp2l = 0.f;
    temp3l = 0.f;

    for (l=0;l<NGLLX;l++) {
        hp1 = hprime_xx[l*NGLLX+I];
        offset1 = K*NGLL2+J*NGLLX+l;
        temp1l += s_dummy_loc[offset1]*hp1;

        //no more assumes that hprime_xx = hprime_yy = hprime_zz
        hp2 = hprime_yy[l*NGLLX+J];
        offset2 = K*NGLL2+l*NGLLX+I;
        temp2l += s_dummy_loc[offset2]*hp2;

        hp3 = hprime_zz[l*NGLLX+K];
        offset3 = l*NGLL2+J*NGLLX+I;
        temp3l += s_dummy_loc[offset3]*hp3;
    }
#else

    temp1l = s_dummy_loc[K*NGLL2+J*NGLLX]*hprime_xx[I]
            + s_dummy_loc[K*NGLL2+J*NGLLX+1]*hprime_xx[NGLLX+I]
            + s_dummy_loc[K*NGLL2+J*NGLLX+2]*hprime_xx[2*NGLLX+I]
            + s_dummy_loc[K*NGLL2+J*NGLLX+3]*hprime_xx[3*NGLLX+I]
            + s_dummy_loc[K*NGLL2+J*NGLLX+4]*hprime_xx[4*NGLLX+I];

    temp2l = s_dummy_loc[K*NGLL2+I]*hprime_yy[J]
            + s_dummy_loc[K*NGLL2+NGLLX+I]*hprime_yy[NGLLX+J]
            + s_dummy_loc[K*NGLL2+2*NGLLX+I]*hprime_yy[2*NGLLX+J]
            + s_dummy_loc[K*NGLL2+3*NGLLX+I]*hprime_yy[3*NGLLX+J]
            + s_dummy_loc[K*NGLL2+4*NGLLX+I]*hprime_yy[4*NGLLX+J];

    temp3l = s_dummy_loc[J*NGLLX+I]*hprime_zz[K]
            + s_dummy_loc[NGLL2+J*NGLLX+I]*hprime_zz[NGLLX+K]
            + s_dummy_loc[2*NGLL2+J*NGLLX+I]*hprime_zz[2*NGLLX+K]
            + s_dummy_loc[3*NGLL2+J*NGLLX+I]*hprime_zz[3*NGLLX+K]
            + s_dummy_loc[4*NGLL2+J*NGLLX+I]*hprime_zz[4*NGLLX+K];

#endif

    // compute derivatives of ux, uy and uz with respect to x, y and z
    offset = working_element*NGLL3_ALIGN + tx;

    xixl = d_xix[offset];
    xiyl = d_xiy[offset];
    xizl = d_xiz[offset];
    etaxl = d_etax[offset];
    etayl = d_etay[offset];
    etazl = d_etaz[offset];
    gammaxl = d_gammax[offset];
    gammayl = d_gammay[offset];
    gammazl = d_gammaz[offset];

    //  compute the jacobian
    jacobianl = 1.f / (xixl*(etayl*gammazl-etazl*gammayl)
                      -xiyl*(etaxl*gammazl-etazl*gammaxl)
                      +xizl*(etaxl*gammayl-etayl*gammaxl));

    // derivatives of potential
    dpotentialdxl = xixl*temp1l + etaxl*temp2l + gammaxl*temp3l;
    dpotentialdyl = xiyl*temp1l + etayl*temp2l + gammayl*temp3l;
    dpotentialdzl = xizl*temp1l + etazl*temp2l + gammazl*temp3l;

    // compute contribution of rotation and add to gradient of potential
    // this term has no Z component
    if(ROTATION){
      compute_element_oc_rotation(tx,working_element,time,two_omega_earth,deltat,
                                  d_A_array_rotation,d_B_array_rotation,
                                  dpotentialdxl,dpotentialdyl,
                                  &dpotentialdx_with_rot,&dpotentialdy_with_rot);

    }else{
      dpotentialdx_with_rot = dpotentialdxl;
      dpotentialdy_with_rot = dpotentialdyl;
    }

    // pre-computes gravity terms

    // use mesh coordinates to get theta and phi
    // x y z contain r theta phi
    radius = d_xstore[iglob];
    theta = d_ystore[iglob];
    phi = d_zstore[iglob];

    cos_theta = cos(theta);
    sin_theta = sin(theta);
    cos_phi = cos(phi);
    sin_phi = sin(phi);

    // for efficiency replace with lookup table every 100 m in radial direction
    // note: radius in outer core should never be zero,
    //          and arrays in C start from 0, thus we need to subtract -1
    int_radius = rint(radius * R_EARTH_KM * 10.0f ) - 1;

    // depending on gravity or not, different potential definitions are used
    if( ! GRAVITY ){
      // add (chi/rho)grad(rho) term in no gravity case

      // grad(rho)/rho in Cartesian components
      grad_x_ln_rho = sin_theta * cos_phi * d_d_ln_density_dr_table[int_radius];
      grad_y_ln_rho = sin_theta * sin_phi * d_d_ln_density_dr_table[int_radius];
      grad_z_ln_rho = cos_theta * d_d_ln_density_dr_table[int_radius];

      // adding (chi/rho)grad(rho)
      dpotentialdx_with_rot = dpotentialdx_with_rot + s_dummy_loc[tx] * grad_x_ln_rho;
      dpotentialdy_with_rot = dpotentialdy_with_rot + s_dummy_loc[tx] * grad_y_ln_rho;
      dpotentialdzl = dpotentialdzl + s_dummy_loc[tx] * grad_z_ln_rho;

    }else{

      // compute divergence of displacement
      // precompute and store gravity term
      //
      // get g, rho and dg/dr=dg
      // spherical components of the gravitational acceleration
      //
      // Cartesian components of the gravitational acceleration
      // integrate and multiply by rho / Kappa
      gxl = sin_theta*cos_phi;
      gyl = sin_theta*sin_phi;
      gzl = cos_theta;

      // uses potential definition: s = grad(chi)
      // gravity term: - rho * g * 1/kappa grad(chi)

      gravity_term = d_minus_rho_g_over_kappa_fluid[int_radius] * jacobianl * wgll_cube[tx] *
                    ( dpotentialdx_with_rot * gxl + dpotentialdy_with_rot * gyl + dpotentialdzl * gzl);

      // divergence of displacement field with gravity on
      // note: these calculations are only considered for SIMULATION_TYPE == 1 .and. SAVE_FORWARD
      //          and one has set MOVIE_VOLUME_TYPE == 4 when MOVIE_VOLUME is .true.;
      //         in case of SIMULATION_TYPE == 3, it gets overwritten by compute_kernels_outer_core()
      //if (NSPEC_OUTER_CORE_ADJOINT /= 1 && MOVIE_VOLUME ){
      //  div_displfluid(i,j,k,ispec) =  d_minus_rho_g_over_kappa_fluid[int_radius] *
      //        (dpotentialdx_with_rot * gxl + dpotentialdy_with_rot * gyl + dpotentialdzl * gzl);
      //}

    }

    // form the dot product with the test vector
    s_temp1[tx] = jacobianl*(xixl*dpotentialdx_with_rot + xiyl*dpotentialdy_with_rot + xizl*dpotentialdzl);
    s_temp2[tx] = jacobianl*(etaxl*dpotentialdx_with_rot + etayl*dpotentialdy_with_rot + etazl*dpotentialdzl);
    s_temp3[tx] = jacobianl*(gammaxl*dpotentialdx_with_rot + gammayl*dpotentialdy_with_rot + gammazl*dpotentialdzl);
  }

// synchronize all the threads (one thread for each of the NGLL grid points of the
// current spectral element) because we need the whole element to be ready in order
// to be able to compute the matrix products along cut planes of the 3D element below
  __syncthreads();

  if (active) {

#ifndef MANUALLY_UNROLLED_LOOPS

    temp1l = 0.f;
    temp2l = 0.f;
    temp3l = 0.f;

    for (l=0;l<NGLLX;l++) {
        fac1 = hprimewgll_xx[I*NGLLX+l];
        offset1 = K*NGLL2+J*NGLLX+l;
        temp1l += s_temp1[offset1]*fac1;

        //no more assumes hprimewgll_xx = hprimewgll_yy = hprimewgll_zz
        fac2 = hprimewgll_yy[J*NGLLX+l];
        offset2 = K*NGLL2+l*NGLLX+I;
        temp2l += s_temp2[offset2]*fac2;

        fac3 = hprimewgll_zz[K*NGLLX+l];
        offset3 = l*NGLL2+J*NGLLX+I;
        temp3l += s_temp3[offset3]*fac3;
    }
#else

    temp1l = s_temp1[K*NGLL2+J*NGLLX]*hprimewgll_xx[I*NGLLX]
            + s_temp1[K*NGLL2+J*NGLLX+1]*hprimewgll_xx[I*NGLLX+1]
            + s_temp1[K*NGLL2+J*NGLLX+2]*hprimewgll_xx[I*NGLLX+2]
            + s_temp1[K*NGLL2+J*NGLLX+3]*hprimewgll_xx[I*NGLLX+3]
            + s_temp1[K*NGLL2+J*NGLLX+4]*hprimewgll_xx[I*NGLLX+4];


    temp2l = s_temp2[K*NGLL2+I]*hprimewgll_yy[J*NGLLX]
            + s_temp2[K*NGLL2+NGLLX+I]*hprimewgll_yy[J*NGLLX+1]
            + s_temp2[K*NGLL2+2*NGLLX+I]*hprimewgll_yy[J*NGLLX+2]
            + s_temp2[K*NGLL2+3*NGLLX+I]*hprimewgll_yy[J*NGLLX+3]
            + s_temp2[K*NGLL2+4*NGLLX+I]*hprimewgll_yy[J*NGLLX+4];


    temp3l = s_temp3[J*NGLLX+I]*hprimewgll_zz[K*NGLLX]
            + s_temp3[NGLL2+J*NGLLX+I]*hprimewgll_zz[K*NGLLX+1]
            + s_temp3[2*NGLL2+J*NGLLX+I]*hprimewgll_zz[K*NGLLX+2]
            + s_temp3[3*NGLL2+J*NGLLX+I]*hprimewgll_zz[K*NGLLX+3]
            + s_temp3[4*NGLL2+J*NGLLX+I]*hprimewgll_zz[K*NGLLX+4];


#endif

    fac1 = wgllwgll_yz[K*NGLLX+J];
    fac2 = wgllwgll_xz[K*NGLLX+I];
    fac3 = wgllwgll_xy[J*NGLLX+I];

    sum_terms = -(fac1*temp1l + fac2*temp2l + fac3*temp3l);
    if( GRAVITY ) sum_terms += gravity_term;

    iglob = d_ibool[working_element*NGLL3 + tx]-1;

#ifdef USE_TEXTURES
    d_potential_dot_dot[iglob] = tex1Dfetch(tex_potential_dot_dot, iglob)
                                            + sum_terms;
#else

#ifdef USE_MESH_COLORING_GPU
    // no atomic operation needed, colors don't share global points between elements
    d_potential_dot_dot[iglob] += sum_terms;
#else
    //mesh coloring
    if( use_mesh_coloring_gpu ){

      // no atomic operation needed, colors don't share global points between elements
      d_potential_dot_dot[iglob] += sum_terms;

    }else{

      atomicAdd(&d_potential_dot_dot[iglob],sum_terms);

    }
#endif

#endif
  }

#else  // of #ifndef MAKE_KERNEL2_BECOME_STUPID_FOR_TESTS
  d_potential_dot_dot[iglob] = 123.123f;
#endif // of #ifndef MAKE_KERNEL2_BECOME_STUPID_FOR_TESTS
}


/* ----------------------------------------------------------------------------------------------- */

void Kernel_2_outer_core(int nb_blocks_to_compute, Mesh* mp,
                         int d_iphase,
                         int* d_ibool,
                         realw* d_xix,realw* d_xiy,realw* d_xiz,
                         realw* d_etax,realw* d_etay,realw* d_etaz,
                         realw* d_gammax,realw* d_gammay,realw* d_gammaz,
                         realw time, realw b_time,
                         realw* d_A_array_rotation,realw* d_B_array_rotation,
                         realw* d_b_A_array_rotation,realw* d_b_B_array_rotation
                         ){

#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
  exit_on_cuda_error("before outer_core kernel Kernel_2");
#endif

  /* if the grid can handle the number of blocks, we let it be 1D */
  /* grid_2_x = nb_elem_color; */
  /* nb_elem_color is just how many blocks we are computing now */

  int num_blocks_x = nb_blocks_to_compute;
  int num_blocks_y = 1;
  while(num_blocks_x > 65535) {
    num_blocks_x = (int) ceil(num_blocks_x*0.5f);
    num_blocks_y = num_blocks_y*2;
  }

  int threads_2 = NGLL3_PADDED;//BLOCK_SIZE_K2;
  dim3 grid_2(num_blocks_x,num_blocks_y);

  // Cuda timing
  // hipEvent_t start, stop;
  // realw time;
  // hipEventCreate(&start);
  // hipEventCreate(&stop);
  // hipEventRecord( start, 0 );

  Kernel_2_outer_core_impl<<< grid_2, threads_2, 0, 0 >>>(nb_blocks_to_compute,
                                                        mp->NGLOB_OUTER_CORE,
                                                        d_ibool,
                                                        mp->d_phase_ispec_inner_outer_core,
                                                        mp->num_phase_ispec_outer_core,
                                                        d_iphase,
                                                        mp->use_mesh_coloring_gpu,
                                                        mp->d_displ_outer_core,
                                                        mp->d_accel_outer_core,
                                                        d_xix, d_xiy, d_xiz,
                                                        d_etax, d_etay, d_etaz,
                                                        d_gammax, d_gammay, d_gammaz,
                                                        mp->d_hprime_xx, mp->d_hprime_yy, mp->d_hprime_zz,
                                                        mp->d_hprimewgll_xx, mp->d_hprimewgll_yy, mp->d_hprimewgll_zz,
                                                        mp->d_wgllwgll_xy, mp->d_wgllwgll_xz, mp->d_wgllwgll_yz,
                                                        mp->gravity,
                                                        mp->d_xstore_outer_core,mp->d_ystore_outer_core,mp->d_zstore_outer_core,
                                                        mp->d_d_ln_density_dr_table,
                                                        mp->d_minus_rho_g_over_kappa_fluid,
                                                        mp->d_wgll_cube,
                                                        mp->rotation,
                                                        time,
                                                        mp->d_two_omega_earth,
                                                        mp->d_deltat,
                                                        d_A_array_rotation,d_B_array_rotation);

  if(mp->simulation_type == 3) {
    Kernel_2_outer_core_impl<<< grid_2, threads_2, 0, 0 >>>(nb_blocks_to_compute,
                                                          mp->NGLOB_OUTER_CORE,
                                                          d_ibool,
                                                          mp->d_phase_ispec_inner_outer_core,
                                                          mp->num_phase_ispec_outer_core,
                                                          d_iphase,
                                                          mp->use_mesh_coloring_gpu,
                                                          mp->d_b_displ_outer_core,
                                                          mp->d_b_accel_outer_core,
                                                          d_xix, d_xiy, d_xiz,
                                                          d_etax, d_etay, d_etaz,
                                                          d_gammax, d_gammay, d_gammaz,
                                                          mp->d_hprime_xx, mp->d_hprime_yy, mp->d_hprime_zz,
                                                          mp->d_hprimewgll_xx, mp->d_hprimewgll_yy, mp->d_hprimewgll_zz,
                                                          mp->d_wgllwgll_xy, mp->d_wgllwgll_xz, mp->d_wgllwgll_yz,
                                                          mp->gravity,
                                                          mp->d_xstore_outer_core,mp->d_ystore_outer_core,mp->d_zstore_outer_core,
                                                          mp->d_d_ln_density_dr_table,
                                                          mp->d_minus_rho_g_over_kappa_fluid,
                                                          mp->d_wgll_cube,
                                                          mp->rotation,
                                                          b_time,
                                                          mp->d_b_two_omega_earth,
                                                          mp->d_b_deltat,
                                                          d_b_A_array_rotation,d_b_B_array_rotation);
  }

  // hipEventRecord( stop, 0 );
  // hipEventSynchronize( stop );
  // hipEventElapsedTime( &time, start, stop );
  // hipEventDestroy( start );
  // hipEventDestroy( stop );
  // printf("Kernel2 Execution Time: %f ms\n",time);

  /* hipDeviceSynchronize(); */
  /* TRACE("Kernel 2 finished"); */
#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
  //printf("Tried to start with %dx1 blocks\n",nb_blocks_to_compute);
  exit_on_cuda_error("kernel Kernel_2_outer_core");
#endif
}

/* ----------------------------------------------------------------------------------------------- */

// main compute_forces_outer_core CUDA routine

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(compute_forces_outer_core_cuda,
              COMPUTE_FORCES_OUTER_CORE_CUDA)(long* Mesh_pointer_f,
                                            int* iphase,
                                            realw* time_f,
                                            realw* b_time_f) {

  TRACE("compute_forces_outer_core_cuda");

//daniel: debug
  //printf("Running compute_forces_outer_core_cuda\n");
  //double start_time = get_time();

  Mesh* mp = (Mesh*)(*Mesh_pointer_f); // get Mesh from fortran integer wrapper

  int num_elements;
  realw time = *time_f;
  realw b_time = *b_time_f;

  if( *iphase == 1 )
    num_elements = mp->nspec_outer_outer_core;
  else
    num_elements = mp->nspec_inner_outer_core;

  if( num_elements == 0 ) return;

  // mesh coloring
  if( mp->use_mesh_coloring_gpu ){

    // note: array offsets require sorted arrays, such that e.g. ibool starts with elastic elements
    //         and followed by acoustic ones.
    //         acoustic elements also start with outer than inner element ordering

    int nb_colors,nb_blocks_to_compute;
    int istart;
    int color_offset,color_offset_nonpadded;

    // sets up color loop
    if( *iphase == 1 ){
      // outer elements
      nb_colors = mp->num_colors_outer_outer_core;
      istart = 0;

      // array offsets
      color_offset = 0;
      color_offset_nonpadded = 0;
    }else{
      // inner element colors (start after outer elements)
      nb_colors = mp->num_colors_outer_outer_core + mp->num_colors_inner_outer_core;
      istart = mp->num_colors_outer_outer_core;

      // array offsets (inner elements start after outer ones)
      color_offset = mp->nspec_outer_outer_core * NGLL3_PADDED;
      color_offset_nonpadded = mp->nspec_outer_outer_core * NGLL3;
    }

    // loops over colors
    for(int icolor = istart; icolor < nb_colors; icolor++){

      nb_blocks_to_compute = mp->h_num_elem_colors_outer_core[icolor];

      Kernel_2_outer_core(nb_blocks_to_compute,mp,
                          *iphase,
                          mp->d_ibool_outer_core + color_offset_nonpadded,
                          mp->d_xix_outer_core + color_offset,
                          mp->d_xiy_outer_core + color_offset,
                          mp->d_xiz_outer_core + color_offset,
                          mp->d_etax_outer_core + color_offset,
                          mp->d_etay_outer_core + color_offset,
                          mp->d_etaz_outer_core + color_offset,
                          mp->d_gammax_outer_core + color_offset,
                          mp->d_gammay_outer_core + color_offset,
                          mp->d_gammaz_outer_core + color_offset,
                          time,b_time,
                          mp->d_A_array_rotation + color_offset_nonpadded,
                          mp->d_B_array_rotation + color_offset_nonpadded,
                          mp->d_b_A_array_rotation + color_offset_nonpadded,
                          mp->d_b_B_array_rotation + color_offset_nonpadded
                         );

      // for padded and aligned arrays
      color_offset += nb_blocks_to_compute * NGLL3_PADDED;
      // for no-aligned arrays
      color_offset_nonpadded += nb_blocks_to_compute * NGLL3;
    }

  }else{

    // no mesh coloring: uses atomic updates
    Kernel_2_outer_core(num_elements, mp,
                        *iphase,
                        mp->d_ibool_outer_core,
                        mp->d_xix_outer_core,mp->d_xiy_outer_core,mp->d_xiz_outer_core,
                        mp->d_etax_outer_core,mp->d_etay_outer_core,mp->d_etaz_outer_core,
                        mp->d_gammax_outer_core,mp->d_gammay_outer_core,mp->d_gammaz_outer_core,
                        time,b_time,
                        mp->d_A_array_rotation,mp->d_B_array_rotation,
                        mp->d_b_A_array_rotation,mp->d_b_B_array_rotation
                        );

  }

#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
  //double end_time = get_time();
  //printf("Elapsed time: %e\n",end_time-start_time);
  exit_on_cuda_error("compute_forces_outer_core_cuda");
#endif
}

