#include "hip/hip_runtime.h"
/*
 !=====================================================================
 !
 !               S p e c f e m 3 D  V e r s i o n  2 . 0
 !               ---------------------------------------
 !
 !          Main authors: Dimitri Komatitsch and Jeroen Tromp
 !    Princeton University, USA and University of Pau / CNRS / INRIA
 ! (c) Princeton University / California Institute of Technology and University of Pau / CNRS / INRIA
 !                            April 2011
 !
 ! This program is free software; you can redistribute it and/or modify
 ! it under the terms of the GNU General Public License as published by
 ! the Free Software Foundation; either version 2 of the License, or
 ! (at your option) any later version.
 !
 ! This program is distributed in the hope that it will be useful,
 ! but WITHOUT ANY WARRANTY; without even the implied warranty of
 ! MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 ! GNU General Public License for more details.
 !
 ! You should have received a copy of the GNU General Public License along
 ! with this program; if not, write to the Free Software Foundation, Inc.,
 ! 51 Franklin Street, Fifth Floor, Boston, MA 02110-1301 USA.
 !
 !=====================================================================
 */

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

#include <sys/types.h>
#include <unistd.h>
#include <sys/time.h>
#include <sys/resource.h>

#include "config.h"
#include "mesh_constants_cuda.h"


/* ----------------------------------------------------------------------------------------------- */

// ELASTIC SIMULATIONS

/* ----------------------------------------------------------------------------------------------- */

__global__ void compute_kernels_cudakernel(int* ibool,
                                           realw* accel,
                                           realw* b_displ,
                                           realw* epsilondev_xx,
                                           realw* epsilondev_yy,
                                           realw* epsilondev_xy,
                                           realw* epsilondev_xz,
                                           realw* epsilondev_yz,
                                           realw* epsilon_trace_over_3,
                                           realw* b_epsilondev_xx,
                                           realw* b_epsilondev_yy,
                                           realw* b_epsilondev_xy,
                                           realw* b_epsilondev_xz,
                                           realw* b_epsilondev_yz,
                                           realw* b_epsilon_trace_over_3,
                                           realw* rho_kl,
                                           realw* mu_kl,
                                           realw* kappa_kl,
                                           int NSPEC,
                                           realw deltat,
                                           int ANISOTROPIC_KL) {

  int ispec = blockIdx.x + blockIdx.y*gridDim.x;

  // handles case when there is 1 extra block (due to rectangular grid)
  if(ispec < NSPEC) {

    int ijk = threadIdx.x;
    int ijk_ispec = ijk + NGLL3*ispec;
    int iglob = ibool[ijk_ispec] - 1 ;

    // isotropic kernels:
    // density kernel
    rho_kl[ijk_ispec] += deltat * (accel[3*iglob]*b_displ[3*iglob]+
                                   accel[3*iglob+1]*b_displ[3*iglob+1]+
                                   accel[3*iglob+2]*b_displ[3*iglob+2]);

    // isotropic kernel contributions
    if( ! ANISOTROPIC_KL ){
      // shear modulus kernel
      mu_kl[ijk_ispec] += deltat * (epsilondev_xx[ijk_ispec]*b_epsilondev_xx[ijk_ispec]+
                                    epsilondev_yy[ijk_ispec]*b_epsilondev_yy[ijk_ispec]+
                                    (epsilondev_xx[ijk_ispec]+epsilondev_yy[ijk_ispec])*
                                      (b_epsilondev_xx[ijk_ispec]+b_epsilondev_yy[ijk_ispec])+
                                      2*(epsilondev_xy[ijk_ispec]*b_epsilondev_xy[ijk_ispec]+
                                         epsilondev_xz[ijk_ispec]*b_epsilondev_xz[ijk_ispec]+
                                         epsilondev_yz[ijk_ispec]*b_epsilondev_yz[ijk_ispec]));

      // bulk modulus kernel
      kappa_kl[ijk_ispec] += deltat*(9*epsilon_trace_over_3[ijk_ispec]*
                                       b_epsilon_trace_over_3[ijk_ispec]);
    }
  }
}

/* ----------------------------------------------------------------------------------------------- */

__device__ void compute_strain_product(realw* prod,
                                       realw eps_trace_over_3,
                                       realw* epsdev,
                                       realw b_eps_trace_over_3,
                                       realw* b_epsdev){

  realw eps[6],b_eps[6];

  // Building of the local matrix of the strain tensor
  // for the adjoint field and the regular backward field

  // note: indices are -1 compared to fortran routine because of fortran -> C array indexing

  // eps11 et eps22
  eps[0] = epsdev[0] + eps_trace_over_3;
  eps[1] = epsdev[1] + eps_trace_over_3;
  //eps33
  eps[2] = - (eps[0] + eps[1]) + 3.0f*eps_trace_over_3;
  //eps23
  eps[3] = epsdev[4];
  //eps13
  eps[4] = epsdev[3];
  //eps12
  eps[5] = epsdev[2];

  b_eps[0] = b_epsdev[0] + b_eps_trace_over_3;
  b_eps[1] = b_epsdev[1] + b_eps_trace_over_3;
  b_eps[2] = - (b_eps[0] + b_eps[1]) + 3.0f*b_eps_trace_over_3;
  b_eps[3] = b_epsdev[4];
  b_eps[4] = b_epsdev[3];
  b_eps[5] = b_epsdev[2];

  // Computing the 21 strain products without assuming eps(i)*b_eps(j) = eps(j)*b_eps(i)
  int p = 0;
  for(int i=0; i<6; i++){
    for(int j=i; j<6; j++){
      prod[p]=eps[i]*b_eps[j];

      if(j>i){
        prod[p]=prod[p]+eps[j]*b_eps[i];
        if(j>2 && i<3){ prod[p] = prod[p]*2.0f;}
      }

      if(i>2){ prod[p]=prod[p]*4.0f;}

      p=p+1;
    }
  }
}

/* ----------------------------------------------------------------------------------------------- */

__global__ void compute_kernels_ani_cudakernel(int* ibool,
                                               realw* epsilondev_xx,
                                               realw* epsilondev_yy,
                                               realw* epsilondev_xy,
                                               realw* epsilondev_xz,
                                               realw* epsilondev_yz,
                                               realw* epsilon_trace_over_3,
                                               realw* b_epsilondev_xx,
                                               realw* b_epsilondev_yy,
                                               realw* b_epsilondev_xy,
                                               realw* b_epsilondev_xz,
                                               realw* b_epsilondev_yz,
                                               realw* b_epsilon_trace_over_3,
                                               realw* cijkl_kl,
                                               int NSPEC,
                                               realw deltat) {

  int ispec = blockIdx.x + blockIdx.y*gridDim.x;

  // handles case when there is 1 extra block (due to rectangular grid)
  if(ispec < NSPEC) {

    int ijk = threadIdx.x;
    int ijk_ispec = ijk + NGLL3*ispec;

    // fully anisotropic kernel contributions

    realw prod[21];
    realw epsdev[5];
    realw b_epsdev[5];

    epsdev[0] = epsilondev_xx[ijk_ispec];
    epsdev[1] = epsilondev_yy[ijk_ispec];
    epsdev[2] = epsilondev_xy[ijk_ispec];
    epsdev[3] = epsilondev_xz[ijk_ispec];
    epsdev[4] = epsilondev_yz[ijk_ispec];

    b_epsdev[0] = b_epsilondev_xx[ijk_ispec];
    b_epsdev[1] = b_epsilondev_yy[ijk_ispec];
    b_epsdev[2] = b_epsilondev_xy[ijk_ispec];
    b_epsdev[3] = b_epsilondev_xz[ijk_ispec];
    b_epsdev[4] = b_epsilondev_yz[ijk_ispec];

    // fully anisotropic kernel contributions
    compute_strain_product(prod,epsilon_trace_over_3[ijk_ispec],epsdev,
                           b_epsilon_trace_over_3[ijk_ispec],b_epsdev);

    for(int i=0;i<21;i++){
      cijkl_kl[i + 21*ijk_ispec] += deltat * prod[i];
    }
  }
}


/* ----------------------------------------------------------------------------------------------- */


// crust_mantle

extern "C"
void FC_FUNC_(compute_kernels_cm_cuda,
              COMPUTE_KERNELS_CM_CUDA)(long* Mesh_pointer,realw* deltat_f) {

TRACE("compute_kernels_cm_cuda");

  Mesh* mp = (Mesh*)(*Mesh_pointer); //get mesh pointer out of fortran integer container

  int blocksize = NGLL3;
  realw deltat = *deltat_f;

  int num_blocks_x = mp->NSPEC_CRUST_MANTLE;
  int num_blocks_y = 1;
  while(num_blocks_x > 65535) {
    num_blocks_x = (int) ceil(num_blocks_x*0.5f);
    num_blocks_y = num_blocks_y*2;
  }
  dim3 grid(num_blocks_x,num_blocks_y);
  dim3 threads(blocksize,1,1);

  compute_kernels_cudakernel<<<grid,threads>>>(mp->d_ibool_crust_mantle,
                                               mp->d_accel_crust_mantle,
                                               mp->d_b_displ_crust_mantle,
                                               mp->d_epsilondev_xx_crust_mantle,
                                               mp->d_epsilondev_yy_crust_mantle,
                                               mp->d_epsilondev_xy_crust_mantle,
                                               mp->d_epsilondev_xz_crust_mantle,
                                               mp->d_epsilondev_yz_crust_mantle,
                                               mp->d_eps_trace_over_3_crust_mantle,
                                               mp->d_b_epsilondev_xx_crust_mantle,
                                               mp->d_b_epsilondev_yy_crust_mantle,
                                               mp->d_b_epsilondev_xy_crust_mantle,
                                               mp->d_b_epsilondev_xz_crust_mantle,
                                               mp->d_b_epsilondev_yz_crust_mantle,
                                               mp->d_b_eps_trace_over_3_crust_mantle,
                                               mp->d_rho_kl_crust_mantle,
                                               mp->d_beta_kl_crust_mantle,
                                               mp->d_alpha_kl_crust_mantle,
                                               mp->NSPEC_CRUST_MANTLE,
                                               deltat,
                                               mp->anisotropic_kl);

  if(mp->anisotropic_kl){
    compute_kernels_ani_cudakernel<<<grid,threads>>>(mp->d_ibool_crust_mantle,
                                                    mp->d_epsilondev_xx_crust_mantle,
                                                    mp->d_epsilondev_yy_crust_mantle,
                                                    mp->d_epsilondev_xy_crust_mantle,
                                                    mp->d_epsilondev_xz_crust_mantle,
                                                    mp->d_epsilondev_yz_crust_mantle,
                                                    mp->d_eps_trace_over_3_crust_mantle,
                                                    mp->d_b_epsilondev_xx_crust_mantle,
                                                    mp->d_b_epsilondev_yy_crust_mantle,
                                                    mp->d_b_epsilondev_xy_crust_mantle,
                                                    mp->d_b_epsilondev_xz_crust_mantle,
                                                    mp->d_b_epsilondev_yz_crust_mantle,
                                                    mp->d_b_eps_trace_over_3_crust_mantle,
                                                    mp->d_cijkl_kl_crust_mantle,
                                                    mp->NSPEC_CRUST_MANTLE,
                                                    deltat);

  }

#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
  exit_on_cuda_error("compute_kernels_elastic_cuda");
#endif
}


/* ----------------------------------------------------------------------------------------------- */


// inner_core

extern "C"
void FC_FUNC_(compute_kernels_ic_cuda,
              COMPUTE_KERNELS_IC_CUDA)(long* Mesh_pointer,realw* deltat_f) {

  TRACE("compute_kernels_cm_cuda");

  Mesh* mp = (Mesh*)(*Mesh_pointer); //get mesh pointer out of fortran integer container

  int blocksize = NGLL3;
  realw deltat = *deltat_f;

  int num_blocks_x = mp->NSPEC_INNER_CORE;
  int num_blocks_y = 1;
  while(num_blocks_x > 65535) {
    num_blocks_x = (int) ceil(num_blocks_x*0.5f);
    num_blocks_y = num_blocks_y*2;
  }
  dim3 grid(num_blocks_x,num_blocks_y);
  dim3 threads(blocksize,1,1);

  // only isotropic kernels in inner core so far implemented
  int aniso_flag = 0;

  compute_kernels_cudakernel<<<grid,threads>>>(mp->d_ibool_inner_core,
                                               mp->d_accel_inner_core,
                                               mp->d_b_displ_inner_core,
                                               mp->d_epsilondev_xx_inner_core,
                                               mp->d_epsilondev_yy_inner_core,
                                               mp->d_epsilondev_xy_inner_core,
                                               mp->d_epsilondev_xz_inner_core,
                                               mp->d_epsilondev_yz_inner_core,
                                               mp->d_eps_trace_over_3_inner_core,
                                               mp->d_b_epsilondev_xx_inner_core,
                                               mp->d_b_epsilondev_yy_inner_core,
                                               mp->d_b_epsilondev_xy_inner_core,
                                               mp->d_b_epsilondev_xz_inner_core,
                                               mp->d_b_epsilondev_yz_inner_core,
                                               mp->d_b_eps_trace_over_3_inner_core,
                                               mp->d_rho_kl_inner_core,
                                               mp->d_beta_kl_inner_core,
                                               mp->d_alpha_kl_inner_core,
                                               mp->NSPEC_INNER_CORE,
                                               deltat,
                                               aniso_flag);


#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
  exit_on_cuda_error("compute_kernels_elastic_cuda");
#endif
}


/* ----------------------------------------------------------------------------------------------- */

// ACOUSTIC SIMULATIONS

// for outer core region

/* ----------------------------------------------------------------------------------------------- */


__device__ void compute_gradient_kernel(int ijk,
                                        int ispec,
                                        realw* scalar_field,
                                        realw* vector_field_element,
                                        realw* hprime_xx,
                                        realw* d_xix,
                                        realw* d_xiy,
                                        realw* d_xiz,
                                        realw* d_etax,
                                        realw* d_etay,
                                        realw* d_etaz,
                                        realw* d_gammax,
                                        realw* d_gammay,
                                        realw* d_gammaz) {

  realw temp1l,temp2l,temp3l;
  realw hp1,hp2,hp3;
  realw xixl,xiyl,xizl,etaxl,etayl,etazl,gammaxl,gammayl,gammazl;
  int l,offset,offset1,offset2,offset3;

  int K = (ijk/NGLL2);
  int J = ((ijk-K*NGLL2)/NGLLX);
  int I = (ijk-K*NGLL2-J*NGLLX);

  // derivative along x
  temp1l = 0.f;
  for( l=0; l<NGLLX;l++){
    hp1 = hprime_xx[l*NGLLX+I];
    offset1 = K*NGLL2+J*NGLLX+l;
    temp1l += scalar_field[offset1]*hp1;
  }

  // derivative along y
  temp2l = 0.f;
  for( l=0; l<NGLLX;l++){
    //assumes that hprime_xx = hprime_yy = hprime_zz
    hp2 = hprime_xx[l*NGLLX+J];
    offset2 = K*NGLL2+l*NGLLX+I;
    temp2l += scalar_field[offset2]*hp2;
  }

  // derivative along z
  temp3l = 0.f;
  for( l=0; l<NGLLX;l++){
    //assumes that hprime_xx = hprime_yy = hprime_zz
    hp3 = hprime_xx[l*NGLLX+K];
    offset3 = l*NGLL2+J*NGLLX+I;
    temp3l += scalar_field[offset3]*hp3;
  }

  offset = ispec*NGLL3_PADDED + ijk;

  xixl = d_xix[offset];
  xiyl = d_xiy[offset];
  xizl = d_xiz[offset];
  etaxl = d_etax[offset];
  etayl = d_etay[offset];
  etazl = d_etaz[offset];
  gammaxl = d_gammax[offset];
  gammayl = d_gammay[offset];
  gammazl = d_gammaz[offset];

  // note: global version uses a different potential definition, no need to divide by rho
  //rho_invl = 1.0f / rhol;

  // derivatives of acoustic scalar potential field on GLL points
  vector_field_element[0] = temp1l*xixl + temp2l*etaxl + temp3l*gammaxl;
  vector_field_element[1] = temp1l*xiyl + temp2l*etayl + temp3l*gammayl;
  vector_field_element[2] = temp1l*xizl + temp2l*etazl + temp3l*gammazl;

}

/* ----------------------------------------------------------------------------------------------- */


__global__ void compute_kernels_acoustic_kernel(int* ibool,
                                                realw* rhostore,
                                                realw* kappastore,
                                                realw* hprime_xx,
                                                realw* d_xix,
                                                realw* d_xiy,
                                                realw* d_xiz,
                                                realw* d_etax,
                                                realw* d_etay,
                                                realw* d_etaz,
                                                realw* d_gammax,
                                                realw* d_gammay,
                                                realw* d_gammaz,
                                                realw* potential_dot_dot_acoustic,
                                                realw* b_potential_acoustic,
                                                realw* b_potential_dot_dot_acoustic,
                                                realw* rho_ac_kl,
                                                realw* kappa_ac_kl,
                                                realw deltat,
                                                int NSPEC) {

  int ispec = blockIdx.x + blockIdx.y*gridDim.x;

  // handles case when there is 1 extra block (due to rectangular grid)
  if( ispec < NSPEC ){

    int ijk = threadIdx.x;

    // local and global indices
    int ijk_ispec = ijk + NGLL3*ispec;
    int ijk_ispec_padded = ijk + NGLL3_PADDED*ispec;
    int iglob = ibool[ijk_ispec] - 1;

    realw accel_elm[3];
    realw b_displ_elm[3];
    realw rhol,kappal;
    realw div_displ,b_div_displ;

    // shared memory between all threads within this block
    __shared__ realw scalar_field_displ[NGLL3];
    __shared__ realw scalar_field_accel[NGLL3];

    // copy field values
    scalar_field_displ[ijk] = b_potential_acoustic[iglob];
    scalar_field_accel[ijk] = potential_dot_dot_acoustic[iglob];
    __syncthreads();

    // displacement vector from backward field
    compute_gradient_kernel(ijk,ispec,scalar_field_displ,b_displ_elm,
                            hprime_xx,
                            d_xix,d_xiy,d_xiz,d_etax,d_etay,d_etaz,d_gammax,d_gammay,d_gammaz);

    // acceleration vector
    compute_gradient_kernel(ijk,ispec,scalar_field_accel,accel_elm,
                            hprime_xx,
                            d_xix,d_xiy,d_xiz,d_etax,d_etay,d_etaz,d_gammax,d_gammay,d_gammaz);

    // gets material parameter
    rhol = rhostore[ijk_ispec_padded];

    // density kernel
    rho_ac_kl[ijk_ispec] += deltat * rhol * (accel_elm[0]*b_displ_elm[0] +
                                             accel_elm[1]*b_displ_elm[1] +
                                             accel_elm[2]*b_displ_elm[2]);

    // bulk modulus kernel
    kappal = rhol/ kappastore[ijk_ispec_padded];

    div_displ = kappal * potential_dot_dot_acoustic[iglob];
    b_div_displ = kappal * b_potential_dot_dot_acoustic[iglob];

    kappa_ac_kl[ijk_ispec] += deltat * div_displ * b_div_displ;
  }
}

/* ----------------------------------------------------------------------------------------------- */


extern "C"
void FC_FUNC_(compute_kernels_oc_cuda,
              COMPUTE_KERNELS_OC_CUDA)(long* Mesh_pointer,realw* deltat_f) {

TRACE("compute_kernels_oc_cuda");

  Mesh* mp = (Mesh*)(*Mesh_pointer); //get mesh pointer out of fortran integer container

  int blocksize = NGLL3; // NGLLX*NGLLY*NGLLZ
  realw deltat = *deltat_f;

  int num_blocks_x = mp->NSPEC_OUTER_CORE;
  int num_blocks_y = 1;
  while(num_blocks_x > 65535) {
    num_blocks_x = (int) ceil(num_blocks_x*0.5f);
    num_blocks_y = num_blocks_y*2;
  }

  dim3 grid(num_blocks_x,num_blocks_y);
  dim3 threads(blocksize,1,1);

  compute_kernels_acoustic_kernel<<<grid,threads>>>(mp->d_ibool_outer_core,
                                                    mp->d_rhostore_outer_core,
                                                    mp->d_kappavstore_outer_core,
                                                    mp->d_hprime_xx,
                                                    mp->d_xix_outer_core,
                                                    mp->d_xiy_outer_core,
                                                    mp->d_xiz_outer_core,
                                                    mp->d_etax_outer_core,
                                                    mp->d_etay_outer_core,
                                                    mp->d_etaz_outer_core,
                                                    mp->d_gammax_outer_core,
                                                    mp->d_gammay_outer_core,
                                                    mp->d_gammaz_outer_core,
                                                    mp->d_accel_outer_core,
                                                    mp->d_b_displ_outer_core,
                                                    mp->d_b_accel_outer_core,
                                                    mp->d_rho_kl_outer_core,
                                                    mp->d_alpha_kl_outer_core,
                                                    deltat,
                                                    mp->NSPEC_OUTER_CORE);

#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
  exit_on_cuda_error("compute_kernels_oc_kernel");
#endif
}

/* ----------------------------------------------------------------------------------------------- */

// NOISE SIMULATIONS

/* ----------------------------------------------------------------------------------------------- */


__global__ void compute_kernels_strength_noise_cuda_kernel(realw* displ,
                                                           int* ibelm_top,
                                                           int* ibool,
                                                           realw* noise_surface_movie,
                                                           realw* normal_x_noise,
                                                           realw* normal_y_noise,
                                                           realw* normal_z_noise,
                                                           realw* Sigma_kl,
                                                           realw deltat,
                                                           int nspec_top) {
  int iface = blockIdx.x + blockIdx.y*gridDim.x;

  if(iface < nspec_top) {

    int ispec = ibelm_top[iface]-1;
    int igll = threadIdx.x;
    int ipoin = igll + NGLL2*iface;

    int k = NGLLX-1;
    int j = (igll/NGLLX);
    int i = (igll-j*NGLLX);

    int iglob = ibool[INDEX4(NGLLX,NGLLX,NGLLX,i,j,k,ispec)] - 1 ;

    realw eta = ( noise_surface_movie[INDEX3(NDIM,NGLL2,0,igll,iface)]*normal_x_noise[ipoin]+
                 noise_surface_movie[INDEX3(NDIM,NGLL2,1,igll,iface)]*normal_y_noise[ipoin]+
                 noise_surface_movie[INDEX3(NDIM,NGLL2,2,igll,iface)]*normal_z_noise[ipoin]);

    Sigma_kl[INDEX4(NGLLX,NGLLX,NGLLX,i,j,k,ispec)] += deltat*eta*
                                                      (normal_x_noise[ipoin]*displ[3*iglob]+
                                                       normal_y_noise[ipoin]*displ[1+3*iglob]+
                                                       normal_z_noise[ipoin]*displ[2+3*iglob]);
  }
}

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(compute_kernels_strgth_noise_cu,
              COMPUTE_KERNELS_STRGTH_NOISE_CU)(long* Mesh_pointer,
                                               realw* h_noise_surface_movie,
                                               realw* deltat_f) {

  TRACE("compute_kernels_strgth_noise_cu");

  Mesh* mp = (Mesh*)(*Mesh_pointer); //get mesh pointer out of fortran integer container

  realw deltat = *deltat_f;

  int num_blocks_x = mp->nspec2D_top_crust_mantle;
  int num_blocks_y = 1;
  while(num_blocks_x > 65535) {
    num_blocks_x = (int) ceil(num_blocks_x*0.5f);
    num_blocks_y = num_blocks_y*2;
  }

  dim3 grid(num_blocks_x,num_blocks_y);
  dim3 threads(NGLL2,1,1);

  // copies surface buffer to GPU
  print_CUDA_error_if_any(hipMemcpy(mp->d_noise_surface_movie,h_noise_surface_movie,
                                     NDIM*NGLL2*(mp->nspec2D_top_crust_mantle)*sizeof(realw),
                                     hipMemcpyHostToDevice),90900);

  // calculates noise strength kernel
  compute_kernels_strength_noise_cuda_kernel<<<grid,threads>>>(mp->d_displ_crust_mantle,
                                                               mp->d_ibelm_top_crust_mantle,
                                                               mp->d_ibool_crust_mantle,
                                                               mp->d_noise_surface_movie,
                                                               mp->d_normal_x_noise,
                                                               mp->d_normal_y_noise,
                                                               mp->d_normal_z_noise,
                                                               mp->d_Sigma_kl,
                                                               deltat,
                                                               mp->nspec2D_top_crust_mantle);

#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
  exit_on_cuda_error("compute_kernels_strength_noise_cuda_kernel");
#endif
}


/* ----------------------------------------------------------------------------------------------- */

// preconditioner (approximate Hessian kernel)

/* ----------------------------------------------------------------------------------------------- */

__global__ void compute_kernels_hess_cudakernel(int* ibool,
                                                realw* accel,
                                                realw* b_accel,
                                                realw* hess_kl,
                                                realw deltat,
                                                int NSPEC_AB) {

  int ispec = blockIdx.x + blockIdx.y*gridDim.x;

  // handles case when there is 1 extra block (due to rectangular grid)
  if(ispec < NSPEC_AB) {

    int ijk = threadIdx.x;
    int ijk_ispec = ijk + NGLL3*ispec;
    int iglob = ibool[ijk_ispec] - 1 ;

    // approximate hessian
    hess_kl[ijk_ispec] += deltat * (accel[3*iglob]*b_accel[3*iglob] +
                                    accel[3*iglob+1]*b_accel[3*iglob+1] +
                                    accel[3*iglob+2]*b_accel[3*iglob+2]);
  }
}

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(compute_kernels_hess_cuda,
              COMPUTE_KERNELS_HESS_CUDA)(long* Mesh_pointer,
                                         realw* deltat_f) {
  TRACE("compute_kernels_hess_cuda");

  Mesh* mp = (Mesh*)(*Mesh_pointer); //get mesh pointer out of fortran integer container

  // checks
  if( ! mp->approximate_hess_kl ){exit_on_cuda_error("approximate_hess_kl flag not properly initialized");}

  int blocksize = NGLL3; // NGLLX*NGLLY*NGLLZ
  realw deltat = *deltat_f;

  int num_blocks_x = mp->NSPEC_CRUST_MANTLE;
  int num_blocks_y = 1;
  while(num_blocks_x > 65535) {
    num_blocks_x = (int) ceil(num_blocks_x*0.5f);
    num_blocks_y = num_blocks_y*2;
  }

  dim3 grid(num_blocks_x,num_blocks_y);
  dim3 threads(blocksize,1,1);

  compute_kernels_hess_cudakernel<<<grid,threads>>>(mp->d_ibool_crust_mantle,
                                                    mp->d_accel_crust_mantle,
                                                    mp->d_b_accel_crust_mantle,
                                                    mp->d_hess_kl_crust_mantle,
                                                    deltat,
                                                    mp->NSPEC_CRUST_MANTLE);

#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
  exit_on_cuda_error("compute_kernels_hess_cuda");
#endif
}

