/*
 !=====================================================================
 !
 !               S p e c f e m 3 D  V e r s i o n  2 . 0
 !               ---------------------------------------
 !
 !          Main authors: Dimitri Komatitsch and Jeroen Tromp
 !    Princeton University, USA and University of Pau / CNRS / INRIA
 ! (c) Princeton University / California Institute of Technology and University of Pau / CNRS / INRIA
 !                            April 2011
 !
 ! This program is free software; you can redistribute it and/or modify
 ! it under the terms of the GNU General Public License as published by
 ! the Free Software Foundation; either version 2 of the License, or
 ! (at your option) any later version.
 !
 ! This program is distributed in the hope that it will be useful,
 ! but WITHOUT ANY WARRANTY; without even the implied warranty of
 ! MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 ! GNU General Public License for more details.
 !
 ! You should have received a copy of the GNU General Public License along
 ! with this program; if not, write to the Free Software Foundation, Inc.,
 ! 51 Franklin Street, Fifth Floor, Boston, MA 02110-1301 USA.
 !
 !=====================================================================
 */

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

#include <sys/time.h>
#include <sys/resource.h>

#include "config.h"
#include "mesh_constants_cuda.h"

/* ----------------------------------------------------------------------------------------------- */

// Transfer functions

/* ----------------------------------------------------------------------------------------------- */



/* ----------------------------------------------------------------------------------------------- */

// transfer memory from CPU host to GPU device

/* ----------------------------------------------------------------------------------------------- */

// crust_mantle
extern "C"
void FC_FUNC_(transfer_fields_cm_to_device,
              TRANSFER_FIELDS_CM_TO_DEVICE)(int* size, realw* displ, realw* veloc, realw* accel,long* Mesh_pointer_f) {

TRACE("transfer_fields_cm_to_device");

  Mesh* mp = (Mesh*)(*Mesh_pointer_f); //get mesh pointer out of fortran integer container

  print_CUDA_error_if_any(hipMemcpy(mp->d_displ_crust_mantle,displ,sizeof(realw)*(*size),hipMemcpyHostToDevice),40003);
  print_CUDA_error_if_any(hipMemcpy(mp->d_veloc_crust_mantle,veloc,sizeof(realw)*(*size),hipMemcpyHostToDevice),40004);
  print_CUDA_error_if_any(hipMemcpy(mp->d_accel_crust_mantle,accel,sizeof(realw)*(*size),hipMemcpyHostToDevice),40005);

}

// inner_core
extern "C"
void FC_FUNC_(transfer_fields_ic_to_device,
              TRANSFER_FIELDS_IC_TO_DEVICE)(int* size, realw* displ, realw* veloc, realw* accel,long* Mesh_pointer_f) {

  TRACE("transfer_fields_ic_to_device");

  Mesh* mp = (Mesh*)(*Mesh_pointer_f); //get mesh pointer out of fortran integer container

  print_CUDA_error_if_any(hipMemcpy(mp->d_displ_inner_core,displ,sizeof(realw)*(*size),hipMemcpyHostToDevice),40003);
  print_CUDA_error_if_any(hipMemcpy(mp->d_veloc_inner_core,veloc,sizeof(realw)*(*size),hipMemcpyHostToDevice),40004);
  print_CUDA_error_if_any(hipMemcpy(mp->d_accel_inner_core,accel,sizeof(realw)*(*size),hipMemcpyHostToDevice),40005);

}

// outer_core
extern "C"
void FC_FUNC_(transfer_fields_oc_to_device,
              TRANSFER_FIELDS_OC_TO_DEVICE)(int* size, realw* displ, realw* veloc, realw* accel,long* Mesh_pointer_f) {

  TRACE("transfer_fields_oc_to_device");

  Mesh* mp = (Mesh*)(*Mesh_pointer_f); //get mesh pointer out of fortran integer container

  print_CUDA_error_if_any(hipMemcpy(mp->d_displ_outer_core,displ,sizeof(realw)*(*size),hipMemcpyHostToDevice),40003);
  print_CUDA_error_if_any(hipMemcpy(mp->d_veloc_outer_core,veloc,sizeof(realw)*(*size),hipMemcpyHostToDevice),40004);
  print_CUDA_error_if_any(hipMemcpy(mp->d_accel_outer_core,accel,sizeof(realw)*(*size),hipMemcpyHostToDevice),40005);

}

/* ----------------------------------------------------------------------------------------------- */

// backward/reconstructed fields

// crust_mantle
extern "C"
void FC_FUNC_(transfer_b_fields_cm_to_device,
              TRANSFER_FIELDS_B_CM_TO_DEVICE)(int* size, realw* b_displ, realw* b_veloc, realw* b_accel,
                                              long* Mesh_pointer_f) {

  TRACE("transfer_fields_b_cm_to_device");

  Mesh* mp = (Mesh*)(*Mesh_pointer_f); //get mesh pointer out of fortran integer container
  print_CUDA_error_if_any(hipMemcpy(mp->d_b_displ_crust_mantle,b_displ,sizeof(realw)*(*size),hipMemcpyHostToDevice),40003);
  print_CUDA_error_if_any(hipMemcpy(mp->d_b_veloc_crust_mantle,b_veloc,sizeof(realw)*(*size),hipMemcpyHostToDevice),40004);
  print_CUDA_error_if_any(hipMemcpy(mp->d_b_accel_crust_mantle,b_accel,sizeof(realw)*(*size),hipMemcpyHostToDevice),40005);

}

// inner_core
extern "C"
void FC_FUNC_(transfer_b_fields_ic_to_device,
              TRANSFER_FIELDS_B_IC_TO_DEVICE)(int* size, realw* b_displ, realw* b_veloc, realw* b_accel,
                                              long* Mesh_pointer_f) {

  TRACE("transfer_fields_b_ic_to_device");

  Mesh* mp = (Mesh*)(*Mesh_pointer_f); //get mesh pointer out of fortran integer container
  print_CUDA_error_if_any(hipMemcpy(mp->d_b_displ_inner_core,b_displ,sizeof(realw)*(*size),hipMemcpyHostToDevice),40003);
  print_CUDA_error_if_any(hipMemcpy(mp->d_b_veloc_inner_core,b_veloc,sizeof(realw)*(*size),hipMemcpyHostToDevice),40004);
  print_CUDA_error_if_any(hipMemcpy(mp->d_b_accel_inner_core,b_accel,sizeof(realw)*(*size),hipMemcpyHostToDevice),40005);

}

// outer_core
extern "C"
void FC_FUNC_(transfer_b_fields_oc_to_device,
              TRANSFER_FIELDS_B_OC_TO_DEVICE)(int* size, realw* b_displ, realw* b_veloc, realw* b_accel,
                                              long* Mesh_pointer_f) {

  TRACE("transfer_fields_b_oc_to_device");

  Mesh* mp = (Mesh*)(*Mesh_pointer_f); //get mesh pointer out of fortran integer container
  print_CUDA_error_if_any(hipMemcpy(mp->d_b_displ_outer_core,b_displ,sizeof(realw)*(*size),hipMemcpyHostToDevice),40003);
  print_CUDA_error_if_any(hipMemcpy(mp->d_b_veloc_outer_core,b_veloc,sizeof(realw)*(*size),hipMemcpyHostToDevice),40004);
  print_CUDA_error_if_any(hipMemcpy(mp->d_b_accel_outer_core,b_accel,sizeof(realw)*(*size),hipMemcpyHostToDevice),40005);

}

/* ----------------------------------------------------------------------------------------------- */

// transfer memory from GPU device to CPU host

/* ----------------------------------------------------------------------------------------------- */

// crust_mantle
extern "C"
void FC_FUNC_(transfer_fields_cm_from_device,
              TRANSFER_FIELDS_CM_FROM_DEVICE)(int* size, realw* displ, realw* veloc, realw* accel,long* Mesh_pointer_f) {

  TRACE("transfer_fields_cm_from_device");

  Mesh* mp = (Mesh*)(*Mesh_pointer_f); //get mesh pointer out of fortran integer container

  print_CUDA_error_if_any(hipMemcpy(displ,mp->d_displ_crust_mantle,sizeof(realw)*(*size),hipMemcpyDeviceToHost),40006);
  print_CUDA_error_if_any(hipMemcpy(veloc,mp->d_veloc_crust_mantle,sizeof(realw)*(*size),hipMemcpyDeviceToHost),40007);
  print_CUDA_error_if_any(hipMemcpy(accel,mp->d_accel_crust_mantle,sizeof(realw)*(*size),hipMemcpyDeviceToHost),40008);

}

/* ----------------------------------------------------------------------------------------------- */

// inner_core
extern "C"
void FC_FUNC_(transfer_fields_ic_from_device,
              TRANSFER_FIELDS_IC_FROM_DEVICE)(int* size, realw* displ, realw* veloc, realw* accel,long* Mesh_pointer_f) {

  TRACE("transfer_fields_ic_from_device");

  Mesh* mp = (Mesh*)(*Mesh_pointer_f); //get mesh pointer out of fortran integer container

  print_CUDA_error_if_any(hipMemcpy(displ,mp->d_displ_inner_core,sizeof(realw)*(*size),hipMemcpyDeviceToHost),40006);
  print_CUDA_error_if_any(hipMemcpy(veloc,mp->d_veloc_inner_core,sizeof(realw)*(*size),hipMemcpyDeviceToHost),40007);
  print_CUDA_error_if_any(hipMemcpy(accel,mp->d_accel_inner_core,sizeof(realw)*(*size),hipMemcpyDeviceToHost),40008);

}

/* ----------------------------------------------------------------------------------------------- */

// outer_core
extern "C"
void FC_FUNC_(transfer_fields_oc_from_device,
              TRANSFER_FIELDS_OC_FROM_DEVICE)(int* size, realw* displ, realw* veloc, realw* accel,long* Mesh_pointer_f) {

  TRACE("transfer_fields_oc_from_device");

  Mesh* mp = (Mesh*)(*Mesh_pointer_f); //get mesh pointer out of fortran integer container

  print_CUDA_error_if_any(hipMemcpy(displ,mp->d_displ_outer_core,sizeof(realw)*(*size),hipMemcpyDeviceToHost),40006);
  print_CUDA_error_if_any(hipMemcpy(veloc,mp->d_veloc_outer_core,sizeof(realw)*(*size),hipMemcpyDeviceToHost),40007);
  print_CUDA_error_if_any(hipMemcpy(accel,mp->d_accel_outer_core,sizeof(realw)*(*size),hipMemcpyDeviceToHost),40008);

}

/* ----------------------------------------------------------------------------------------------- */

// backward/reconstructed fields

// crust_mantle
extern "C"
void FC_FUNC_(transfer_b_fields_cm_from_device,
              TRANSFER_B_FIELDS_CM_FROM_DEVICE)(int* size, realw* b_displ, realw* b_veloc, realw* b_accel,
                                                long* Mesh_pointer_f) {

TRACE("transfer_b_fields_cm_from_device");

  Mesh* mp = (Mesh*)(*Mesh_pointer_f); //get mesh pointer out of fortran integer container

  print_CUDA_error_if_any(hipMemcpy(b_displ,mp->d_b_displ_crust_mantle,sizeof(realw)*(*size),hipMemcpyDeviceToHost),40006);
  print_CUDA_error_if_any(hipMemcpy(b_veloc,mp->d_b_veloc_crust_mantle,sizeof(realw)*(*size),hipMemcpyDeviceToHost),40007);
  print_CUDA_error_if_any(hipMemcpy(b_accel,mp->d_b_accel_crust_mantle,sizeof(realw)*(*size),hipMemcpyDeviceToHost),40008);

}

/* ----------------------------------------------------------------------------------------------- */

// inner_core
extern "C"
void FC_FUNC_(transfer_b_fields_ic_from_device,
              TRANSFER_B_FIELDS_IC_FROM_DEVICE)(int* size, realw* b_displ, realw* b_veloc, realw* b_accel,
                                                long* Mesh_pointer_f) {
  TRACE("transfer_fields_b_ic_from_device");

  Mesh* mp = (Mesh*)(*Mesh_pointer_f); //get mesh pointer out of fortran integer container

  print_CUDA_error_if_any(hipMemcpy(b_displ,mp->d_b_displ_inner_core,sizeof(realw)*(*size),hipMemcpyDeviceToHost),40006);
  print_CUDA_error_if_any(hipMemcpy(b_veloc,mp->d_b_veloc_inner_core,sizeof(realw)*(*size),hipMemcpyDeviceToHost),40007);
  print_CUDA_error_if_any(hipMemcpy(b_accel,mp->d_b_accel_inner_core,sizeof(realw)*(*size),hipMemcpyDeviceToHost),40008);

}

/* ----------------------------------------------------------------------------------------------- */

// outer_core
extern "C"
void FC_FUNC_(transfer_b_fields_oc_from_device,
              TRANSFER_B_FIELDS_OC_FROM_DEVICE)(int* size, realw* b_displ, realw* b_veloc, realw* b_accel,
                                                long* Mesh_pointer_f) {

  TRACE("transfer_b_fields_oc_from_device");

  Mesh* mp = (Mesh*)(*Mesh_pointer_f); //get mesh pointer out of fortran integer container

  print_CUDA_error_if_any(hipMemcpy(b_displ,mp->d_b_displ_outer_core,sizeof(realw)*(*size),hipMemcpyDeviceToHost),40006);
  print_CUDA_error_if_any(hipMemcpy(b_veloc,mp->d_b_veloc_outer_core,sizeof(realw)*(*size),hipMemcpyDeviceToHost),40007);
  print_CUDA_error_if_any(hipMemcpy(b_accel,mp->d_b_accel_outer_core,sizeof(realw)*(*size),hipMemcpyDeviceToHost),40008);

}

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(transfer_accel_cm_to_device,
              TRANSFER_ACCEL_CM_TO_DEVICE)(int* size, realw* accel,long* Mesh_pointer_f) {

TRACE("transfer_accel_cm_to_device");

  Mesh* mp = (Mesh*)(*Mesh_pointer_f); //get mesh pointer out of fortran integer container

  print_CUDA_error_if_any(hipMemcpy(mp->d_accel_crust_mantle,accel,sizeof(realw)*(*size),hipMemcpyHostToDevice),40016);

}


/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(transfer_displ_cm_from_device,
              TRANSFER_DISPL_CM_FROM_DEVICE)(int* size, realw* displ, long* Mesh_pointer_f) {

  TRACE("transfer_displ_cm_from_device");

  Mesh* mp = (Mesh*)(*Mesh_pointer_f); //get mesh pointer out of fortran integer container

  print_CUDA_error_if_any(hipMemcpy(displ,mp->d_displ_crust_mantle,sizeof(realw)*(*size),hipMemcpyDeviceToHost),40006);
}

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(transfer_b_displ_cm_from_device,
              TRANSFER_B_DISPL_CM_FROM_DEVICE)(int* size, realw* displ, long* Mesh_pointer_f) {

  TRACE("transfer_b_displ_cm_from_device");

  Mesh* mp = (Mesh*)(*Mesh_pointer_f); //get mesh pointer out of fortran integer container

  print_CUDA_error_if_any(hipMemcpy(displ,mp->d_b_displ_crust_mantle,sizeof(realw)*(*size),hipMemcpyDeviceToHost),40006);
}

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(transfer_displ_ic_from_device,
              TRANSFER_DISPL_IC_FROM_DEVICE)(int* size, realw* displ, long* Mesh_pointer_f) {

  TRACE("transfer_displ_ic_from_device");

  Mesh* mp = (Mesh*)(*Mesh_pointer_f); //get mesh pointer out of fortran integer container

  print_CUDA_error_if_any(hipMemcpy(displ,mp->d_displ_inner_core,sizeof(realw)*(*size),hipMemcpyDeviceToHost),40006);
}

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(transfer_b_displ_ic_from_device,
              TRANSFER_B_DISPL_IC_FROM_DEVICE)(int* size, realw* displ, long* Mesh_pointer_f) {

  TRACE("transfer_b_displ_ic_from_device");

  Mesh* mp = (Mesh*)(*Mesh_pointer_f); //get mesh pointer out of fortran integer container

  print_CUDA_error_if_any(hipMemcpy(displ,mp->d_b_displ_inner_core,sizeof(realw)*(*size),hipMemcpyDeviceToHost),40006);
}

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(transfer_displ_oc_from_device,
              TRANSFER_DISPL_OC_FROM_DEVICE)(int* size, realw* displ, long* Mesh_pointer_f) {

  TRACE("transfer_displ_oc_from_device");

  Mesh* mp = (Mesh*)(*Mesh_pointer_f); //get mesh pointer out of fortran integer container

  print_CUDA_error_if_any(hipMemcpy(displ,mp->d_displ_outer_core,sizeof(realw)*(*size),hipMemcpyDeviceToHost),40006);
}

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(transfer_b_displ_oc_from_device,
              TRANSFER_B_DISPL_OC_FROM_DEVICE)(int* size, realw* displ, long* Mesh_pointer_f) {

  TRACE("transfer_b_displ_oc_from_device");

  Mesh* mp = (Mesh*)(*Mesh_pointer_f); //get mesh pointer out of fortran integer container

  print_CUDA_error_if_any(hipMemcpy(displ,mp->d_b_displ_outer_core,sizeof(realw)*(*size),hipMemcpyDeviceToHost),40006);
}

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(transfer_veloc_cm_from_device,
              TRANSFER_DISPL_CM_FROM_DEVICE)(int* size, realw* veloc, long* Mesh_pointer_f) {

  TRACE("transfer_veloc_cm_from_device");

  Mesh* mp = (Mesh*)(*Mesh_pointer_f); //get mesh pointer out of fortran integer container

  print_CUDA_error_if_any(hipMemcpy(veloc,mp->d_veloc_crust_mantle,sizeof(realw)*(*size),hipMemcpyDeviceToHost),40007);
}

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(transfer_accel_cm_from_device,
              TRANSFER_ACCEL_CM_FROM_DEVICE)(int* size, realw* accel,long* Mesh_pointer_f) {

  TRACE("transfer_accel_cm_from_device");

  Mesh* mp = (Mesh*)(*Mesh_pointer_f); //get mesh pointer out of fortran integer container

  print_CUDA_error_if_any(hipMemcpy(accel,mp->d_accel_crust_mantle,sizeof(realw)*(*size),hipMemcpyDeviceToHost),40026);

}

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(transfer_b_accel_cm_from_device,
              TRANSFER_B_ACCEL_CM_FROM_DEVICE)(int* size, realw* b_accel,long* Mesh_pointer_f) {

TRACE("transfer_b_accel_cm_from_device");

  Mesh* mp = (Mesh*)(*Mesh_pointer_f); //get mesh pointer out of fortran integer container

  print_CUDA_error_if_any(hipMemcpy(b_accel,mp->d_b_accel_crust_mantle,sizeof(realw)*(*size),hipMemcpyDeviceToHost),40036);

}

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(transfer_accel_ic_from_device,
              TRANSFER_ACCEL_IC_FROM_DEVICE)(int* size, realw* accel,long* Mesh_pointer_f) {

  TRACE("transfer_accel_ic_from_device");

  Mesh* mp = (Mesh*)(*Mesh_pointer_f); //get mesh pointer out of fortran integer container

  print_CUDA_error_if_any(hipMemcpy(accel,mp->d_accel_inner_core,sizeof(realw)*(*size),hipMemcpyDeviceToHost),40026);

}

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(transfer_accel_oc_from_device,
              TRANSFER_ACCEL_OC_FROM_DEVICE)(int* size, realw* accel,long* Mesh_pointer_f) {

  TRACE("transfer_accel_oc_from_device");

  Mesh* mp = (Mesh*)(*Mesh_pointer_f); //get mesh pointer out of fortran integer container

  print_CUDA_error_if_any(hipMemcpy(accel,mp->d_accel_outer_core,sizeof(realw)*(*size),hipMemcpyDeviceToHost),40026);

}

/* ----------------------------------------------------------------------------------------------- */

// strain fields

/* ----------------------------------------------------------------------------------------------- */

// crust/mantle
extern "C"
void FC_FUNC_(transfer_strain_cm_from_device,
              TRANSFER_STRAIN_CM_FROM_DEVICE)(long* Mesh_pointer,
                                                  realw* eps_trace_over_3,
                                                  realw* epsilondev_xx,
                                                  realw* epsilondev_yy,
                                                  realw* epsilondev_xy,
                                                  realw* epsilondev_xz,
                                                  realw* epsilondev_yz) {
  TRACE("transfer_strain_cm_from_device");
  //get mesh pointer out of fortran integer container
  Mesh* mp = (Mesh*)(*Mesh_pointer);

  int size = NGLL3*mp->NSPEC_CRUST_MANTLE;

  hipMemcpy(eps_trace_over_3,mp->d_eps_trace_over_3_crust_mantle,size*sizeof(realw),hipMemcpyDeviceToHost);

  hipMemcpy(epsilondev_xx,mp->d_epsilondev_xx_crust_mantle,size*sizeof(realw),hipMemcpyDeviceToHost);
  hipMemcpy(epsilondev_yy,mp->d_epsilondev_yy_crust_mantle,size*sizeof(realw),hipMemcpyDeviceToHost);
  hipMemcpy(epsilondev_xy,mp->d_epsilondev_xy_crust_mantle,size*sizeof(realw),hipMemcpyDeviceToHost);
  hipMemcpy(epsilondev_xz,mp->d_epsilondev_xz_crust_mantle,size*sizeof(realw),hipMemcpyDeviceToHost);
  hipMemcpy(epsilondev_yz,mp->d_epsilondev_yz_crust_mantle,size*sizeof(realw),hipMemcpyDeviceToHost);


#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
  exit_on_cuda_error("after transfer_strain_cm_from_device");
#endif
}

/* ----------------------------------------------------------------------------------------------- */

// crust/mantle

extern "C"
void FC_FUNC_(transfer_b_strain_cm_to_device,
              TRANSFER_B_STRAIN_CM_TO_DEVICE)(long* Mesh_pointer,
                                              realw* epsilondev_xx,
                                              realw* epsilondev_yy,
                                              realw* epsilondev_xy,
                                              realw* epsilondev_xz,
                                              realw* epsilondev_yz) {
  TRACE("transfer_b_strain_cm_to_device");
  //get mesh pointer out of fortran integer container
  Mesh* mp = (Mesh*)(*Mesh_pointer);

  int size = NGLL3*mp->NSPEC_CRUST_MANTLE;

  hipMemcpy(mp->d_b_epsilondev_xx_crust_mantle,epsilondev_xx,size*sizeof(realw),hipMemcpyHostToDevice);
  hipMemcpy(mp->d_b_epsilondev_yy_crust_mantle,epsilondev_yy,size*sizeof(realw),hipMemcpyHostToDevice);
  hipMemcpy(mp->d_b_epsilondev_xy_crust_mantle,epsilondev_xy,size*sizeof(realw),hipMemcpyHostToDevice);
  hipMemcpy(mp->d_b_epsilondev_xz_crust_mantle,epsilondev_xz,size*sizeof(realw),hipMemcpyHostToDevice);
  hipMemcpy(mp->d_b_epsilondev_yz_crust_mantle,epsilondev_yz,size*sizeof(realw),hipMemcpyHostToDevice);

#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
  exit_on_cuda_error("after transfer_b_strain_cm_to_device");
#endif
}

/* ----------------------------------------------------------------------------------------------- */

// inner core

extern "C"
void FC_FUNC_(transfer_strain_ic_from_device,
              TRANSFER_STRAIN_IC_FROM_DEVICE)(long* Mesh_pointer,
                                              realw* eps_trace_over_3,
                                              realw* epsilondev_xx,
                                              realw* epsilondev_yy,
                                              realw* epsilondev_xy,
                                              realw* epsilondev_xz,
                                              realw* epsilondev_yz) {
  TRACE("transfer_strain_ic_from_device");
  //get mesh pointer out of fortran integer container
  Mesh* mp = (Mesh*)(*Mesh_pointer);

  int size = NGLL3*mp->NSPEC_INNER_CORE;

  hipMemcpy(eps_trace_over_3,mp->d_eps_trace_over_3_inner_core,size*sizeof(realw),hipMemcpyDeviceToHost);

  hipMemcpy(epsilondev_xx,mp->d_epsilondev_xx_inner_core,size*sizeof(realw),hipMemcpyDeviceToHost);
  hipMemcpy(epsilondev_yy,mp->d_epsilondev_yy_inner_core,size*sizeof(realw),hipMemcpyDeviceToHost);
  hipMemcpy(epsilondev_xy,mp->d_epsilondev_xy_inner_core,size*sizeof(realw),hipMemcpyDeviceToHost);
  hipMemcpy(epsilondev_xz,mp->d_epsilondev_xz_inner_core,size*sizeof(realw),hipMemcpyDeviceToHost);
  hipMemcpy(epsilondev_yz,mp->d_epsilondev_yz_inner_core,size*sizeof(realw),hipMemcpyDeviceToHost);


#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
  exit_on_cuda_error("after transfer_strain_ic_from_device");
#endif
}

/* ----------------------------------------------------------------------------------------------- */

// inner_core

extern "C"
void FC_FUNC_(transfer_b_strain_ic_to_device,
              TRANSFER_B_STRAIN_IC_TO_DEVICE)(long* Mesh_pointer,
                                              realw* epsilondev_xx,
                                              realw* epsilondev_yy,
                                              realw* epsilondev_xy,
                                              realw* epsilondev_xz,
                                              realw* epsilondev_yz) {
  TRACE("transfer_b_strain_cm_to_device");
  //get mesh pointer out of fortran integer container
  Mesh* mp = (Mesh*)(*Mesh_pointer);

  int size = NGLL3*mp->NSPEC_INNER_CORE;

  hipMemcpy(mp->d_b_epsilondev_xx_inner_core,epsilondev_xx,size*sizeof(realw),hipMemcpyHostToDevice);
  hipMemcpy(mp->d_b_epsilondev_yy_inner_core,epsilondev_yy,size*sizeof(realw),hipMemcpyHostToDevice);
  hipMemcpy(mp->d_b_epsilondev_xy_inner_core,epsilondev_xy,size*sizeof(realw),hipMemcpyHostToDevice);
  hipMemcpy(mp->d_b_epsilondev_xz_inner_core,epsilondev_xz,size*sizeof(realw),hipMemcpyHostToDevice);
  hipMemcpy(mp->d_b_epsilondev_yz_inner_core,epsilondev_yz,size*sizeof(realw),hipMemcpyHostToDevice);

#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
  exit_on_cuda_error("after transfer_b_strain_ic_to_device");
#endif
}


/* ----------------------------------------------------------------------------------------------- */

// rotation arrays

/* ----------------------------------------------------------------------------------------------- */

// for outer core

extern "C"
void FC_FUNC_(transfer_rotation_from_device,
              TRANSFER_ROTATION_FROM_DEVICE)(long* Mesh_pointer,
                                             realw* A_array_rotation,
                                             realw* B_array_rotation) {
  TRACE("transfer_rotation_from_device");

  //get mesh pointer out of fortran integer container
  Mesh* mp = (Mesh*)(*Mesh_pointer);

  int size = NGLL3*mp->NSPEC_OUTER_CORE;

  hipMemcpy(A_array_rotation,mp->d_A_array_rotation,size*sizeof(realw),hipMemcpyDeviceToHost);
  hipMemcpy(B_array_rotation,mp->d_B_array_rotation,size*sizeof(realw),hipMemcpyDeviceToHost);

#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
  exit_on_cuda_error("after transfer_rotation_from_device");
#endif
}

/* ----------------------------------------------------------------------------------------------- */

// for outer core

extern "C"
void FC_FUNC_(transfer_b_rotation_to_device,
              TRANSFER_B_ROTATION_TO_DEVICE)(long* Mesh_pointer,
                                              realw* A_array_rotation,
                                              realw* B_array_rotation) {
  TRACE("transfer_b_rotation_to_device");
  //get mesh pointer out of fortran integer container
  Mesh* mp = (Mesh*)(*Mesh_pointer);

  int size = NGLL3*mp->NSPEC_OUTER_CORE;

  hipMemcpy(mp->d_b_A_array_rotation,A_array_rotation,size*sizeof(realw),hipMemcpyHostToDevice);
  hipMemcpy(mp->d_b_B_array_rotation,B_array_rotation,size*sizeof(realw),hipMemcpyHostToDevice);

#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
  exit_on_cuda_error("after transfer_b_rotation_to_device");
#endif
}


/* ----------------------------------------------------------------------------------------------- */

// KERNEL transfers

/* ----------------------------------------------------------------------------------------------- */

// crust/mantle

extern "C"
void FC_FUNC_(transfer_kernels_cm_to_host,
              TRANSFER_KERNELS_CM_TO_HOST)(long* Mesh_pointer,
                                           realw* h_rho_kl,
                                           realw* h_alpha_kl,
                                           realw* h_beta_kl,
                                           realw* h_cijkl_kl,
                                           int* NSPEC) {
  TRACE("transfer_kernels_cm_to_host");

  //get mesh pointer out of fortran integer container
  Mesh* mp = (Mesh*)(*Mesh_pointer);

  int size = (*NSPEC)*NGLL3;

  print_CUDA_error_if_any(hipMemcpy(h_rho_kl,mp->d_rho_kl_crust_mantle,
                                     size*sizeof(realw),hipMemcpyDeviceToHost),40101);

  if( ! mp->anisotropic_kl){
    print_CUDA_error_if_any(hipMemcpy(h_alpha_kl,mp->d_alpha_kl_crust_mantle,
                                       size*sizeof(realw),hipMemcpyDeviceToHost),40102);
    print_CUDA_error_if_any(hipMemcpy(h_beta_kl,mp->d_beta_kl_crust_mantle,
                                     size*sizeof(realw),hipMemcpyDeviceToHost),40103);
  }else{
    print_CUDA_error_if_any(hipMemcpy(h_cijkl_kl,mp->d_cijkl_kl_crust_mantle,
                                       21*size*sizeof(realw),hipMemcpyDeviceToHost),40102);
  }
}

/* ----------------------------------------------------------------------------------------------- */

// inner core

extern "C"
void FC_FUNC_(transfer_kernels_ic_to_host,
              TRANSFER_KERNELS_IC_TO_HOST)(long* Mesh_pointer,
                                                    realw* h_rho_kl,
                                                    realw* h_alpha_kl,
                                                    realw* h_beta_kl,
                                                    int* NSPEC) {
TRACE("transfer_kernels_ic_to_host");

  //get mesh pointer out of fortran integer container
  Mesh* mp = (Mesh*)(*Mesh_pointer);

  int size = (*NSPEC)*NGLL3;

  print_CUDA_error_if_any(hipMemcpy(h_rho_kl,mp->d_rho_kl_inner_core,
                                     size*sizeof(realw),hipMemcpyDeviceToHost),40101);
  print_CUDA_error_if_any(hipMemcpy(h_alpha_kl,mp->d_alpha_kl_inner_core,
                                     size*sizeof(realw),hipMemcpyDeviceToHost),40102);
  print_CUDA_error_if_any(hipMemcpy(h_beta_kl,mp->d_beta_kl_inner_core,
                                     size*sizeof(realw),hipMemcpyDeviceToHost),40103);
}


/* ----------------------------------------------------------------------------------------------- */

// outer core

extern "C"
void FC_FUNC_(transfer_kernels_oc_to_host,
              TRANSFER_KERNELS_OC_TO_HOST)(long* Mesh_pointer,
                                           realw* h_rho_kl,
                                           realw* h_alpha_kl,
                                           int* NSPEC) {

  TRACE("transfer_kernels_oc_to_host");

  //get mesh pointer out of fortran integer container
  Mesh* mp = (Mesh*)(*Mesh_pointer);

  int size = (*NSPEC)*NGLL3;

  // copies kernel values over to CPU host
  print_CUDA_error_if_any(hipMemcpy(h_rho_kl,mp->d_rho_kl_outer_core,
                                     size*sizeof(realw),hipMemcpyDeviceToHost),54101);
  print_CUDA_error_if_any(hipMemcpy(h_alpha_kl,mp->d_alpha_kl_outer_core,
                                     size*sizeof(realw),hipMemcpyDeviceToHost),54102);
}

/* ----------------------------------------------------------------------------------------------- */

// for NOISE simulations

extern "C"
void FC_FUNC_(transfer_kernels_noise_to_host,
              TRANSFER_KERNELS_NOISE_TO_HOST)(long* Mesh_pointer,
                                              realw* h_Sigma_kl,
                                              int* NSPEC) {
  TRACE("transfer_kernels_noise_to_host");

  Mesh* mp = (Mesh*)(*Mesh_pointer); //get mesh pointer out of fortran integer container

  print_CUDA_error_if_any(hipMemcpy(h_Sigma_kl,mp->d_Sigma_kl,NGLL3*(*NSPEC)*sizeof(realw),
                                     hipMemcpyDeviceToHost),40201);
}

/* ----------------------------------------------------------------------------------------------- */

// for Hess kernel calculations

extern "C"
void FC_FUNC_(transfer_kernels_hess_cm_tohost,
              TRANSFER_KERNELS_HESS_CM_TOHOST)(long* Mesh_pointer,
                                              realw* h_hess_kl,
                                              int* NSPEC) {
TRACE("transfer_kernels_hess_cm_tohost");

  Mesh* mp = (Mesh*)(*Mesh_pointer); //get mesh pointer out of fortran integer container

  print_CUDA_error_if_any(hipMemcpy(h_hess_kl,mp->d_hess_kl_crust_mantle,NGLL3*(*NSPEC)*sizeof(realw),
                                     hipMemcpyDeviceToHost),70201);
}

