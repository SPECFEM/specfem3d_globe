/*
 !=====================================================================
 !
 !               S p e c f e m 3 D  V e r s i o n  2 . 0
 !               ---------------------------------------
 !
 !          Main authors: Dimitri Komatitsch and Jeroen Tromp
 !    Princeton University, USA and University of Pau / CNRS / INRIA
 ! (c) Princeton University / California Institute of Technology and University of Pau / CNRS / INRIA
 !                            August 2013
 !
 ! This program is free software; you can redistribute it and/or modify
 ! it under the terms of the GNU General Public License as published by
 ! the Free Software Foundation; either version 2 of the License, or
 ! (at your option) any later version.
 !
 ! This program is distributed in the hope that it will be useful,
 ! but WITHOUT ANY WARRANTY; without even the implied warranty of
 ! MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 ! GNU General Public License for more details.
 !
 ! You should have received a copy of the GNU General Public License along
 ! with this program; if not, write to the Free Software Foundation, Inc.,
 ! 51 Franklin Street, Fifth Floor, Boston, MA 02110-1301 USA.
 !
 !=====================================================================
 */

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

#include <sys/time.h>
#include <sys/resource.h>

#include "config.h"
#include "mesh_constants_cuda.h"

/* ----------------------------------------------------------------------------------------------- */

// Transfer functions

/* ----------------------------------------------------------------------------------------------- */



/* ----------------------------------------------------------------------------------------------- */

// transfer memory from CPU host to GPU device

/* ----------------------------------------------------------------------------------------------- */

// crust_mantle
extern "C"
void FC_FUNC_(transfer_fields_cm_to_device,
              TRANSFER_FIELDS_CM_TO_DEVICE)(int* size, realw* displ, realw* veloc, realw* accel,long* Mesh_pointer_f) {

TRACE("transfer_fields_cm_to_device");

  Mesh* mp = (Mesh*)(*Mesh_pointer_f); //get mesh pointer out of fortran integer container

  print_CUDA_error_if_any(hipMemcpy(mp->d_displ_crust_mantle,displ,sizeof(realw)*(*size),hipMemcpyHostToDevice),40003);
  print_CUDA_error_if_any(hipMemcpy(mp->d_veloc_crust_mantle,veloc,sizeof(realw)*(*size),hipMemcpyHostToDevice),40004);
  print_CUDA_error_if_any(hipMemcpy(mp->d_accel_crust_mantle,accel,sizeof(realw)*(*size),hipMemcpyHostToDevice),40005);

}

// inner_core
extern "C"
void FC_FUNC_(transfer_fields_ic_to_device,
              TRANSFER_FIELDS_IC_TO_DEVICE)(int* size, realw* displ, realw* veloc, realw* accel,long* Mesh_pointer_f) {

  TRACE("transfer_fields_ic_to_device");

  Mesh* mp = (Mesh*)(*Mesh_pointer_f); //get mesh pointer out of fortran integer container

  print_CUDA_error_if_any(hipMemcpy(mp->d_displ_inner_core,displ,sizeof(realw)*(*size),hipMemcpyHostToDevice),40003);
  print_CUDA_error_if_any(hipMemcpy(mp->d_veloc_inner_core,veloc,sizeof(realw)*(*size),hipMemcpyHostToDevice),40004);
  print_CUDA_error_if_any(hipMemcpy(mp->d_accel_inner_core,accel,sizeof(realw)*(*size),hipMemcpyHostToDevice),40005);

}

// outer_core
extern "C"
void FC_FUNC_(transfer_fields_oc_to_device,
              TRANSFER_FIELDS_OC_TO_DEVICE)(int* size, realw* displ, realw* veloc, realw* accel,long* Mesh_pointer_f) {

  TRACE("transfer_fields_oc_to_device");

  Mesh* mp = (Mesh*)(*Mesh_pointer_f); //get mesh pointer out of fortran integer container

  print_CUDA_error_if_any(hipMemcpy(mp->d_displ_outer_core,displ,sizeof(realw)*(*size),hipMemcpyHostToDevice),40003);
  print_CUDA_error_if_any(hipMemcpy(mp->d_veloc_outer_core,veloc,sizeof(realw)*(*size),hipMemcpyHostToDevice),40004);
  print_CUDA_error_if_any(hipMemcpy(mp->d_accel_outer_core,accel,sizeof(realw)*(*size),hipMemcpyHostToDevice),40005);

}

/* ----------------------------------------------------------------------------------------------- */

// backward/reconstructed fields

// crust_mantle
extern "C"
void FC_FUNC_(transfer_b_fields_cm_to_device,
              TRANSFER_FIELDS_B_CM_TO_DEVICE)(int* size, realw* b_displ, realw* b_veloc, realw* b_accel,
                                              long* Mesh_pointer_f) {

  TRACE("transfer_fields_b_cm_to_device");
  // debug
  DEBUG_BACKWARD_TRANSFER();

  Mesh* mp = (Mesh*)(*Mesh_pointer_f); //get mesh pointer out of fortran integer container
  print_CUDA_error_if_any(hipMemcpy(mp->d_b_displ_crust_mantle,b_displ,sizeof(realw)*(*size),hipMemcpyHostToDevice),40003);
  print_CUDA_error_if_any(hipMemcpy(mp->d_b_veloc_crust_mantle,b_veloc,sizeof(realw)*(*size),hipMemcpyHostToDevice),40004);
  print_CUDA_error_if_any(hipMemcpy(mp->d_b_accel_crust_mantle,b_accel,sizeof(realw)*(*size),hipMemcpyHostToDevice),40005);

}

// inner_core
extern "C"
void FC_FUNC_(transfer_b_fields_ic_to_device,
              TRANSFER_FIELDS_B_IC_TO_DEVICE)(int* size, realw* b_displ, realw* b_veloc, realw* b_accel,
                                              long* Mesh_pointer_f) {

  TRACE("transfer_fields_b_ic_to_device");
  // debug
  DEBUG_BACKWARD_TRANSFER();

  Mesh* mp = (Mesh*)(*Mesh_pointer_f); //get mesh pointer out of fortran integer container
  print_CUDA_error_if_any(hipMemcpy(mp->d_b_displ_inner_core,b_displ,sizeof(realw)*(*size),hipMemcpyHostToDevice),40003);
  print_CUDA_error_if_any(hipMemcpy(mp->d_b_veloc_inner_core,b_veloc,sizeof(realw)*(*size),hipMemcpyHostToDevice),40004);
  print_CUDA_error_if_any(hipMemcpy(mp->d_b_accel_inner_core,b_accel,sizeof(realw)*(*size),hipMemcpyHostToDevice),40005);

}

// outer_core
extern "C"
void FC_FUNC_(transfer_b_fields_oc_to_device,
              TRANSFER_FIELDS_B_OC_TO_DEVICE)(int* size, realw* b_displ, realw* b_veloc, realw* b_accel,
                                              long* Mesh_pointer_f) {

  TRACE("transfer_fields_b_oc_to_device");
  // debug
  DEBUG_BACKWARD_TRANSFER();

  Mesh* mp = (Mesh*)(*Mesh_pointer_f); //get mesh pointer out of fortran integer container
  print_CUDA_error_if_any(hipMemcpy(mp->d_b_displ_outer_core,b_displ,sizeof(realw)*(*size),hipMemcpyHostToDevice),40003);
  print_CUDA_error_if_any(hipMemcpy(mp->d_b_veloc_outer_core,b_veloc,sizeof(realw)*(*size),hipMemcpyHostToDevice),40004);
  print_CUDA_error_if_any(hipMemcpy(mp->d_b_accel_outer_core,b_accel,sizeof(realw)*(*size),hipMemcpyHostToDevice),40005);

}

/* ----------------------------------------------------------------------------------------------- */

// transfer memory from GPU device to CPU host

/* ----------------------------------------------------------------------------------------------- */

// crust_mantle
extern "C"
void FC_FUNC_(transfer_fields_cm_from_device,
              TRANSFER_FIELDS_CM_FROM_DEVICE)(int* size, realw* displ, realw* veloc, realw* accel,long* Mesh_pointer_f) {

  TRACE("transfer_fields_cm_from_device");

  Mesh* mp = (Mesh*)(*Mesh_pointer_f); //get mesh pointer out of fortran integer container

  print_CUDA_error_if_any(hipMemcpy(displ,mp->d_displ_crust_mantle,sizeof(realw)*(*size),hipMemcpyDeviceToHost),40006);
  print_CUDA_error_if_any(hipMemcpy(veloc,mp->d_veloc_crust_mantle,sizeof(realw)*(*size),hipMemcpyDeviceToHost),40007);
  print_CUDA_error_if_any(hipMemcpy(accel,mp->d_accel_crust_mantle,sizeof(realw)*(*size),hipMemcpyDeviceToHost),40008);

}

/* ----------------------------------------------------------------------------------------------- */

// inner_core
extern "C"
void FC_FUNC_(transfer_fields_ic_from_device,
              TRANSFER_FIELDS_IC_FROM_DEVICE)(int* size, realw* displ, realw* veloc, realw* accel,long* Mesh_pointer_f) {

  TRACE("transfer_fields_ic_from_device");

  Mesh* mp = (Mesh*)(*Mesh_pointer_f); //get mesh pointer out of fortran integer container

  print_CUDA_error_if_any(hipMemcpy(displ,mp->d_displ_inner_core,sizeof(realw)*(*size),hipMemcpyDeviceToHost),40006);
  print_CUDA_error_if_any(hipMemcpy(veloc,mp->d_veloc_inner_core,sizeof(realw)*(*size),hipMemcpyDeviceToHost),40007);
  print_CUDA_error_if_any(hipMemcpy(accel,mp->d_accel_inner_core,sizeof(realw)*(*size),hipMemcpyDeviceToHost),40008);

}

/* ----------------------------------------------------------------------------------------------- */

// outer_core
extern "C"
void FC_FUNC_(transfer_fields_oc_from_device,
              TRANSFER_FIELDS_OC_FROM_DEVICE)(int* size, realw* displ, realw* veloc, realw* accel,long* Mesh_pointer_f) {

  TRACE("transfer_fields_oc_from_device");

  Mesh* mp = (Mesh*)(*Mesh_pointer_f); //get mesh pointer out of fortran integer container

  print_CUDA_error_if_any(hipMemcpy(displ,mp->d_displ_outer_core,sizeof(realw)*(*size),hipMemcpyDeviceToHost),40006);
  print_CUDA_error_if_any(hipMemcpy(veloc,mp->d_veloc_outer_core,sizeof(realw)*(*size),hipMemcpyDeviceToHost),40007);
  print_CUDA_error_if_any(hipMemcpy(accel,mp->d_accel_outer_core,sizeof(realw)*(*size),hipMemcpyDeviceToHost),40008);

}

/* ----------------------------------------------------------------------------------------------- */

// backward/reconstructed fields

// crust_mantle
extern "C"
void FC_FUNC_(transfer_b_fields_cm_from_device,
              TRANSFER_B_FIELDS_CM_FROM_DEVICE)(int* size, realw* b_displ, realw* b_veloc, realw* b_accel,
                                                long* Mesh_pointer_f) {

TRACE("transfer_b_fields_cm_from_device");

  Mesh* mp = (Mesh*)(*Mesh_pointer_f); //get mesh pointer out of fortran integer container

  print_CUDA_error_if_any(hipMemcpy(b_displ,mp->d_b_displ_crust_mantle,sizeof(realw)*(*size),hipMemcpyDeviceToHost),40006);
  print_CUDA_error_if_any(hipMemcpy(b_veloc,mp->d_b_veloc_crust_mantle,sizeof(realw)*(*size),hipMemcpyDeviceToHost),40007);
  print_CUDA_error_if_any(hipMemcpy(b_accel,mp->d_b_accel_crust_mantle,sizeof(realw)*(*size),hipMemcpyDeviceToHost),40008);

}

/* ----------------------------------------------------------------------------------------------- */

// inner_core
extern "C"
void FC_FUNC_(transfer_b_fields_ic_from_device,
              TRANSFER_B_FIELDS_IC_FROM_DEVICE)(int* size, realw* b_displ, realw* b_veloc, realw* b_accel,
                                                long* Mesh_pointer_f) {
  TRACE("transfer_fields_b_ic_from_device");

  Mesh* mp = (Mesh*)(*Mesh_pointer_f); //get mesh pointer out of fortran integer container

  print_CUDA_error_if_any(hipMemcpy(b_displ,mp->d_b_displ_inner_core,sizeof(realw)*(*size),hipMemcpyDeviceToHost),40006);
  print_CUDA_error_if_any(hipMemcpy(b_veloc,mp->d_b_veloc_inner_core,sizeof(realw)*(*size),hipMemcpyDeviceToHost),40007);
  print_CUDA_error_if_any(hipMemcpy(b_accel,mp->d_b_accel_inner_core,sizeof(realw)*(*size),hipMemcpyDeviceToHost),40008);

}

/* ----------------------------------------------------------------------------------------------- */

// outer_core
extern "C"
void FC_FUNC_(transfer_b_fields_oc_from_device,
              TRANSFER_B_FIELDS_OC_FROM_DEVICE)(int* size, realw* b_displ, realw* b_veloc, realw* b_accel,
                                                long* Mesh_pointer_f) {

  TRACE("transfer_b_fields_oc_from_device");

  Mesh* mp = (Mesh*)(*Mesh_pointer_f); //get mesh pointer out of fortran integer container

  print_CUDA_error_if_any(hipMemcpy(b_displ,mp->d_b_displ_outer_core,sizeof(realw)*(*size),hipMemcpyDeviceToHost),40006);
  print_CUDA_error_if_any(hipMemcpy(b_veloc,mp->d_b_veloc_outer_core,sizeof(realw)*(*size),hipMemcpyDeviceToHost),40007);
  print_CUDA_error_if_any(hipMemcpy(b_accel,mp->d_b_accel_outer_core,sizeof(realw)*(*size),hipMemcpyDeviceToHost),40008);

}

/* ----------------------------------------------------------------------------------------------- */

// single wavefield transfers

/* ----------------------------------------------------------------------------------------------- */

/* ----------------------------------------------------------------------------------------------- */

// displacements

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(transfer_displ_cm_from_device,
              TRANSFER_DISPL_CM_FROM_DEVICE)(int* size, realw* displ, long* Mesh_pointer_f) {

  TRACE("transfer_displ_cm_from_device");

  Mesh* mp = (Mesh*)(*Mesh_pointer_f); //get mesh pointer out of fortran integer container

  print_CUDA_error_if_any(hipMemcpy(displ,mp->d_displ_crust_mantle,sizeof(realw)*(*size),hipMemcpyDeviceToHost),40006);
}

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(transfer_b_displ_cm_from_device,
              TRANSFER_B_DISPL_CM_FROM_DEVICE)(int* size, realw* displ, long* Mesh_pointer_f) {

  TRACE("transfer_b_displ_cm_from_device");

  Mesh* mp = (Mesh*)(*Mesh_pointer_f); //get mesh pointer out of fortran integer container

  print_CUDA_error_if_any(hipMemcpy(displ,mp->d_b_displ_crust_mantle,sizeof(realw)*(*size),hipMemcpyDeviceToHost),40006);
}

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(transfer_displ_ic_from_device,
              TRANSFER_DISPL_IC_FROM_DEVICE)(int* size, realw* displ, long* Mesh_pointer_f) {

  TRACE("transfer_displ_ic_from_device");

  Mesh* mp = (Mesh*)(*Mesh_pointer_f); //get mesh pointer out of fortran integer container

  print_CUDA_error_if_any(hipMemcpy(displ,mp->d_displ_inner_core,sizeof(realw)*(*size),hipMemcpyDeviceToHost),40006);
}

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(transfer_b_displ_ic_from_device,
              TRANSFER_B_DISPL_IC_FROM_DEVICE)(int* size, realw* displ, long* Mesh_pointer_f) {

  TRACE("transfer_b_displ_ic_from_device");

  Mesh* mp = (Mesh*)(*Mesh_pointer_f); //get mesh pointer out of fortran integer container

  print_CUDA_error_if_any(hipMemcpy(displ,mp->d_b_displ_inner_core,sizeof(realw)*(*size),hipMemcpyDeviceToHost),40006);
}

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(transfer_displ_oc_from_device,
              TRANSFER_DISPL_OC_FROM_DEVICE)(int* size, realw* displ, long* Mesh_pointer_f) {

  TRACE("transfer_displ_oc_from_device");

  Mesh* mp = (Mesh*)(*Mesh_pointer_f); //get mesh pointer out of fortran integer container

  print_CUDA_error_if_any(hipMemcpy(displ,mp->d_displ_outer_core,sizeof(realw)*(*size),hipMemcpyDeviceToHost),40006);
}

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(transfer_b_displ_oc_from_device,
              TRANSFER_B_DISPL_OC_FROM_DEVICE)(int* size, realw* displ, long* Mesh_pointer_f) {

  TRACE("transfer_b_displ_oc_from_device");

  Mesh* mp = (Mesh*)(*Mesh_pointer_f); //get mesh pointer out of fortran integer container

  print_CUDA_error_if_any(hipMemcpy(displ,mp->d_b_displ_outer_core,sizeof(realw)*(*size),hipMemcpyDeviceToHost),40006);
}


/* ----------------------------------------------------------------------------------------------- */

// velocities

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(transfer_veloc_cm_from_device,
              TRANSFER_VELOC_CM_FROM_DEVICE)(int* size, realw* veloc, long* Mesh_pointer_f) {

  TRACE("transfer_veloc_cm_from_device");

  Mesh* mp = (Mesh*)(*Mesh_pointer_f); //get mesh pointer out of fortran integer container

  print_CUDA_error_if_any(hipMemcpy(veloc,mp->d_veloc_crust_mantle,sizeof(realw)*(*size),hipMemcpyDeviceToHost),40007);
}

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(transfer_veloc_ic_from_device,
              TRANSFER_VELOC_IC_FROM_DEVICE)(int* size, realw* veloc, long* Mesh_pointer_f) {

  TRACE("transfer_veloc_ic_from_device");

  Mesh* mp = (Mesh*)(*Mesh_pointer_f); //get mesh pointer out of fortran integer container

  print_CUDA_error_if_any(hipMemcpy(veloc,mp->d_veloc_inner_core,sizeof(realw)*(*size),hipMemcpyDeviceToHost),40007);
}

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(transfer_veloc_oc_from_device,
              TRANSFER_VELOC_OC_FROM_DEVICE)(int* size, realw* veloc, long* Mesh_pointer_f) {

  TRACE("transfer_veloc_oc_from_device");

  Mesh* mp = (Mesh*)(*Mesh_pointer_f); //get mesh pointer out of fortran integer container

  print_CUDA_error_if_any(hipMemcpy(veloc,mp->d_veloc_outer_core,sizeof(realw)*(*size),hipMemcpyDeviceToHost),40007);
}


/* ----------------------------------------------------------------------------------------------- */

// accelerations

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(transfer_accel_cm_to_device,
              TRANSFER_ACCEL_CM_TO_DEVICE)(int* size, realw* accel,long* Mesh_pointer_f) {

TRACE("transfer_accel_cm_to_device");

  Mesh* mp = (Mesh*)(*Mesh_pointer_f); //get mesh pointer out of fortran integer container

  print_CUDA_error_if_any(hipMemcpy(mp->d_accel_crust_mantle,accel,sizeof(realw)*(*size),hipMemcpyHostToDevice),40016);

}

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(transfer_accel_cm_from_device,
              TRANSFER_ACCEL_CM_FROM_DEVICE)(int* size, realw* accel,long* Mesh_pointer_f) {

  TRACE("transfer_accel_cm_from_device");

  Mesh* mp = (Mesh*)(*Mesh_pointer_f); //get mesh pointer out of fortran integer container

  print_CUDA_error_if_any(hipMemcpy(accel,mp->d_accel_crust_mantle,sizeof(realw)*(*size),hipMemcpyDeviceToHost),40026);

}

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(transfer_b_accel_cm_from_device,
              TRANSFER_B_ACCEL_CM_FROM_DEVICE)(int* size, realw* b_accel,long* Mesh_pointer_f) {

TRACE("transfer_b_accel_cm_from_device");

  Mesh* mp = (Mesh*)(*Mesh_pointer_f); //get mesh pointer out of fortran integer container

  print_CUDA_error_if_any(hipMemcpy(b_accel,mp->d_b_accel_crust_mantle,sizeof(realw)*(*size),hipMemcpyDeviceToHost),40036);

}

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(transfer_accel_ic_from_device,
              TRANSFER_ACCEL_IC_FROM_DEVICE)(int* size, realw* accel,long* Mesh_pointer_f) {

  TRACE("transfer_accel_ic_from_device");

  Mesh* mp = (Mesh*)(*Mesh_pointer_f); //get mesh pointer out of fortran integer container

  print_CUDA_error_if_any(hipMemcpy(accel,mp->d_accel_inner_core,sizeof(realw)*(*size),hipMemcpyDeviceToHost),40026);

}

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(transfer_accel_oc_from_device,
              TRANSFER_ACCEL_OC_FROM_DEVICE)(int* size, realw* accel,long* Mesh_pointer_f) {

  TRACE("transfer_accel_oc_from_device");

  Mesh* mp = (Mesh*)(*Mesh_pointer_f); //get mesh pointer out of fortran integer container

  print_CUDA_error_if_any(hipMemcpy(accel,mp->d_accel_outer_core,sizeof(realw)*(*size),hipMemcpyDeviceToHost),40026);

}

/* ----------------------------------------------------------------------------------------------- */

// strain fields

/* ----------------------------------------------------------------------------------------------- */

// crust/mantle
extern "C"
void FC_FUNC_(transfer_strain_cm_from_device,
              TRANSFER_STRAIN_CM_FROM_DEVICE)(long* Mesh_pointer,
                                                  realw* eps_trace_over_3,
                                                  realw* epsilondev_xx,
                                                  realw* epsilondev_yy,
                                                  realw* epsilondev_xy,
                                                  realw* epsilondev_xz,
                                                  realw* epsilondev_yz) {
  TRACE("transfer_strain_cm_from_device");
  //get mesh pointer out of fortran integer container
  Mesh* mp = (Mesh*)(*Mesh_pointer);

  int size = NGLL3*mp->NSPEC_CRUST_MANTLE;

  print_CUDA_error_if_any(hipMemcpy(eps_trace_over_3,mp->d_eps_trace_over_3_crust_mantle,size*sizeof(realw),hipMemcpyDeviceToHost),320001);

  print_CUDA_error_if_any(hipMemcpy(epsilondev_xx,mp->d_epsilondev_xx_crust_mantle,size*sizeof(realw),hipMemcpyDeviceToHost),320002);
  print_CUDA_error_if_any(hipMemcpy(epsilondev_yy,mp->d_epsilondev_yy_crust_mantle,size*sizeof(realw),hipMemcpyDeviceToHost),320003);
  print_CUDA_error_if_any(hipMemcpy(epsilondev_xy,mp->d_epsilondev_xy_crust_mantle,size*sizeof(realw),hipMemcpyDeviceToHost),320004);
  print_CUDA_error_if_any(hipMemcpy(epsilondev_xz,mp->d_epsilondev_xz_crust_mantle,size*sizeof(realw),hipMemcpyDeviceToHost),320005);
  print_CUDA_error_if_any(hipMemcpy(epsilondev_yz,mp->d_epsilondev_yz_crust_mantle,size*sizeof(realw),hipMemcpyDeviceToHost),320006);


#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
  exit_on_cuda_error("after transfer_strain_cm_from_device");
#endif
}

/* ----------------------------------------------------------------------------------------------- */

// crust/mantle

extern "C"
void FC_FUNC_(transfer_b_strain_cm_to_device,
              TRANSFER_B_STRAIN_CM_TO_DEVICE)(long* Mesh_pointer,
                                              realw* epsilondev_xx,
                                              realw* epsilondev_yy,
                                              realw* epsilondev_xy,
                                              realw* epsilondev_xz,
                                              realw* epsilondev_yz) {
  TRACE("transfer_b_strain_cm_to_device");
  // debug
  DEBUG_BACKWARD_TRANSFER();

  //get mesh pointer out of fortran integer container
  Mesh* mp = (Mesh*)(*Mesh_pointer);

  int size = NGLL3*mp->NSPEC_CRUST_MANTLE;

  if( ! mp->undo_attenuation ){
    print_CUDA_error_if_any(hipMemcpy(mp->d_b_epsilondev_xx_crust_mantle,epsilondev_xx,size*sizeof(realw),hipMemcpyHostToDevice),330001);
    print_CUDA_error_if_any(hipMemcpy(mp->d_b_epsilondev_yy_crust_mantle,epsilondev_yy,size*sizeof(realw),hipMemcpyHostToDevice),330002);
    print_CUDA_error_if_any(hipMemcpy(mp->d_b_epsilondev_xy_crust_mantle,epsilondev_xy,size*sizeof(realw),hipMemcpyHostToDevice),330003);
    print_CUDA_error_if_any(hipMemcpy(mp->d_b_epsilondev_xz_crust_mantle,epsilondev_xz,size*sizeof(realw),hipMemcpyHostToDevice),330004);
    print_CUDA_error_if_any(hipMemcpy(mp->d_b_epsilondev_yz_crust_mantle,epsilondev_yz,size*sizeof(realw),hipMemcpyHostToDevice),330005);
  }

#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
  exit_on_cuda_error("after transfer_b_strain_cm_to_device");
#endif
}

/* ----------------------------------------------------------------------------------------------- */

// inner core

extern "C"
void FC_FUNC_(transfer_strain_ic_from_device,
              TRANSFER_STRAIN_IC_FROM_DEVICE)(long* Mesh_pointer,
                                              realw* eps_trace_over_3,
                                              realw* epsilondev_xx,
                                              realw* epsilondev_yy,
                                              realw* epsilondev_xy,
                                              realw* epsilondev_xz,
                                              realw* epsilondev_yz) {
  TRACE("transfer_strain_ic_from_device");
  //get mesh pointer out of fortran integer container
  Mesh* mp = (Mesh*)(*Mesh_pointer);

  int size = NGLL3*mp->NSPEC_INNER_CORE;

  print_CUDA_error_if_any(hipMemcpy(eps_trace_over_3,mp->d_eps_trace_over_3_inner_core,size*sizeof(realw),hipMemcpyDeviceToHost),340001);

  print_CUDA_error_if_any(hipMemcpy(epsilondev_xx,mp->d_epsilondev_xx_inner_core,size*sizeof(realw),hipMemcpyDeviceToHost),340002);
  print_CUDA_error_if_any(hipMemcpy(epsilondev_yy,mp->d_epsilondev_yy_inner_core,size*sizeof(realw),hipMemcpyDeviceToHost),340003);
  print_CUDA_error_if_any(hipMemcpy(epsilondev_xy,mp->d_epsilondev_xy_inner_core,size*sizeof(realw),hipMemcpyDeviceToHost),340004);
  print_CUDA_error_if_any(hipMemcpy(epsilondev_xz,mp->d_epsilondev_xz_inner_core,size*sizeof(realw),hipMemcpyDeviceToHost),340005);
  print_CUDA_error_if_any(hipMemcpy(epsilondev_yz,mp->d_epsilondev_yz_inner_core,size*sizeof(realw),hipMemcpyDeviceToHost),340006);


#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
  exit_on_cuda_error("after transfer_strain_ic_from_device");
#endif
}

/* ----------------------------------------------------------------------------------------------- */

// inner_core

extern "C"
void FC_FUNC_(transfer_b_strain_ic_to_device,
              TRANSFER_B_STRAIN_IC_TO_DEVICE)(long* Mesh_pointer,
                                              realw* epsilondev_xx,
                                              realw* epsilondev_yy,
                                              realw* epsilondev_xy,
                                              realw* epsilondev_xz,
                                              realw* epsilondev_yz) {
  TRACE("transfer_b_strain_cm_to_device");
  // debug
  DEBUG_BACKWARD_TRANSFER();

  //get mesh pointer out of fortran integer container
  Mesh* mp = (Mesh*)(*Mesh_pointer);

  int size = NGLL3*mp->NSPEC_INNER_CORE;

  if( ! mp->undo_attenuation ){
    print_CUDA_error_if_any(hipMemcpy(mp->d_b_epsilondev_xx_inner_core,epsilondev_xx,size*sizeof(realw),hipMemcpyHostToDevice),350001);
    print_CUDA_error_if_any(hipMemcpy(mp->d_b_epsilondev_yy_inner_core,epsilondev_yy,size*sizeof(realw),hipMemcpyHostToDevice),350002);
    print_CUDA_error_if_any(hipMemcpy(mp->d_b_epsilondev_xy_inner_core,epsilondev_xy,size*sizeof(realw),hipMemcpyHostToDevice),350003);
    print_CUDA_error_if_any(hipMemcpy(mp->d_b_epsilondev_xz_inner_core,epsilondev_xz,size*sizeof(realw),hipMemcpyHostToDevice),350004);
    print_CUDA_error_if_any(hipMemcpy(mp->d_b_epsilondev_yz_inner_core,epsilondev_yz,size*sizeof(realw),hipMemcpyHostToDevice),350005);
  }

#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
  exit_on_cuda_error("after transfer_b_strain_ic_to_device");
#endif
}

/* ----------------------------------------------------------------------------------------------- */

// R memory variables

/* ----------------------------------------------------------------------------------------------- */

// crust/mantle

extern "C"
void FC_FUNC_(transfer_b_rmemory_cm_to_device,
              TRANSFER_B_RMEMORY_CM_TO_DEVICE)(long* Mesh_pointer,
                                               realw* b_R_xx,
                                               realw* b_R_yy,
                                               realw* b_R_xy,
                                               realw* b_R_xz,
                                               realw* b_R_yz) {
  TRACE("transfer_b_Rmemory_cm_to_device");
  // debug
  DEBUG_BACKWARD_TRANSFER();

  //get mesh pointer out of fortran integer container
  Mesh* mp = (Mesh*)(*Mesh_pointer);

  int size = N_SLS*NGLL3*mp->NSPEC_CRUST_MANTLE;

  if( ! mp->partial_phys_dispersion_only ){
    print_CUDA_error_if_any(hipMemcpy(mp->d_b_R_xx_crust_mantle,b_R_xx,size*sizeof(realw),hipMemcpyHostToDevice),360001);
    print_CUDA_error_if_any(hipMemcpy(mp->d_b_R_yy_crust_mantle,b_R_yy,size*sizeof(realw),hipMemcpyHostToDevice),360002);
    print_CUDA_error_if_any(hipMemcpy(mp->d_b_R_xy_crust_mantle,b_R_xy,size*sizeof(realw),hipMemcpyHostToDevice),360003);
    print_CUDA_error_if_any(hipMemcpy(mp->d_b_R_xz_crust_mantle,b_R_xz,size*sizeof(realw),hipMemcpyHostToDevice),360004);
    print_CUDA_error_if_any(hipMemcpy(mp->d_b_R_yz_crust_mantle,b_R_yz,size*sizeof(realw),hipMemcpyHostToDevice),360005);
  }

#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
  exit_on_cuda_error("after transfer_b_rmemory_cm_to_device");
#endif
}


/* ----------------------------------------------------------------------------------------------- */

// inner core

extern "C"
void FC_FUNC_(transfer_b_rmemory_ic_to_device,
              TRANSFER_B_RMEMORY_IC_TO_DEVICE)(long* Mesh_pointer,
                                               realw* b_R_xx,
                                               realw* b_R_yy,
                                               realw* b_R_xy,
                                               realw* b_R_xz,
                                               realw* b_R_yz) {
  TRACE("transfer_b_rmemory_ic_to_device");
  // debug
  DEBUG_BACKWARD_TRANSFER();

  //get mesh pointer out of fortran integer container
  Mesh* mp = (Mesh*)(*Mesh_pointer);

  int size = N_SLS*NGLL3*mp->NSPEC_INNER_CORE;

  if( ! mp->partial_phys_dispersion_only ){
    print_CUDA_error_if_any(hipMemcpy(mp->d_b_R_xx_inner_core,b_R_xx,size*sizeof(realw),hipMemcpyHostToDevice),370001);
    print_CUDA_error_if_any(hipMemcpy(mp->d_b_R_yy_inner_core,b_R_yy,size*sizeof(realw),hipMemcpyHostToDevice),370002);
    print_CUDA_error_if_any(hipMemcpy(mp->d_b_R_xy_inner_core,b_R_xy,size*sizeof(realw),hipMemcpyHostToDevice),370003);
    print_CUDA_error_if_any(hipMemcpy(mp->d_b_R_xz_inner_core,b_R_xz,size*sizeof(realw),hipMemcpyHostToDevice),370004);
    print_CUDA_error_if_any(hipMemcpy(mp->d_b_R_yz_inner_core,b_R_yz,size*sizeof(realw),hipMemcpyHostToDevice),370005);
  }

#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
  exit_on_cuda_error("after transfer_b_rmemory_ic_to_device");
#endif
}


/* ----------------------------------------------------------------------------------------------- */

// rotation arrays

/* ----------------------------------------------------------------------------------------------- */

// for outer core

extern "C"
void FC_FUNC_(transfer_rotation_from_device,
              TRANSFER_ROTATION_FROM_DEVICE)(long* Mesh_pointer,
                                             realw* A_array_rotation,
                                             realw* B_array_rotation) {
  TRACE("transfer_rotation_from_device");

  //get mesh pointer out of fortran integer container
  Mesh* mp = (Mesh*)(*Mesh_pointer);

  int size = NGLL3*mp->NSPEC_OUTER_CORE;

  print_CUDA_error_if_any(hipMemcpy(A_array_rotation,mp->d_A_array_rotation,
                                     size*sizeof(realw),hipMemcpyDeviceToHost),380001);
  print_CUDA_error_if_any(hipMemcpy(B_array_rotation,mp->d_B_array_rotation,
                                     size*sizeof(realw),hipMemcpyDeviceToHost),380002);
}

/* ----------------------------------------------------------------------------------------------- */

// for outer core

extern "C"
void FC_FUNC_(transfer_b_rotation_to_device,
              TRANSFER_B_ROTATION_TO_DEVICE)(long* Mesh_pointer,
                                              realw* A_array_rotation,
                                              realw* B_array_rotation) {
  TRACE("transfer_b_rotation_to_device");
  // debug
  DEBUG_BACKWARD_TRANSFER();

  //get mesh pointer out of fortran integer container
  Mesh* mp = (Mesh*)(*Mesh_pointer);

  int size = NGLL3*mp->NSPEC_OUTER_CORE;

  print_CUDA_error_if_any(hipMemcpy(mp->d_b_A_array_rotation,A_array_rotation,
                                     size*sizeof(realw),hipMemcpyHostToDevice),390001);
  print_CUDA_error_if_any(hipMemcpy(mp->d_b_B_array_rotation,B_array_rotation,
                                     size*sizeof(realw),hipMemcpyHostToDevice),390002);
}


/* ----------------------------------------------------------------------------------------------- */

// KERNEL transfers

/* ----------------------------------------------------------------------------------------------- */

// crust/mantle

extern "C"
void FC_FUNC_(transfer_kernels_cm_to_host,
              TRANSFER_KERNELS_CM_TO_HOST)(long* Mesh_pointer,
                                           realw* h_rho_kl,
                                           realw* h_alpha_kl,
                                           realw* h_beta_kl,
                                           realw* h_cijkl_kl,
                                           int* NSPEC) {
  TRACE("transfer_kernels_cm_to_host");

  //get mesh pointer out of fortran integer container
  Mesh* mp = (Mesh*)(*Mesh_pointer);

  int size = (*NSPEC)*NGLL3;

  // density kernel
  print_CUDA_error_if_any(hipMemcpy(h_rho_kl,mp->d_rho_kl_crust_mantle,
                                     size*sizeof(realw),hipMemcpyDeviceToHost),40101);

  if( ! mp->anisotropic_kl){
    // isotropic kernels
    print_CUDA_error_if_any(hipMemcpy(h_alpha_kl,mp->d_alpha_kl_crust_mantle,
                                       size*sizeof(realw),hipMemcpyDeviceToHost),40102);
    print_CUDA_error_if_any(hipMemcpy(h_beta_kl,mp->d_beta_kl_crust_mantle,
                                       size*sizeof(realw),hipMemcpyDeviceToHost),40103);
  }else{
    // anisotropic kernels
    print_CUDA_error_if_any(hipMemcpy(h_cijkl_kl,mp->d_cijkl_kl_crust_mantle,
                                       21*size*sizeof(realw),hipMemcpyDeviceToHost),40102);
  }
}

/* ----------------------------------------------------------------------------------------------- */

// inner core

extern "C"
void FC_FUNC_(transfer_kernels_ic_to_host,
              TRANSFER_KERNELS_IC_TO_HOST)(long* Mesh_pointer,
                                           realw* h_rho_kl,
                                           realw* h_alpha_kl,
                                           realw* h_beta_kl,
                                           int* NSPEC) {
TRACE("transfer_kernels_ic_to_host");

  //get mesh pointer out of fortran integer container
  Mesh* mp = (Mesh*)(*Mesh_pointer);

  int size = (*NSPEC)*NGLL3;

  print_CUDA_error_if_any(hipMemcpy(h_rho_kl,mp->d_rho_kl_inner_core,
                                     size*sizeof(realw),hipMemcpyDeviceToHost),40101);
  print_CUDA_error_if_any(hipMemcpy(h_alpha_kl,mp->d_alpha_kl_inner_core,
                                     size*sizeof(realw),hipMemcpyDeviceToHost),40102);
  print_CUDA_error_if_any(hipMemcpy(h_beta_kl,mp->d_beta_kl_inner_core,
                                     size*sizeof(realw),hipMemcpyDeviceToHost),40103);
}


/* ----------------------------------------------------------------------------------------------- */

// outer core

extern "C"
void FC_FUNC_(transfer_kernels_oc_to_host,
              TRANSFER_KERNELS_OC_TO_HOST)(long* Mesh_pointer,
                                           realw* h_rho_kl,
                                           realw* h_alpha_kl,
                                           int* NSPEC) {

  TRACE("transfer_kernels_oc_to_host");

  //get mesh pointer out of fortran integer container
  Mesh* mp = (Mesh*)(*Mesh_pointer);

  int size = (*NSPEC)*NGLL3;

  // copies kernel values over to CPU host
  print_CUDA_error_if_any(hipMemcpy(h_rho_kl,mp->d_rho_kl_outer_core,
                                     size*sizeof(realw),hipMemcpyDeviceToHost),54101);
  print_CUDA_error_if_any(hipMemcpy(h_alpha_kl,mp->d_alpha_kl_outer_core,
                                     size*sizeof(realw),hipMemcpyDeviceToHost),54102);
}

/* ----------------------------------------------------------------------------------------------- */

// for NOISE simulations

extern "C"
void FC_FUNC_(transfer_kernels_noise_to_host,
              TRANSFER_KERNELS_NOISE_TO_HOST)(long* Mesh_pointer,
                                              realw* h_Sigma_kl,
                                              int* NSPEC) {
  TRACE("transfer_kernels_noise_to_host");

  Mesh* mp = (Mesh*)(*Mesh_pointer); //get mesh pointer out of fortran integer container

  print_CUDA_error_if_any(hipMemcpy(h_Sigma_kl,mp->d_Sigma_kl,NGLL3*(*NSPEC)*sizeof(realw),
                                     hipMemcpyDeviceToHost),40201);
}

/* ----------------------------------------------------------------------------------------------- */

// for Hess kernel calculations

extern "C"
void FC_FUNC_(transfer_kernels_hess_cm_tohost,
              TRANSFER_KERNELS_HESS_CM_TOHOST)(long* Mesh_pointer,
                                              realw* h_hess_kl,
                                              int* NSPEC) {
TRACE("transfer_kernels_hess_cm_tohost");

  Mesh* mp = (Mesh*)(*Mesh_pointer); //get mesh pointer out of fortran integer container

  print_CUDA_error_if_any(hipMemcpy(h_hess_kl,mp->d_hess_kl_crust_mantle,NGLL3*(*NSPEC)*sizeof(realw),
                                     hipMemcpyDeviceToHost),70201);
}

