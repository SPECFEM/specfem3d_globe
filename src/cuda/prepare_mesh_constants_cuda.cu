/*
 !=====================================================================
 !
 !               S p e c f e m 3 D  V e r s i o n  2 . 0
 !               ---------------------------------------
 !
 !          Main authors: Dimitri Komatitsch and Jeroen Tromp
 !    Princeton University, USA and University of Pau / CNRS / INRIA
 ! (c) Princeton University / California Institute of Technology and University of Pau / CNRS / INRIA
 !                            April 2011
 !
 ! This program is free software; you can redistribute it and/or modify
 ! it under the terms of the GNU General Public License as published by
 ! the Free Software Foundation; either version 2 of the License, or
 ! (at your option) any later version.
 !
 ! This program is distributed in the hope that it will be useful,
 ! but WITHOUT ANY WARRANTY; without even the implied warranty of
 ! MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 ! GNU General Public License for more details.
 !
 ! You should have received a copy of the GNU General Public License along
 ! with this program; if not, write to the Free Software Foundation, Inc.,
 ! 51 Franklin Street, Fifth Floor, Boston, MA 02110-1301 USA.
 !
 !=====================================================================
 */

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

#ifdef WITH_MPI
#include <mpi.h>
#endif

#include <sys/time.h>
#include <sys/resource.h>

#include "config.h"
#include "mesh_constants_cuda.h"
#include "prepare_constants_cuda.h"



/* ----------------------------------------------------------------------------------------------- */

// GPU preparation

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(prepare_cuda_device,
              PREPARE_CUDA_DEVICE)(int* myrank_f,int* ncuda_devices) {
  TRACE("prepare_cuda_device");

  // Gets rank number of MPI process
  int myrank = *myrank_f;

  // cuda initialization (needs -lcuda library)
  // note:   hipInit initializes the driver API.
  //             it is needed for any following CUDA driver API function call (format cuFUNCTION(..) )
  //             however, for the CUDA runtime API functions (format cudaFUNCTION(..) )
  //             the initialization is implicit, thus hipInit() here would not be needed...
  hipError_t status = hipInit(0);
  if ( hipSuccess != status ) exit_on_error("CUDA driver API device initialization failed\n");

  // returns a handle to the first cuda compute device
  hipDevice_t dev;
  status = hipDeviceGet(&dev, 0);
  if ( hipSuccess != status ) exit_on_error("CUDA device not found\n");

  // gets device properties
  int major,minor;
  status = hipDeviceComputeCapability(&major,&minor,dev);
  if ( hipSuccess != status ) exit_on_error("CUDA device information not found\n");

  // make sure that the device has compute capability >= 1.3
  if (major < 1){
    fprintf(stderr,"Compute capability major number should be at least 1, got: %d \nexiting...\n",major);
    exit_on_error("CUDA Compute capability major number should be at least 1\n");
  }
  if (major == 1 && minor < 3){
    fprintf(stderr,"Compute capability should be at least 1.3, got: %d.%d \nexiting...\n",major,minor);
    exit_on_error("CUDA Compute capability major number should be at least 1.3\n");
  }

  // note: from here on we use the runtime API  ...
  // Gets number of GPU devices
  int device_count = 0;
  hipGetDeviceCount(&device_count);
  exit_on_cuda_error("CUDA runtime hipGetDeviceCount: check if driver and runtime libraries work together\nexiting...\n");

  // returns device count to fortran
  if (device_count == 0) exit_on_error("CUDA runtime error: there is no device supporting CUDA\n");
  *ncuda_devices = device_count;


  // Sets the active device
  if(device_count > 1) {
    // generalized for more GPUs per node
    // note: without previous context release, hipSetDevice will complain with the cuda error
    //         "setting the device when a process is active is not allowed"
    // releases previous contexts
    hipDeviceReset();

    //printf("rank %d: cuda device count = %d sets device = %d \n",myrank,device_count,myrank % device_count);
    //MPI_Barrier(MPI_COMM_WORLD);

    // sets active device
    hipSetDevice( myrank % device_count );
    exit_on_cuda_error("hipSetDevice");
  }

  // returns a handle to the active device
  int device;
  hipGetDevice(&device);

  // get device properties
  struct hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp,device);

  // exit if the machine has no CUDA-enabled device
  if (deviceProp.major == 9999 && deviceProp.minor == 9999){
    fprintf(stderr,"No CUDA-enabled device found, exiting...\n\n");
    exit_on_error("CUDA runtime error: there is no CUDA-enabled device found\n");
  }

  // outputs device infos to file
  char filename[BUFSIZ];
  FILE* fp;
  sprintf(filename,"OUTPUT_FILES/gpu_device_info_proc_%06d.txt",myrank);
  fp = fopen(filename,"a+");
  if (fp != NULL){
    // display device properties
    fprintf(fp,"Device Name = %s\n",deviceProp.name);
    fprintf(fp,"multiProcessorCount: %d\n",deviceProp.multiProcessorCount);
    fprintf(fp,"totalGlobalMem (in MB): %f\n",(unsigned long) deviceProp.totalGlobalMem / (1024.f * 1024.f));
    fprintf(fp,"totalGlobalMem (in GB): %f\n",(unsigned long) deviceProp.totalGlobalMem / (1024.f * 1024.f * 1024.f));
    fprintf(fp,"sharedMemPerBlock (in bytes): %lu\n",(unsigned long) deviceProp.sharedMemPerBlock);
    fprintf(fp,"Maximum number of threads per block: %d\n",deviceProp.maxThreadsPerBlock);
    fprintf(fp,"Maximum size of each dimension of a block: %d x %d x %d\n",
            deviceProp.maxThreadsDim[0],deviceProp.maxThreadsDim[1],deviceProp.maxThreadsDim[2]);
    fprintf(fp,"Maximum sizes of each dimension of a grid: %d x %d x %d\n",
            deviceProp.maxGridSize[0],deviceProp.maxGridSize[1],deviceProp.maxGridSize[2]);
    fprintf(fp,"Compute capability of the device = %d.%d\n", deviceProp.major, deviceProp.minor);
    if(deviceProp.canMapHostMemory){
      fprintf(fp,"canMapHostMemory: TRUE\n");
    }else{
      fprintf(fp,"canMapHostMemory: FALSE\n");
    }
    if(deviceProp.deviceOverlap){
      fprintf(fp,"deviceOverlap: TRUE\n");
    }else{
      fprintf(fp,"deviceOverlap: FALSE\n");
    }

    // make sure that the device has compute capability >= 1.3
    //if (deviceProp.major < 1){
    //  fprintf(stderr,"Compute capability major number should be at least 1, exiting...\n\n");
    //  exit_on_error("CUDA Compute capability major number should be at least 1");
    //}
    //if (deviceProp.major == 1 && deviceProp.minor < 3){
    //  fprintf(stderr,"Compute capability should be at least 1.3, exiting...\n");
    //  exit_on_error("CUDA Compute capability major number should be at least 1.3");
    //}

    // outputs initial memory infos via hipMemGetInfo()
    double free_db,used_db,total_db;
    get_free_memory(&free_db,&used_db,&total_db);
    fprintf(fp,"%d: GPU memory usage: used = %f MB, free = %f MB, total = %f MB\n",myrank,
            used_db/1024.0/1024.0, free_db/1024.0/1024.0, total_db/1024.0/1024.0);

    fclose(fp);
  }
}

/* ----------------------------------------------------------------------------------------------- */

// SIMULATION constants

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(prepare_constants_device,
              PREPARE_CONSTANTS_DEVICE)(long* Mesh_pointer,
                                        int* myrank_f,
                                        int* h_NGLLX,
                                        realw* h_hprime_xx,realw* h_hprime_yy,realw* h_hprime_zz,
                                        realw* h_hprimewgll_xx,realw* h_hprimewgll_yy,realw* h_hprimewgll_zz,
                                        realw* h_wgllwgll_xy,realw* h_wgllwgll_xz,realw* h_wgllwgll_yz,
                                        int* NSOURCES,int* nsources_local,
                                        realw* h_sourcearrays,
                                        int* h_islice_selected_source,int* h_ispec_selected_source,
                                        int* h_number_receiver_global,
                                        int* h_islice_selected_rec,int* h_ispec_selected_rec,
                                        int* nrec,int* nrec_local, int* nadj_rec_local,
                                        int* NSPEC_CRUST_MANTLE, int* NGLOB_CRUST_MANTLE,
                                        int* NSPEC_CRUST_MANTLE_STRAIN_ONLY,
                                        int* NGLOB_CRUST_MANTLE_OCEANS,
                                        int* NSPEC_OUTER_CORE, int* NGLOB_OUTER_CORE,
                                        int* NSPEC_INNER_CORE, int* NGLOB_INNER_CORE,
                                        int* SIMULATION_TYPE,
                                        int* NOISE_TOMOGRAPHY,
                                        int* SAVE_FORWARD_f,
                                        int* ABSORBING_CONDITIONS_f,
                                        int* OCEANS_f,
                                        int* GRAVITY_f,
                                        int* ROTATION_f,
                                        int* ATTENUATION_f,
                                        int* ATTENUATION_NEW_f,
                                        int* USE_ATTENUATION_MIMIC_f,
                                        int* COMPUTE_AND_STORE_STRAIN_f,
                                        int* ANISOTROPIC_3D_MANTLE_f,
                                        int* ANISOTROPIC_INNER_CORE_f,
                                        int* SAVE_BOUNDARY_MESH_f,
                                        int* USE_MESH_COLORING_GPU_f,
                                        int* ANISOTROPIC_KL_f,
                                        int* APPROXIMATE_HESS_KL_f) {

TRACE("prepare_constants_device");

  // allocates mesh parameter structure
  Mesh* mp = (Mesh*) malloc( sizeof(Mesh) );
  if (mp == NULL) exit_on_error("error allocating mesh pointer");
  *Mesh_pointer = (long)mp;

  // checks if NGLLX == 5
  if( *h_NGLLX != NGLLX ){
    exit_on_error("NGLLX must be 5 for CUDA devices");
  }

  // sets constant arrays
  setConst_hprime_xx(h_hprime_xx,mp);
  setConst_hprime_yy(h_hprime_yy,mp);
  setConst_hprime_zz(h_hprime_zz,mp);
  setConst_hprimewgll_xx(h_hprimewgll_xx,mp);
  setConst_hprimewgll_yy(h_hprimewgll_yy,mp);
  setConst_hprimewgll_zz(h_hprimewgll_zz,mp);
  setConst_wgllwgll_xy(h_wgllwgll_xy,mp);
  setConst_wgllwgll_xz(h_wgllwgll_xz,mp);
  setConst_wgllwgll_yz(h_wgllwgll_yz,mp);

  // sets global parameters
  mp->NSPEC_CRUST_MANTLE = *NSPEC_CRUST_MANTLE;
  mp->NGLOB_CRUST_MANTLE = *NGLOB_CRUST_MANTLE;
  mp->NSPEC_CRUST_MANTLE_STRAIN_ONLY = *NSPEC_CRUST_MANTLE_STRAIN_ONLY;
  mp->NGLOB_CRUST_MANTLE_OCEANS = *NGLOB_CRUST_MANTLE_OCEANS;
  mp->NSPEC_OUTER_CORE = *NSPEC_OUTER_CORE;
  mp->NGLOB_OUTER_CORE = *NGLOB_OUTER_CORE;
  mp->NSPEC_INNER_CORE = *NSPEC_INNER_CORE;
  mp->NGLOB_INNER_CORE = *NGLOB_INNER_CORE;

  // simulation type
  mp->simulation_type = *SIMULATION_TYPE;
  mp->noise_tomography = *NOISE_TOMOGRAPHY;

  // simulation flags initialization
  mp->save_forward = *SAVE_FORWARD_f;
  mp->absorbing_conditions = *ABSORBING_CONDITIONS_f;
  mp->oceans = *OCEANS_f;
  mp->gravity = *GRAVITY_f;
  mp->rotation = *ROTATION_f;
  mp->attenuation = *ATTENUATION_f;
  mp->attenuation_new = *ATTENUATION_NEW_f;
  mp->use_attenuation_mimic = *USE_ATTENUATION_MIMIC_f;
  mp->compute_and_store_strain = *COMPUTE_AND_STORE_STRAIN_f;
  mp->anisotropic_3D_mantle = *ANISOTROPIC_3D_MANTLE_f;
  mp->anisotropic_inner_core = *ANISOTROPIC_INNER_CORE_f;
  mp->save_boundary_mesh = *SAVE_BOUNDARY_MESH_f;

  mp->anisotropic_kl = *ANISOTROPIC_KL_f;
  mp->approximate_hess_kl = *APPROXIMATE_HESS_KL_f;

  // mpi process rank
  mp->myrank = *myrank_f;

  // mesh coloring flag
#ifdef USE_MESH_COLORING_GPU
  mp->use_mesh_coloring_gpu = 1;
  if( ! *USE_MESH_COLORING_GPU_f ){exit_on_error("error with USE_MESH_COLORING_GPU constant; please re-compile\n");}
#else
  // mesh coloring
  // note: this here passes the coloring as an option to the kernel routines
  //          the performance seems to be the same if one uses the pre-processing directives above or not
  mp->use_mesh_coloring_gpu = *USE_MESH_COLORING_GPU_f;
#endif

  // sources
  mp->nsources_local = *nsources_local;
  if( mp->simulation_type == 1  || mp->simulation_type == 3 ){
    // not needed in case of pure adjoint simulations (SIMULATION_TYPE == 2)
    print_CUDA_error_if_any(hipMalloc((void**)&mp->d_sourcearrays,
                                       sizeof(realw)* *NSOURCES*3*NGLL3),1301);
    print_CUDA_error_if_any(hipMemcpy(mp->d_sourcearrays, h_sourcearrays,
                                       sizeof(realw)* *NSOURCES*3*NGLL3,hipMemcpyHostToDevice),1302);
    // buffer for source time function values
    print_CUDA_error_if_any(hipMalloc((void**)&mp->d_stf_pre_compute,
                                       *NSOURCES*sizeof(double)),1303);
  }

  print_CUDA_error_if_any(hipMalloc((void**)&mp->d_islice_selected_source,
                                     sizeof(int) * *NSOURCES),1401);
  print_CUDA_error_if_any(hipMemcpy(mp->d_islice_selected_source, h_islice_selected_source,
                                     sizeof(int)* *NSOURCES,hipMemcpyHostToDevice),1402);

  print_CUDA_error_if_any(hipMalloc((void**)&mp->d_ispec_selected_source,
                                     sizeof(int)* *NSOURCES),1403);
  print_CUDA_error_if_any(hipMemcpy(mp->d_ispec_selected_source, h_ispec_selected_source,
                                     sizeof(int)* *NSOURCES,hipMemcpyHostToDevice),1404);


  // receiver stations
  // note that:   size(number_receiver_global) = nrec_local
  //                   size(ispec_selected_rec) = nrec
  // number of receiver located in this partition
  mp->nrec_local = *nrec_local;
  if( mp->nrec_local > 0 ){
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_number_receiver_global),mp->nrec_local*sizeof(int)),1);
    print_CUDA_error_if_any(hipMemcpy(mp->d_number_receiver_global,h_number_receiver_global,
                                     mp->nrec_local*sizeof(int),hipMemcpyHostToDevice),1512);

    // for seismograms
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_station_seismo_field),
                                       3*NGLL3*(mp->nrec_local)*sizeof(realw)),4015);

    mp->h_station_seismo_field = (realw*) malloc( 3*NGLL3*(mp->nrec_local)*sizeof(realw) );
    if( mp->h_station_seismo_field == NULL) exit_on_error("h_station_seismo_field not allocated \n");

  }
  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_ispec_selected_rec),(*nrec)*sizeof(int)),1513);
  print_CUDA_error_if_any(hipMemcpy(mp->d_ispec_selected_rec,h_ispec_selected_rec,
                                     (*nrec)*sizeof(int),hipMemcpyHostToDevice),1514);

  // receiver adjoint source arrays only used for noise and adjoint simulations
  // adjoint source arrays
  mp->nadj_rec_local = *nadj_rec_local;
  if( mp->nadj_rec_local > 0 ){
    print_CUDA_error_if_any(hipMalloc((void**)&mp->d_adj_sourcearrays,
                                       (mp->nadj_rec_local)*3*NGLL3*sizeof(realw)),6003);

    print_CUDA_error_if_any(hipMalloc((void**)&mp->d_pre_computed_irec,
                                       (mp->nadj_rec_local)*sizeof(int)),6004);

    // prepares local irec array:
    // the irec_local variable needs to be precomputed (as
    // h_pre_comp..), because normally it is in the loop updating accel,
    // and due to how it's incremented, it cannot be parallelized
    int* h_pre_computed_irec = (int*) malloc( (mp->nadj_rec_local)*sizeof(int) );
    if( h_pre_computed_irec == NULL ) exit_on_error("h_pre_computed_irec not allocated\n");

    int irec_local = 0;
    for(int irec = 0; irec < *nrec; irec++) {
      if(mp->myrank == h_islice_selected_rec[irec]) {
        irec_local++;
        h_pre_computed_irec[irec_local-1] = irec;
      }
    }
    if( irec_local != mp->nadj_rec_local ) exit_on_error("prepare_sim2_or_3_const_device: irec_local not equal\n");
    // copies values onto GPU
    print_CUDA_error_if_any(hipMemcpy(mp->d_pre_computed_irec,h_pre_computed_irec,
                                       (mp->nadj_rec_local)*sizeof(int),hipMemcpyHostToDevice),6010);
    free(h_pre_computed_irec);

    // temporary array to prepare extracted source array values
    mp->h_adj_sourcearrays_slice = (realw*) malloc( (mp->nadj_rec_local)*3*NGLL3*sizeof(realw) );
    if( mp->h_adj_sourcearrays_slice == NULL ) exit_on_error("h_adj_sourcearrays_slice not allocated\n");
  }

#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
  exit_on_cuda_error("prepare_constants_device");
#endif
}

/* ----------------------------------------------------------------------------------------------- */

// ROTATION simulations

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(prepare_fields_rotation_device,
              PREPARE_FIELDS_ROTATION_DEVICE)(long* Mesh_pointer_f,
                                              realw* two_omega_earth,
                                              realw* deltat,
                                              realw* A_array_rotation,
                                              realw* B_array_rotation,
                                              realw* b_two_omega_earth,
                                              realw* b_deltat,
                                              realw* b_A_array_rotation,
                                              realw* b_B_array_rotation,
                                              int* NSPEC_OUTER_CORE_ROTATION
                                              ) {

  TRACE("prepare_fields_rotation_device");

  Mesh* mp = (Mesh*)(*Mesh_pointer_f);

  // arrays only needed when rotation is required
  if( ! mp->rotation ){ exit_on_cuda_error("prepare_fields_rotation_device rotation not properly initialized"); }

  // rotation arrays (needed only for outer core region)
  mp->d_two_omega_earth = *two_omega_earth;
  mp->d_deltat = *deltat;

  print_CUDA_error_if_any(hipMalloc((void**) &mp->d_A_array_rotation,
                                     NGLL3*(*NSPEC_OUTER_CORE_ROTATION)*sizeof(realw)),9000);
  print_CUDA_error_if_any(hipMemcpy(mp->d_A_array_rotation, A_array_rotation,
                                     NGLL3*(*NSPEC_OUTER_CORE_ROTATION)*sizeof(realw),hipMemcpyHostToDevice),9001);

  print_CUDA_error_if_any(hipMalloc((void**) &mp->d_B_array_rotation,
                                     NGLL3*(*NSPEC_OUTER_CORE_ROTATION)*sizeof(realw)),9002);
  print_CUDA_error_if_any(hipMemcpy(mp->d_B_array_rotation, B_array_rotation,
                                     NGLL3*(*NSPEC_OUTER_CORE_ROTATION)*sizeof(realw),hipMemcpyHostToDevice),9003);

  // backward/reconstructed fields
  if( mp->simulation_type == 3 ){
    mp->d_b_two_omega_earth = *b_two_omega_earth;
    mp->d_b_deltat = *b_deltat;

    print_CUDA_error_if_any(hipMalloc((void**) &mp->d_b_A_array_rotation,
                                       NGLL3*(*NSPEC_OUTER_CORE_ROTATION)*sizeof(realw)),9000);
    print_CUDA_error_if_any(hipMemcpy(mp->d_b_A_array_rotation, b_A_array_rotation,
                                       NGLL3*(*NSPEC_OUTER_CORE_ROTATION)*sizeof(realw),hipMemcpyHostToDevice),9001);

    print_CUDA_error_if_any(hipMalloc((void**) &mp->d_b_B_array_rotation,
                                       NGLL3*(*NSPEC_OUTER_CORE_ROTATION)*sizeof(realw)),9002);
    print_CUDA_error_if_any(hipMemcpy(mp->d_b_B_array_rotation, b_B_array_rotation,
                                       NGLL3*(*NSPEC_OUTER_CORE_ROTATION)*sizeof(realw),hipMemcpyHostToDevice),9003);
  }
}


/* ----------------------------------------------------------------------------------------------- */

// GRAVITY simulations

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(prepare_fields_gravity_device,
              PREPARE_FIELDS_gravity_DEVICE)(long* Mesh_pointer_f,
                                             realw* d_ln_density_dr_table,
                                             realw* minus_rho_g_over_kappa_fluid,
                                             realw* minus_gravity_table,
                                             realw* minus_deriv_gravity_table,
                                             realw* density_table,
                                             realw* h_wgll_cube,
                                             int* NRAD_GRAVITY
                                             ) {

  TRACE("prepare_fields_gravity_device");

  Mesh* mp = (Mesh*)(*Mesh_pointer_f);

  if( ! mp->gravity ){
    // no gravity case

    // d ln(rho)/dr needed for the no gravity fluid potential
    print_CUDA_error_if_any(hipMalloc((void**) &mp->d_d_ln_density_dr_table,
                                       (*NRAD_GRAVITY)*sizeof(realw)),8000);
    print_CUDA_error_if_any(hipMemcpy(mp->d_d_ln_density_dr_table, d_ln_density_dr_table,
                                       (*NRAD_GRAVITY)*sizeof(realw),hipMemcpyHostToDevice),8001);

  }else{
    // gravity case

    // sets up gll weights cubed
    setConst_wgll_cube(h_wgll_cube,mp);

    // prepares gravity arrays
    print_CUDA_error_if_any(hipMalloc((void**) &mp->d_minus_rho_g_over_kappa_fluid,
                                       (*NRAD_GRAVITY)*sizeof(realw)),8000);
    print_CUDA_error_if_any(hipMemcpy(mp->d_minus_rho_g_over_kappa_fluid, minus_rho_g_over_kappa_fluid,
                                       (*NRAD_GRAVITY)*sizeof(realw),hipMemcpyHostToDevice),8001);

    print_CUDA_error_if_any(hipMalloc((void**) &mp->d_minus_gravity_table,
                                       (*NRAD_GRAVITY)*sizeof(realw)),8000);
    print_CUDA_error_if_any(hipMemcpy(mp->d_minus_gravity_table, minus_gravity_table,
                                       (*NRAD_GRAVITY)*sizeof(realw),hipMemcpyHostToDevice),8001);

    print_CUDA_error_if_any(hipMalloc((void**) &mp->d_minus_deriv_gravity_table,
                                       (*NRAD_GRAVITY)*sizeof(realw)),8000);
    print_CUDA_error_if_any(hipMemcpy(mp->d_minus_deriv_gravity_table, minus_deriv_gravity_table,
                                       (*NRAD_GRAVITY)*sizeof(realw),hipMemcpyHostToDevice),8001);

    print_CUDA_error_if_any(hipMalloc((void**) &mp->d_density_table,
                                       (*NRAD_GRAVITY)*sizeof(realw)),8000);
    print_CUDA_error_if_any(hipMemcpy(mp->d_density_table, density_table,
                                       (*NRAD_GRAVITY)*sizeof(realw),hipMemcpyHostToDevice),8001);
  }
}



/* ----------------------------------------------------------------------------------------------- */

// ATTENUATION simulations

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(prepare_fields_attenuat_device,
              PREPARE_FIELDS_ATTENUAT_DEVICE)(long* Mesh_pointer_f,
                                                 realw* R_xx_crust_mantle,
                                                 realw* R_yy_crust_mantle,
                                                 realw* R_xy_crust_mantle,
                                                 realw* R_xz_crust_mantle,
                                                 realw* R_yz_crust_mantle,
                                                 realw* factor_common_crust_mantle,
                                                 realw* one_minus_sum_beta_crust_mantle,
                                                 realw* R_xx_inner_core,
                                                 realw* R_yy_inner_core,
                                                 realw* R_xy_inner_core,
                                                 realw* R_xz_inner_core,
                                                 realw* R_yz_inner_core,
                                                 realw* factor_common_inner_core,
                                                 realw* one_minus_sum_beta_inner_core,
                                                 realw* alphaval,realw* betaval,realw* gammaval,
                                                 realw* b_alphaval,realw* b_betaval,realw* b_gammaval
                                                 ) {

  TRACE("prepare_fields_attenuat_device");
  int R_size1,R_size2,R_size3;

  Mesh* mp = (Mesh*)(*Mesh_pointer_f);

  // checks flag
  if( ! mp->attenuation ){ exit_on_cuda_error("prepare_fields_attenuat_device attenuation not properly initialized"); }

  // crust_mantle
  R_size1 = N_SLS*NGLL3*mp->NSPEC_CRUST_MANTLE;
  R_size2 = NGLL3*mp->NSPEC_CRUST_MANTLE;
  R_size3 = N_SLS*NGLL3*mp->NSPEC_CRUST_MANTLE;

  print_CUDA_error_if_any(hipMalloc((void**) &mp->d_one_minus_sum_beta_crust_mantle,
                                     R_size2*sizeof(realw)),4430);
  print_CUDA_error_if_any(hipMemcpy(mp->d_one_minus_sum_beta_crust_mantle,one_minus_sum_beta_crust_mantle,
                                     R_size2*sizeof(realw),hipMemcpyHostToDevice),4431);

  if( ! mp->use_attenuation_mimic ){
    // common factor
    print_CUDA_error_if_any(hipMalloc((void**) &mp->d_factor_common_crust_mantle,
                                         R_size3*sizeof(realw)),4432);
    print_CUDA_error_if_any(hipMemcpy(mp->d_factor_common_crust_mantle,factor_common_crust_mantle,
                                         R_size3*sizeof(realw),hipMemcpyHostToDevice),4433);

    // memory variables
    print_CUDA_error_if_any(hipMalloc((void**) &mp->d_R_xx_crust_mantle,
                                       R_size1*sizeof(realw)),4401);
    print_CUDA_error_if_any(hipMalloc((void**) &mp->d_R_yy_crust_mantle,
                                       R_size1*sizeof(realw)),4401);
    print_CUDA_error_if_any(hipMalloc((void**) &mp->d_R_xy_crust_mantle,
                                       R_size1*sizeof(realw)),4401);
    print_CUDA_error_if_any(hipMalloc((void**) &mp->d_R_xz_crust_mantle,
                                       R_size1*sizeof(realw)),4401);
    print_CUDA_error_if_any(hipMalloc((void**) &mp->d_R_yz_crust_mantle,
                                       R_size1*sizeof(realw)),4401);

    print_CUDA_error_if_any(hipMemcpy(mp->d_R_xx_crust_mantle,R_xx_crust_mantle,
                                         R_size1*sizeof(realw),hipMemcpyHostToDevice),4800);
    print_CUDA_error_if_any(hipMemcpy(mp->d_R_yy_crust_mantle,R_yy_crust_mantle,
                                         R_size1*sizeof(realw),hipMemcpyHostToDevice),4800);
    print_CUDA_error_if_any(hipMemcpy(mp->d_R_xy_crust_mantle,R_xy_crust_mantle,
                                         R_size1*sizeof(realw),hipMemcpyHostToDevice),4800);
    print_CUDA_error_if_any(hipMemcpy(mp->d_R_xz_crust_mantle,R_xz_crust_mantle,
                                         R_size1*sizeof(realw),hipMemcpyHostToDevice),4800);
    print_CUDA_error_if_any(hipMemcpy(mp->d_R_yz_crust_mantle,R_yz_crust_mantle,
                                         R_size1*sizeof(realw),hipMemcpyHostToDevice),4800);
  }

  // inner_core
  R_size1 = 5*N_SLS*NGLL3*mp->NSPEC_INNER_CORE;
  R_size2 = NGLL3*mp->NSPEC_INNER_CORE;
  R_size3 = N_SLS*NGLL3*mp->NSPEC_INNER_CORE;

  print_CUDA_error_if_any(hipMalloc((void**) &mp->d_one_minus_sum_beta_inner_core,
                                     R_size2*sizeof(realw)),4430);
  print_CUDA_error_if_any(hipMemcpy(mp->d_one_minus_sum_beta_inner_core,one_minus_sum_beta_inner_core,
                                     R_size2*sizeof(realw),hipMemcpyHostToDevice),4431);

  if( ! mp->use_attenuation_mimic ){
    // common factor
    print_CUDA_error_if_any(hipMalloc((void**) &mp->d_factor_common_inner_core,
                                       R_size3*sizeof(realw)),4432);
    print_CUDA_error_if_any(hipMemcpy(mp->d_factor_common_inner_core,factor_common_inner_core,
                                       R_size3*sizeof(realw),hipMemcpyHostToDevice),4433);

    // memory variables
    print_CUDA_error_if_any(hipMalloc((void**) &mp->d_R_xx_inner_core,
                                     R_size1*sizeof(realw)),4401);
    print_CUDA_error_if_any(hipMalloc((void**) &mp->d_R_yy_inner_core,
                                       R_size1*sizeof(realw)),4401);
    print_CUDA_error_if_any(hipMalloc((void**) &mp->d_R_xy_inner_core,
                                       R_size1*sizeof(realw)),4401);
    print_CUDA_error_if_any(hipMalloc((void**) &mp->d_R_xz_inner_core,
                                       R_size1*sizeof(realw)),4401);
    print_CUDA_error_if_any(hipMalloc((void**) &mp->d_R_yz_inner_core,
                                       R_size1*sizeof(realw)),4401);

    print_CUDA_error_if_any(hipMemcpy(mp->d_R_xx_inner_core,R_xx_inner_core,
                                     R_size1*sizeof(realw),hipMemcpyHostToDevice),4402);
    print_CUDA_error_if_any(hipMemcpy(mp->d_R_yy_inner_core,R_yy_inner_core,
                                       R_size1*sizeof(realw),hipMemcpyHostToDevice),4402);
    print_CUDA_error_if_any(hipMemcpy(mp->d_R_xy_inner_core,R_xy_inner_core,
                                       R_size1*sizeof(realw),hipMemcpyHostToDevice),4402);
    print_CUDA_error_if_any(hipMemcpy(mp->d_R_xz_inner_core,R_xz_inner_core,
                                       R_size1*sizeof(realw),hipMemcpyHostToDevice),4402);
    print_CUDA_error_if_any(hipMemcpy(mp->d_R_yz_inner_core,R_yz_inner_core,
                                       R_size1*sizeof(realw),hipMemcpyHostToDevice),4402);
  }

  // alpha,beta,gamma factors
  print_CUDA_error_if_any(hipMalloc((void**) &(mp->d_alphaval),
                                     N_SLS*sizeof(realw)),4434);
  print_CUDA_error_if_any(hipMemcpy(mp->d_alphaval ,alphaval,
                                     N_SLS*sizeof(realw),hipMemcpyHostToDevice),4435);

  print_CUDA_error_if_any(hipMalloc((void**) &(mp->d_betaval),
                                     N_SLS*sizeof(realw)),4436);
  print_CUDA_error_if_any(hipMemcpy(mp->d_betaval ,betaval,
                                     N_SLS*sizeof(realw),hipMemcpyHostToDevice),4437);

  print_CUDA_error_if_any(hipMalloc((void**) &(mp->d_gammaval),
                                     N_SLS*sizeof(realw)),4438);
  print_CUDA_error_if_any(hipMemcpy(mp->d_gammaval ,gammaval,
                                     N_SLS*sizeof(realw),hipMemcpyHostToDevice),4439);

  if( mp->simulation_type == 3 ){
    // alpha,beta,gamma factors for backward fields
    print_CUDA_error_if_any(hipMalloc((void**) &(mp->d_b_alphaval),
                                       N_SLS*sizeof(realw)),5434);
    print_CUDA_error_if_any(hipMemcpy(mp->d_b_alphaval ,b_alphaval,
                                       N_SLS*sizeof(realw),hipMemcpyHostToDevice),5435);

    print_CUDA_error_if_any(hipMalloc((void**) &(mp->d_b_betaval),
                                       N_SLS*sizeof(realw)),5436);
    print_CUDA_error_if_any(hipMemcpy(mp->d_b_betaval ,b_betaval,
                                       N_SLS*sizeof(realw),hipMemcpyHostToDevice),5437);

    print_CUDA_error_if_any(hipMalloc((void**) &(mp->d_b_gammaval),
                                       N_SLS*sizeof(realw)),5438);
    print_CUDA_error_if_any(hipMemcpy(mp->d_b_gammaval ,b_gammaval,
                                       N_SLS*sizeof(realw),hipMemcpyHostToDevice),5439);
  }
}

/* ----------------------------------------------------------------------------------------------- */

// STRAIN simulations

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(prepare_fields_strain_device,
              PREPARE_FIELDS_STRAIN_DEVICE)(long* Mesh_pointer_f,
                                            realw* epsilondev_xx_crust_mantle,
                                            realw* epsilondev_yy_crust_mantle,
                                            realw* epsilondev_xy_crust_mantle,
                                            realw* epsilondev_xz_crust_mantle,
                                            realw* epsilondev_yz_crust_mantle,
                                            realw* b_epsilondev_xx_crust_mantle,
                                            realw* b_epsilondev_yy_crust_mantle,
                                            realw* b_epsilondev_xy_crust_mantle,
                                            realw* b_epsilondev_xz_crust_mantle,
                                            realw* b_epsilondev_yz_crust_mantle,
                                            realw* eps_trace_over_3_crust_mantle,
                                            realw* b_eps_trace_over_3_crust_mantle,
                                            realw* epsilondev_xx_inner_core,
                                            realw* epsilondev_yy_inner_core,
                                            realw* epsilondev_xy_inner_core,
                                            realw* epsilondev_xz_inner_core,
                                            realw* epsilondev_yz_inner_core,
                                            realw* b_epsilondev_xx_inner_core,
                                            realw* b_epsilondev_yy_inner_core,
                                            realw* b_epsilondev_xy_inner_core,
                                            realw* b_epsilondev_xz_inner_core,
                                            realw* b_epsilondev_yz_inner_core,
                                            realw* eps_trace_over_3_inner_core,
                                            realw* b_eps_trace_over_3_inner_core
                                            ) {

  TRACE("prepare_fields_strain_device");
  int R_size,size_strain_only;

  Mesh* mp = (Mesh*)(*Mesh_pointer_f);

  // checks flag
  if( ! mp->compute_and_store_strain ){ exit_on_cuda_error("prepare_fields_strain_device strain not properly initialized"); }

  // crust_mantle
  R_size = NGLL3*mp->NSPEC_CRUST_MANTLE;
  print_CUDA_error_if_any(hipMalloc((void**) &mp->d_epsilondev_xx_crust_mantle,
                                     R_size*sizeof(realw)),4432);
  print_CUDA_error_if_any(hipMalloc((void**) &mp->d_epsilondev_yy_crust_mantle,
                                     R_size*sizeof(realw)),4432);
  print_CUDA_error_if_any(hipMalloc((void**) &mp->d_epsilondev_xy_crust_mantle,
                                     R_size*sizeof(realw)),4432);
  print_CUDA_error_if_any(hipMalloc((void**) &mp->d_epsilondev_xz_crust_mantle,
                                     R_size*sizeof(realw)),4432);
  print_CUDA_error_if_any(hipMalloc((void**) &mp->d_epsilondev_yz_crust_mantle,
                                     R_size*sizeof(realw)),4432);

  print_CUDA_error_if_any(hipMemcpy(mp->d_epsilondev_xx_crust_mantle,epsilondev_xx_crust_mantle,
                                     R_size*sizeof(realw),hipMemcpyHostToDevice),4433);
  print_CUDA_error_if_any(hipMemcpy(mp->d_epsilondev_yy_crust_mantle,epsilondev_yy_crust_mantle,
                                     R_size*sizeof(realw),hipMemcpyHostToDevice),4433);
  print_CUDA_error_if_any(hipMemcpy(mp->d_epsilondev_xy_crust_mantle,epsilondev_xy_crust_mantle,
                                     R_size*sizeof(realw),hipMemcpyHostToDevice),4433);
  print_CUDA_error_if_any(hipMemcpy(mp->d_epsilondev_xz_crust_mantle,epsilondev_xz_crust_mantle,
                                     R_size*sizeof(realw),hipMemcpyHostToDevice),4433);
  print_CUDA_error_if_any(hipMemcpy(mp->d_epsilondev_yz_crust_mantle,epsilondev_yz_crust_mantle,
                                     R_size*sizeof(realw),hipMemcpyHostToDevice),4433);

  // strain
  size_strain_only = NGLL3*(mp->NSPEC_CRUST_MANTLE_STRAIN_ONLY);
  print_CUDA_error_if_any(hipMalloc((void**) &mp->d_eps_trace_over_3_crust_mantle,
                                      size_strain_only*sizeof(realw)),4401);
  print_CUDA_error_if_any(hipMemcpy(mp->d_eps_trace_over_3_crust_mantle,eps_trace_over_3_crust_mantle,
                                      size_strain_only*sizeof(realw),hipMemcpyHostToDevice),4402);

  // backward/reconstructed fields
  if( mp->simulation_type == 3 ){
    print_CUDA_error_if_any(hipMalloc((void**) &mp->d_b_epsilondev_xx_crust_mantle,
                                       R_size*sizeof(realw)),4432);
    print_CUDA_error_if_any(hipMalloc((void**) &mp->d_b_epsilondev_yy_crust_mantle,
                                       R_size*sizeof(realw)),4432);
    print_CUDA_error_if_any(hipMalloc((void**) &mp->d_b_epsilondev_xy_crust_mantle,
                                       R_size*sizeof(realw)),4432);
    print_CUDA_error_if_any(hipMalloc((void**) &mp->d_b_epsilondev_xz_crust_mantle,
                                       R_size*sizeof(realw)),4432);
    print_CUDA_error_if_any(hipMalloc((void**) &mp->d_b_epsilondev_yz_crust_mantle,
                                       R_size*sizeof(realw)),4432);

    print_CUDA_error_if_any(hipMemcpy(mp->d_b_epsilondev_xx_crust_mantle,b_epsilondev_xx_crust_mantle,
                                       R_size*sizeof(realw),hipMemcpyHostToDevice),4433);
    print_CUDA_error_if_any(hipMemcpy(mp->d_b_epsilondev_yy_crust_mantle,b_epsilondev_yy_crust_mantle,
                                       R_size*sizeof(realw),hipMemcpyHostToDevice),4433);
    print_CUDA_error_if_any(hipMemcpy(mp->d_b_epsilondev_xy_crust_mantle,b_epsilondev_xy_crust_mantle,
                                       R_size*sizeof(realw),hipMemcpyHostToDevice),4433);
    print_CUDA_error_if_any(hipMemcpy(mp->d_b_epsilondev_xz_crust_mantle,b_epsilondev_xz_crust_mantle,
                                       R_size*sizeof(realw),hipMemcpyHostToDevice),4433);
    print_CUDA_error_if_any(hipMemcpy(mp->d_b_epsilondev_yz_crust_mantle,b_epsilondev_yz_crust_mantle,
                                       R_size*sizeof(realw),hipMemcpyHostToDevice),4433);


    print_CUDA_error_if_any(hipMalloc((void**) &mp->d_b_eps_trace_over_3_crust_mantle,
                                         R_size*sizeof(realw)),4401);
    print_CUDA_error_if_any(hipMemcpy(mp->d_b_eps_trace_over_3_crust_mantle,b_eps_trace_over_3_crust_mantle,
                                         R_size*sizeof(realw),hipMemcpyHostToDevice),4402);
  }

  // inner_core
  R_size = NGLL3*mp->NSPEC_INNER_CORE;
  print_CUDA_error_if_any(hipMalloc((void**) &mp->d_epsilondev_xx_inner_core,
                                     R_size*sizeof(realw)),4432);
  print_CUDA_error_if_any(hipMalloc((void**) &mp->d_epsilondev_yy_inner_core,
                                     R_size*sizeof(realw)),4432);
  print_CUDA_error_if_any(hipMalloc((void**) &mp->d_epsilondev_xy_inner_core,
                                     R_size*sizeof(realw)),4432);
  print_CUDA_error_if_any(hipMalloc((void**) &mp->d_epsilondev_xz_inner_core,
                                     R_size*sizeof(realw)),4432);
  print_CUDA_error_if_any(hipMalloc((void**) &mp->d_epsilondev_yz_inner_core,
                                     R_size*sizeof(realw)),4432);

  print_CUDA_error_if_any(hipMemcpy(mp->d_epsilondev_xx_inner_core,epsilondev_xx_inner_core,
                                     R_size*sizeof(realw),hipMemcpyHostToDevice),4433);
  print_CUDA_error_if_any(hipMemcpy(mp->d_epsilondev_yy_inner_core,epsilondev_yy_inner_core,
                                     R_size*sizeof(realw),hipMemcpyHostToDevice),4433);
  print_CUDA_error_if_any(hipMemcpy(mp->d_epsilondev_xy_inner_core,epsilondev_xy_inner_core,
                                     R_size*sizeof(realw),hipMemcpyHostToDevice),4433);
  print_CUDA_error_if_any(hipMemcpy(mp->d_epsilondev_xz_inner_core,epsilondev_xz_inner_core,
                                     R_size*sizeof(realw),hipMemcpyHostToDevice),4433);
  print_CUDA_error_if_any(hipMemcpy(mp->d_epsilondev_yz_inner_core,epsilondev_yz_inner_core,
                                     R_size*sizeof(realw),hipMemcpyHostToDevice),4433);


  print_CUDA_error_if_any(hipMalloc((void**) &mp->d_eps_trace_over_3_inner_core,
                                     R_size*sizeof(realw)),4401);
  print_CUDA_error_if_any(hipMemcpy(mp->d_eps_trace_over_3_inner_core,eps_trace_over_3_inner_core,
                                     R_size*sizeof(realw),hipMemcpyHostToDevice),4402);
  // backward/reconstructed fields
  if( mp->simulation_type == 3 ){
    print_CUDA_error_if_any(hipMalloc((void**) &mp->d_b_epsilondev_xx_inner_core,
                                       R_size*sizeof(realw)),4432);
    print_CUDA_error_if_any(hipMalloc((void**) &mp->d_b_epsilondev_yy_inner_core,
                                       R_size*sizeof(realw)),4432);
    print_CUDA_error_if_any(hipMalloc((void**) &mp->d_b_epsilondev_xy_inner_core,
                                       R_size*sizeof(realw)),4432);
    print_CUDA_error_if_any(hipMalloc((void**) &mp->d_b_epsilondev_xz_inner_core,
                                       R_size*sizeof(realw)),4432);
    print_CUDA_error_if_any(hipMalloc((void**) &mp->d_b_epsilondev_yz_inner_core,
                                       R_size*sizeof(realw)),4432);

    print_CUDA_error_if_any(hipMemcpy(mp->d_b_epsilondev_xx_inner_core,b_epsilondev_xx_inner_core,
                                       R_size*sizeof(realw),hipMemcpyHostToDevice),4433);
    print_CUDA_error_if_any(hipMemcpy(mp->d_b_epsilondev_yy_inner_core,b_epsilondev_yy_inner_core,
                                       R_size*sizeof(realw),hipMemcpyHostToDevice),4433);
    print_CUDA_error_if_any(hipMemcpy(mp->d_b_epsilondev_xy_inner_core,b_epsilondev_xy_inner_core,
                                       R_size*sizeof(realw),hipMemcpyHostToDevice),4433);
    print_CUDA_error_if_any(hipMemcpy(mp->d_b_epsilondev_xz_inner_core,b_epsilondev_xz_inner_core,
                                       R_size*sizeof(realw),hipMemcpyHostToDevice),4433);
    print_CUDA_error_if_any(hipMemcpy(mp->d_b_epsilondev_yz_inner_core,b_epsilondev_yz_inner_core,
                                       R_size*sizeof(realw),hipMemcpyHostToDevice),4433);


    print_CUDA_error_if_any(hipMalloc((void**) &mp->d_b_eps_trace_over_3_inner_core,
                                       R_size*sizeof(realw)),4401);
    print_CUDA_error_if_any(hipMemcpy(mp->d_b_eps_trace_over_3_inner_core,b_eps_trace_over_3_inner_core,
                                       R_size*sizeof(realw),hipMemcpyHostToDevice),4402);
  }
}

/* ----------------------------------------------------------------------------------------------- */

// STRAIN simulations

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(prepare_fields_absorb_device,
              PREPARE_FIELDS_ABSORB_DEVICE)(long* Mesh_pointer_f,
                                            int* nspec2D_xmin_crust_mantle,int* nspec2D_xmax_crust_mantle,
                                            int* nspec2D_ymin_crust_mantle,int* nspec2D_ymax_crust_mantle,
                                            int* NSPEC2DMAX_XMIN_XMAX_CM,int* NSPEC2DMAX_YMIN_YMAX_CM,
                                            int* nimin_crust_mantle,int* nimax_crust_mantle,
                                            int* njmin_crust_mantle,int* njmax_crust_mantle,
                                            int* nkmin_xi_crust_mantle,int* nkmin_eta_crust_mantle,
                                            int* ibelm_xmin_crust_mantle,int* ibelm_xmax_crust_mantle,
                                            int* ibelm_ymin_crust_mantle,int* ibelm_ymax_crust_mantle,
                                            realw* normal_xmin_crust_mantle,realw* normal_xmax_crust_mantle,
                                            realw* normal_ymin_crust_mantle,realw* normal_ymax_crust_mantle,
                                            realw* jacobian2D_xmin_crust_mantle, realw* jacobian2D_xmax_crust_mantle,
                                            realw* jacobian2D_ymin_crust_mantle, realw* jacobian2D_ymax_crust_mantle,
                                            realw* rho_vp_crust_mantle,
                                            realw* rho_vs_crust_mantle,
                                            int* nspec2D_xmin_outer_core,int* nspec2D_xmax_outer_core,
                                            int* nspec2D_ymin_outer_core,int* nspec2D_ymax_outer_core,
                                            int* nspec2D_zmin_outer_core,
                                            int* NSPEC2DMAX_XMIN_XMAX_OC,int* NSPEC2DMAX_YMIN_YMAX_OC,
                                            int* nimin_outer_core,int* nimax_outer_core,
                                            int* njmin_outer_core,int* njmax_outer_core,
                                            int* nkmin_xi_outer_core,int* nkmin_eta_outer_core,
                                            int* ibelm_xmin_outer_core,int* ibelm_xmax_outer_core,
                                            int* ibelm_ymin_outer_core,int* ibelm_ymax_outer_core,
                                            int* ibelm_bottom_outer_core,
                                            realw* jacobian2D_xmin_outer_core, realw* jacobian2D_xmax_outer_core,
                                            realw* jacobian2D_ymin_outer_core, realw* jacobian2D_ymax_outer_core,
                                            realw* jacobian2D_bottom_outer_core,
                                            realw* vp_outer_core
                                            ) {

  TRACE("prepare_fields_absorb_device");
  int size;

  Mesh* mp = (Mesh*)(*Mesh_pointer_f);

  // checks flag
  if( ! mp->absorbing_conditions ){ exit_on_cuda_error("prepare_fields_absorb_device absorbing_conditions not properly initialized"); }

  // crust_mantle
  mp->nspec2D_xmin_crust_mantle = *nspec2D_xmin_crust_mantle;
  mp->nspec2D_xmax_crust_mantle = *nspec2D_xmax_crust_mantle;
  mp->nspec2D_ymin_crust_mantle = *nspec2D_ymin_crust_mantle;
  mp->nspec2D_ymax_crust_mantle = *nspec2D_ymax_crust_mantle;

  // vp & vs
  size = NGLL3*(mp->NSPEC_CRUST_MANTLE);
  print_CUDA_error_if_any(hipMalloc((void**) &mp->d_rho_vp_crust_mantle,
                                     size*sizeof(realw)),2201);
  print_CUDA_error_if_any(hipMemcpy(mp->d_rho_vp_crust_mantle,rho_vp_crust_mantle,
                                     size*sizeof(realw),hipMemcpyHostToDevice),2202);

  print_CUDA_error_if_any(hipMalloc((void**) &mp->d_rho_vs_crust_mantle,
                                     size*sizeof(realw)),2201);
  print_CUDA_error_if_any(hipMemcpy(mp->d_rho_vs_crust_mantle,rho_vs_crust_mantle,
                                     size*sizeof(realw),hipMemcpyHostToDevice),2202);

  // ijk index arrays
  print_CUDA_error_if_any(hipMalloc((void**) &mp->d_nkmin_xi_crust_mantle,
                                     2*(*NSPEC2DMAX_XMIN_XMAX_CM)*sizeof(int)),1201);
  print_CUDA_error_if_any(hipMemcpy(mp->d_nkmin_xi_crust_mantle,nkmin_xi_crust_mantle,
                                     2*(*NSPEC2DMAX_XMIN_XMAX_CM)*sizeof(int),hipMemcpyHostToDevice),1202);

  print_CUDA_error_if_any(hipMalloc((void**) &mp->d_nkmin_eta_crust_mantle,
                                     2*(*NSPEC2DMAX_YMIN_YMAX_CM)*sizeof(int)),1201);
  print_CUDA_error_if_any(hipMemcpy(mp->d_nkmin_eta_crust_mantle,nkmin_eta_crust_mantle,
                                     2*(*NSPEC2DMAX_YMIN_YMAX_CM)*sizeof(int),hipMemcpyHostToDevice),1202);

  print_CUDA_error_if_any(hipMalloc((void**) &mp->d_njmin_crust_mantle,
                                     2*(*NSPEC2DMAX_XMIN_XMAX_CM)*sizeof(int)),1201);
  print_CUDA_error_if_any(hipMemcpy(mp->d_njmin_crust_mantle,njmin_crust_mantle,
                                     2*(*NSPEC2DMAX_XMIN_XMAX_CM)*sizeof(int),hipMemcpyHostToDevice),1202);

  print_CUDA_error_if_any(hipMalloc((void**) &mp->d_njmax_crust_mantle,
                                     2*(*NSPEC2DMAX_XMIN_XMAX_CM)*sizeof(int)),1201);
  print_CUDA_error_if_any(hipMemcpy(mp->d_njmax_crust_mantle,njmax_crust_mantle,
                                     2*(*NSPEC2DMAX_XMIN_XMAX_CM)*sizeof(int),hipMemcpyHostToDevice),1202);

  print_CUDA_error_if_any(hipMalloc((void**) &mp->d_nimin_crust_mantle,
                                     2*(*NSPEC2DMAX_YMIN_YMAX_CM)*sizeof(int)),1201);
  print_CUDA_error_if_any(hipMemcpy(mp->d_nimin_crust_mantle,nimin_crust_mantle,
                                     2*(*NSPEC2DMAX_YMIN_YMAX_CM)*sizeof(int),hipMemcpyHostToDevice),1202);

  print_CUDA_error_if_any(hipMalloc((void**) &mp->d_nimax_crust_mantle,
                                     2*(*NSPEC2DMAX_YMIN_YMAX_CM)*sizeof(int)),1201);
  print_CUDA_error_if_any(hipMemcpy(mp->d_nimax_crust_mantle,nimax_crust_mantle,
                                     2*(*NSPEC2DMAX_YMIN_YMAX_CM)*sizeof(int),hipMemcpyHostToDevice),1202);


  // xmin
  if( mp->nspec2D_xmin_crust_mantle > 0 ){
    print_CUDA_error_if_any(hipMalloc((void**) &mp->d_ibelm_xmin_crust_mantle,
                                       (mp->nspec2D_xmin_crust_mantle)*sizeof(int)),1201);
    print_CUDA_error_if_any(hipMemcpy(mp->d_ibelm_xmin_crust_mantle,ibelm_xmin_crust_mantle,
                                       (mp->nspec2D_xmin_crust_mantle)*sizeof(int),hipMemcpyHostToDevice),1202);

    print_CUDA_error_if_any(hipMalloc((void**) &mp->d_normal_xmin_crust_mantle,
                            NDIM*NGLL2*(mp->nspec2D_xmin_crust_mantle)*sizeof(realw)),1201);
    print_CUDA_error_if_any(hipMemcpy(mp->d_normal_xmin_crust_mantle,normal_xmin_crust_mantle,
                            NDIM*NGLL2*(mp->nspec2D_xmin_crust_mantle)*sizeof(realw),hipMemcpyHostToDevice),1202);

    print_CUDA_error_if_any(hipMalloc((void**) &mp->d_jacobian2D_xmin_crust_mantle,
                            NGLL2*(mp->nspec2D_xmin_crust_mantle)*sizeof(realw)),1201);
    print_CUDA_error_if_any(hipMemcpy(mp->d_jacobian2D_xmin_crust_mantle,jacobian2D_xmin_crust_mantle,
                            NGLL2*(mp->nspec2D_xmin_crust_mantle)*sizeof(realw),hipMemcpyHostToDevice),1202);

    // boundary buffer
    if( (mp->simulation_type == 1 && mp->save_forward ) || (mp->simulation_type == 3) ){
      print_CUDA_error_if_any(hipMalloc((void**) &mp->d_absorb_xmin_crust_mantle,
                              NDIM*NGLL2*(mp->nspec2D_xmin_crust_mantle)*sizeof(realw)),1202);
    }
  }

  // xmax
  if( mp->nspec2D_xmax_crust_mantle > 0 ){
    print_CUDA_error_if_any(hipMalloc((void**) &mp->d_ibelm_xmax_crust_mantle,
                                       (mp->nspec2D_xmax_crust_mantle)*sizeof(int)),1201);
    print_CUDA_error_if_any(hipMemcpy(mp->d_ibelm_xmax_crust_mantle,ibelm_xmax_crust_mantle,
                                       (mp->nspec2D_xmax_crust_mantle)*sizeof(int),hipMemcpyHostToDevice),1202);

    print_CUDA_error_if_any(hipMalloc((void**) &mp->d_normal_xmax_crust_mantle,
                                       NDIM*NGLL2*(mp->nspec2D_xmax_crust_mantle)*sizeof(realw)),1201);
    print_CUDA_error_if_any(hipMemcpy(mp->d_normal_xmax_crust_mantle,normal_xmax_crust_mantle,
                                       NDIM*NGLL2*(mp->nspec2D_xmax_crust_mantle)*sizeof(realw),hipMemcpyHostToDevice),1202);

    print_CUDA_error_if_any(hipMalloc((void**) &mp->d_jacobian2D_xmax_crust_mantle,
                                       NGLL2*(mp->nspec2D_xmax_crust_mantle)*sizeof(realw)),1201);
    print_CUDA_error_if_any(hipMemcpy(mp->d_jacobian2D_xmax_crust_mantle,jacobian2D_xmax_crust_mantle,
                                       NGLL2*(mp->nspec2D_xmax_crust_mantle)*sizeof(realw),hipMemcpyHostToDevice),1202);

    // boundary buffer
    if( (mp->simulation_type == 1 && mp->save_forward ) || (mp->simulation_type == 3) ){
      print_CUDA_error_if_any(hipMalloc((void**) &mp->d_absorb_xmax_crust_mantle,
                                         NDIM*NGLL2*(mp->nspec2D_xmax_crust_mantle)*sizeof(realw)),1202);
    }
  }

  // ymin
  if( mp->nspec2D_ymin_crust_mantle > 0 ){
    print_CUDA_error_if_any(hipMalloc((void**) &mp->d_ibelm_ymin_crust_mantle,
                                       (mp->nspec2D_ymin_crust_mantle)*sizeof(int)),1201);
    print_CUDA_error_if_any(hipMemcpy(mp->d_ibelm_ymin_crust_mantle,ibelm_ymin_crust_mantle,
                                       (mp->nspec2D_ymin_crust_mantle)*sizeof(int),hipMemcpyHostToDevice),1202);

    print_CUDA_error_if_any(hipMalloc((void**) &mp->d_normal_ymin_crust_mantle,
                                       NDIM*NGLL2*(mp->nspec2D_ymin_crust_mantle)*sizeof(realw)),1201);
    print_CUDA_error_if_any(hipMemcpy(mp->d_normal_ymin_crust_mantle,normal_ymin_crust_mantle,
                                       NDIM*NGLL2*(mp->nspec2D_ymin_crust_mantle)*sizeof(realw),hipMemcpyHostToDevice),1202);

    print_CUDA_error_if_any(hipMalloc((void**) &mp->d_jacobian2D_ymin_crust_mantle,
                                       NGLL2*(mp->nspec2D_ymin_crust_mantle)*sizeof(realw)),1201);
    print_CUDA_error_if_any(hipMemcpy(mp->d_jacobian2D_ymin_crust_mantle,jacobian2D_ymin_crust_mantle,
                                       NGLL2*(mp->nspec2D_ymin_crust_mantle)*sizeof(realw),hipMemcpyHostToDevice),1202);

    // boundary buffer
    if( (mp->simulation_type == 1 && mp->save_forward ) || (mp->simulation_type == 3) ){
      print_CUDA_error_if_any(hipMalloc((void**) &mp->d_absorb_ymin_crust_mantle,
                                         NDIM*NGLL2*(mp->nspec2D_ymin_crust_mantle)*sizeof(realw)),1202);
    }
  }

  // ymax
  if( mp->nspec2D_ymax_crust_mantle > 0 ){
    print_CUDA_error_if_any(hipMalloc((void**) &mp->d_ibelm_ymax_crust_mantle,
                                       (mp->nspec2D_ymax_crust_mantle)*sizeof(int)),1201);
    print_CUDA_error_if_any(hipMemcpy(mp->d_ibelm_ymax_crust_mantle,ibelm_ymax_crust_mantle,
                                       (mp->nspec2D_ymax_crust_mantle)*sizeof(int),hipMemcpyHostToDevice),1202);

    print_CUDA_error_if_any(hipMalloc((void**) &mp->d_normal_ymax_crust_mantle,
                                       NDIM*NGLL2*(mp->nspec2D_ymax_crust_mantle)*sizeof(realw)),1201);
    print_CUDA_error_if_any(hipMemcpy(mp->d_normal_ymax_crust_mantle,normal_ymax_crust_mantle,
                                       NDIM*NGLL2*(mp->nspec2D_ymax_crust_mantle)*sizeof(realw),hipMemcpyHostToDevice),1202);

    print_CUDA_error_if_any(hipMalloc((void**) &mp->d_jacobian2D_ymax_crust_mantle,
                                       NGLL2*(mp->nspec2D_ymax_crust_mantle)*sizeof(realw)),1201);
    print_CUDA_error_if_any(hipMemcpy(mp->d_jacobian2D_ymax_crust_mantle,jacobian2D_ymax_crust_mantle,
                                       NGLL2*(mp->nspec2D_ymax_crust_mantle)*sizeof(realw),hipMemcpyHostToDevice),1202);

    // boundary buffer
    if( (mp->simulation_type == 1 && mp->save_forward ) || (mp->simulation_type == 3) ){
      print_CUDA_error_if_any(hipMalloc((void**) &mp->d_absorb_ymax_crust_mantle,
                                         NDIM*NGLL2*(mp->nspec2D_ymax_crust_mantle)*sizeof(realw)),1202);
    }
  }


  // outer_core
  mp->nspec2D_xmin_outer_core = *nspec2D_xmin_outer_core;
  mp->nspec2D_xmax_outer_core = *nspec2D_xmax_outer_core;
  mp->nspec2D_ymin_outer_core = *nspec2D_ymin_outer_core;
  mp->nspec2D_ymax_outer_core = *nspec2D_ymax_outer_core;
  mp->nspec2D_zmin_outer_core = *nspec2D_zmin_outer_core;

  // vp
  size = NGLL3*(mp->NSPEC_OUTER_CORE);
  print_CUDA_error_if_any(hipMalloc((void**) &mp->d_vp_outer_core,
                                     size*sizeof(realw)),2201);
  print_CUDA_error_if_any(hipMemcpy(mp->d_vp_outer_core,vp_outer_core,
                                     size*sizeof(realw),hipMemcpyHostToDevice),2202);

  // ijk index arrays
  print_CUDA_error_if_any(hipMalloc((void**) &mp->d_nkmin_xi_outer_core,
                                     2*(*NSPEC2DMAX_XMIN_XMAX_OC)*sizeof(int)),1201);
  print_CUDA_error_if_any(hipMemcpy(mp->d_nkmin_xi_outer_core,nkmin_xi_outer_core,
                                     2*(*NSPEC2DMAX_XMIN_XMAX_OC)*sizeof(int),hipMemcpyHostToDevice),1202);

  print_CUDA_error_if_any(hipMalloc((void**) &mp->d_nkmin_eta_outer_core,
                                     2*(*NSPEC2DMAX_YMIN_YMAX_OC)*sizeof(int)),1201);
  print_CUDA_error_if_any(hipMemcpy(mp->d_nkmin_eta_outer_core,nkmin_eta_outer_core,
                                     2*(*NSPEC2DMAX_YMIN_YMAX_OC)*sizeof(int),hipMemcpyHostToDevice),1202);

  print_CUDA_error_if_any(hipMalloc((void**) &mp->d_njmin_outer_core,
                                     2*(*NSPEC2DMAX_XMIN_XMAX_OC)*sizeof(int)),1201);
  print_CUDA_error_if_any(hipMemcpy(mp->d_njmin_outer_core,njmin_outer_core,
                                     2*(*NSPEC2DMAX_XMIN_XMAX_OC)*sizeof(int),hipMemcpyHostToDevice),1202);

  print_CUDA_error_if_any(hipMalloc((void**) &mp->d_njmax_outer_core,
                                     2*(*NSPEC2DMAX_XMIN_XMAX_OC)*sizeof(int)),1201);
  print_CUDA_error_if_any(hipMemcpy(mp->d_njmax_outer_core,njmax_outer_core,
                                     2*(*NSPEC2DMAX_XMIN_XMAX_OC)*sizeof(int),hipMemcpyHostToDevice),1202);

  print_CUDA_error_if_any(hipMalloc((void**) &mp->d_nimin_outer_core,
                                     2*(*NSPEC2DMAX_YMIN_YMAX_OC)*sizeof(int)),1201);
  print_CUDA_error_if_any(hipMemcpy(mp->d_nimin_outer_core,nimin_outer_core,
                                     2*(*NSPEC2DMAX_YMIN_YMAX_OC)*sizeof(int),hipMemcpyHostToDevice),1202);

  print_CUDA_error_if_any(hipMalloc((void**) &mp->d_nimax_outer_core,
                                     2*(*NSPEC2DMAX_YMIN_YMAX_OC)*sizeof(int)),1201);
  print_CUDA_error_if_any(hipMemcpy(mp->d_nimax_outer_core,nimax_outer_core,
                                     2*(*NSPEC2DMAX_YMIN_YMAX_OC)*sizeof(int),hipMemcpyHostToDevice),1202);

  // xmin
  if( mp->nspec2D_xmin_outer_core > 0 ){
    print_CUDA_error_if_any(hipMalloc((void**) &mp->d_ibelm_xmin_outer_core,
                                       (mp->nspec2D_xmin_outer_core)*sizeof(int)),1201);
    print_CUDA_error_if_any(hipMemcpy(mp->d_ibelm_xmin_outer_core,ibelm_xmin_outer_core,
                                       (mp->nspec2D_xmin_outer_core)*sizeof(int),hipMemcpyHostToDevice),1202);

    print_CUDA_error_if_any(hipMalloc((void**) &mp->d_jacobian2D_xmin_outer_core,
                                       NGLL2*(mp->nspec2D_xmin_outer_core)*sizeof(realw)),1201);
    print_CUDA_error_if_any(hipMemcpy(mp->d_jacobian2D_xmin_outer_core,jacobian2D_xmin_outer_core,
                                       NGLL2*(mp->nspec2D_xmin_outer_core)*sizeof(realw),hipMemcpyHostToDevice),1202);

    // boundary buffer
    if( (mp->simulation_type == 1 && mp->save_forward ) || (mp->simulation_type == 3) ){
      print_CUDA_error_if_any(hipMalloc((void**) &mp->d_absorb_xmin_outer_core,
                                         NGLL2*(mp->nspec2D_xmin_outer_core)*sizeof(realw)),1202);
    }
  }

  // xmax
  if( mp->nspec2D_xmax_outer_core > 0 ){
    print_CUDA_error_if_any(hipMalloc((void**) &mp->d_ibelm_xmax_outer_core,
                                       (mp->nspec2D_xmax_outer_core)*sizeof(int)),1201);
    print_CUDA_error_if_any(hipMemcpy(mp->d_ibelm_xmax_outer_core,ibelm_xmax_outer_core,
                                       (mp->nspec2D_xmax_outer_core)*sizeof(int),hipMemcpyHostToDevice),1202);

    print_CUDA_error_if_any(hipMalloc((void**) &mp->d_jacobian2D_xmax_outer_core,
                                       NGLL2*(mp->nspec2D_xmax_outer_core)*sizeof(realw)),1201);
    print_CUDA_error_if_any(hipMemcpy(mp->d_jacobian2D_xmax_outer_core,jacobian2D_xmax_outer_core,
                                       NGLL2*(mp->nspec2D_xmax_outer_core)*sizeof(realw),hipMemcpyHostToDevice),1202);

    // boundary buffer
    if( (mp->simulation_type == 1 && mp->save_forward ) || (mp->simulation_type == 3) ){
      print_CUDA_error_if_any(hipMalloc((void**) &mp->d_absorb_xmax_outer_core,
                                         NGLL2*(mp->nspec2D_xmax_outer_core)*sizeof(realw)),1202);
    }
  }

  // ymin
  if( mp->nspec2D_ymin_outer_core > 0 ){
    print_CUDA_error_if_any(hipMalloc((void**) &mp->d_ibelm_ymin_outer_core,
                                       (mp->nspec2D_ymin_outer_core)*sizeof(int)),1201);
    print_CUDA_error_if_any(hipMemcpy(mp->d_ibelm_ymin_outer_core,ibelm_ymin_outer_core,
                                       (mp->nspec2D_ymin_outer_core)*sizeof(int),hipMemcpyHostToDevice),1202);

    print_CUDA_error_if_any(hipMalloc((void**) &mp->d_jacobian2D_ymin_outer_core,
                                       NGLL2*(mp->nspec2D_ymin_outer_core)*sizeof(realw)),1201);
    print_CUDA_error_if_any(hipMemcpy(mp->d_jacobian2D_ymin_outer_core,jacobian2D_ymin_outer_core,
                                       NGLL2*(mp->nspec2D_ymin_outer_core)*sizeof(realw),hipMemcpyHostToDevice),1202);

    // boundary buffer
    if( (mp->simulation_type == 1 && mp->save_forward ) || (mp->simulation_type == 3) ){
      print_CUDA_error_if_any(hipMalloc((void**) &mp->d_absorb_ymin_outer_core,
                                         NGLL2*(mp->nspec2D_ymin_outer_core)*sizeof(realw)),1202);
    }
  }

  // ymax
  if( mp->nspec2D_ymax_outer_core > 0 ){
    print_CUDA_error_if_any(hipMalloc((void**) &mp->d_ibelm_ymax_outer_core,
                                       (mp->nspec2D_ymax_outer_core)*sizeof(int)),1201);
    print_CUDA_error_if_any(hipMemcpy(mp->d_ibelm_ymax_outer_core,ibelm_ymax_outer_core,
                                       (mp->nspec2D_ymax_outer_core)*sizeof(int),hipMemcpyHostToDevice),1202);

    print_CUDA_error_if_any(hipMalloc((void**) &mp->d_jacobian2D_ymax_outer_core,
                                       NGLL2*(mp->nspec2D_ymax_outer_core)*sizeof(realw)),1201);
    print_CUDA_error_if_any(hipMemcpy(mp->d_jacobian2D_ymax_outer_core,jacobian2D_ymax_outer_core,
                                       NGLL2*(mp->nspec2D_ymax_outer_core)*sizeof(realw),hipMemcpyHostToDevice),1202);

    // boundary buffer
    if( (mp->simulation_type == 1 && mp->save_forward ) || (mp->simulation_type == 3) ){
      print_CUDA_error_if_any(hipMalloc((void**) &mp->d_absorb_ymax_outer_core,
                                         NGLL2*(mp->nspec2D_ymax_outer_core)*sizeof(realw)),1202);
    }
  }

  // zmin
  if( mp->nspec2D_zmin_outer_core > 0 ){
    print_CUDA_error_if_any(hipMalloc((void**) &mp->d_ibelm_zmin_outer_core,
                                       (mp->nspec2D_zmin_outer_core)*sizeof(int)),1201);
    print_CUDA_error_if_any(hipMemcpy(mp->d_ibelm_zmin_outer_core,ibelm_bottom_outer_core,
                                       (mp->nspec2D_zmin_outer_core)*sizeof(int),hipMemcpyHostToDevice),1202);

    print_CUDA_error_if_any(hipMalloc((void**) &mp->d_jacobian2D_zmin_outer_core,
                                       NGLL2*(mp->nspec2D_zmin_outer_core)*sizeof(realw)),1201);
    print_CUDA_error_if_any(hipMemcpy(mp->d_jacobian2D_zmin_outer_core,jacobian2D_bottom_outer_core,
                                       NGLL2*(mp->nspec2D_zmin_outer_core)*sizeof(realw),hipMemcpyHostToDevice),1202);

    // boundary buffer
    if( (mp->simulation_type == 1 && mp->save_forward ) || (mp->simulation_type == 3) ){
      print_CUDA_error_if_any(hipMalloc((void**) &mp->d_absorb_zmin_outer_core,
                                         NGLL2*(mp->nspec2D_zmin_outer_core)*sizeof(realw)),1202);
    }
  }

}

/* ----------------------------------------------------------------------------------------------- */

// MPI interfaces

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(prepare_mpi_buffers_device,
              PREPARE_MPI_BUFFERS_DEVICE)(long* Mesh_pointer_f,
                                          int* num_interfaces_crust_mantle,
                                          int* max_nibool_interfaces_crust_mantle,
                                          int* nibool_interfaces_crust_mantle,
                                          int* ibool_interfaces_crust_mantle,
                                          int* num_interfaces_inner_core,
                                          int* max_nibool_interfaces_inner_core,
                                          int* nibool_interfaces_inner_core,
                                          int* ibool_interfaces_inner_core,
                                          int* num_interfaces_outer_core,
                                          int* max_nibool_interfaces_outer_core,
                                          int* nibool_interfaces_outer_core,
                                          int* ibool_interfaces_outer_core
                                          ){

  TRACE("prepare_mpi_buffers_device");

  Mesh* mp = (Mesh*)(*Mesh_pointer_f);

  // prepares interprocess-edge exchange information

  // crust/mantle mesh
  mp->num_interfaces_crust_mantle = *num_interfaces_crust_mantle;
  mp->max_nibool_interfaces_crust_mantle = *max_nibool_interfaces_crust_mantle;
  if( mp->num_interfaces_crust_mantle > 0 ){
    // number of ibool entries array
    print_CUDA_error_if_any(hipMalloc((void**) &mp->d_nibool_interfaces_crust_mantle,
                                       (mp->num_interfaces_crust_mantle)*sizeof(int)),1201);
    print_CUDA_error_if_any(hipMemcpy(mp->d_nibool_interfaces_crust_mantle,nibool_interfaces_crust_mantle,
                                       (mp->num_interfaces_crust_mantle)*sizeof(int),hipMemcpyHostToDevice),1202);
    // ibool entries (iglob indices) values on interface
    print_CUDA_error_if_any(hipMalloc((void**) &mp->d_ibool_interfaces_crust_mantle,
                                       (mp->num_interfaces_crust_mantle)*(mp->max_nibool_interfaces_crust_mantle)*sizeof(int)),1203);
    print_CUDA_error_if_any(hipMemcpy(mp->d_ibool_interfaces_crust_mantle,ibool_interfaces_crust_mantle,
                                       (mp->num_interfaces_crust_mantle)*(mp->max_nibool_interfaces_crust_mantle)*sizeof(int),
                                       hipMemcpyHostToDevice),1204);
    // allocates mpi buffer for exchange with cpu
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_send_accel_buffer_crust_mantle),
                                       3*(mp->max_nibool_interfaces_crust_mantle)*(mp->num_interfaces_crust_mantle)*sizeof(realw)),4004);
  }

  // inner core mesh
  mp->num_interfaces_inner_core = *num_interfaces_inner_core;
  mp->max_nibool_interfaces_inner_core = *max_nibool_interfaces_inner_core;
  if( mp->num_interfaces_inner_core > 0 ){
    // number of ibool entries array
    print_CUDA_error_if_any(hipMalloc((void**) &mp->d_nibool_interfaces_inner_core,
                                       (mp->num_interfaces_inner_core)*sizeof(int)),1201);
    print_CUDA_error_if_any(hipMemcpy(mp->d_nibool_interfaces_inner_core,nibool_interfaces_inner_core,
                                       (mp->num_interfaces_inner_core)*sizeof(int),hipMemcpyHostToDevice),1202);
    // ibool entries (iglob indices) values on interface
    print_CUDA_error_if_any(hipMalloc((void**) &mp->d_ibool_interfaces_inner_core,
                                       (mp->num_interfaces_inner_core)*(mp->max_nibool_interfaces_inner_core)*sizeof(int)),1203);
    print_CUDA_error_if_any(hipMemcpy(mp->d_ibool_interfaces_inner_core,ibool_interfaces_inner_core,
                                       (mp->num_interfaces_inner_core)*(mp->max_nibool_interfaces_inner_core)*sizeof(int),
                                       hipMemcpyHostToDevice),1204);
    // allocates mpi buffer for exchange with cpu
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_send_accel_buffer_inner_core),
                                       3*(mp->max_nibool_interfaces_inner_core)*(mp->num_interfaces_inner_core)*sizeof(realw)),4004);
  }

  // outer core mesh
  // note: uses only scalar wavefield arrays
  mp->num_interfaces_outer_core = *num_interfaces_outer_core;
  mp->max_nibool_interfaces_outer_core = *max_nibool_interfaces_outer_core;
  if( mp->num_interfaces_outer_core > 0 ){
    // number of ibool entries array
    print_CUDA_error_if_any(hipMalloc((void**) &mp->d_nibool_interfaces_outer_core,
                                       (mp->num_interfaces_outer_core)*sizeof(int)),1201);
    print_CUDA_error_if_any(hipMemcpy(mp->d_nibool_interfaces_outer_core,nibool_interfaces_outer_core,
                                       (mp->num_interfaces_outer_core)*sizeof(int),hipMemcpyHostToDevice),1202);
    // ibool entries (iglob indices) values on interface
    print_CUDA_error_if_any(hipMalloc((void**) &mp->d_ibool_interfaces_outer_core,
                                       (mp->num_interfaces_outer_core)*(mp->max_nibool_interfaces_outer_core)*sizeof(int)),1203);
    print_CUDA_error_if_any(hipMemcpy(mp->d_ibool_interfaces_outer_core,ibool_interfaces_outer_core,
                                       (mp->num_interfaces_outer_core)*(mp->max_nibool_interfaces_outer_core)*sizeof(int),
                                       hipMemcpyHostToDevice),1204);
    // allocates mpi buffer for exchange with cpu
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_send_accel_buffer_outer_core),
                                       (mp->max_nibool_interfaces_outer_core)*(mp->num_interfaces_outer_core)*sizeof(realw)),4004);
  }


}

/* ----------------------------------------------------------------------------------------------- */

// for NOISE simulations

/* ----------------------------------------------------------------------------------------------- */


extern "C"
void FC_FUNC_(prepare_fields_noise_device,
              PREPARE_FIELDS_NOISE_DEVICE)(long* Mesh_pointer_f,
                                           int* nspec_top,
                                           int* ibelm_top_crust_mantle,
                                           int* NSTEP,
                                           realw* noise_sourcearray,
                                           realw* normal_x_noise,
                                           realw* normal_y_noise,
                                           realw* normal_z_noise,
                                           realw* mask_noise,
                                           realw* jacobian2D_top_crust_mantle) {

  TRACE("prepare_fields_noise_device");

  Mesh* mp = (Mesh*)(*Mesh_pointer_f);

  // free surface
  mp->nspec_top = *nspec_top;
  if( mp->nspec_top > 0 ){
    print_CUDA_error_if_any(hipMalloc((void**) &mp->d_ibelm_top_crust_mantle,
                                       mp->nspec_top*sizeof(int)),7001);
    print_CUDA_error_if_any(hipMemcpy(mp->d_ibelm_top_crust_mantle,ibelm_top_crust_mantle,
                                       mp->nspec_top*sizeof(int),hipMemcpyHostToDevice),7002);

    // alloc storage for the surface buffer to be copied
    print_CUDA_error_if_any(hipMalloc((void**) &mp->d_noise_surface_movie,
                                       NDIM*NGLL2*(mp->nspec_top)*sizeof(realw)),7005);
  }else{
    // for global mesh: each crust/mantle slice should have at top a free surface
    exit_on_cuda_error("prepare_fields_noise_device nspec_top not properly initialized");
  }


  // prepares noise source array
  if( mp->noise_tomography == 1 ){
    print_CUDA_error_if_any(hipMalloc((void**)&mp->d_noise_sourcearray,
                                       NDIM*NGLL3*(*NSTEP)*sizeof(realw)),7101);
    print_CUDA_error_if_any(hipMemcpy(mp->d_noise_sourcearray,noise_sourcearray,
                                       NDIM*NGLL3*(*NSTEP)*sizeof(realw),hipMemcpyHostToDevice),7102);
  }

  // prepares noise directions
  if( mp->noise_tomography > 1 ){
    int nface_size = NGLL2*(mp->nspec_top);
    // allocates memory on GPU
    print_CUDA_error_if_any(hipMalloc((void**)&mp->d_normal_x_noise,
                                       nface_size*sizeof(realw)),7301);
    print_CUDA_error_if_any(hipMemcpy(mp->d_normal_x_noise, normal_x_noise,
                                       nface_size*sizeof(realw),hipMemcpyHostToDevice),7306);

    print_CUDA_error_if_any(hipMalloc((void**)&mp->d_normal_y_noise,
                                       nface_size*sizeof(realw)),7302);
    print_CUDA_error_if_any(hipMemcpy(mp->d_normal_y_noise, normal_y_noise,
                                       nface_size*sizeof(realw),hipMemcpyHostToDevice),7307);

    print_CUDA_error_if_any(hipMalloc((void**)&mp->d_normal_z_noise,
                                       nface_size*sizeof(realw)),7303);
    print_CUDA_error_if_any(hipMemcpy(mp->d_normal_z_noise, normal_z_noise,
                                       nface_size*sizeof(realw),hipMemcpyHostToDevice),7308);

    print_CUDA_error_if_any(hipMalloc((void**)&mp->d_mask_noise,
                                       nface_size*sizeof(realw)),7304);
    print_CUDA_error_if_any(hipMemcpy(mp->d_mask_noise, mask_noise,
                                       nface_size*sizeof(realw),hipMemcpyHostToDevice),7309);

    print_CUDA_error_if_any(hipMalloc((void**)&mp->d_jacobian2D_top_crust_mantle,
                                       nface_size*sizeof(realw)),7305);
    print_CUDA_error_if_any(hipMemcpy(mp->d_jacobian2D_top_crust_mantle, jacobian2D_top_crust_mantle,
                                       nface_size*sizeof(realw),hipMemcpyHostToDevice),7310);
  }

  // prepares noise strength kernel
  if( mp->noise_tomography == 3 ){
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_Sigma_kl),
                                       NGLL3*(mp->NSPEC_CRUST_MANTLE)*sizeof(realw)),7401);
    // initializes kernel values to zero
    print_CUDA_error_if_any(hipMemset(mp->d_Sigma_kl,0,
                                       NGLL3*mp->NSPEC_CRUST_MANTLE*sizeof(realw)),7403);

  }

#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
  exit_on_cuda_error("prepare_fields_noise_device");
#endif
}


/* ----------------------------------------------------------------------------------------------- */

// Earth regions

// CRUST / MANTLE

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(prepare_crust_mantle_device,
             PREPARE_CRUST_MANTLE_DEVICE)(long* Mesh_pointer_f,
             realw* h_xix, realw* h_xiy, realw* h_xiz,
             realw* h_etax, realw* h_etay, realw* h_etaz,
             realw* h_gammax, realw* h_gammay, realw* h_gammaz,
             realw* h_rho,
             realw* h_kappav, realw* h_muv,
             realw* h_kappah, realw* h_muh,
             realw* h_eta_aniso,
             realw* h_rmassx,
             realw* h_rmassy,
             realw* h_rmassz,
             realw* h_normal_top_crust_mantle,
             int* h_ibelm_top_crust_mantle,
             int* h_ibelm_bottom_crust_mantle,
             int* h_ibool,
             realw* h_xstore, realw* h_ystore, realw* h_zstore,
             int* h_ispec_is_tiso,
             realw *c11store,realw *c12store,realw *c13store,
             realw *c14store,realw *c15store,realw *c16store,
             realw *c22store,realw *c23store,realw *c24store,
             realw *c25store,realw *c26store,realw *c33store,
             realw *c34store,realw *c35store,realw *c36store,
             realw *c44store,realw *c45store,realw *c46store,
             realw *c55store,realw *c56store,realw *c66store,
             int* num_phase_ispec,
             int* phase_ispec_inner,
             int* nspec_outer,
             int* nspec_inner,
             int* NSPEC2D_TOP_CM,
	     int* NSPEC2D_BOTTOM_CM,
	     int* NCHUNKS_VAL
             ) {

  TRACE("prepare_crust_mantle_device");

  Mesh* mp = (Mesh*)(*Mesh_pointer_f);

  /* Assuming NGLLX=5. Padded is then 128 (5^3+3) */
  int size_padded = NGLL3_PADDED * (mp->NSPEC_CRUST_MANTLE);
  int size_glob = mp->NGLOB_CRUST_MANTLE;

  // mesh
  print_CUDA_error_if_any(hipMalloc((void**) &mp->d_xix_crust_mantle, size_padded*sizeof(realw)),1001);
  print_CUDA_error_if_any(hipMalloc((void**) &mp->d_xiy_crust_mantle, size_padded*sizeof(realw)),1002);
  print_CUDA_error_if_any(hipMalloc((void**) &mp->d_xiz_crust_mantle, size_padded*sizeof(realw)),1003);
  print_CUDA_error_if_any(hipMalloc((void**) &mp->d_etax_crust_mantle, size_padded*sizeof(realw)),1004);
  print_CUDA_error_if_any(hipMalloc((void**) &mp->d_etay_crust_mantle, size_padded*sizeof(realw)),1005);
  print_CUDA_error_if_any(hipMalloc((void**) &mp->d_etaz_crust_mantle, size_padded*sizeof(realw)),1006);
  print_CUDA_error_if_any(hipMalloc((void**) &mp->d_gammax_crust_mantle, size_padded*sizeof(realw)),1007);
  print_CUDA_error_if_any(hipMalloc((void**) &mp->d_gammay_crust_mantle, size_padded*sizeof(realw)),1008);
  print_CUDA_error_if_any(hipMalloc((void**) &mp->d_gammaz_crust_mantle, size_padded*sizeof(realw)),1009);
  // muv needed in case for attenuation (only Q_mu shear attenuation)
  print_CUDA_error_if_any(hipMalloc((void**) &mp->d_muvstore_crust_mantle, size_padded*sizeof(realw)),1011);

  // transfer constant element data with padding
  for(int i=0;i < mp->NSPEC_CRUST_MANTLE;i++) {
    print_CUDA_error_if_any(hipMemcpy(mp->d_xix_crust_mantle + i*NGLL3_PADDED, &h_xix[i*NGLL3],
                                       NGLL3*sizeof(realw),hipMemcpyHostToDevice),1501);
    print_CUDA_error_if_any(hipMemcpy(mp->d_xiy_crust_mantle+i*NGLL3_PADDED,   &h_xiy[i*NGLL3],
                                       NGLL3*sizeof(realw),hipMemcpyHostToDevice),1502);
    print_CUDA_error_if_any(hipMemcpy(mp->d_xiz_crust_mantle+i*NGLL3_PADDED,   &h_xiz[i*NGLL3],
                                       NGLL3*sizeof(realw),hipMemcpyHostToDevice),1503);
    print_CUDA_error_if_any(hipMemcpy(mp->d_etax_crust_mantle+i*NGLL3_PADDED,  &h_etax[i*NGLL3],
                                       NGLL3*sizeof(realw),hipMemcpyHostToDevice),1504);
    print_CUDA_error_if_any(hipMemcpy(mp->d_etay_crust_mantle+i*NGLL3_PADDED,  &h_etay[i*NGLL3],
                                       NGLL3*sizeof(realw),hipMemcpyHostToDevice),1505);
    print_CUDA_error_if_any(hipMemcpy(mp->d_etaz_crust_mantle+i*NGLL3_PADDED,  &h_etaz[i*NGLL3],
                                       NGLL3*sizeof(realw),hipMemcpyHostToDevice),1506);
    print_CUDA_error_if_any(hipMemcpy(mp->d_gammax_crust_mantle+i*NGLL3_PADDED,&h_gammax[i*NGLL3],
                                       NGLL3*sizeof(realw),hipMemcpyHostToDevice),1507);
    print_CUDA_error_if_any(hipMemcpy(mp->d_gammay_crust_mantle+i*NGLL3_PADDED,&h_gammay[i*NGLL3],
                                       NGLL3*sizeof(realw),hipMemcpyHostToDevice),1508);
    print_CUDA_error_if_any(hipMemcpy(mp->d_gammaz_crust_mantle+i*NGLL3_PADDED,&h_gammaz[i*NGLL3],
                                       NGLL3*sizeof(realw),hipMemcpyHostToDevice),1509);
    // muvstore
    print_CUDA_error_if_any(hipMemcpy(mp->d_muvstore_crust_mantle+i*NGLL3_PADDED,   &h_muv[i*NGLL3],
                                       NGLL3*sizeof(realw),hipMemcpyHostToDevice),1511);
  }

  // global indexing
  print_CUDA_error_if_any(hipMalloc((void**) &mp->d_ibool_crust_mantle, size_padded*sizeof(int)),1021);
  print_CUDA_error_if_any(hipMemcpy(mp->d_ibool_crust_mantle, h_ibool,
                                     NGLL3*(mp->NSPEC_CRUST_MANTLE)*sizeof(int),hipMemcpyHostToDevice),1022);

  // transverse isotropic elements
  // only needed if not anisotropic 3D mantle
  if( ! mp->anisotropic_3D_mantle ){
    // no anisotropy

    // transverse isotropy flag
    print_CUDA_error_if_any(hipMalloc((void**) &mp->d_ispec_is_tiso_crust_mantle, (mp->NSPEC_CRUST_MANTLE)*sizeof(int)),1025);
    print_CUDA_error_if_any(hipMemcpy(mp->d_ispec_is_tiso_crust_mantle, h_ispec_is_tiso,
                                       (mp->NSPEC_CRUST_MANTLE)*sizeof(int),hipMemcpyHostToDevice),1025);

    // kappavstore, kappahstore
    print_CUDA_error_if_any(hipMalloc((void**) &mp->d_kappavstore_crust_mantle, size_padded*sizeof(realw)),1010);
    print_CUDA_error_if_any(hipMalloc((void**) &mp->d_kappahstore_crust_mantle, size_padded*sizeof(realw)),1010);
    // muhstore
    print_CUDA_error_if_any(hipMalloc((void**) &mp->d_muhstore_crust_mantle, size_padded*sizeof(realw)),1010);
    // eta_anisostore
    print_CUDA_error_if_any(hipMalloc((void**) &mp->d_eta_anisostore_crust_mantle, size_padded*sizeof(realw)),1010);

    // transfer with padding
    for(int i=0;i < mp->NSPEC_CRUST_MANTLE;i++) {
      print_CUDA_error_if_any(hipMemcpy(mp->d_kappavstore_crust_mantle+i*NGLL3_PADDED,&h_kappav[i*NGLL3],
                                       NGLL3*sizeof(realw),hipMemcpyHostToDevice),1510);
      print_CUDA_error_if_any(hipMemcpy(mp->d_kappahstore_crust_mantle+i*NGLL3_PADDED,&h_kappah[i*NGLL3],
                                         NGLL3*sizeof(realw),hipMemcpyHostToDevice),1510);
      print_CUDA_error_if_any(hipMemcpy(mp->d_muhstore_crust_mantle+i*NGLL3_PADDED,&h_muh[i*NGLL3],
                                         NGLL3*sizeof(realw),hipMemcpyHostToDevice),1510);
      print_CUDA_error_if_any(hipMemcpy(mp->d_eta_anisostore_crust_mantle+i*NGLL3_PADDED,&h_eta_aniso[i*NGLL3],
                                         NGLL3*sizeof(realw),hipMemcpyHostToDevice),1510);
    }
  }else{
    // anisotropic 3D mantle

    // allocates memory on GPU
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_c11store_crust_mantle),
                                       size_padded*sizeof(realw)),4700);
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_c12store_crust_mantle),
                                       size_padded*sizeof(realw)),4701);
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_c13store_crust_mantle),
                                       size_padded*sizeof(realw)),4702);
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_c14store_crust_mantle),
                                       size_padded*sizeof(realw)),4703);
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_c15store_crust_mantle),
                                       size_padded*sizeof(realw)),4704);
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_c16store_crust_mantle),
                                       size_padded*sizeof(realw)),4705);
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_c22store_crust_mantle),
                                       size_padded*sizeof(realw)),4706);
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_c23store_crust_mantle),
                                       size_padded*sizeof(realw)),4707);
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_c24store_crust_mantle),
                                       size_padded*sizeof(realw)),4708);
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_c25store_crust_mantle),
                                       size_padded*sizeof(realw)),4709);
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_c26store_crust_mantle),
                                       size_padded*sizeof(realw)),4710);
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_c33store_crust_mantle),
                                       size_padded*sizeof(realw)),4711);
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_c34store_crust_mantle),
                                       size_padded*sizeof(realw)),4712);
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_c35store_crust_mantle),
                                       size_padded*sizeof(realw)),4713);
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_c36store_crust_mantle),
                                       size_padded*sizeof(realw)),4714);
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_c44store_crust_mantle),
                                       size_padded*sizeof(realw)),4715);
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_c45store_crust_mantle),
                                       size_padded*sizeof(realw)),4716);
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_c46store_crust_mantle),
                                       size_padded*sizeof(realw)),4717);
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_c55store_crust_mantle),
                                       size_padded*sizeof(realw)),4718);
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_c56store_crust_mantle),
                                       size_padded*sizeof(realw)),4719);
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_c66store_crust_mantle),
                                       size_padded*sizeof(realw)),4720);

    // transfer constant element data with padding
    for(int i=0;i < mp->NSPEC_CRUST_MANTLE;i++) {
      print_CUDA_error_if_any(hipMemcpy(mp->d_c11store_crust_mantle + i*NGLL3_PADDED, &c11store[i*NGLL3],
                                         NGLL3*sizeof(realw),hipMemcpyHostToDevice),4800);
      print_CUDA_error_if_any(hipMemcpy(mp->d_c12store_crust_mantle + i*NGLL3_PADDED, &c12store[i*NGLL3],
                                         NGLL3*sizeof(realw),hipMemcpyHostToDevice),4801);
      print_CUDA_error_if_any(hipMemcpy(mp->d_c13store_crust_mantle + i*NGLL3_PADDED, &c13store[i*NGLL3],
                                         NGLL3*sizeof(realw),hipMemcpyHostToDevice),4802);
      print_CUDA_error_if_any(hipMemcpy(mp->d_c14store_crust_mantle + i*NGLL3_PADDED, &c14store[i*NGLL3],
                                         NGLL3*sizeof(realw),hipMemcpyHostToDevice),4803);
      print_CUDA_error_if_any(hipMemcpy(mp->d_c15store_crust_mantle + i*NGLL3_PADDED, &c15store[i*NGLL3],
                                         NGLL3*sizeof(realw),hipMemcpyHostToDevice),4804);
      print_CUDA_error_if_any(hipMemcpy(mp->d_c16store_crust_mantle + i*NGLL3_PADDED, &c16store[i*NGLL3],
                                         NGLL3*sizeof(realw),hipMemcpyHostToDevice),4805);
      print_CUDA_error_if_any(hipMemcpy(mp->d_c22store_crust_mantle + i*NGLL3_PADDED, &c22store[i*NGLL3],
                                         NGLL3*sizeof(realw),hipMemcpyHostToDevice),4806);
      print_CUDA_error_if_any(hipMemcpy(mp->d_c23store_crust_mantle + i*NGLL3_PADDED, &c23store[i*NGLL3],
                                         NGLL3*sizeof(realw),hipMemcpyHostToDevice),4807);
      print_CUDA_error_if_any(hipMemcpy(mp->d_c24store_crust_mantle + i*NGLL3_PADDED, &c24store[i*NGLL3],
                                         NGLL3*sizeof(realw),hipMemcpyHostToDevice),4808);
      print_CUDA_error_if_any(hipMemcpy(mp->d_c25store_crust_mantle + i*NGLL3_PADDED, &c25store[i*NGLL3],
                                         NGLL3*sizeof(realw),hipMemcpyHostToDevice),4809);
      print_CUDA_error_if_any(hipMemcpy(mp->d_c26store_crust_mantle + i*NGLL3_PADDED, &c26store[i*NGLL3],
                                         NGLL3*sizeof(realw),hipMemcpyHostToDevice),4810);
      print_CUDA_error_if_any(hipMemcpy(mp->d_c33store_crust_mantle + i*NGLL3_PADDED, &c33store[i*NGLL3],
                                         NGLL3*sizeof(realw),hipMemcpyHostToDevice),4811);
      print_CUDA_error_if_any(hipMemcpy(mp->d_c34store_crust_mantle + i*NGLL3_PADDED, &c34store[i*NGLL3],
                                         NGLL3*sizeof(realw),hipMemcpyHostToDevice),4812);
      print_CUDA_error_if_any(hipMemcpy(mp->d_c35store_crust_mantle + i*NGLL3_PADDED, &c35store[i*NGLL3],
                                         NGLL3*sizeof(realw),hipMemcpyHostToDevice),4813);
      print_CUDA_error_if_any(hipMemcpy(mp->d_c36store_crust_mantle + i*NGLL3_PADDED, &c36store[i*NGLL3],
                                         NGLL3*sizeof(realw),hipMemcpyHostToDevice),4814);
      print_CUDA_error_if_any(hipMemcpy(mp->d_c44store_crust_mantle + i*NGLL3_PADDED, &c44store[i*NGLL3],
                                         NGLL3*sizeof(realw),hipMemcpyHostToDevice),4815);
      print_CUDA_error_if_any(hipMemcpy(mp->d_c45store_crust_mantle + i*NGLL3_PADDED, &c45store[i*NGLL3],
                                         NGLL3*sizeof(realw),hipMemcpyHostToDevice),4816);
      print_CUDA_error_if_any(hipMemcpy(mp->d_c46store_crust_mantle + i*NGLL3_PADDED, &c46store[i*NGLL3],
                                         NGLL3*sizeof(realw),hipMemcpyHostToDevice),4817);
      print_CUDA_error_if_any(hipMemcpy(mp->d_c55store_crust_mantle + i*NGLL3_PADDED, &c55store[i*NGLL3],
                                         NGLL3*sizeof(realw),hipMemcpyHostToDevice),4818);
      print_CUDA_error_if_any(hipMemcpy(mp->d_c56store_crust_mantle + i*NGLL3_PADDED, &c56store[i*NGLL3],
                                         NGLL3*sizeof(realw),hipMemcpyHostToDevice),4819);
      print_CUDA_error_if_any(hipMemcpy(mp->d_c66store_crust_mantle + i*NGLL3_PADDED, &c66store[i*NGLL3],
                                         NGLL3*sizeof(realw),hipMemcpyHostToDevice),4820);
    }
  }

  // needed for boundary kernel calculations
  if( mp->simulation_type == 3 && mp->save_boundary_mesh ){
    print_CUDA_error_if_any(hipMalloc((void**) &mp->d_rhostore_crust_mantle, size_padded*sizeof(realw)),1010);
    for(int i=0;i < mp->NSPEC_CRUST_MANTLE;i++) {
      print_CUDA_error_if_any(hipMemcpy(mp->d_rhostore_crust_mantle+i*NGLL3_PADDED, &h_rho[i*NGLL3],
                                         NGLL3*sizeof(realw),hipMemcpyHostToDevice),2106);
    }
  }

  // mesh locations
  // ystore & zstore needed for tiso elements
  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_ystore_crust_mantle),sizeof(realw)*size_glob),2005);
  print_CUDA_error_if_any(hipMemcpy(mp->d_ystore_crust_mantle,h_ystore,
                                     sizeof(realw)*size_glob,hipMemcpyHostToDevice),2100);
  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_zstore_crust_mantle),sizeof(realw)*size_glob),2005);
  print_CUDA_error_if_any(hipMemcpy(mp->d_zstore_crust_mantle,h_zstore,
                                     sizeof(realw)*size_glob,hipMemcpyHostToDevice),2100);

  // xstore only needed when gravity is on
  if( mp->gravity ){
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_xstore_crust_mantle),sizeof(realw)*size_glob),2005);
    print_CUDA_error_if_any(hipMemcpy(mp->d_xstore_crust_mantle,h_xstore,
                                     sizeof(realw)*size_glob,hipMemcpyHostToDevice),2100);
  }

  // inner/outer elements
  mp->num_phase_ispec_crust_mantle = *num_phase_ispec;
  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_phase_ispec_inner_crust_mantle),
                                     mp->num_phase_ispec_crust_mantle*2*sizeof(int)),2008);
  print_CUDA_error_if_any(hipMemcpy(mp->d_phase_ispec_inner_crust_mantle,phase_ispec_inner,
                                     mp->num_phase_ispec_crust_mantle*2*sizeof(int),hipMemcpyHostToDevice),2101);

  mp->nspec_outer_crust_mantle = *nspec_outer;
  mp->nspec_inner_crust_mantle = *nspec_inner;

  // CMB/ocean coupling
  mp->nspec2D_top_crust_mantle = *NSPEC2D_TOP_CM;
  mp->nspec2D_bottom_crust_mantle = *NSPEC2D_BOTTOM_CM;
  int size_tcm = NGLL2*(mp->nspec2D_top_crust_mantle);

  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_normal_top_crust_mantle),sizeof(realw)*NDIM*size_tcm),40020);
  print_CUDA_error_if_any(hipMemcpy(mp->d_normal_top_crust_mantle,h_normal_top_crust_mantle,sizeof(realw)*NDIM*size_tcm,hipMemcpyHostToDevice),40030);

  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_ibelm_top_crust_mantle),sizeof(int)*(mp->nspec2D_top_crust_mantle)),40021);
  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_ibelm_bottom_crust_mantle),sizeof(int)*(mp->nspec2D_bottom_crust_mantle)),40021);
  print_CUDA_error_if_any(hipMemcpy(mp->d_ibelm_top_crust_mantle,h_ibelm_top_crust_mantle,sizeof(int)*(mp->nspec2D_top_crust_mantle),hipMemcpyHostToDevice),40031);
  print_CUDA_error_if_any(hipMemcpy(mp->d_ibelm_bottom_crust_mantle,h_ibelm_bottom_crust_mantle,sizeof(int)*(mp->nspec2D_bottom_crust_mantle),hipMemcpyHostToDevice),40031);

  // wavefield
  int size = NDIM * mp->NGLOB_CRUST_MANTLE;

  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_displ_crust_mantle),sizeof(realw)*size),4001);
  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_veloc_crust_mantle),sizeof(realw)*size),4002);
  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_accel_crust_mantle),sizeof(realw)*size),4003);
  // backward/reconstructed wavefield
  if( mp->simulation_type == 3 ){
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_b_displ_crust_mantle),sizeof(realw)*size),4001);
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_b_veloc_crust_mantle),sizeof(realw)*size),4002);
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_b_accel_crust_mantle),sizeof(realw)*size),4003);
  }

  // mass matrices
  if( *NCHUNKS_VAL != 6 && mp->absorbing_conditions){
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_rmassx_crust_mantle),sizeof(realw)*size_glob),2005);
    print_CUDA_error_if_any(hipMemcpy(mp->d_rmassx_crust_mantle,h_rmassx,
				       sizeof(realw)*size_glob,hipMemcpyHostToDevice),2100);
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_rmassy_crust_mantle),sizeof(realw)*size_glob),2005);
    print_CUDA_error_if_any(hipMemcpy(mp->d_rmassy_crust_mantle,h_rmassy,
				       sizeof(realw)*size_glob,hipMemcpyHostToDevice),2100);
  }

  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_rmassz_crust_mantle),sizeof(realw)*size_glob),2005);
  print_CUDA_error_if_any(hipMemcpy(mp->d_rmassz_crust_mantle,h_rmassz,
                                     sizeof(realw)*size_glob,hipMemcpyHostToDevice),2100);

  // kernels
  if( mp->simulation_type == 3 ){

    size = NGLL3*(mp->NSPEC_CRUST_MANTLE);

    // density kernel
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_rho_kl_crust_mantle),
                                       size*sizeof(realw)),5204);
    // initializes kernel values to zero
    print_CUDA_error_if_any(hipMemset(mp->d_rho_kl_crust_mantle,0,size*sizeof(realw)),5207);

    if( ! mp->anisotropic_kl){
      // isotropic kernels
      print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_alpha_kl_crust_mantle),
                                         size*sizeof(realw)),5205);
      print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_beta_kl_crust_mantle),
                                         size*sizeof(realw)),5206);
      print_CUDA_error_if_any(hipMemset(mp->d_alpha_kl_crust_mantle,0,size*sizeof(realw)),5208);
      print_CUDA_error_if_any(hipMemset(mp->d_beta_kl_crust_mantle,0,size*sizeof(realw)),5209);
    }else{
      // anisotropic kernels
      print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_cijkl_kl_crust_mantle),
                                         21*size*sizeof(realw)),5206);
      print_CUDA_error_if_any(hipMemset(mp->d_cijkl_kl_crust_mantle,0,size*sizeof(realw)),5209);
    }

    // preconditioner
    if( mp->approximate_hess_kl ){
      print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_hess_kl_crust_mantle),
                                         size*sizeof(realw)),3030);
      print_CUDA_error_if_any(hipMemset(mp->d_hess_kl_crust_mantle,0,size*sizeof(realw)),3031);
    }
  }

#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
  exit_on_cuda_error("prepare_crust_mantle_device");
#endif
}


/* ----------------------------------------------------------------------------------------------- */

// OUTER CORE

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(prepare_outer_core_device,
              PREPARE_OUTER_CORE_DEVICE)(long* Mesh_pointer_f,
                                         realw* h_xix, realw* h_xiy, realw* h_xiz,
                                         realw* h_etax, realw* h_etay, realw* h_etaz,
                                         realw* h_gammax, realw* h_gammay, realw* h_gammaz,
                                         realw* h_rho, realw* h_kappav,
                                         realw* h_rmass,
					 realw* h_normal_top_outer_core,
					 realw* h_normal_bottom_outer_core,
					 realw* h_jacobian2D_top_outer_core,
					 realw* h_jacobian2D_bottom_outer_core,
					 int* h_ibelm_top_outer_core,
					 int* h_ibelm_bottom_outer_core,
                                         int* h_ibool,
                                         realw* h_xstore, realw* h_ystore, realw* h_zstore,
                                         int* num_phase_ispec,
                                         int* phase_ispec_inner,
                                         int* nspec_outer,
                                         int* nspec_inner,
					 int* NSPEC2D_TOP_OC,
					 int* NSPEC2D_BOTTOM_OC
                                         ) {

  TRACE("prepare_outer_core_device");

  Mesh* mp = (Mesh*)(*Mesh_pointer_f);

  /* Assuming NGLLX=5. Padded is then 128 (5^3+3) */
  int size_padded = NGLL3_PADDED * (mp->NSPEC_OUTER_CORE);
  int size_glob = mp->NGLOB_OUTER_CORE;

  // mesh
  print_CUDA_error_if_any(hipMalloc((void**) &mp->d_xix_outer_core, size_padded*sizeof(realw)),1001);
  print_CUDA_error_if_any(hipMalloc((void**) &mp->d_xiy_outer_core, size_padded*sizeof(realw)),1002);
  print_CUDA_error_if_any(hipMalloc((void**) &mp->d_xiz_outer_core, size_padded*sizeof(realw)),1003);
  print_CUDA_error_if_any(hipMalloc((void**) &mp->d_etax_outer_core, size_padded*sizeof(realw)),1004);
  print_CUDA_error_if_any(hipMalloc((void**) &mp->d_etay_outer_core, size_padded*sizeof(realw)),1005);
  print_CUDA_error_if_any(hipMalloc((void**) &mp->d_etaz_outer_core, size_padded*sizeof(realw)),1006);
  print_CUDA_error_if_any(hipMalloc((void**) &mp->d_gammax_outer_core, size_padded*sizeof(realw)),1007);
  print_CUDA_error_if_any(hipMalloc((void**) &mp->d_gammay_outer_core, size_padded*sizeof(realw)),1008);
  print_CUDA_error_if_any(hipMalloc((void**) &mp->d_gammaz_outer_core, size_padded*sizeof(realw)),1009);

  print_CUDA_error_if_any(hipMalloc((void**) &mp->d_kappavstore_outer_core, size_padded*sizeof(realw)),1010);

  // transfer constant element data with padding
  for(int i=0;i < mp->NSPEC_OUTER_CORE;i++) {
    print_CUDA_error_if_any(hipMemcpy(mp->d_xix_outer_core + i*NGLL3_PADDED, &h_xix[i*NGLL3],
                                       NGLL3*sizeof(realw),hipMemcpyHostToDevice),1501);
    print_CUDA_error_if_any(hipMemcpy(mp->d_xiy_outer_core+i*NGLL3_PADDED,   &h_xiy[i*NGLL3],
                                       NGLL3*sizeof(realw),hipMemcpyHostToDevice),1502);
    print_CUDA_error_if_any(hipMemcpy(mp->d_xiz_outer_core+i*NGLL3_PADDED,   &h_xiz[i*NGLL3],
                                       NGLL3*sizeof(realw),hipMemcpyHostToDevice),1503);
    print_CUDA_error_if_any(hipMemcpy(mp->d_etax_outer_core+i*NGLL3_PADDED,  &h_etax[i*NGLL3],
                                       NGLL3*sizeof(realw),hipMemcpyHostToDevice),1504);
    print_CUDA_error_if_any(hipMemcpy(mp->d_etay_outer_core+i*NGLL3_PADDED,  &h_etay[i*NGLL3],
                                       NGLL3*sizeof(realw),hipMemcpyHostToDevice),1505);
    print_CUDA_error_if_any(hipMemcpy(mp->d_etaz_outer_core+i*NGLL3_PADDED,  &h_etaz[i*NGLL3],
                                       NGLL3*sizeof(realw),hipMemcpyHostToDevice),1506);
    print_CUDA_error_if_any(hipMemcpy(mp->d_gammax_outer_core+i*NGLL3_PADDED,&h_gammax[i*NGLL3],
                                       NGLL3*sizeof(realw),hipMemcpyHostToDevice),1507);
    print_CUDA_error_if_any(hipMemcpy(mp->d_gammay_outer_core+i*NGLL3_PADDED,&h_gammay[i*NGLL3],
                                       NGLL3*sizeof(realw),hipMemcpyHostToDevice),1508);
    print_CUDA_error_if_any(hipMemcpy(mp->d_gammaz_outer_core+i*NGLL3_PADDED,&h_gammaz[i*NGLL3],
                                       NGLL3*sizeof(realw),hipMemcpyHostToDevice),1509);

    print_CUDA_error_if_any(hipMemcpy(mp->d_kappavstore_outer_core+i*NGLL3_PADDED,&h_kappav[i*NGLL3],
                                       NGLL3*sizeof(realw),hipMemcpyHostToDevice),1510);
  }

  // needed for kernel calculations
  if( mp->simulation_type == 3 ){
    print_CUDA_error_if_any(hipMalloc((void**) &mp->d_rhostore_outer_core, size_padded*sizeof(realw)),1010);
    for(int i=0;i < mp->NSPEC_OUTER_CORE;i++) {
      print_CUDA_error_if_any(hipMemcpy(mp->d_rhostore_outer_core+i*NGLL3_PADDED, &h_rho[i*NGLL3],
                                       NGLL3*sizeof(realw),hipMemcpyHostToDevice),2106);
    }
  }

  // global indexing
  print_CUDA_error_if_any(hipMalloc((void**) &mp->d_ibool_outer_core, size_padded*sizeof(int)),1021);
  print_CUDA_error_if_any(hipMemcpy(mp->d_ibool_outer_core, h_ibool,
                                     NGLL3*(mp->NSPEC_OUTER_CORE)*sizeof(int),hipMemcpyHostToDevice),1022);

  // mesh locations
  // always needed
  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_xstore_outer_core),sizeof(realw)*size_glob),2005);
  print_CUDA_error_if_any(hipMemcpy(mp->d_xstore_outer_core,h_xstore,
                                     sizeof(realw)*size_glob,hipMemcpyHostToDevice),2100);
  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_ystore_outer_core),sizeof(realw)*size_glob),2005);
  print_CUDA_error_if_any(hipMemcpy(mp->d_ystore_outer_core,h_ystore,
                                     sizeof(realw)*size_glob,hipMemcpyHostToDevice),2100);
  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_zstore_outer_core),sizeof(realw)*size_glob),2005);
  print_CUDA_error_if_any(hipMemcpy(mp->d_zstore_outer_core,h_zstore,
                                     sizeof(realw)*size_glob,hipMemcpyHostToDevice),2100);

  // inner/outer elements
  mp->num_phase_ispec_outer_core = *num_phase_ispec;
  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_phase_ispec_inner_outer_core),
                                     mp->num_phase_ispec_outer_core*2*sizeof(int)),2008);
  print_CUDA_error_if_any(hipMemcpy(mp->d_phase_ispec_inner_outer_core,phase_ispec_inner,
                                     mp->num_phase_ispec_outer_core*2*sizeof(int),hipMemcpyHostToDevice),2101);

  mp->nspec_outer_outer_core = *nspec_outer;
  mp->nspec_inner_outer_core = *nspec_inner;

  // CMB/ICB coupling
  mp->nspec2D_top_outer_core = *NSPEC2D_TOP_OC;
  mp->nspec2D_bottom_outer_core = *NSPEC2D_BOTTOM_OC;
  int size_toc = NGLL2*(mp->nspec2D_top_outer_core);
  int size_boc = NGLL2*(mp->nspec2D_bottom_outer_core);

  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_normal_top_outer_core),sizeof(realw)*NDIM*size_toc),40020);
  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_normal_bottom_outer_core),sizeof(realw)*NDIM*size_boc),40021);
  print_CUDA_error_if_any(hipMemcpy(mp->d_normal_top_outer_core,h_normal_top_outer_core,sizeof(realw)*NDIM*size_toc,hipMemcpyHostToDevice),40030);
  print_CUDA_error_if_any(hipMemcpy(mp->d_normal_bottom_outer_core,h_normal_bottom_outer_core,sizeof(realw)*NDIM*size_boc,hipMemcpyHostToDevice),40031);

  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_jacobian2D_top_outer_core),sizeof(realw)*size_toc),40022);
  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_jacobian2D_bottom_outer_core),sizeof(realw)*size_boc),40023);
  print_CUDA_error_if_any(hipMemcpy(mp->d_jacobian2D_top_outer_core,h_jacobian2D_top_outer_core,sizeof(realw)*size_toc,hipMemcpyHostToDevice),40032);
  print_CUDA_error_if_any(hipMemcpy(mp->d_jacobian2D_bottom_outer_core,h_jacobian2D_bottom_outer_core,sizeof(realw)*size_boc,hipMemcpyHostToDevice),40033);

  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_ibelm_top_outer_core),sizeof(int)*(mp->nspec2D_top_outer_core)),40024);
  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_ibelm_bottom_outer_core),sizeof(int)*(mp->nspec2D_bottom_outer_core)),40025);
  print_CUDA_error_if_any(hipMemcpy(mp->d_ibelm_top_outer_core,h_ibelm_top_outer_core,sizeof(int)*(mp->nspec2D_top_outer_core),hipMemcpyHostToDevice),40034);
  print_CUDA_error_if_any(hipMemcpy(mp->d_ibelm_bottom_outer_core,h_ibelm_bottom_outer_core,sizeof(int)*(mp->nspec2D_bottom_outer_core),hipMemcpyHostToDevice),40035);

  // wavefield
  int size = mp->NGLOB_OUTER_CORE;
  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_displ_outer_core),sizeof(realw)*size),4001);
  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_veloc_outer_core),sizeof(realw)*size),4002);
  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_accel_outer_core),sizeof(realw)*size),4003);
  // backward/reconstructed wavefield
  if( mp->simulation_type == 3 ){
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_b_displ_outer_core),sizeof(realw)*size),4001);
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_b_veloc_outer_core),sizeof(realw)*size),4002);
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_b_accel_outer_core),sizeof(realw)*size),4003);
  }

  // mass matrix
  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_rmass_outer_core),sizeof(realw)*size_glob),2005);
  print_CUDA_error_if_any(hipMemcpy(mp->d_rmass_outer_core,h_rmass,
                                     sizeof(realw)*size_glob,hipMemcpyHostToDevice),2100);

  // kernels
  if( mp->simulation_type == 3 ){

    size = NGLL3*(mp->NSPEC_OUTER_CORE);

    // density kernel
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_rho_kl_outer_core),
                                       size*sizeof(realw)),5204);
    print_CUDA_error_if_any(hipMemset(mp->d_rho_kl_outer_core,0,size*sizeof(realw)),5207);

    // isotropic kernel
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_alpha_kl_outer_core),
                                        size*sizeof(realw)),5205);
    print_CUDA_error_if_any(hipMemset(mp->d_alpha_kl_outer_core,0,size*sizeof(realw)),5208);
  }

#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
  exit_on_cuda_error("prepare_outer_core_device");
#endif
}


/* ----------------------------------------------------------------------------------------------- */

// INNER CORE

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(prepare_inner_core_device,
              PREPARE_INNER_CORE_DEVICE)(long* Mesh_pointer_f,
					 realw* h_xix, realw* h_xiy, realw* h_xiz,
					 realw* h_etax, realw* h_etay, realw* h_etaz,
					 realw* h_gammax, realw* h_gammay, realw* h_gammaz,
					 realw* h_rho, realw* h_kappav, realw* h_muv,
					 realw* h_rmass,
					 int* h_ibelm_top_inner_core,
					 int* h_ibool,
					 realw* h_xstore, realw* h_ystore, realw* h_zstore,
					 realw *c11store,realw *c12store,realw *c13store,
					 realw *c33store,realw *c44store,
					 int* h_idoubling_inner_core,
					 int* num_phase_ispec,
					 int* phase_ispec_inner,
					 int* nspec_outer,
					 int* nspec_inner,
					 int* NSPEC2D_TOP_IC) {
  
  TRACE("prepare_inner_core_device");

  Mesh* mp = (Mesh*)(*Mesh_pointer_f);

  /* Assuming NGLLX=5. Padded is then 128 (5^3+3) */
  int size_padded = NGLL3_PADDED * (mp->NSPEC_INNER_CORE);
  int size_glob = mp->NGLOB_INNER_CORE;

  // mesh
  print_CUDA_error_if_any(hipMalloc((void**) &mp->d_xix_inner_core, size_padded*sizeof(realw)),1001);
  print_CUDA_error_if_any(hipMalloc((void**) &mp->d_xiy_inner_core, size_padded*sizeof(realw)),1002);
  print_CUDA_error_if_any(hipMalloc((void**) &mp->d_xiz_inner_core, size_padded*sizeof(realw)),1003);
  print_CUDA_error_if_any(hipMalloc((void**) &mp->d_etax_inner_core, size_padded*sizeof(realw)),1004);
  print_CUDA_error_if_any(hipMalloc((void**) &mp->d_etay_inner_core, size_padded*sizeof(realw)),1005);
  print_CUDA_error_if_any(hipMalloc((void**) &mp->d_etaz_inner_core, size_padded*sizeof(realw)),1006);
  print_CUDA_error_if_any(hipMalloc((void**) &mp->d_gammax_inner_core, size_padded*sizeof(realw)),1007);
  print_CUDA_error_if_any(hipMalloc((void**) &mp->d_gammay_inner_core, size_padded*sizeof(realw)),1008);
  print_CUDA_error_if_any(hipMalloc((void**) &mp->d_gammaz_inner_core, size_padded*sizeof(realw)),1009);

  print_CUDA_error_if_any(hipMalloc((void**) &mp->d_muvstore_inner_core, size_padded*sizeof(realw)),1011);

  // transfer constant element data with padding
  for(int i=0;i < mp->NSPEC_INNER_CORE;i++) {
    print_CUDA_error_if_any(hipMemcpy(mp->d_xix_inner_core + i*NGLL3_PADDED, &h_xix[i*NGLL3],
                                       NGLL3*sizeof(realw),hipMemcpyHostToDevice),1501);
    print_CUDA_error_if_any(hipMemcpy(mp->d_xiy_inner_core+i*NGLL3_PADDED,   &h_xiy[i*NGLL3],
                                       NGLL3*sizeof(realw),hipMemcpyHostToDevice),1502);
    print_CUDA_error_if_any(hipMemcpy(mp->d_xiz_inner_core+i*NGLL3_PADDED,   &h_xiz[i*NGLL3],
                                       NGLL3*sizeof(realw),hipMemcpyHostToDevice),1503);
    print_CUDA_error_if_any(hipMemcpy(mp->d_etax_inner_core+i*NGLL3_PADDED,  &h_etax[i*NGLL3],
                                       NGLL3*sizeof(realw),hipMemcpyHostToDevice),1504);
    print_CUDA_error_if_any(hipMemcpy(mp->d_etay_inner_core+i*NGLL3_PADDED,  &h_etay[i*NGLL3],
                                       NGLL3*sizeof(realw),hipMemcpyHostToDevice),1505);
    print_CUDA_error_if_any(hipMemcpy(mp->d_etaz_inner_core+i*NGLL3_PADDED,  &h_etaz[i*NGLL3],
                                       NGLL3*sizeof(realw),hipMemcpyHostToDevice),1506);
    print_CUDA_error_if_any(hipMemcpy(mp->d_gammax_inner_core+i*NGLL3_PADDED,&h_gammax[i*NGLL3],
                                       NGLL3*sizeof(realw),hipMemcpyHostToDevice),1507);
    print_CUDA_error_if_any(hipMemcpy(mp->d_gammay_inner_core+i*NGLL3_PADDED,&h_gammay[i*NGLL3],
                                       NGLL3*sizeof(realw),hipMemcpyHostToDevice),1508);
    print_CUDA_error_if_any(hipMemcpy(mp->d_gammaz_inner_core+i*NGLL3_PADDED,&h_gammaz[i*NGLL3],
                                       NGLL3*sizeof(realw),hipMemcpyHostToDevice),1509);

    print_CUDA_error_if_any(hipMemcpy(mp->d_muvstore_inner_core+i*NGLL3_PADDED,   &h_muv[i*NGLL3],
                                       NGLL3*sizeof(realw),hipMemcpyHostToDevice),1511);
  }

  // anisotropy
  if( ! mp->anisotropic_inner_core ){
    // no anisotropy (uses kappav and muv in inner core)
    // kappavstore needed
    print_CUDA_error_if_any(hipMalloc((void**) &mp->d_kappavstore_inner_core, size_padded*sizeof(realw)),1010);
    for(int i=0;i < mp->NSPEC_INNER_CORE;i++) {
      print_CUDA_error_if_any(hipMemcpy(mp->d_kappavstore_inner_core+i*NGLL3_PADDED,&h_kappav[i*NGLL3],
                                         NGLL3*sizeof(realw),hipMemcpyHostToDevice),1510);
    }
  }else{
    // anisotropic inner core
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_c11store_inner_core),
                                     size_padded*sizeof(realw)),4700);
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_c12store_inner_core),
                                     size_padded*sizeof(realw)),4701);
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_c13store_inner_core),
                                     size_padded*sizeof(realw)),4702);
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_c33store_inner_core),
                                     size_padded*sizeof(realw)),4703);
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_c44store_inner_core),
                                     size_padded*sizeof(realw)),4704);

    // transfer constant element data with padding
    for(int i=0;i < mp->NSPEC_INNER_CORE;i++) {
      print_CUDA_error_if_any(hipMemcpy(mp->d_c11store_inner_core + i*NGLL3_PADDED, &c11store[i*NGLL3],
                                         NGLL3*sizeof(realw),hipMemcpyHostToDevice),4800);
      print_CUDA_error_if_any(hipMemcpy(mp->d_c12store_inner_core + i*NGLL3_PADDED, &c12store[i*NGLL3],
                                         NGLL3*sizeof(realw),hipMemcpyHostToDevice),4801);
      print_CUDA_error_if_any(hipMemcpy(mp->d_c13store_inner_core + i*NGLL3_PADDED, &c13store[i*NGLL3],
                                         NGLL3*sizeof(realw),hipMemcpyHostToDevice),4802);
      print_CUDA_error_if_any(hipMemcpy(mp->d_c33store_inner_core + i*NGLL3_PADDED, &c33store[i*NGLL3],
                                         NGLL3*sizeof(realw),hipMemcpyHostToDevice),4803);
      print_CUDA_error_if_any(hipMemcpy(mp->d_c44store_inner_core + i*NGLL3_PADDED, &c44store[i*NGLL3],
                                         NGLL3*sizeof(realw),hipMemcpyHostToDevice),4804);
    }
  }

  // needed for boundary kernel calculations
  if( mp->simulation_type == 3 && mp->save_boundary_mesh ){
    print_CUDA_error_if_any(hipMalloc((void**) &mp->d_rhostore_inner_core, size_padded*sizeof(realw)),1010);
    for(int i=0;i < mp->NSPEC_INNER_CORE;i++) {
      print_CUDA_error_if_any(hipMemcpy(mp->d_rhostore_inner_core+i*NGLL3_PADDED, &h_rho[i*NGLL3],
                                         NGLL3*sizeof(realw),hipMemcpyHostToDevice),2106);
    }
  }

  // global indexing
  print_CUDA_error_if_any(hipMalloc((void**) &mp->d_ibool_inner_core, size_padded*sizeof(int)),1021);
  print_CUDA_error_if_any(hipMemcpy(mp->d_ibool_inner_core, h_ibool,
                                     NGLL3*(mp->NSPEC_INNER_CORE)*sizeof(int),hipMemcpyHostToDevice),1022);

  // fictious element flags
  print_CUDA_error_if_any(hipMalloc((void**) &mp->d_idoubling_inner_core,
                                     mp->NSPEC_INNER_CORE*sizeof(int)),2010);
  print_CUDA_error_if_any(hipMemcpy(mp->d_idoubling_inner_core, h_idoubling_inner_core,
                                     mp->NSPEC_INNER_CORE*sizeof(int),hipMemcpyHostToDevice),2011);

  // mesh locations
  // only needed when gravity is on
  if( mp->gravity ){
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_xstore_inner_core),sizeof(realw)*size_glob),2005);
    print_CUDA_error_if_any(hipMemcpy(mp->d_xstore_inner_core,h_xstore,
                                       sizeof(realw)*size_glob,hipMemcpyHostToDevice),2100);
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_ystore_inner_core),sizeof(realw)*size_glob),2005);
    print_CUDA_error_if_any(hipMemcpy(mp->d_ystore_inner_core,h_ystore,
                                       sizeof(realw)*size_glob,hipMemcpyHostToDevice),2100);
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_zstore_inner_core),sizeof(realw)*size_glob),2005);
    print_CUDA_error_if_any(hipMemcpy(mp->d_zstore_inner_core,h_zstore,
                                       sizeof(realw)*size_glob,hipMemcpyHostToDevice),2100);
  }

  // inner/outer elements
  mp->num_phase_ispec_inner_core = *num_phase_ispec;
  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_phase_ispec_inner_inner_core),
                                     mp->num_phase_ispec_inner_core*2*sizeof(int)),2008);
  print_CUDA_error_if_any(hipMemcpy(mp->d_phase_ispec_inner_inner_core,phase_ispec_inner,
                                     mp->num_phase_ispec_inner_core*2*sizeof(int),hipMemcpyHostToDevice),2101);

  mp->nspec_outer_inner_core = *nspec_outer;
  mp->nspec_inner_inner_core = *nspec_inner;
  mp->nspec2D_top_inner_core = *NSPEC2D_TOP_IC;

  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_ibelm_top_inner_core),sizeof(int)*(mp->nspec2D_top_inner_core)),40021);
  print_CUDA_error_if_any(hipMemcpy(mp->d_ibelm_top_inner_core,h_ibelm_top_inner_core,sizeof(int)*(mp->nspec2D_top_inner_core),hipMemcpyHostToDevice),40031);

  // wavefield
  int size = NDIM * mp->NGLOB_INNER_CORE;
  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_displ_inner_core),sizeof(realw)*size),4001);
  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_veloc_inner_core),sizeof(realw)*size),4002);
  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_accel_inner_core),sizeof(realw)*size),4003);
  // backward/reconstructed wavefield
  if( mp->simulation_type == 3 ){
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_b_displ_inner_core),sizeof(realw)*size),4001);
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_b_veloc_inner_core),sizeof(realw)*size),4002);
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_b_accel_inner_core),sizeof(realw)*size),4003);
  }

  // mass matrix
  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_rmass_inner_core),sizeof(realw)*size_glob),2005);
  print_CUDA_error_if_any(hipMemcpy(mp->d_rmass_inner_core,h_rmass,
                                     sizeof(realw)*size_glob,hipMemcpyHostToDevice),2100);

  // kernels
  if( mp->simulation_type == 3 ){

    size = NGLL3*(mp->NSPEC_INNER_CORE);

    // density kernel
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_rho_kl_inner_core),
                                       size*sizeof(realw)),5204);
    print_CUDA_error_if_any(hipMemset(mp->d_rho_kl_inner_core,0,size*sizeof(realw)),5207);

    // isotropic kernel
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_alpha_kl_inner_core),
                                       size*sizeof(realw)),5205);
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_beta_kl_inner_core),
                                       size*sizeof(realw)),5205);
    print_CUDA_error_if_any(hipMemset(mp->d_alpha_kl_inner_core,0,size*sizeof(realw)),5208);
    print_CUDA_error_if_any(hipMemset(mp->d_beta_kl_inner_core,0,size*sizeof(realw)),5208);
  }


#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
  exit_on_cuda_error("prepare_inner_core_device");
#endif
}

/* ----------------------------------------------------------------------------------------------- */

// OCEANS

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(prepare_oceans_device,
              PREPARE_OCEANS_DEVICE)(long* Mesh_pointer_f,
             realw* h_rmass_ocean_load) {

  TRACE("prepare_oceans_device");

  Mesh* mp = (Mesh*)(*Mesh_pointer_f);

  // mass matrix
  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_rmass_ocean_load),
                                     sizeof(realw)*mp->NGLOB_CRUST_MANTLE_OCEANS),4501);
  print_CUDA_error_if_any(hipMemcpy(mp->d_rmass_ocean_load,h_rmass_ocean_load,
                                     sizeof(realw)*mp->NGLOB_CRUST_MANTLE_OCEANS,hipMemcpyHostToDevice),4502);

  // temporary global array: used to synchronize updates on global accel array
  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_updated_dof_ocean_load),
                                     sizeof(int)*mp->NGLOB_CRUST_MANTLE_OCEANS),4502);


#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
  exit_on_cuda_error("prepare_oceans_device");
#endif
}

/* ----------------------------------------------------------------------------------------------- */

// for ELASTIC simulations

/* ----------------------------------------------------------------------------------------------- */

/*
extern "C"
void FC_FUNC_(prepare_fields_elastic_device,
              PREPARE_FIELDS_ELASTIC_DEVICE)(long* Mesh_pointer_f,
                                             int* size,
                                             realw* rmass,
                                             realw* rho_vp,
                                             realw* rho_vs,
                                             int* num_phase_ispec_elastic,
                                             int* phase_ispec_inner_elastic,
                                             int* ispec_is_elastic,
                                             int* ABSORBING_CONDITIONS,
                                             realw* h_b_absorb_field,
                                             int* h_b_reclen_field,
                                             int* SIMULATION_TYPE,int* SAVE_FORWARD,
                                             int* COMPUTE_AND_STORE_STRAIN,
                                             realw* epsilondev_xx,realw* epsilondev_yy,realw* epsilondev_xy,
                                             realw* epsilondev_xz,realw* epsilondev_yz,
                                             int* ATTENUATION,
                                             int* R_size,
                                             realw* R_xx,realw* R_yy,realw* R_xy,realw* R_xz,realw* R_yz,
                                             realw* one_minus_sum_beta,realw* factor_common,
                                             realw* alphaval,realw* betaval,realw* gammaval,
                                             int* OCEANS,
                                             realw* rmass_ocean_load,
                                             int* NOISE_TOMOGRAPHY,
                                             realw* free_surface_normal,
                                             int* free_surface_ispec,
                                             int* free_surface_ijk,
                                             int* num_free_surface_faces,
                                             int* ACOUSTIC_SIMULATION,
                                             int* num_colors_outer_elastic,
                                             int* num_colors_inner_elastic,
                                             int* num_elem_colors_elastic,
                                             int* ANISOTROPY,
                                             realw *c11store,
                                             realw *c12store,
                                             realw *c13store,
                                             realw *c14store,
                                             realw *c15store,
                                             realw *c16store,
                                             realw *c22store,
                                             realw *c23store,
                                             realw *c24store,
                                             realw *c25store,
                                             realw *c26store,
                                             realw *c33store,
                                             realw *c34store,
                                             realw *c35store,
                                             realw *c36store,
                                             realw *c44store,
                                             realw *c45store,
                                             realw *c46store,
                                             realw *c55store,
                                             realw *c56store,
                                             realw *c66store){

TRACE("prepare_fields_elastic_device");

  Mesh* mp = (Mesh*)(*Mesh_pointer_f);
  // Assuming NGLLX==5. Padded is then 128 (5^3+3)
  int size_padded = NGLL3_PADDED * (mp->NSPEC_AB);
  int size_nonpadded = NGLL3 * (mp->NSPEC_AB);

  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_displ),sizeof(realw)*(*size)),4001);
  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_veloc),sizeof(realw)*(*size)),4002);
  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_accel),sizeof(realw)*(*size)),4003);

  // mpi buffer
  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_send_accel_buffer),
                        3*(mp->max_nibool_interfaces_ext_mesh)*(mp->num_interfaces_ext_mesh)*sizeof(realw)),4004);

  // mass matrix
  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_rmass),sizeof(realw)*mp->NGLOB_AB),4005);
  // transfer element data
  print_CUDA_error_if_any(hipMemcpy(mp->d_rmass,rmass,
                                     sizeof(realw)*mp->NGLOB_AB,hipMemcpyHostToDevice),4010);


  // element indices
  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_ispec_is_elastic),mp->NSPEC_AB*sizeof(int)),4009);
  print_CUDA_error_if_any(hipMemcpy(mp->d_ispec_is_elastic,ispec_is_elastic,
                                     mp->NSPEC_AB*sizeof(int),hipMemcpyHostToDevice),4012);

  // phase elements
  mp->num_phase_ispec_elastic = *num_phase_ispec_elastic;
  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_phase_ispec_inner_elastic),
                                     mp->num_phase_ispec_elastic*2*sizeof(int)),4008);
  print_CUDA_error_if_any(hipMemcpy(mp->d_phase_ispec_inner_elastic,phase_ispec_inner_elastic,
                                     mp->num_phase_ispec_elastic*2*sizeof(int),hipMemcpyHostToDevice),4011);

  // for seismograms
  if( mp->nrec_local > 0 ){
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_station_seismo_field),
                                     3*NGLL3*(mp->nrec_local)*sizeof(realw)),4015);

    mp->h_station_seismo_field = (realw*) malloc( 3*NGLL3*(mp->nrec_local)*sizeof(realw) );
    if( mp->h_station_seismo_field == NULL) exit_on_error("h_station_seismo_field not allocated \n");
  }

  // absorbing conditions
  if( *ABSORBING_CONDITIONS && mp->d_num_abs_boundary_faces > 0){
    // non-padded arrays
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_rho_vp),size_nonpadded*sizeof(realw)),4006);
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_rho_vs),size_nonpadded*sizeof(realw)),4007);

    // rho_vp, rho_vs non-padded; they are needed for stacey boundary condition
    print_CUDA_error_if_any(hipMemcpy(mp->d_rho_vp, rho_vp,
                                       NGLL3*mp->NSPEC_AB*sizeof(realw),hipMemcpyHostToDevice),4013);
    print_CUDA_error_if_any(hipMemcpy(mp->d_rho_vs, rho_vs,
                                       NGLL3*mp->NSPEC_AB*sizeof(realw),hipMemcpyHostToDevice),4014);

    // absorb_field array used for file i/o
    if(*SIMULATION_TYPE == 3 || ( *SIMULATION_TYPE == 1 && *SAVE_FORWARD )){
      mp->d_b_reclen_field = *h_b_reclen_field;
      print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_b_absorb_field),
                                       mp->d_b_reclen_field),4016);
      print_CUDA_error_if_any(hipMemcpy(mp->d_b_absorb_field, h_b_absorb_field,
                                       mp->d_b_reclen_field,hipMemcpyHostToDevice),4017);
    }
  }

  // strains used for attenuation and kernel simulations
  if( *COMPUTE_AND_STORE_STRAIN ){
    // strains
    int epsilondev_size = NGLL3*mp->NSPEC_AB; // note: non-aligned; if align, check memcpy below and indexing

    print_CUDA_error_if_any(hipMalloc((void**)&mp->d_epsilondev_xx,
                                       epsilondev_size*sizeof(realw)),4301);
    print_CUDA_error_if_any(hipMemcpy(mp->d_epsilondev_xx,epsilondev_xx,epsilondev_size*sizeof(realw),
                                       hipMemcpyHostToDevice),4302);
    print_CUDA_error_if_any(hipMalloc((void**)&mp->d_epsilondev_yy,
                                       epsilondev_size*sizeof(realw)),4302);
    print_CUDA_error_if_any(hipMemcpy(mp->d_epsilondev_yy,epsilondev_yy,epsilondev_size*sizeof(realw),
                                       hipMemcpyHostToDevice),4303);
    print_CUDA_error_if_any(hipMalloc((void**)&mp->d_epsilondev_xy,
                                       epsilondev_size*sizeof(realw)),4304);
    print_CUDA_error_if_any(hipMemcpy(mp->d_epsilondev_xy,epsilondev_xy,epsilondev_size*sizeof(realw),
                                       hipMemcpyHostToDevice),4305);
    print_CUDA_error_if_any(hipMalloc((void**)&mp->d_epsilondev_xz,
                                       epsilondev_size*sizeof(realw)),4306);
    print_CUDA_error_if_any(hipMemcpy(mp->d_epsilondev_xz,epsilondev_xz,epsilondev_size*sizeof(realw),
                                       hipMemcpyHostToDevice),4307);
    print_CUDA_error_if_any(hipMalloc((void**)&mp->d_epsilondev_yz,
                                       epsilondev_size*sizeof(realw)),4308);
    print_CUDA_error_if_any(hipMemcpy(mp->d_epsilondev_yz,epsilondev_yz,epsilondev_size*sizeof(realw),
                                       hipMemcpyHostToDevice),4309);

  }

  // attenuation memory variables
  if( *ATTENUATION ){
    // memory arrays
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_R_xx),
                                       (*R_size)*sizeof(realw)),4401);
    print_CUDA_error_if_any(hipMemcpy(mp->d_R_xx,R_xx,(*R_size)*sizeof(realw),
                                       hipMemcpyHostToDevice),4402);

    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_R_yy),
                                       (*R_size)*sizeof(realw)),4403);
    print_CUDA_error_if_any(hipMemcpy(mp->d_R_yy,R_yy,(*R_size)*sizeof(realw),
                                       hipMemcpyHostToDevice),4404);

    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_R_xy),
                                       (*R_size)*sizeof(realw)),4405);
    print_CUDA_error_if_any(hipMemcpy(mp->d_R_xy,R_xy,(*R_size)*sizeof(realw),
                                       hipMemcpyHostToDevice),4406);

    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_R_xz),
                                       (*R_size)*sizeof(realw)),4407);
    print_CUDA_error_if_any(hipMemcpy(mp->d_R_xz,R_xz,(*R_size)*sizeof(realw),
                                       hipMemcpyHostToDevice),4408);

    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_R_yz),
                                       (*R_size)*sizeof(realw)),4409);
    print_CUDA_error_if_any(hipMemcpy(mp->d_R_yz,R_yz,(*R_size)*sizeof(realw),
                                       hipMemcpyHostToDevice),4410);

    // attenuation factors
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_one_minus_sum_beta),
                                       NGLL3*mp->NSPEC_AB*sizeof(realw)),4430);
    print_CUDA_error_if_any(hipMemcpy(mp->d_one_minus_sum_beta ,one_minus_sum_beta,
                                       NGLL3*mp->NSPEC_AB*sizeof(realw),hipMemcpyHostToDevice),4431);

    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_factor_common),
                                       N_SLS*NGLL3*mp->NSPEC_AB*sizeof(realw)),4432);
    print_CUDA_error_if_any(hipMemcpy(mp->d_factor_common ,factor_common,
                                       N_SLS*NGLL3*mp->NSPEC_AB*sizeof(realw),hipMemcpyHostToDevice),4433);

    // alpha,beta,gamma factors
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_alphaval),
                                       N_SLS*sizeof(realw)),4434);
    print_CUDA_error_if_any(hipMemcpy(mp->d_alphaval ,alphaval,
                                       N_SLS*sizeof(realw),hipMemcpyHostToDevice),4435);

    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_betaval),
                                       N_SLS*sizeof(realw)),4436);
    print_CUDA_error_if_any(hipMemcpy(mp->d_betaval ,betaval,
                                       N_SLS*sizeof(realw),hipMemcpyHostToDevice),4437);

    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_gammaval),
                                       N_SLS*sizeof(realw)),4438);
    print_CUDA_error_if_any(hipMemcpy(mp->d_gammaval ,gammaval,
                                       N_SLS*sizeof(realw),hipMemcpyHostToDevice),4439);

  }

  // anisotropy
  if( *ANISOTROPY ){
    // allocates memory on GPU
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_c11store),
                                       size_padded*sizeof(realw)),4700);
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_c12store),
                                       size_padded*sizeof(realw)),4701);
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_c13store),
                                       size_padded*sizeof(realw)),4702);
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_c14store),
                                       size_padded*sizeof(realw)),4703);
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_c15store),
                                       size_padded*sizeof(realw)),4704);
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_c16store),
                                       size_padded*sizeof(realw)),4705);
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_c22store),
                                       size_padded*sizeof(realw)),4706);
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_c23store),
                                       size_padded*sizeof(realw)),4707);
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_c24store),
                                       size_padded*sizeof(realw)),4708);
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_c25store),
                                       size_padded*sizeof(realw)),4709);
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_c26store),
                                       size_padded*sizeof(realw)),4710);
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_c33store),
                                       size_padded*sizeof(realw)),4711);
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_c34store),
                                       size_padded*sizeof(realw)),4712);
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_c35store),
                                       size_padded*sizeof(realw)),4713);
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_c36store),
                                       size_padded*sizeof(realw)),4714);
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_c44store),
                                       size_padded*sizeof(realw)),4715);
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_c45store),
                                       size_padded*sizeof(realw)),4716);
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_c46store),
                                       size_padded*sizeof(realw)),4717);
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_c55store),
                                       size_padded*sizeof(realw)),4718);
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_c56store),
                                       size_padded*sizeof(realw)),4719);
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_c66store),
                                       size_padded*sizeof(realw)),4720);

    // transfer constant element data with padding
    for(int i=0;i < mp->NSPEC_AB;i++) {
      print_CUDA_error_if_any(hipMemcpy(mp->d_c11store + i*NGLL3_PADDED, &c11store[i*NGLL3],
                                         NGLL3*sizeof(realw),hipMemcpyHostToDevice),4800);
      print_CUDA_error_if_any(hipMemcpy(mp->d_c12store + i*NGLL3_PADDED, &c12store[i*NGLL3],
                                         NGLL3*sizeof(realw),hipMemcpyHostToDevice),4801);
      print_CUDA_error_if_any(hipMemcpy(mp->d_c13store + i*NGLL3_PADDED, &c13store[i*NGLL3],
                                         NGLL3*sizeof(realw),hipMemcpyHostToDevice),4802);
      print_CUDA_error_if_any(hipMemcpy(mp->d_c14store + i*NGLL3_PADDED, &c14store[i*NGLL3],
                                         NGLL3*sizeof(realw),hipMemcpyHostToDevice),4803);
      print_CUDA_error_if_any(hipMemcpy(mp->d_c15store + i*NGLL3_PADDED, &c15store[i*NGLL3],
                                         NGLL3*sizeof(realw),hipMemcpyHostToDevice),4804);
      print_CUDA_error_if_any(hipMemcpy(mp->d_c16store + i*NGLL3_PADDED, &c16store[i*NGLL3],
                                         NGLL3*sizeof(realw),hipMemcpyHostToDevice),4805);
      print_CUDA_error_if_any(hipMemcpy(mp->d_c22store + i*NGLL3_PADDED, &c22store[i*NGLL3],
                                         NGLL3*sizeof(realw),hipMemcpyHostToDevice),4806);
      print_CUDA_error_if_any(hipMemcpy(mp->d_c23store + i*NGLL3_PADDED, &c23store[i*NGLL3],
                                         NGLL3*sizeof(realw),hipMemcpyHostToDevice),4807);
      print_CUDA_error_if_any(hipMemcpy(mp->d_c24store + i*NGLL3_PADDED, &c24store[i*NGLL3],
                                         NGLL3*sizeof(realw),hipMemcpyHostToDevice),4808);
      print_CUDA_error_if_any(hipMemcpy(mp->d_c25store + i*NGLL3_PADDED, &c25store[i*NGLL3],
                                         NGLL3*sizeof(realw),hipMemcpyHostToDevice),4809);
      print_CUDA_error_if_any(hipMemcpy(mp->d_c26store + i*NGLL3_PADDED, &c26store[i*NGLL3],
                                         NGLL3*sizeof(realw),hipMemcpyHostToDevice),4810);
      print_CUDA_error_if_any(hipMemcpy(mp->d_c33store + i*NGLL3_PADDED, &c33store[i*NGLL3],
                                         NGLL3*sizeof(realw),hipMemcpyHostToDevice),4811);
      print_CUDA_error_if_any(hipMemcpy(mp->d_c34store + i*NGLL3_PADDED, &c34store[i*NGLL3],
                                         NGLL3*sizeof(realw),hipMemcpyHostToDevice),4812);
      print_CUDA_error_if_any(hipMemcpy(mp->d_c35store + i*NGLL3_PADDED, &c35store[i*NGLL3],
                                         NGLL3*sizeof(realw),hipMemcpyHostToDevice),4813);
      print_CUDA_error_if_any(hipMemcpy(mp->d_c36store + i*NGLL3_PADDED, &c36store[i*NGLL3],
                                         NGLL3*sizeof(realw),hipMemcpyHostToDevice),4814);
      print_CUDA_error_if_any(hipMemcpy(mp->d_c44store + i*NGLL3_PADDED, &c44store[i*NGLL3],
                                         NGLL3*sizeof(realw),hipMemcpyHostToDevice),4815);
      print_CUDA_error_if_any(hipMemcpy(mp->d_c45store + i*NGLL3_PADDED, &c45store[i*NGLL3],
                                         NGLL3*sizeof(realw),hipMemcpyHostToDevice),4816);
      print_CUDA_error_if_any(hipMemcpy(mp->d_c46store + i*NGLL3_PADDED, &c46store[i*NGLL3],
                                         NGLL3*sizeof(realw),hipMemcpyHostToDevice),4817);
      print_CUDA_error_if_any(hipMemcpy(mp->d_c55store + i*NGLL3_PADDED, &c55store[i*NGLL3],
                                         NGLL3*sizeof(realw),hipMemcpyHostToDevice),4818);
      print_CUDA_error_if_any(hipMemcpy(mp->d_c56store + i*NGLL3_PADDED, &c56store[i*NGLL3],
                                         NGLL3*sizeof(realw),hipMemcpyHostToDevice),4819);
      print_CUDA_error_if_any(hipMemcpy(mp->d_c66store + i*NGLL3_PADDED, &c66store[i*NGLL3],
                                         NGLL3*sizeof(realw),hipMemcpyHostToDevice),4820);
    }
  }

  // ocean load approximation
  if( *OCEANS ){
    // oceans needs a free surface
    mp->num_free_surface_faces = *num_free_surface_faces;
    if( mp->num_free_surface_faces > 0 ){
      // mass matrix
      print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_rmass_ocean_load),
                                         sizeof(realw)*mp->NGLOB_AB),4501);
      print_CUDA_error_if_any(hipMemcpy(mp->d_rmass_ocean_load,rmass_ocean_load,
                                         sizeof(realw)*mp->NGLOB_AB,hipMemcpyHostToDevice),4502);
      // surface normal
      print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_free_surface_normal),
                                         3*NGLL2*(mp->num_free_surface_faces)*sizeof(realw)),4503);
      print_CUDA_error_if_any(hipMemcpy(mp->d_free_surface_normal,free_surface_normal,
                                         3*NGLL2*(mp->num_free_surface_faces)*sizeof(realw),hipMemcpyHostToDevice),4504);

      // temporary global array: used to synchronize updates on global accel array
      print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_updated_dof_ocean_load),
                                         sizeof(int)*mp->NGLOB_AB),4505);

      if( *NOISE_TOMOGRAPHY == 0 && *ACOUSTIC_SIMULATION == 0 ){
        print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_free_surface_ispec),
                                          mp->num_free_surface_faces*sizeof(int)),4601);
        print_CUDA_error_if_any(hipMemcpy(mp->d_free_surface_ispec,free_surface_ispec,
                                          mp->num_free_surface_faces*sizeof(int),hipMemcpyHostToDevice),4603);
        print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_free_surface_ijk),
                                          3*NGLL2*mp->num_free_surface_faces*sizeof(int)),4602);
        print_CUDA_error_if_any(hipMemcpy(mp->d_free_surface_ijk,free_surface_ijk,
                                          3*NGLL2*mp->num_free_surface_faces*sizeof(int),hipMemcpyHostToDevice),4604);
      }
    }
  }

  // mesh coloring
  if( mp->use_mesh_coloring_gpu ){
    mp->num_colors_outer_elastic = *num_colors_outer_elastic;
    mp->num_colors_inner_elastic = *num_colors_inner_elastic;
    mp->h_num_elem_colors_elastic = (int*) num_elem_colors_elastic;
  }

#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
  exit_on_cuda_error("prepare_fields_elastic_device");
#endif
}
*/



/* ----------------------------------------------------------------------------------------------- */

// cleanup

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(prepare_cleanup_device,
              PREPARE_CLEANUP_DEVICE)(long* Mesh_pointer_f,
				      int* NCHUNKS_VAL) {

TRACE("prepare_cleanup_device");

  // frees allocated memory arrays
  Mesh* mp = (Mesh*)(*Mesh_pointer_f);

  // frees memory on GPU

  //------------------------------------------
  // sources
  //------------------------------------------
  if( mp->simulation_type == 1  || mp->simulation_type == 3 ){
    hipFree(mp->d_sourcearrays);
    hipFree(mp->d_stf_pre_compute);
  }

  hipFree(mp->d_islice_selected_source);
  hipFree(mp->d_ispec_selected_source);

  //------------------------------------------
  // receivers
  //------------------------------------------
  if( mp->nrec_local > 0 ) {
    hipFree(mp->d_number_receiver_global);
    hipFree(mp->d_station_seismo_field);
    free(mp->h_station_seismo_field);
  }
  hipFree(mp->d_ispec_selected_rec);

  if( mp->nadj_rec_local > 0 ){
    hipFree(mp->d_adj_sourcearrays);
    hipFree(mp->d_pre_computed_irec);
    free(mp->h_adj_sourcearrays_slice);
  }

  //------------------------------------------
  // rotation arrays
  //------------------------------------------
  if( mp->rotation ){
    hipFree(mp->d_A_array_rotation);
    hipFree(mp->d_B_array_rotation);
    if( mp->simulation_type == 3 ){
      hipFree(mp->d_b_A_array_rotation);
      hipFree(mp->d_b_B_array_rotation);
    }
  }

  //------------------------------------------
  // gravity arrays
  //------------------------------------------
  if( ! mp->gravity ){
    hipFree(mp->d_d_ln_density_dr_table);
  }else{
    hipFree(mp->d_minus_rho_g_over_kappa_fluid);
    hipFree(mp->d_minus_gravity_table);
    hipFree(mp->d_minus_deriv_gravity_table);
    hipFree(mp->d_density_table);
  }

  //------------------------------------------
  // attenuation arrays
  //------------------------------------------
  if( mp->attenuation ){
    hipFree(mp->d_one_minus_sum_beta_crust_mantle);
    hipFree(mp->d_one_minus_sum_beta_inner_core);
    if( ! mp->use_attenuation_mimic ){
      hipFree(mp->d_factor_common_crust_mantle);
      hipFree(mp->d_R_xx_crust_mantle);
      hipFree(mp->d_R_yy_crust_mantle);
      hipFree(mp->d_R_xy_crust_mantle);
      hipFree(mp->d_R_xz_crust_mantle);
      hipFree(mp->d_R_yz_crust_mantle);
      hipFree(mp->d_factor_common_inner_core);
      hipFree(mp->d_R_xx_inner_core);
      hipFree(mp->d_R_yy_inner_core);
      hipFree(mp->d_R_xy_inner_core);
      hipFree(mp->d_R_xz_inner_core);
      hipFree(mp->d_R_yz_inner_core);
    }
    hipFree(mp->d_alphaval);
    hipFree(mp->d_betaval);
    hipFree(mp->d_gammaval);
    if( mp->simulation_type == 3 ){
      hipFree(mp->d_b_alphaval);
      hipFree(mp->d_b_betaval);
      hipFree(mp->d_b_gammaval);
    }
  }

  //------------------------------------------
  // strain
  //------------------------------------------
  if( mp->compute_and_store_strain ){
    hipFree(mp->d_epsilondev_xx_crust_mantle);
    hipFree(mp->d_epsilondev_yy_crust_mantle);
    hipFree(mp->d_epsilondev_xy_crust_mantle);
    hipFree(mp->d_epsilondev_xz_crust_mantle);
    hipFree(mp->d_epsilondev_yz_crust_mantle);

    hipFree(mp->d_epsilondev_xx_inner_core);
    hipFree(mp->d_epsilondev_yy_inner_core);
    hipFree(mp->d_epsilondev_xy_inner_core);
    hipFree(mp->d_epsilondev_xz_inner_core);
    hipFree(mp->d_epsilondev_yz_inner_core);

    hipFree(mp->d_eps_trace_over_3_crust_mantle);
    hipFree(mp->d_eps_trace_over_3_inner_core);
    if( mp->simulation_type == 3 ){
      hipFree(mp->d_b_epsilondev_xx_crust_mantle);
      hipFree(mp->d_b_epsilondev_yy_crust_mantle);
      hipFree(mp->d_b_epsilondev_xy_crust_mantle);
      hipFree(mp->d_b_epsilondev_xz_crust_mantle);
      hipFree(mp->d_b_epsilondev_yz_crust_mantle);

      hipFree(mp->d_b_epsilondev_xx_inner_core);
      hipFree(mp->d_b_epsilondev_yy_inner_core);
      hipFree(mp->d_b_epsilondev_xy_inner_core);
      hipFree(mp->d_b_epsilondev_xz_inner_core);
      hipFree(mp->d_b_epsilondev_yz_inner_core);

      hipFree(mp->d_b_eps_trace_over_3_crust_mantle);
      hipFree(mp->d_b_eps_trace_over_3_inner_core);
    }
  }

  //------------------------------------------
  // absorbing boundaries arrays
  //------------------------------------------
  if( mp->absorbing_conditions){
    hipFree(mp->d_rho_vp_crust_mantle);
    hipFree(mp->d_rho_vs_crust_mantle);
    hipFree(mp->d_nkmin_xi_crust_mantle);
    hipFree(mp->d_nkmin_eta_crust_mantle);
    hipFree(mp->d_njmin_crust_mantle);
    hipFree(mp->d_njmax_crust_mantle);
    hipFree(mp->d_nimin_crust_mantle);
    hipFree(mp->d_nimax_crust_mantle);
    if( mp->nspec2D_xmin_crust_mantle > 0 ){
      hipFree(mp->d_ibelm_xmin_crust_mantle);
      hipFree(mp->d_normal_xmin_crust_mantle);
      hipFree(mp->d_jacobian2D_xmin_crust_mantle);
      if( (mp->simulation_type == 1 && mp->save_forward ) || (mp->simulation_type == 3) ){
        hipFree(mp->d_absorb_xmin_crust_mantle);
      }
    }
    if( mp->nspec2D_xmax_crust_mantle > 0 ){
      hipFree(mp->d_ibelm_xmax_crust_mantle);
      hipFree(mp->d_normal_xmax_crust_mantle);
      hipFree(mp->d_jacobian2D_xmax_crust_mantle);
      if( (mp->simulation_type == 1 && mp->save_forward ) || (mp->simulation_type == 3) ){
        hipFree(mp->d_absorb_xmax_crust_mantle);
      }
    }
    if( mp->nspec2D_ymin_crust_mantle > 0 ){
      hipFree(mp->d_ibelm_ymin_crust_mantle);
      hipFree(mp->d_normal_ymin_crust_mantle);
      hipFree(mp->d_jacobian2D_ymin_crust_mantle);
      if( (mp->simulation_type == 1 && mp->save_forward ) || (mp->simulation_type == 3) ){
        hipFree(mp->d_absorb_ymin_crust_mantle);
      }
    }
    if( mp->nspec2D_ymax_crust_mantle > 0 ){
      hipFree(mp->d_ibelm_ymax_crust_mantle);
      hipFree(mp->d_normal_ymax_crust_mantle);
      hipFree(mp->d_jacobian2D_ymax_crust_mantle);
      if( (mp->simulation_type == 1 && mp->save_forward ) || (mp->simulation_type == 3) ){
        hipFree(mp->d_absorb_ymax_crust_mantle);
      }
    }

    hipFree(mp->d_vp_outer_core);
    hipFree(mp->d_nkmin_xi_outer_core);
    hipFree(mp->d_nkmin_eta_outer_core);
    hipFree(mp->d_njmin_outer_core);
    hipFree(mp->d_njmax_outer_core);
    hipFree(mp->d_nimin_outer_core);
    hipFree(mp->d_nimax_outer_core);
    if( mp->nspec2D_xmin_outer_core > 0 ){
      hipFree(mp->d_ibelm_xmin_outer_core);
      hipFree(mp->d_jacobian2D_xmin_outer_core);
      if( (mp->simulation_type == 1 && mp->save_forward ) || (mp->simulation_type == 3) ){
        hipFree(mp->d_absorb_xmin_outer_core);
      }
    }
    if( mp->nspec2D_xmax_outer_core > 0 ){
      hipFree(mp->d_ibelm_xmax_outer_core);
      hipFree(mp->d_jacobian2D_xmax_outer_core);
      if( (mp->simulation_type == 1 && mp->save_forward ) || (mp->simulation_type == 3) ){
        hipFree(mp->d_absorb_xmax_outer_core);
      }
    }
    if( mp->nspec2D_ymin_outer_core > 0 ){
      hipFree(mp->d_ibelm_ymin_outer_core);
      hipFree(mp->d_jacobian2D_ymin_outer_core);
      if( (mp->simulation_type == 1 && mp->save_forward ) || (mp->simulation_type == 3) ){
        hipFree(mp->d_absorb_ymin_outer_core);
      }
    }
    if( mp->nspec2D_ymax_outer_core > 0 ){
      hipFree(mp->d_ibelm_ymax_outer_core);
      hipFree(mp->d_jacobian2D_ymax_outer_core);
      if( (mp->simulation_type == 1 && mp->save_forward ) || (mp->simulation_type == 3) ){
        hipFree(mp->d_absorb_ymax_outer_core);
      }
    }
    if( mp->nspec2D_zmin_outer_core > 0 ){
      hipFree(mp->d_ibelm_zmin_outer_core);
      hipFree(mp->d_jacobian2D_zmin_outer_core);
      if( (mp->simulation_type == 1 && mp->save_forward ) || (mp->simulation_type == 3) ){
        hipFree(mp->d_absorb_zmin_outer_core);
      }
    }

  }

  //------------------------------------------
  // mpi buffers
  //------------------------------------------
  if( mp->num_interfaces_crust_mantle > 0 ){
    hipFree(mp->d_nibool_interfaces_crust_mantle);
    hipFree(mp->d_ibool_interfaces_crust_mantle);
    hipFree(mp->d_send_accel_buffer_crust_mantle);
  }
  if( mp->num_interfaces_inner_core > 0 ){
    hipFree(mp->d_nibool_interfaces_inner_core);
    hipFree(mp->d_ibool_interfaces_inner_core);
    hipFree(mp->d_send_accel_buffer_inner_core);
  }
  if( mp->num_interfaces_outer_core > 0 ){
    hipFree(mp->d_nibool_interfaces_outer_core);
    hipFree(mp->d_ibool_interfaces_outer_core);
    hipFree(mp->d_send_accel_buffer_outer_core);
  }

  //------------------------------------------
  // NOISE arrays
  //------------------------------------------
  if( mp->noise_tomography > 0 ){
    hipFree(mp->d_ibelm_top_crust_mantle);
    hipFree(mp->d_noise_surface_movie);
    if( mp->noise_tomography == 1 ) hipFree(mp->d_noise_sourcearray);
    if( mp->noise_tomography > 1 ){
      hipFree(mp->d_normal_x_noise);
      hipFree(mp->d_normal_y_noise);
      hipFree(mp->d_normal_z_noise);
      hipFree(mp->d_mask_noise);
      hipFree(mp->d_jacobian2D_top_crust_mantle);
    }
    if( mp->noise_tomography == 3 ) hipFree(mp->d_Sigma_kl);
  }

  //------------------------------------------
  // crust_mantle
  //------------------------------------------
  hipFree(mp->d_xix_crust_mantle);
  hipFree(mp->d_xiy_crust_mantle);
  hipFree(mp->d_xiz_crust_mantle);
  hipFree(mp->d_etax_crust_mantle);
  hipFree(mp->d_etay_crust_mantle);
  hipFree(mp->d_etaz_crust_mantle);
  hipFree(mp->d_gammax_crust_mantle);
  hipFree(mp->d_gammay_crust_mantle);
  hipFree(mp->d_gammaz_crust_mantle);

  hipFree(mp->d_muvstore_crust_mantle);
  hipFree(mp->d_ibool_crust_mantle);

  if( ! mp->anisotropic_3D_mantle ){
    hipFree(mp->d_kappavstore_crust_mantle);
    hipFree(mp->d_kappahstore_crust_mantle);
    hipFree(mp->d_muhstore_crust_mantle);
    hipFree(mp->d_eta_anisostore_crust_mantle);
    hipFree(mp->d_ispec_is_tiso_crust_mantle);
  }else{
    hipFree(mp->d_c11store_crust_mantle);
    hipFree(mp->d_c12store_crust_mantle);
    hipFree(mp->d_c13store_crust_mantle);
    hipFree(mp->d_c14store_crust_mantle);
    hipFree(mp->d_c15store_crust_mantle);
    hipFree(mp->d_c16store_crust_mantle);
    hipFree(mp->d_c22store_crust_mantle);
    hipFree(mp->d_c23store_crust_mantle);
    hipFree(mp->d_c24store_crust_mantle);
    hipFree(mp->d_c25store_crust_mantle);
    hipFree(mp->d_c26store_crust_mantle);
    hipFree(mp->d_c33store_crust_mantle);
    hipFree(mp->d_c34store_crust_mantle);
    hipFree(mp->d_c35store_crust_mantle);
    hipFree(mp->d_c36store_crust_mantle);
    hipFree(mp->d_c44store_crust_mantle);
    hipFree(mp->d_c45store_crust_mantle);
    hipFree(mp->d_c46store_crust_mantle);
    hipFree(mp->d_c55store_crust_mantle);
    hipFree(mp->d_c56store_crust_mantle);
    hipFree(mp->d_c66store_crust_mantle);
  }

  if( mp->simulation_type == 3 && mp->save_boundary_mesh ){
    hipFree(mp->d_rhostore_crust_mantle);
  }

  hipFree(mp->d_ystore_crust_mantle);
  hipFree(mp->d_zstore_crust_mantle);
  if( mp->gravity ){
    hipFree(mp->d_xstore_crust_mantle);
  }
  hipFree(mp->d_phase_ispec_inner_crust_mantle);

  hipFree(mp->d_normal_top_crust_mantle);
  hipFree(mp->d_ibelm_top_crust_mantle);
  hipFree(mp->d_ibelm_bottom_crust_mantle);

  hipFree(mp->d_displ_crust_mantle);
  hipFree(mp->d_veloc_crust_mantle);
  hipFree(mp->d_accel_crust_mantle);
  if( mp->simulation_type == 3 ){
    hipFree(mp->d_b_displ_crust_mantle);
    hipFree(mp->d_b_veloc_crust_mantle);
    hipFree(mp->d_b_accel_crust_mantle);
    hipFree(mp->d_rho_kl_crust_mantle);
    if(mp->anisotropic_kl){
      hipFree(mp->d_cijkl_kl_crust_mantle);
    }else{
      hipFree(mp->d_alpha_kl_crust_mantle);
      hipFree(mp->d_beta_kl_crust_mantle);
    }
    if(mp->approximate_hess_kl){ hipFree(mp->d_hess_kl_crust_mantle);}
  }
  if( *NCHUNKS_VAL != 6 && mp->absorbing_conditions){
    hipFree(mp->d_rmassx_crust_mantle);
    hipFree(mp->d_rmassy_crust_mantle);
  }
  hipFree(mp->d_rmassz_crust_mantle);

  //------------------------------------------
  // outer_core
  //------------------------------------------
  hipFree(mp->d_xix_outer_core);
  hipFree(mp->d_xiy_outer_core);
  hipFree(mp->d_xiz_outer_core);
  hipFree(mp->d_etax_outer_core);
  hipFree(mp->d_etay_outer_core);
  hipFree(mp->d_etaz_outer_core);
  hipFree(mp->d_gammax_outer_core);
  hipFree(mp->d_gammay_outer_core);
  hipFree(mp->d_gammaz_outer_core);

  hipFree(mp->d_kappavstore_outer_core);
  if( mp->simulation_type == 3 ){
    hipFree(mp->d_rhostore_outer_core);
  }

  hipFree(mp->d_xstore_outer_core);
  hipFree(mp->d_ystore_outer_core);
  hipFree(mp->d_zstore_outer_core);

  hipFree(mp->d_ibool_outer_core);
  hipFree(mp->d_phase_ispec_inner_outer_core);

  hipFree(mp->d_ibelm_top_outer_core);
  hipFree(mp->d_ibelm_bottom_outer_core);

  hipFree(mp->d_normal_top_outer_core);
  hipFree(mp->d_normal_bottom_outer_core);

  hipFree(mp->d_jacobian2D_top_outer_core);
  hipFree(mp->d_jacobian2D_bottom_outer_core);

  hipFree(mp->d_displ_outer_core);
  hipFree(mp->d_veloc_outer_core);
  hipFree(mp->d_accel_outer_core);
  if( mp->simulation_type == 3 ){
    hipFree(mp->d_b_displ_outer_core);
    hipFree(mp->d_b_veloc_outer_core);
    hipFree(mp->d_b_accel_outer_core);
    hipFree(mp->d_rho_kl_outer_core);
    hipFree(mp->d_alpha_kl_outer_core);
  }
  hipFree(mp->d_rmass_outer_core);

  //------------------------------------------
  // inner_core
  //------------------------------------------
  hipFree(mp->d_xix_inner_core);
  hipFree(mp->d_xiy_inner_core);
  hipFree(mp->d_xiz_inner_core);
  hipFree(mp->d_etax_inner_core);
  hipFree(mp->d_etay_inner_core);
  hipFree(mp->d_etaz_inner_core);
  hipFree(mp->d_gammax_inner_core);
  hipFree(mp->d_gammay_inner_core);
  hipFree(mp->d_gammaz_inner_core);


  hipFree(mp->d_muvstore_inner_core);
  hipFree(mp->d_ibool_inner_core);

  if( mp->oceans ){
    hipFree(mp->d_rmass_ocean_load);
    hipFree(mp->d_updated_dof_ocean_load);
  }

  if( mp->gravity ){
    hipFree(mp->d_xstore_inner_core);
    hipFree(mp->d_ystore_inner_core);
    hipFree(mp->d_zstore_inner_core);
  }

  hipFree(mp->d_ibelm_top_inner_core);

  if( ! mp->anisotropic_inner_core ){
    hipFree(mp->d_kappavstore_inner_core);
  }else{
    hipFree(mp->d_c11store_inner_core);
    hipFree(mp->d_c12store_inner_core);
    hipFree(mp->d_c13store_inner_core);
    hipFree(mp->d_c33store_inner_core);
    hipFree(mp->d_c44store_inner_core);
  }

  if( mp->simulation_type == 3 && mp->save_boundary_mesh ){
    hipFree(mp->d_rhostore_inner_core);
  }
  hipFree(mp->d_idoubling_inner_core);
  if( mp->gravity ){
    hipFree(mp->d_xstore_inner_core);
    hipFree(mp->d_ystore_inner_core);
    hipFree(mp->d_zstore_inner_core);
  }
  hipFree(mp->d_phase_ispec_inner_inner_core);

  hipFree(mp->d_displ_inner_core);
  hipFree(mp->d_veloc_inner_core);
  hipFree(mp->d_accel_inner_core);
  if( mp->simulation_type == 3 ) {
    hipFree(mp->d_b_displ_inner_core);
    hipFree(mp->d_b_veloc_inner_core);
    hipFree(mp->d_b_accel_inner_core);

    hipFree(mp->d_rho_kl_inner_core);
    hipFree(mp->d_alpha_kl_inner_core);
    hipFree(mp->d_beta_kl_inner_core);
  }
  hipFree(mp->d_rmass_inner_core);

/*

    if( *OCEANS ){
      if( mp->num_free_surface_faces > 0 ){
        hipFree(mp->d_rmass_ocean_load);
        hipFree(mp->d_free_surface_normal);
        hipFree(mp->d_updated_dof_ocean_load);
        if( *NOISE_TOMOGRAPHY == 0){
          hipFree(mp->d_free_surface_ispec);
          hipFree(mp->d_free_surface_ijk);
        }
      }
    }
  } // ELASTIC_SIMULATION

*/

  // releases previous contexts
  hipDeviceReset();

  // mesh pointer - not needed anymore
  free(mp);
}

