#include "hip/hip_runtime.h"
/*
 !=====================================================================
 !
 !               S p e c f e m 3 D  V e r s i o n  2 . 0
 !               ---------------------------------------
 !
 !          Main authors: Dimitri Komatitsch and Jeroen Tromp
 !    Princeton University, USA and University of Pau / CNRS / INRIA
 ! (c) Princeton University / California Institute of Technology and University of Pau / CNRS / INRIA
 !                            August 2013
 !
 ! This program is free software; you can redistribute it and/or modify
 ! it under the terms of the GNU General Public License as published by
 ! the Free Software Foundation; either version 2 of the License, or
 ! (at your option) any later version.
 !
 ! This program is distributed in the hope that it will be useful,
 ! but WITHOUT ANY WARRANTY; without even the implied warranty of
 ! MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 ! GNU General Public License for more details.
 !
 ! You should have received a copy of the GNU General Public License along
 ! with this program; if not, write to the Free Software Foundation, Inc.,
 ! 51 Franklin Street, Fifth Floor, Boston, MA 02110-1301 USA.
 !
 !=====================================================================
 */

#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hipblas.h>

#include <sys/time.h>
#include <sys/resource.h>

#include "config.h"
#include "mesh_constants_cuda.h"


#ifdef USE_TEXTURES_FIELDS
texture<realw, hipTextureType1D, hipReadModeElementType> d_displ_cm_tex;
texture<realw, hipTextureType1D, hipReadModeElementType> d_accel_cm_tex;
#endif

#ifdef USE_TEXTURES_CONSTANTS
texture<realw, hipTextureType1D, hipReadModeElementType> d_hprime_xx_cm_tex;
#endif


/* ----------------------------------------------------------------------------------------------- */

// elemental routines

/* ----------------------------------------------------------------------------------------------- */

// updates stress

__device__ void compute_element_cm_att_stress(int tx,int working_element,
                                              realw* R_xx,
                                              realw* R_yy,
                                              realw* R_xy,
                                              realw* R_xz,
                                              realw* R_yz,
                                              realw* sigma_xx,
                                              realw* sigma_yy,
                                              realw* sigma_zz,
                                              realw* sigma_xy,
                                              realw* sigma_xz,
                                              realw* sigma_yz) {

  realw R_xx_val,R_yy_val;

  for(int i_sls = 0; i_sls < N_SLS; i_sls++){
    // index
    // note: index for R_xx,.. here is (i_sls,i,j,k,ispec) and not (i,j,k,ispec,i_sls) as in local version
    //          local version: offset_sls = tx + NGLL3*(working_element + NSPEC*i_sls);
    R_xx_val = R_xx[i_sls + N_SLS*(tx + NGLL3*working_element)];
    R_yy_val = R_yy[i_sls + N_SLS*(tx + NGLL3*working_element)];

    *sigma_xx = *sigma_xx - R_xx_val;
    *sigma_yy = *sigma_yy - R_yy_val;
    *sigma_zz = *sigma_zz + R_xx_val + R_yy_val;
    *sigma_xy = *sigma_xy - R_xy[i_sls + N_SLS*(tx + NGLL3*working_element)];
    *sigma_xz = *sigma_xz - R_xz[i_sls + N_SLS*(tx + NGLL3*working_element)];
    *sigma_yz = *sigma_yz - R_yz[i_sls + N_SLS*(tx + NGLL3*working_element)];
  }
}

/* ----------------------------------------------------------------------------------------------- */

// updates R_memory

__device__ void compute_element_cm_att_memory(int tx,int working_element,
                                              realw* d_muvstore,
                                              realw* factor_common,
                                              realw* alphaval,realw* betaval,realw* gammaval,
                                              realw* R_xx,realw* R_yy,realw* R_xy,realw* R_xz,realw* R_yz,
                                              realw* epsilondev_xx,realw* epsilondev_yy,realw* epsilondev_xy,
                                              realw* epsilondev_xz,realw* epsilondev_yz,
                                              realw epsilondev_xx_loc,realw epsilondev_yy_loc,realw epsilondev_xy_loc,
                                              realw epsilondev_xz_loc,realw epsilondev_yz_loc,
                                              realw* d_c44store,
                                              int ANISOTROPY,
                                              int USE_3D_ATTENUATION_ARRAYS) {

  realw fac;
  realw alphaval_loc,betaval_loc,gammaval_loc;
  realw factor_loc,Sn,Snp1;

  // shear moduli for common factor (only Q_mu attenuation)
  if( ANISOTROPY ){
    fac = d_c44store[tx + NGLL3_PADDED * working_element];
  }else{
    fac = d_muvstore[tx + NGLL3_PADDED * working_element];
  }

  // use Runge-Kutta scheme to march in time
  for(int i_sls = 0; i_sls < N_SLS; i_sls++){
    // indices
    // note: index for R_xx,... here is (i_sls,i,j,k,ispec) and not (i,j,k,ispec,i_sls) as in local version
    //          local version: offset_sls = tx + NGLL3*(working_element + NSPEC*i_sls);
    //
    // either mustore(i,j,k,ispec) * factor_common(i_sls,i,j,k,ispec)
    // or       factor_common(i_sls,:,:,:,ispec) * c44store(:,:,:,ispec)
    if( USE_3D_ATTENUATION_ARRAYS ){
      // array dimension: factor_common(N_SLS,NGLLX,NGLLY,NGLLZ,NSPEC)
      factor_loc = fac * factor_common[i_sls + N_SLS*(tx + NGLL3*working_element)];
    }else{
      // array dimension: factor_common(N_SLS,1,1,1,NSPEC)
      factor_loc = fac * factor_common[i_sls + N_SLS*working_element];
    }

    alphaval_loc = alphaval[i_sls]; // (i_sls)
    betaval_loc = betaval[i_sls];
    gammaval_loc = gammaval[i_sls];

    // term in xx
    Sn   = factor_loc * epsilondev_xx[tx + NGLL3 * working_element]; //(i,j,k,ispec)
    Snp1   = factor_loc * epsilondev_xx_loc; //(i,j,k)
    R_xx[i_sls + N_SLS*(tx + NGLL3*working_element)] =
      alphaval_loc * R_xx[i_sls + N_SLS*(tx + NGLL3*working_element)] + betaval_loc * Sn + gammaval_loc * Snp1;

    // term in yy
    Sn   = factor_loc * epsilondev_yy[tx + NGLL3 * working_element];
    Snp1   = factor_loc * epsilondev_yy_loc;
    R_yy[i_sls + N_SLS*(tx + NGLL3*working_element)] =
      alphaval_loc * R_yy[i_sls + N_SLS*(tx + NGLL3*working_element)] + betaval_loc * Sn + gammaval_loc * Snp1;
    // term in zz not computed since zero trace

    // term in xy
    Sn   = factor_loc * epsilondev_xy[tx + NGLL3 * working_element];
    Snp1   = factor_loc * epsilondev_xy_loc;
    R_xy[i_sls + N_SLS*(tx + NGLL3*working_element)] =
      alphaval_loc * R_xy[i_sls + N_SLS*(tx + NGLL3*working_element)] + betaval_loc * Sn + gammaval_loc * Snp1;

    // term in xz
    Sn   = factor_loc * epsilondev_xz[tx + NGLL3 * working_element];
    Snp1   = factor_loc * epsilondev_xz_loc;
    R_xz[i_sls + N_SLS*(tx + NGLL3*working_element)] =
      alphaval_loc * R_xz[i_sls + N_SLS*(tx + NGLL3*working_element)] + betaval_loc * Sn + gammaval_loc * Snp1;

    // term in yz
    Sn   = factor_loc * epsilondev_yz[tx + NGLL3 * working_element];
    Snp1   = factor_loc * epsilondev_yz_loc;
    R_yz[i_sls + N_SLS*(tx + NGLL3*working_element)] =
      alphaval_loc * R_yz[i_sls + N_SLS*(tx + NGLL3*working_element)] + betaval_loc * Sn + gammaval_loc * Snp1;
  }
}

/* ----------------------------------------------------------------------------------------------- */

// pre-computes gravity term

__device__ void compute_element_cm_gravity(int tx,int working_element,
                                          int* d_ibool,
                                          realw* d_xstore,realw* d_ystore,realw* d_zstore,
                                          realw* d_minus_gravity_table,
                                          realw* d_minus_deriv_gravity_table,
                                          realw* d_density_table,
                                          realw* wgll_cube,
                                          realw jacobianl,
                                          realw* s_dummyx_loc,
                                          realw* s_dummyy_loc,
                                          realw* s_dummyz_loc,
                                          realw* sigma_xx,
                                          realw* sigma_yy,
                                          realw* sigma_zz,
                                          realw* sigma_xy,
                                          realw* sigma_yx,
                                          realw* sigma_xz,
                                          realw* sigma_zx,
                                          realw* sigma_yz,
                                          realw* sigma_zy,
                                          realw* rho_s_H1,
                                          realw* rho_s_H2,
                                          realw* rho_s_H3){

  realw radius,theta,phi;
  realw cos_theta,sin_theta,cos_phi,sin_phi;
  realw minus_g,minus_dg;
  realw rho;
  realw gxl,gyl,gzl;
  realw minus_g_over_radius,minus_dg_plus_g_over_radius;
  realw cos_theta_sq,sin_theta_sq,cos_phi_sq,sin_phi_sq;
  realw Hxxl,Hyyl,Hzzl,Hxyl,Hxzl,Hyzl;
  realw sx_l,sy_l,sz_l;
  realw factor;

  // R_EARTH_KM is the radius of the bottom of the oceans (radius of Earth in km)
  //const realw R_EARTH_KM = 6371.0f;
  // uncomment line below for PREM with oceans
  //const realw R_EARTH_KM = 6368.0f;

  // compute non-symmetric terms for gravity

  // use mesh coordinates to get theta and phi
  // x y z contain r theta phi
  int iglob = d_ibool[working_element*NGLL3 + tx]-1;

  radius = d_xstore[iglob];
  theta = d_ystore[iglob];
  phi = d_zstore[iglob];

  if( sizeof( theta ) == sizeof( float ) ){
    // float operations
    // sincos function return sinus and cosine for given value
    sincosf(theta, &sin_theta, &cos_theta);
    sincosf(phi, &sin_phi, &cos_phi);
  }else{
    cos_theta = cos(theta);
    sin_theta = sin(theta);
    cos_phi = cos(phi);
    sin_phi = sin(phi);
  }

  // for efficiency replace with lookup table every 100 m in radial direction
  // note: radius in crust mantle should never be zero,
  //          and arrays in C start from 0, thus we need to subtract -1
  int int_radius = rint(radius * R_EARTH_KM * 10.0f ) - 1;

  // get g, rho and dg/dr=dg
  // spherical components of the gravitational acceleration
  // for efficiency replace with lookup table every 100 m in radial direction
  minus_g = d_minus_gravity_table[int_radius];
  minus_dg = d_minus_deriv_gravity_table[int_radius];
  rho = d_density_table[int_radius];

  // Cartesian components of the gravitational acceleration
  gxl = minus_g*sin_theta*cos_phi;
  gyl = minus_g*sin_theta*sin_phi;
  gzl = minus_g*cos_theta;

  // Cartesian components of gradient of gravitational acceleration
  // obtained from spherical components

  minus_g_over_radius = minus_g / radius;
  minus_dg_plus_g_over_radius = minus_dg - minus_g_over_radius;

  cos_theta_sq = cos_theta*cos_theta;
  sin_theta_sq = sin_theta*sin_theta;
  cos_phi_sq = cos_phi*cos_phi;
  sin_phi_sq = sin_phi*sin_phi;

  Hxxl = minus_g_over_radius*(cos_phi_sq*cos_theta_sq + sin_phi_sq) + cos_phi_sq*minus_dg*sin_theta_sq;
  Hyyl = minus_g_over_radius*(cos_phi_sq + cos_theta_sq*sin_phi_sq) + minus_dg*sin_phi_sq*sin_theta_sq;
  Hzzl = cos_theta_sq*minus_dg + minus_g_over_radius*sin_theta_sq;
  Hxyl = cos_phi*minus_dg_plus_g_over_radius*sin_phi*sin_theta_sq;
  Hxzl = cos_phi*cos_theta*minus_dg_plus_g_over_radius*sin_theta;
  Hyzl = cos_theta*minus_dg_plus_g_over_radius*sin_phi*sin_theta;

  // get displacement and multiply by density to compute G tensor
  sx_l = rho * s_dummyx_loc[tx];
  sy_l = rho * s_dummyy_loc[tx];
  sz_l = rho * s_dummyz_loc[tx];

  // compute G tensor from s . g and add to sigma (not symmetric)
  *sigma_xx = *sigma_xx + sy_l*gyl + sz_l*gzl;
  *sigma_yy = *sigma_yy + sx_l*gxl + sz_l*gzl;
  *sigma_zz = *sigma_zz + sx_l*gxl + sy_l*gyl;

  *sigma_xy = *sigma_xy - sx_l * gyl;
  *sigma_yx = *sigma_yx - sy_l * gxl;

  *sigma_xz = *sigma_xz - sx_l * gzl;
  *sigma_zx = *sigma_zx - sz_l * gxl;

  *sigma_yz = *sigma_yz - sy_l * gzl;
  *sigma_zy = *sigma_zy - sz_l * gyl;

  // precompute vector
  factor = jacobianl * wgll_cube[tx];
  *rho_s_H1 = factor * (sx_l * Hxxl + sy_l * Hxyl + sz_l * Hxzl);
  *rho_s_H2 = factor * (sx_l * Hxyl + sy_l * Hyyl + sz_l * Hyzl);
  *rho_s_H3 = factor * (sx_l * Hxzl + sy_l * Hyzl + sz_l * Hzzl);
}

/* ----------------------------------------------------------------------------------------------- */

// computes stresses for anisotropic element

__device__ void compute_element_cm_aniso(int offset,
                                         realw* d_c11store,realw* d_c12store,realw* d_c13store,
                                         realw* d_c14store,realw* d_c15store,realw* d_c16store,
                                         realw* d_c22store,realw* d_c23store,realw* d_c24store,
                                         realw* d_c25store,realw* d_c26store,realw* d_c33store,
                                         realw* d_c34store,realw* d_c35store,realw* d_c36store,
                                         realw* d_c44store,realw* d_c45store,realw* d_c46store,
                                         realw* d_c55store,realw* d_c56store,realw* d_c66store,
                                         int ATTENUATION,
                                         realw one_minus_sum_beta_use,
                                         realw duxdxl,realw duxdyl,realw duxdzl,
                                         realw duydxl,realw duydyl,realw duydzl,
                                         realw duzdxl,realw duzdyl,realw duzdzl,
                                         realw duxdyl_plus_duydxl,realw duzdxl_plus_duxdzl,realw duzdyl_plus_duydzl,
                                         realw* sigma_xx,realw* sigma_yy,realw* sigma_zz,
                                         realw* sigma_xy,realw* sigma_xz,realw* sigma_yz
                                         ){

  realw c11,c12,c13,c14,c15,c16,c22,c23,c24,c25,c26,c33,c34,c35,c36,c44,c45,c46,c55,c56,c66;
  realw mul,minus_sum_beta;

  c11 = d_c11store[offset];
  c12 = d_c12store[offset];
  c13 = d_c13store[offset];
  c14 = d_c14store[offset];
  c15 = d_c15store[offset];
  c16 = d_c16store[offset];
  c22 = d_c22store[offset];
  c23 = d_c23store[offset];
  c24 = d_c24store[offset];
  c25 = d_c25store[offset];
  c26 = d_c26store[offset];
  c33 = d_c33store[offset];
  c34 = d_c34store[offset];
  c35 = d_c35store[offset];
  c36 = d_c36store[offset];
  c44 = d_c44store[offset];
  c45 = d_c45store[offset];
  c46 = d_c46store[offset];
  c55 = d_c55store[offset];
  c56 = d_c56store[offset];
  c66 = d_c66store[offset];

  // use unrelaxed parameters if attenuation
  if( ATTENUATION){
    minus_sum_beta = one_minus_sum_beta_use - 1.0f;
    mul = c44;

    c11 = c11 + 1.33333333333333333333f * minus_sum_beta * mul;
    c12 = c12 - 0.66666666666666666666f * minus_sum_beta * mul;
    c13 = c13 - 0.66666666666666666666f * minus_sum_beta * mul;
    c22 = c22 + 1.33333333333333333333f * minus_sum_beta * mul;
    c23 = c23 - 0.66666666666666666666f * minus_sum_beta * mul;
    c33 = c33 + 1.33333333333333333333f * minus_sum_beta * mul;
    c44 = c44 + minus_sum_beta * mul;
    c55 = c55 + minus_sum_beta * mul;
    c66 = c66 + minus_sum_beta * mul;
  }

  *sigma_xx = c11*duxdxl + c16*duxdyl_plus_duydxl + c12*duydyl +
             c15*duzdxl_plus_duxdzl + c14*duzdyl_plus_duydzl + c13*duzdzl;
  *sigma_yy = c12*duxdxl + c26*duxdyl_plus_duydxl + c22*duydyl +
             c25*duzdxl_plus_duxdzl + c24*duzdyl_plus_duydzl + c23*duzdzl;
  *sigma_zz = c13*duxdxl + c36*duxdyl_plus_duydxl + c23*duydyl +
             c35*duzdxl_plus_duxdzl + c34*duzdyl_plus_duydzl + c33*duzdzl;
  *sigma_xy = c16*duxdxl + c66*duxdyl_plus_duydxl + c26*duydyl +
             c56*duzdxl_plus_duxdzl + c46*duzdyl_plus_duydzl + c36*duzdzl;
  *sigma_xz = c15*duxdxl + c56*duxdyl_plus_duydxl + c25*duydyl +
             c55*duzdxl_plus_duxdzl + c45*duzdyl_plus_duydzl + c35*duzdzl;
  *sigma_yz = c14*duxdxl + c46*duxdyl_plus_duydxl + c24*duydyl +
             c45*duzdxl_plus_duxdzl + c44*duzdyl_plus_duydzl + c34*duzdzl;
}

/* ----------------------------------------------------------------------------------------------- */

// computes stresses for isotropic element

__device__ void compute_element_cm_iso(int offset,
                                       realw* d_kappavstore,realw* d_muvstore,
                                       int ATTENUATION,
                                       realw one_minus_sum_beta_use,
                                       realw duxdxl,realw duydyl,realw duzdzl,
                                       realw duxdxl_plus_duydyl,realw duxdxl_plus_duzdzl,realw duydyl_plus_duzdzl,
                                       realw duxdyl_plus_duydxl,realw duzdxl_plus_duxdzl,realw duzdyl_plus_duydzl,
                                       realw* sigma_xx,realw* sigma_yy,realw* sigma_zz,
                                       realw* sigma_xy,realw* sigma_xz,realw* sigma_yz){

  realw lambdal,mul,lambdalplus2mul,kappal;

  // compute elements with an elastic isotropic rheology
  kappal = d_kappavstore[offset];
  mul = d_muvstore[offset];

  // use unrelaxed parameters if attenuation
  if( ATTENUATION ){
    mul = mul * one_minus_sum_beta_use;
  }

  lambdalplus2mul = kappal + 1.33333333333333333333f * mul;  // 4./3. = 1.3333333
  lambdal = lambdalplus2mul - 2.0f * mul;

  // compute the six components of the stress tensor sigma
  *sigma_xx = lambdalplus2mul*duxdxl + lambdal*duydyl_plus_duzdzl;
  *sigma_yy = lambdalplus2mul*duydyl + lambdal*duxdxl_plus_duzdzl;
  *sigma_zz = lambdalplus2mul*duzdzl + lambdal*duxdxl_plus_duydyl;

  *sigma_xy = mul*duxdyl_plus_duydxl;
  *sigma_xz = mul*duzdxl_plus_duxdzl;
  *sigma_yz = mul*duzdyl_plus_duydzl;

}

/* ----------------------------------------------------------------------------------------------- */

// computes stresses for transversely isotropic element

__device__ void compute_element_cm_tiso(int offset,
                                        realw* d_kappavstore,realw* d_muvstore,
                                        realw* d_kappahstore,realw* d_muhstore,realw* d_eta_anisostore,
                                        int ATTENUATION,
                                        realw one_minus_sum_beta_use,
                                        realw duxdxl,realw duxdyl,realw duxdzl,
                                        realw duydxl,realw duydyl,realw duydzl,
                                        realw duzdxl,realw duzdyl,realw duzdzl,
                                        realw duxdyl_plus_duydxl,realw duzdxl_plus_duxdzl,realw duzdyl_plus_duydzl,
                                        int iglob,int NGLOB,
                                        realw* d_ystore, realw* d_zstore,
                                        realw* sigma_xx,realw* sigma_yy,realw* sigma_zz,
                                        realw* sigma_xy,realw* sigma_xz,realw* sigma_yz){

  realw kappavl,muvl,kappahl,muhl;
  realw rhovpvsq,rhovphsq,rhovsvsq,rhovshsq,eta_aniso;
  realw costheta,sintheta,cosphi,sinphi;
  realw costhetasq,sinthetasq,cosphisq,sinphisq,costhetafour,sinthetafour,cosphifour,sinphifour;
  realw costwotheta,sintwotheta,costwophi,sintwophi,cosfourtheta,cosfourphi;
  realw costwothetasq,costwophisq,sintwophisq;
  realw etaminone,twoetaminone;
  realw two_eta_aniso,four_eta_aniso,six_eta_aniso;
  realw two_rhovsvsq,two_rhovshsq; // two_rhovpvsq,two_rhovphsq
  realw four_rhovsvsq,four_rhovshsq; // four_rhovpvsq,four_rhovphsq
  realw c11,c12,c13,c14,c15,c16,c22,c23,c24,c25,c26,c33,c34,c35,c36,c44,c45,c46,c55,c56,c66;

  // cosine and sine function in CUDA only supported for float
  realw theta,phi;

  // use Kappa and mu from transversely isotropic model
  kappavl = d_kappavstore[offset];
  muvl = d_muvstore[offset];

  kappahl = d_kappahstore[offset];
  muhl = d_muhstore[offset];

  // use unrelaxed parameters if attenuation
  // eta does not need to be shifted since it is a ratio
  if( ATTENUATION ){
    muvl = muvl * one_minus_sum_beta_use;
    muhl = muhl * one_minus_sum_beta_use;
  }

  rhovpvsq = kappavl + 1.33333333333333333333f * muvl ; //!!! that is C
  rhovphsq = kappahl + 1.33333333333333333333f * muhl ; //!!! that is A

  rhovsvsq = muvl; // !!! that is L
  rhovshsq = muhl; //!!! that is N

  eta_aniso = d_eta_anisostore[offset]; // !!! that is  F / (A - 2 L)

  // use mesh coordinates to get theta and phi
  //ystore and zstore contain theta and phi
  theta = d_ystore[iglob];
  phi = d_zstore[iglob];

  if( sizeof( theta ) == sizeof( float ) ){
    // float operations

    // sincos function return sinus and cosine for given value
    // example:
    //   sincosf(theta, &sintheta, &costheta);
    // or with loss of accuracy:  __sincosf(theta, &sintheta, &costheta);
    // or compile with: -use_fast_math

    //costheta = cosf(theta);
    //sintheta = sinf(theta);
    sincosf(theta, &sintheta, &costheta);

    //cosphi = cosf(phi);
    //sinphi = sinf(phi);
    sincosf(phi, &sinphi, &cosphi);

    //costwotheta = cosf(2.0f * theta);
    //sintwotheta = sinf(2.0f * theta);
    sincosf(2.0f * theta, &sintwotheta, &costwotheta);

    //costwophi = cosf(2.0f * phi);
    //sintwophi = sinf(2.0f * phi);
    sincosf(2.0f * phi, &sintwophi, &costwophi);

    cosfourtheta = cosf(4.0f * theta);
    cosfourphi = cosf(4.0f * phi);

  }else{
    // double operations
    costheta = cos(theta);
    sintheta = sin(theta);

    cosphi = cos(phi);
    sinphi = sin(phi);

    costwotheta = cos(2.0f * theta);
    sintwotheta = sin(2.0f * theta);
    costwophi = cos(2.0f * phi);
    sintwophi = sin(2.0f * phi);

    cosfourtheta = cos(4.0f * theta);
    cosfourphi = cos(4.0f * phi);
  }

  costhetasq = costheta * costheta;
  sinthetasq = sintheta * sintheta;
  cosphisq = cosphi * cosphi;
  sinphisq = sinphi * sinphi;

  costhetafour = costhetasq * costhetasq;
  sinthetafour = sinthetasq * sinthetasq;
  cosphifour = cosphisq * cosphisq;
  sinphifour = sinphisq * sinphisq;

  costwothetasq = costwotheta * costwotheta;

  costwophisq = costwophi * costwophi;
  sintwophisq = sintwophi * sintwophi;

  etaminone = eta_aniso - 1.0f;
  twoetaminone = 2.0f * eta_aniso - 1.0f;

  // precompute some products to reduce the CPU time

  two_eta_aniso = 2.0f * eta_aniso;
  four_eta_aniso = 4.0f * eta_aniso;
  six_eta_aniso = 6.0f * eta_aniso;

  //two_rhovpvsq = 2.0f * rhovpvsq;
  //two_rhovphsq = 2.0f * rhovphsq;
  two_rhovsvsq = 2.0f * rhovsvsq;
  two_rhovshsq = 2.0f * rhovshsq;

  //four_rhovpvsq = 4.0f * rhovpvsq;
  //four_rhovphsq = 4.0f * rhovphsq;
  four_rhovsvsq = 4.0f * rhovsvsq;
  four_rhovshsq = 4.0f * rhovshsq;

  // the 21 anisotropic coefficients computed using Mathematica

  c11 = rhovphsq*sinphifour + 2.0f*cosphisq*sinphisq*
        (rhovphsq*costhetasq + (eta_aniso*rhovphsq + two_rhovsvsq - two_eta_aniso*rhovsvsq)*
        sinthetasq) + cosphifour*
        (rhovphsq*costhetafour + 2.0f*(eta_aniso*rhovphsq + two_rhovsvsq - two_eta_aniso*rhovsvsq)*
        costhetasq*sinthetasq + rhovpvsq*sinthetafour);

  c12 = ((rhovphsq - two_rhovshsq)*(3.0f + cosfourphi)*costhetasq)*0.25f -
        four_rhovshsq*cosphisq*costhetasq*sinphisq +
        (rhovphsq*(11.0f + 4.0f*costwotheta + cosfourtheta)*sintwophisq)*0.03125f +
        eta_aniso*(rhovphsq - two_rhovsvsq)*(cosphifour +
        2.0f*cosphisq*costhetasq*sinphisq + sinphifour)*sinthetasq +
        rhovpvsq*cosphisq*sinphisq*sinthetafour -
        rhovsvsq*sintwophisq*sinthetafour;

  c13 = (cosphisq*(rhovphsq + six_eta_aniso*rhovphsq + rhovpvsq - four_rhovsvsq -
        12.0f*eta_aniso*rhovsvsq + (twoetaminone*rhovphsq - rhovpvsq + four_rhovsvsq -
        four_eta_aniso*rhovsvsq)*cosfourtheta))*0.125f +
        sinphisq*(eta_aniso*(rhovphsq - two_rhovsvsq)*costhetasq +
        (rhovphsq - two_rhovshsq)*sinthetasq);

  c14 = costheta*sinphi*((cosphisq*
        (-rhovphsq + rhovpvsq + four_rhovshsq - four_rhovsvsq +
        (-rhovphsq + two_eta_aniso*rhovphsq - rhovpvsq + four_rhovsvsq -
        four_eta_aniso*rhovsvsq)*costwotheta))*0.5f +
        (etaminone*rhovphsq + 2.0f*(rhovshsq - eta_aniso*rhovsvsq))*sinphisq)* sintheta;

  c15 = cosphi*costheta*((cosphisq* (-rhovphsq + rhovpvsq +
        (twoetaminone*rhovphsq - rhovpvsq + four_rhovsvsq - four_eta_aniso*rhovsvsq)*
        costwotheta))*0.5f + etaminone*(rhovphsq - two_rhovsvsq)*sinphisq)*sintheta;

  c16 = (cosphi*sinphi*(cosphisq* (-rhovphsq + rhovpvsq +
        (-rhovphsq + two_eta_aniso*rhovphsq - rhovpvsq + four_rhovsvsq -
        four_eta_aniso*rhovsvsq)*costwotheta) +
        2.0f*etaminone*(rhovphsq - two_rhovsvsq)*sinphisq)*sinthetasq)*0.5f;

  c22 = rhovphsq*cosphifour + 2.0f*cosphisq*sinphisq*
        (rhovphsq*costhetasq + (eta_aniso*rhovphsq + two_rhovsvsq - two_eta_aniso*rhovsvsq)*
        sinthetasq) + sinphifour*
        (rhovphsq*costhetafour + 2.0f*(eta_aniso*rhovphsq + two_rhovsvsq - two_eta_aniso*rhovsvsq)*
        costhetasq*sinthetasq + rhovpvsq*sinthetafour);

  c23 = ((rhovphsq + six_eta_aniso*rhovphsq + rhovpvsq - four_rhovsvsq - 12.0f*eta_aniso*rhovsvsq +
        (twoetaminone*rhovphsq - rhovpvsq + four_rhovsvsq - four_eta_aniso*rhovsvsq)*
        cosfourtheta)*sinphisq)*0.125f +
        cosphisq*(eta_aniso*(rhovphsq - two_rhovsvsq)*costhetasq +
        (rhovphsq - two_rhovshsq)*sinthetasq);

  c24 = costheta*sinphi*(etaminone*(rhovphsq - two_rhovsvsq)*cosphisq +
        ((-rhovphsq + rhovpvsq + (twoetaminone*rhovphsq - rhovpvsq +
        four_rhovsvsq - four_eta_aniso*rhovsvsq)*costwotheta)*sinphisq)*0.5f)*sintheta;

  c25 = cosphi*costheta*((etaminone*rhovphsq + 2.0f*(rhovshsq - eta_aniso*rhovsvsq))*
        cosphisq + ((-rhovphsq + rhovpvsq + four_rhovshsq - four_rhovsvsq +
        (-rhovphsq + two_eta_aniso*rhovphsq - rhovpvsq + four_rhovsvsq -
        four_eta_aniso*rhovsvsq)*costwotheta)*sinphisq)*0.5f)*sintheta;

  c26 = (cosphi*sinphi*(2.0f*etaminone*(rhovphsq - two_rhovsvsq)*cosphisq +
        (-rhovphsq + rhovpvsq + (-rhovphsq + two_eta_aniso*rhovphsq - rhovpvsq + four_rhovsvsq -
        four_eta_aniso*rhovsvsq)*costwotheta)*sinphisq)*sinthetasq)*0.5f;

  c33 = rhovpvsq*costhetafour + 2.0f*(eta_aniso*(rhovphsq - two_rhovsvsq) + two_rhovsvsq)*
        costhetasq*sinthetasq + rhovphsq*sinthetafour;

  c34 = -((rhovphsq - rhovpvsq + (twoetaminone*rhovphsq - rhovpvsq + four_rhovsvsq
        - four_eta_aniso*rhovsvsq)*costwotheta)*sinphi*sintwotheta)*0.25f;

  c35 = -(cosphi*(rhovphsq - rhovpvsq +
        (twoetaminone*rhovphsq - rhovpvsq + four_rhovsvsq - four_eta_aniso*rhovsvsq)*
        costwotheta)*sintwotheta)*0.25f;

  c36 = -((rhovphsq - rhovpvsq - four_rhovshsq + four_rhovsvsq +
        (twoetaminone*rhovphsq - rhovpvsq + four_rhovsvsq - four_eta_aniso*rhovsvsq)*
        costwotheta)*sintwophi*sinthetasq)*0.25f;

  c44 = cosphisq*(rhovsvsq*costhetasq + rhovshsq*sinthetasq) +
        sinphisq*(rhovsvsq*costwothetasq +
        (rhovphsq - two_eta_aniso*rhovphsq + rhovpvsq + four_eta_aniso*rhovsvsq)*costhetasq* sinthetasq);

  c45 = ((rhovphsq - two_eta_aniso*rhovphsq + rhovpvsq - two_rhovshsq - two_rhovsvsq +
        four_eta_aniso*rhovsvsq + (rhovphsq - two_eta_aniso*rhovphsq + rhovpvsq +
        4.0f*etaminone*rhovsvsq)*costwotheta)*sintwophi*sinthetasq)*0.25f;

  c46 = -(cosphi*costheta*((rhovshsq - rhovsvsq)*cosphisq -
        ((rhovphsq - two_eta_aniso*rhovphsq + rhovpvsq - two_rhovshsq - two_rhovsvsq +
        four_eta_aniso*rhovsvsq + (-rhovphsq + two_eta_aniso*rhovphsq - rhovpvsq +
        four_rhovsvsq - four_eta_aniso*rhovsvsq)*costwotheta)*sinphisq)*0.5f)* sintheta);

  c55 = sinphisq*(rhovsvsq*costhetasq + rhovshsq*sinthetasq) +
        cosphisq*(rhovsvsq*costwothetasq +
        (rhovphsq - two_eta_aniso*rhovphsq + rhovpvsq + four_eta_aniso*rhovsvsq)*costhetasq* sinthetasq);

  c56 = costheta*sinphi*((cosphisq*
        (rhovphsq - two_eta_aniso*rhovphsq + rhovpvsq - two_rhovshsq - two_rhovsvsq +
        four_eta_aniso*rhovsvsq + (-rhovphsq + two_eta_aniso*rhovphsq - rhovpvsq +
        four_rhovsvsq - four_eta_aniso*rhovsvsq)*costwotheta))*0.5f +
        (-rhovshsq + rhovsvsq)*sinphisq)*sintheta;

  c66 = rhovshsq*costwophisq*costhetasq -
        2.0f*(rhovphsq - two_rhovshsq)*cosphisq*costhetasq*sinphisq +
        (rhovphsq*(11.0f + 4.0f*costwotheta + cosfourtheta)*sintwophisq)*0.03125f -
        (rhovsvsq*(-6.0f - 2.0f*cosfourphi + cos(4.0f*phi - 2.0f*theta) - 2.0f*costwotheta +
        cos(2.0f*(2.0f*phi + theta)))*sinthetasq)*0.125f +
        rhovpvsq*cosphisq*sinphisq*sinthetafour -
        (eta_aniso*(rhovphsq - two_rhovsvsq)*sintwophisq*sinthetafour)*0.5f;

  // general expression of stress tensor for full Cijkl with 21 coefficients

  *sigma_xx = c11*duxdxl + c16*duxdyl_plus_duydxl + c12*duydyl +
              c15*duzdxl_plus_duxdzl + c14*duzdyl_plus_duydzl + c13*duzdzl;

  *sigma_yy = c12*duxdxl + c26*duxdyl_plus_duydxl + c22*duydyl +
              c25*duzdxl_plus_duxdzl + c24*duzdyl_plus_duydzl + c23*duzdzl;

  *sigma_zz = c13*duxdxl + c36*duxdyl_plus_duydxl + c23*duydyl +
              c35*duzdxl_plus_duxdzl + c34*duzdyl_plus_duydzl + c33*duzdzl;

  *sigma_xy = c16*duxdxl + c66*duxdyl_plus_duydxl + c26*duydyl +
              c56*duzdxl_plus_duxdzl + c46*duzdyl_plus_duydzl + c36*duzdzl;

  *sigma_xz = c15*duxdxl + c56*duxdyl_plus_duydxl + c25*duydyl +
              c55*duzdxl_plus_duxdzl + c45*duzdyl_plus_duydzl + c35*duzdzl;

  *sigma_yz = c14*duxdxl + c46*duxdyl_plus_duydxl + c24*duydyl +
              c45*duzdxl_plus_duxdzl + c44*duzdyl_plus_duydzl + c34*duzdzl;
}



/* ----------------------------------------------------------------------------------------------- */

// KERNEL 2
//
// for crust_mantle

/* ----------------------------------------------------------------------------------------------- */

__global__ void Kernel_2_crust_mantle_impl(int nb_blocks_to_compute,
                                          int NGLOB,
                                          int* d_ibool,
                                          int* d_ispec_is_tiso,
                                          int* d_phase_ispec_inner,
                                          int num_phase_ispec,
                                          int d_iphase,
                                          realw deltat,
                                          int use_mesh_coloring_gpu,
                                          realw* d_displ,
                                          realw* d_veloc,
                                          realw* d_accel,
                                          realw* d_xix, realw* d_xiy, realw* d_xiz,
                                          realw* d_etax, realw* d_etay, realw* d_etaz,
                                          realw* d_gammax, realw* d_gammay, realw* d_gammaz,
                                          realw* d_hprime_xx,
                                          realw* d_hprimewgll_xx,
                                          realw* d_wgllwgll_xy,realw* d_wgllwgll_xz,realw* d_wgllwgll_yz,
                                          realw* d_kappavstore, realw* d_muvstore,
                                          realw* d_kappahstore, realw* d_muhstore,
                                          realw* d_eta_anisostore,
                                          int COMPUTE_AND_STORE_STRAIN,
                                          realw* epsilondev_xx,realw* epsilondev_yy,realw* epsilondev_xy,
                                          realw* epsilondev_xz,realw* epsilondev_yz,
                                          realw* epsilon_trace_over_3,
                                          int ATTENUATION,
                                          int PARTIAL_PHYS_DISPERSION_ONLY,
                                          int USE_3D_ATTENUATION_ARRAYS,
                                          realw* one_minus_sum_beta,realw* factor_common,
                                          realw* R_xx, realw* R_yy, realw* R_xy, realw* R_xz, realw* R_yz,
                                          realw* alphaval,realw* betaval,realw* gammaval,
                                          int ANISOTROPY,
                                          realw* d_c11store,realw* d_c12store,realw* d_c13store,
                                          realw* d_c14store,realw* d_c15store,realw* d_c16store,
                                          realw* d_c22store,realw* d_c23store,realw* d_c24store,
                                          realw* d_c25store,realw* d_c26store,realw* d_c33store,
                                          realw* d_c34store,realw* d_c35store,realw* d_c36store,
                                          realw* d_c44store,realw* d_c45store,realw* d_c46store,
                                          realw* d_c55store,realw* d_c56store,realw* d_c66store,
                                          int GRAVITY,
                                          realw* d_xstore,realw* d_ystore,realw* d_zstore,
                                          realw* d_minus_gravity_table,
                                          realw* d_minus_deriv_gravity_table,
                                          realw* d_density_table,
                                          realw* wgll_cube,
                                          int NSPEC_CRUST_MANTLE_STRAIN_ONLY){

  // block id == spectral-element id
  int bx = blockIdx.y*gridDim.x+blockIdx.x;
  // thread id == GLL point id
  int tx = threadIdx.x;

  int K = (tx/NGLL2);
  int J = ((tx-K*NGLL2)/NGLLX);
  int I = (tx-K*NGLL2-J*NGLLX);

  int active,offset;
  int iglob;
  int working_element;

  realw tempx1l,tempx2l,tempx3l,tempy1l,tempy2l,tempy3l,tempz1l,tempz2l,tempz3l;
  realw xixl,xiyl,xizl,etaxl,etayl,etazl,gammaxl,gammayl,gammazl,jacobianl;
  realw duxdxl,duxdyl,duxdzl,duydxl,duydyl,duydzl,duzdxl,duzdyl,duzdzl;
  realw duxdxl_plus_duydyl,duxdxl_plus_duzdzl,duydyl_plus_duzdzl;
  realw duxdyl_plus_duydxl,duzdxl_plus_duxdzl,duzdyl_plus_duydzl;
  realw templ;

  realw fac1,fac2,fac3;
  realw one_minus_sum_beta_use;

  realw sigma_xx,sigma_yy,sigma_zz,sigma_xy,sigma_xz,sigma_yz;
  realw epsilondev_xx_loc,epsilondev_yy_loc,epsilondev_xy_loc,epsilondev_xz_loc,epsilondev_yz_loc;
  realw sum_terms1,sum_terms2,sum_terms3;

  // gravity variables
  realw sigma_yx,sigma_zx,sigma_zy;
  realw rho_s_H1,rho_s_H2,rho_s_H3;

#ifndef MANUALLY_UNROLLED_LOOPS
    int l;
#endif

  __shared__ realw s_dummyx_loc[NGLL3];
  __shared__ realw s_dummyy_loc[NGLL3];
  __shared__ realw s_dummyz_loc[NGLL3];

  __shared__ realw s_tempx1[NGLL3];
  __shared__ realw s_tempx2[NGLL3];
  __shared__ realw s_tempx3[NGLL3];

  __shared__ realw s_tempy1[NGLL3];
  __shared__ realw s_tempy2[NGLL3];
  __shared__ realw s_tempy3[NGLL3];

  __shared__ realw s_tempz1[NGLL3];
  __shared__ realw s_tempz2[NGLL3];
  __shared__ realw s_tempz3[NGLL3];

  __shared__ realw sh_hprime_xx[NGLL2];
  __shared__ realw sh_hprimewgll_xx[NGLL2];

// use only NGLL^3 = 125 active threads, plus 3 inactive/ghost threads,
// because we used memory padding from NGLL^3 = 125 to 128 to get coalescent memory accesses
  active = (tx < NGLL3 && bx < nb_blocks_to_compute) ? 1:0;

// copy from global memory to shared memory
// each thread writes one of the NGLL^3 = 125 data points
  if (active) {

#ifdef USE_MESH_COLORING_GPU
    working_element = bx;
#else
    //mesh coloring
    if( use_mesh_coloring_gpu ){
      working_element = bx;
    }else{
      // iphase-1 and working_element-1 for Fortran->C array conventions
      working_element = d_phase_ispec_inner[bx + num_phase_ispec*(d_iphase-1)]-1;
    }
#endif

    // iglob = d_ibool[working_element*NGLL3_PADDED + tx]-1;
    iglob = d_ibool[working_element*NGLL3 + tx]-1;

#ifdef USE_TEXTURES_FIELDS
    s_dummyx_loc[tx] = tex1Dfetch(d_displ_cm_tex, iglob*3);
    s_dummyy_loc[tx] = tex1Dfetch(d_displ_cm_tex, iglob*3 + 1);
    s_dummyz_loc[tx] = tex1Dfetch(d_displ_cm_tex, iglob*3 + 2);
#else
    // changing iglob indexing to match fortran row changes fast style
    s_dummyx_loc[tx] = d_displ[iglob*3];
    s_dummyy_loc[tx] = d_displ[iglob*3 + 1];
    s_dummyz_loc[tx] = d_displ[iglob*3 + 2];
#endif

  } // active

  // gets constant arrays into shared memory
  // (only ghost threads which would be idle anyway)
  if (tx == NGLL3_PADDED-1) {
    for(int m=0; m < NGLL2; m++){
      // hprime
#ifdef USE_TEXTURES_CONSTANTS
      sh_hprime_xx[m] = tex1Dfetch(d_hprime_xx_cm_tex,m);
#else
      sh_hprime_xx[m] = d_hprime_xx[m];
#endif
      // weighted hprime
      sh_hprimewgll_xx[m] = d_hprimewgll_xx[m];
    }
  }

/*
  if (tx < NGLL2) {
    // hprime
#ifdef USE_TEXTURES_CONSTANTS
    sh_hprime_xx[tx] = tex1Dfetch(d_hprime_xx_cm_tex,tx);
#else
    sh_hprime_xx[tx] = d_hprime_xx[tx];
#endif
    // weighted hprime
    sh_hprimewgll_xx[tx] = d_hprimewgll_xx[tx];
  }
*/

// synchronize all the threads (one thread for each of the NGLL grid points of the
// current spectral element) because we need the whole element to be ready in order
// to be able to compute the matrix products along cut planes of the 3D element below
  __syncthreads();

  if (active) {

#ifndef MANUALLY_UNROLLED_LOOPS
    tempx1l = 0.f;
    tempx2l = 0.f;
    tempx3l = 0.f;

    tempy1l = 0.f;
    tempy2l = 0.f;
    tempy3l = 0.f;

    tempz1l = 0.f;
    tempz2l = 0.f;
    tempz3l = 0.f;

    for (l=0;l<NGLLX;l++) {
        fac1 = sh_hprime_xx[l*NGLLX+I];
        tempx1l += s_dummyx_loc[K*NGLL2+J*NGLLX+l]*fac1;
        tempy1l += s_dummyy_loc[K*NGLL2+J*NGLLX+l]*fac1;
        tempz1l += s_dummyz_loc[K*NGLL2+J*NGLLX+l]*fac1;

        fac2 = sh_hprime_xx[l*NGLLX+J];
        tempx2l += s_dummyx_loc[K*NGLL2+l*NGLLX+I]*fac2;
        tempy2l += s_dummyy_loc[K*NGLL2+l*NGLLX+I]*fac2;
        tempz2l += s_dummyz_loc[K*NGLL2+l*NGLLX+I]*fac2;

        fac3 = sh_hprime_xx[l*NGLLX+K];
        tempx3l += s_dummyx_loc[l*NGLL2+J*NGLLX+I]*fac3;
        tempy3l += s_dummyy_loc[l*NGLL2+J*NGLLX+I]*fac3;
        tempz3l += s_dummyz_loc[l*NGLL2+J*NGLLX+I]*fac3;
    }
#else

    tempx1l = s_dummyx_loc[K*NGLL2+J*NGLLX]*sh_hprime_xx[I]
            + s_dummyx_loc[K*NGLL2+J*NGLLX+1]*sh_hprime_xx[NGLLX+I]
            + s_dummyx_loc[K*NGLL2+J*NGLLX+2]*sh_hprime_xx[2*NGLLX+I]
            + s_dummyx_loc[K*NGLL2+J*NGLLX+3]*sh_hprime_xx[3*NGLLX+I]
            + s_dummyx_loc[K*NGLL2+J*NGLLX+4]*sh_hprime_xx[4*NGLLX+I];

    tempy1l = s_dummyy_loc[K*NGLL2+J*NGLLX]*sh_hprime_xx[I]
            + s_dummyy_loc[K*NGLL2+J*NGLLX+1]*sh_hprime_xx[NGLLX+I]
            + s_dummyy_loc[K*NGLL2+J*NGLLX+2]*sh_hprime_xx[2*NGLLX+I]
            + s_dummyy_loc[K*NGLL2+J*NGLLX+3]*sh_hprime_xx[3*NGLLX+I]
            + s_dummyy_loc[K*NGLL2+J*NGLLX+4]*sh_hprime_xx[4*NGLLX+I];

    tempz1l = s_dummyz_loc[K*NGLL2+J*NGLLX]*sh_hprime_xx[I]
            + s_dummyz_loc[K*NGLL2+J*NGLLX+1]*sh_hprime_xx[NGLLX+I]
            + s_dummyz_loc[K*NGLL2+J*NGLLX+2]*sh_hprime_xx[2*NGLLX+I]
            + s_dummyz_loc[K*NGLL2+J*NGLLX+3]*sh_hprime_xx[3*NGLLX+I]
            + s_dummyz_loc[K*NGLL2+J*NGLLX+4]*sh_hprime_xx[4*NGLLX+I];

    tempx2l = s_dummyx_loc[K*NGLL2+I]*sh_hprime_xx[J]
            + s_dummyx_loc[K*NGLL2+NGLLX+I]*sh_hprime_xx[NGLLX+J]
            + s_dummyx_loc[K*NGLL2+2*NGLLX+I]*sh_hprime_xx[2*NGLLX+J]
            + s_dummyx_loc[K*NGLL2+3*NGLLX+I]*sh_hprime_xx[3*NGLLX+J]
            + s_dummyx_loc[K*NGLL2+4*NGLLX+I]*sh_hprime_xx[4*NGLLX+J];

    tempy2l = s_dummyy_loc[K*NGLL2+I]*sh_hprime_xx[J]
            + s_dummyy_loc[K*NGLL2+NGLLX+I]*sh_hprime_xx[NGLLX+J]
            + s_dummyy_loc[K*NGLL2+2*NGLLX+I]*sh_hprime_xx[2*NGLLX+J]
            + s_dummyy_loc[K*NGLL2+3*NGLLX+I]*sh_hprime_xx[3*NGLLX+J]
            + s_dummyy_loc[K*NGLL2+4*NGLLX+I]*sh_hprime_xx[4*NGLLX+J];

    tempz2l = s_dummyz_loc[K*NGLL2+I]*sh_hprime_xx[J]
            + s_dummyz_loc[K*NGLL2+NGLLX+I]*sh_hprime_xx[NGLLX+J]
            + s_dummyz_loc[K*NGLL2+2*NGLLX+I]*sh_hprime_xx[2*NGLLX+J]
            + s_dummyz_loc[K*NGLL2+3*NGLLX+I]*sh_hprime_xx[3*NGLLX+J]
            + s_dummyz_loc[K*NGLL2+4*NGLLX+I]*sh_hprime_xx[4*NGLLX+J];

    tempx3l = s_dummyx_loc[J*NGLLX+I]*sh_hprime_xx[K]
            + s_dummyx_loc[NGLL2+J*NGLLX+I]*sh_hprime_xx[NGLLX+K]
            + s_dummyx_loc[2*NGLL2+J*NGLLX+I]*sh_hprime_xx[2*NGLLX+K]
            + s_dummyx_loc[3*NGLL2+J*NGLLX+I]*sh_hprime_xx[3*NGLLX+K]
            + s_dummyx_loc[4*NGLL2+J*NGLLX+I]*sh_hprime_xx[4*NGLLX+K];

    tempy3l = s_dummyy_loc[J*NGLLX+I]*sh_hprime_xx[K]
            + s_dummyy_loc[NGLL2+J*NGLLX+I]*sh_hprime_xx[NGLLX+K]
            + s_dummyy_loc[2*NGLL2+J*NGLLX+I]*sh_hprime_xx[2*NGLLX+K]
            + s_dummyy_loc[3*NGLL2+J*NGLLX+I]*sh_hprime_xx[3*NGLLX+K]
            + s_dummyy_loc[4*NGLL2+J*NGLLX+I]*sh_hprime_xx[4*NGLLX+K];

    tempz3l = s_dummyz_loc[J*NGLLX+I]*sh_hprime_xx[K]
            + s_dummyz_loc[NGLL2+J*NGLLX+I]*sh_hprime_xx[NGLLX+K]
            + s_dummyz_loc[2*NGLL2+J*NGLLX+I]*sh_hprime_xx[2*NGLLX+K]
            + s_dummyz_loc[3*NGLL2+J*NGLLX+I]*sh_hprime_xx[3*NGLLX+K]
            + s_dummyz_loc[4*NGLL2+J*NGLLX+I]*sh_hprime_xx[4*NGLLX+K];


#endif

    // compute derivatives of ux, uy and uz with respect to x, y and z
    offset = working_element*NGLL3_PADDED + tx;

    xixl = d_xix[offset];
    xiyl = d_xiy[offset];
    xizl = d_xiz[offset];
    etaxl = d_etax[offset];
    etayl = d_etay[offset];
    etazl = d_etaz[offset];
    gammaxl = d_gammax[offset];
    gammayl = d_gammay[offset];
    gammazl = d_gammaz[offset];

    duxdxl = xixl*tempx1l + etaxl*tempx2l + gammaxl*tempx3l;
    duxdyl = xiyl*tempx1l + etayl*tempx2l + gammayl*tempx3l;
    duxdzl = xizl*tempx1l + etazl*tempx2l + gammazl*tempx3l;

    duydxl = xixl*tempy1l + etaxl*tempy2l + gammaxl*tempy3l;
    duydyl = xiyl*tempy1l + etayl*tempy2l + gammayl*tempy3l;
    duydzl = xizl*tempy1l + etazl*tempy2l + gammazl*tempy3l;

    duzdxl = xixl*tempz1l + etaxl*tempz2l + gammaxl*tempz3l;
    duzdyl = xiyl*tempz1l + etayl*tempz2l + gammayl*tempz3l;
    duzdzl = xizl*tempz1l + etazl*tempz2l + gammazl*tempz3l;

    // precompute some sums to save CPU time
    duxdxl_plus_duydyl = duxdxl + duydyl;
    duxdxl_plus_duzdzl = duxdxl + duzdzl;
    duydyl_plus_duzdzl = duydyl + duzdzl;
    duxdyl_plus_duydxl = duxdyl + duydxl;
    duzdxl_plus_duxdzl = duzdxl + duxdzl;
    duzdyl_plus_duydzl = duzdyl + duydzl;

    // computes deviatoric strain attenuation and/or for kernel calculations
    if(COMPUTE_AND_STORE_STRAIN) {
      templ = 0.33333333333333333333f * (duxdxl + duydyl + duzdzl); // 1./3. = 0.33333

      // local storage: stresses at this current time step
      epsilondev_xx_loc = duxdxl - templ;
      epsilondev_yy_loc = duydyl - templ;
      epsilondev_xy_loc = 0.5f * duxdyl_plus_duydxl;
      epsilondev_xz_loc = 0.5f * duzdxl_plus_duxdzl;
      epsilondev_yz_loc = 0.5f * duzdyl_plus_duydzl;

      if(NSPEC_CRUST_MANTLE_STRAIN_ONLY == 1) {
        epsilon_trace_over_3[tx] = templ;
      }else{
        epsilon_trace_over_3[tx + working_element*NGLL3] = templ;
      }
    }

    // attenuation
    if(ATTENUATION){
      // use unrelaxed parameters if attenuation
      if( USE_3D_ATTENUATION_ARRAYS ){
        one_minus_sum_beta_use = one_minus_sum_beta[tx+working_element*NGLL3]; // (i,j,k,ispec)
      }else{
        one_minus_sum_beta_use = one_minus_sum_beta[working_element]; // (1,1,1,ispec)
      }
    }

    // computes stresses
    if(ANISOTROPY){
      // full anisotropic case, stress calculations
      compute_element_cm_aniso(offset,
                            d_c11store,d_c12store,d_c13store,d_c14store,d_c15store,d_c16store,d_c22store,
                            d_c23store,d_c24store,d_c25store,d_c26store,d_c33store,d_c34store,d_c35store,
                            d_c36store,d_c44store,d_c45store,d_c46store,d_c55store,d_c56store,d_c66store,
                            ATTENUATION,
                            one_minus_sum_beta_use,
                            duxdxl,duxdyl,duxdzl,duydxl,duydyl,duydzl,duzdxl,duzdyl,duzdzl,
                            duxdyl_plus_duydxl,duzdxl_plus_duxdzl,duzdyl_plus_duydzl,
                            &sigma_xx,&sigma_yy,&sigma_zz,
                            &sigma_xy,&sigma_xz,&sigma_yz);

    }else{
      if( ! d_ispec_is_tiso[working_element] ){
        // isotropic case
        compute_element_cm_iso(offset,
                            d_kappavstore,d_muvstore,
                            ATTENUATION,
                            one_minus_sum_beta_use,
                            duxdxl,duydyl,duzdzl,
                            duxdxl_plus_duydyl,duxdxl_plus_duzdzl,duydyl_plus_duzdzl,
                            duxdyl_plus_duydxl,duzdxl_plus_duxdzl,duzdyl_plus_duydzl,
                            &sigma_xx,&sigma_yy,&sigma_zz,
                            &sigma_xy,&sigma_xz,&sigma_yz);
      }else{
        // transverse isotropy
        compute_element_cm_tiso(offset,
                              d_kappavstore,d_muvstore,
                              d_kappahstore,d_muhstore,d_eta_anisostore,
                              ATTENUATION,
                              one_minus_sum_beta_use,
                              duxdxl,duxdyl,duxdzl,
                              duydxl,duydyl,duydzl,
                              duzdxl,duzdyl,duzdzl,
                              duxdyl_plus_duydxl,duzdxl_plus_duxdzl,duzdyl_plus_duydzl,
                              iglob, NGLOB,
                              d_ystore,d_zstore,
                              &sigma_xx,&sigma_yy,&sigma_zz,
                              &sigma_xy,&sigma_xz,&sigma_yz);
      }
    } // ! end of test whether isotropic or anisotropic element

    if(ATTENUATION && (! PARTIAL_PHYS_DISPERSION_ONLY ) ){
      // subtracts memory variables if attenuation
      compute_element_cm_att_stress(tx,working_element,
                                    R_xx,R_yy,R_xy,R_xz,R_yz,
                                    &sigma_xx,&sigma_yy,&sigma_zz,&sigma_xy,&sigma_xz,&sigma_yz);
    }

    // define symmetric components (needed for non-symmetric dot product and sigma for gravity)
    sigma_yx = sigma_xy;
    sigma_zx = sigma_xz;
    sigma_zy = sigma_yz;

    // jacobian
    jacobianl = 1.0f / (xixl*(etayl*gammazl-etazl*gammayl)
                     - xiyl*(etaxl*gammazl-etazl*gammaxl)
                     + xizl*(etaxl*gammayl-etayl*gammaxl));

    if( GRAVITY ){
      //  computes non-symmetric terms for gravity
      compute_element_cm_gravity(tx,working_element,
                                 d_ibool,d_xstore,d_ystore,d_zstore,
                                 d_minus_gravity_table,d_minus_deriv_gravity_table,d_density_table,
                                 wgll_cube,jacobianl,
                                 s_dummyx_loc,s_dummyy_loc,s_dummyz_loc,
                                 &sigma_xx,&sigma_yy,&sigma_zz,&sigma_xy,&sigma_yx,
                                 &sigma_xz,&sigma_zx,&sigma_yz,&sigma_zy,
                                 &rho_s_H1,&rho_s_H2,&rho_s_H3);
    }

    // form dot product with test vector, non-symmetric form
    s_tempx1[tx] = jacobianl * (sigma_xx*xixl + sigma_yx*xiyl + sigma_zx*xizl);
    s_tempy1[tx] = jacobianl * (sigma_xy*xixl + sigma_yy*xiyl + sigma_zy*xizl);
    s_tempz1[tx] = jacobianl * (sigma_xz*xixl + sigma_yz*xiyl + sigma_zz*xizl);

    s_tempx2[tx] = jacobianl * (sigma_xx*etaxl + sigma_yx*etayl + sigma_zx*etazl);
    s_tempy2[tx] = jacobianl * (sigma_xy*etaxl + sigma_yy*etayl + sigma_zy*etazl);
    s_tempz2[tx] = jacobianl * (sigma_xz*etaxl + sigma_yz*etayl + sigma_zz*etazl);

    s_tempx3[tx] = jacobianl * (sigma_xx*gammaxl + sigma_yx*gammayl + sigma_zx*gammazl);
    s_tempy3[tx] = jacobianl * (sigma_xy*gammaxl + sigma_yy*gammayl + sigma_zy*gammazl);
    s_tempz3[tx] = jacobianl * (sigma_xz*gammaxl + sigma_yz*gammayl + sigma_zz*gammazl);

  }

// synchronize all the threads (one thread for each of the NGLL grid points of the
// current spectral element) because we need the whole element to be ready in order
// to be able to compute the matrix products along cut planes of the 3D element below
  __syncthreads();

  if (active) {

#ifndef MANUALLY_UNROLLED_LOOPS
    tempx1l = 0.f;
    tempy1l = 0.f;
    tempz1l = 0.f;

    tempx2l = 0.f;
    tempy2l = 0.f;
    tempz2l = 0.f;

    tempx3l = 0.f;
    tempy3l = 0.f;
    tempz3l = 0.f;

    for (l=0;l<NGLLX;l++) {
      fac1 = sh_hprimewgll_xx[I*NGLLX+l];
      tempx1l += s_tempx1[K*NGLL2+J*NGLLX+l]*fac1;
      tempy1l += s_tempy1[K*NGLL2+J*NGLLX+l]*fac1;
      tempz1l += s_tempz1[K*NGLL2+J*NGLLX+l]*fac1;

      // assume hprimewgll_xx == hprimewgll_yy == hprimewgll_zz
      fac2 = sh_hprimewgll_xx[J*NGLLX+l];
      tempx2l += s_tempx2[K*NGLL2+l*NGLLX+I]*fac2;
      tempy2l += s_tempy2[K*NGLL2+l*NGLLX+I]*fac2;
      tempz2l += s_tempz2[K*NGLL2+l*NGLLX+I]*fac2;

      fac3 = sh_hprimewgll_xx[K*NGLLX+l];
      tempx3l += s_tempx3[l*NGLL2+J*NGLLX+I]*fac3;
      tempy3l += s_tempy3[l*NGLL2+J*NGLLX+I]*fac3;
      tempz3l += s_tempz3[l*NGLL2+J*NGLLX+I]*fac3;
    }
#else

    tempx1l = s_tempx1[K*NGLL2+J*NGLLX]*sh_hprimewgll_xx[I*NGLLX]
            + s_tempx1[K*NGLL2+J*NGLLX+1]*sh_hprimewgll_xx[I*NGLLX+1]
            + s_tempx1[K*NGLL2+J*NGLLX+2]*sh_hprimewgll_xx[I*NGLLX+2]
            + s_tempx1[K*NGLL2+J*NGLLX+3]*sh_hprimewgll_xx[I*NGLLX+3]
            + s_tempx1[K*NGLL2+J*NGLLX+4]*sh_hprimewgll_xx[I*NGLLX+4];

    tempy1l = s_tempy1[K*NGLL2+J*NGLLX]*sh_hprimewgll_xx[I*NGLLX]
            + s_tempy1[K*NGLL2+J*NGLLX+1]*sh_hprimewgll_xx[I*NGLLX+1]
            + s_tempy1[K*NGLL2+J*NGLLX+2]*sh_hprimewgll_xx[I*NGLLX+2]
            + s_tempy1[K*NGLL2+J*NGLLX+3]*sh_hprimewgll_xx[I*NGLLX+3]
            + s_tempy1[K*NGLL2+J*NGLLX+4]*sh_hprimewgll_xx[I*NGLLX+4];

    tempz1l = s_tempz1[K*NGLL2+J*NGLLX]*sh_hprimewgll_xx[I*NGLLX]
            + s_tempz1[K*NGLL2+J*NGLLX+1]*sh_hprimewgll_xx[I*NGLLX+1]
            + s_tempz1[K*NGLL2+J*NGLLX+2]*sh_hprimewgll_xx[I*NGLLX+2]
            + s_tempz1[K*NGLL2+J*NGLLX+3]*sh_hprimewgll_xx[I*NGLLX+3]
            + s_tempz1[K*NGLL2+J*NGLLX+4]*sh_hprimewgll_xx[I*NGLLX+4];

    tempx2l = s_tempx2[K*NGLL2+I]*sh_hprimewgll_xx[J*NGLLX]
            + s_tempx2[K*NGLL2+NGLLX+I]*sh_hprimewgll_xx[J*NGLLX+1]
            + s_tempx2[K*NGLL2+2*NGLLX+I]*sh_hprimewgll_xx[J*NGLLX+2]
            + s_tempx2[K*NGLL2+3*NGLLX+I]*sh_hprimewgll_xx[J*NGLLX+3]
            + s_tempx2[K*NGLL2+4*NGLLX+I]*sh_hprimewgll_xx[J*NGLLX+4];

    tempy2l = s_tempy2[K*NGLL2+I]*sh_hprimewgll_xx[J*NGLLX]
            + s_tempy2[K*NGLL2+NGLLX+I]*sh_hprimewgll_xx[J*NGLLX+1]
            + s_tempy2[K*NGLL2+2*NGLLX+I]*sh_hprimewgll_xx[J*NGLLX+2]
            + s_tempy2[K*NGLL2+3*NGLLX+I]*sh_hprimewgll_xx[J*NGLLX+3]
            + s_tempy2[K*NGLL2+4*NGLLX+I]*sh_hprimewgll_xx[J*NGLLX+4];

    tempz2l = s_tempz2[K*NGLL2+I]*sh_hprimewgll_xx[J*NGLLX]
            + s_tempz2[K*NGLL2+NGLLX+I]*sh_hprimewgll_xx[J*NGLLX+1]
            + s_tempz2[K*NGLL2+2*NGLLX+I]*sh_hprimewgll_xx[J*NGLLX+2]
            + s_tempz2[K*NGLL2+3*NGLLX+I]*sh_hprimewgll_xx[J*NGLLX+3]
            + s_tempz2[K*NGLL2+4*NGLLX+I]*sh_hprimewgll_xx[J*NGLLX+4];

    tempx3l = s_tempx3[J*NGLLX+I]*sh_hprimewgll_xx[K*NGLLX]
            + s_tempx3[NGLL2+J*NGLLX+I]*sh_hprimewgll_xx[K*NGLLX+1]
            + s_tempx3[2*NGLL2+J*NGLLX+I]*sh_hprimewgll_xx[K*NGLLX+2]
            + s_tempx3[3*NGLL2+J*NGLLX+I]*sh_hprimewgll_xx[K*NGLLX+3]
            + s_tempx3[4*NGLL2+J*NGLLX+I]*sh_hprimewgll_xx[K*NGLLX+4];

    tempy3l = s_tempy3[J*NGLLX+I]*sh_hprimewgll_xx[K*NGLLX]
            + s_tempy3[NGLL2+J*NGLLX+I]*sh_hprimewgll_xx[K*NGLLX+1]
            + s_tempy3[2*NGLL2+J*NGLLX+I]*sh_hprimewgll_xx[K*NGLLX+2]
            + s_tempy3[3*NGLL2+J*NGLLX+I]*sh_hprimewgll_xx[K*NGLLX+3]
            + s_tempy3[4*NGLL2+J*NGLLX+I]*sh_hprimewgll_xx[K*NGLLX+4];

    tempz3l = s_tempz3[J*NGLLX+I]*sh_hprimewgll_xx[K*NGLLX]
            + s_tempz3[NGLL2+J*NGLLX+I]*sh_hprimewgll_xx[K*NGLLX+1]
            + s_tempz3[2*NGLL2+J*NGLLX+I]*sh_hprimewgll_xx[K*NGLLX+2]
            + s_tempz3[3*NGLL2+J*NGLLX+I]*sh_hprimewgll_xx[K*NGLLX+3]
            + s_tempz3[4*NGLL2+J*NGLLX+I]*sh_hprimewgll_xx[K*NGLLX+4];

#endif

    fac1 = d_wgllwgll_yz[K*NGLLX+J];
    fac2 = d_wgllwgll_xz[K*NGLLX+I];
    fac3 = d_wgllwgll_xy[J*NGLLX+I];

    sum_terms1 = - (fac1*tempx1l + fac2*tempx2l + fac3*tempx3l);
    sum_terms2 = - (fac1*tempy1l + fac2*tempy2l + fac3*tempy3l);
    sum_terms3 = - (fac1*tempz1l + fac2*tempz2l + fac3*tempz3l);

    // adds gravity term
    if( GRAVITY ){
      sum_terms1 += rho_s_H1;
      sum_terms2 += rho_s_H2;
      sum_terms3 += rho_s_H3;
    }


#ifdef USE_MESH_COLORING_GPU
    // no atomic operation needed, colors don't share global points between elements

#ifdef USE_TEXTURES_FIELDS
    d_accel[iglob*3]     = tex1Dfetch(d_accel_cm_tex, iglob*3) + sum_terms1;
    d_accel[iglob*3 + 1] = tex1Dfetch(d_accel_cm_tex, iglob*3 + 1) + sum_terms2;
    d_accel[iglob*3 + 2] = tex1Dfetch(d_accel_cm_tex, iglob*3 + 2) + sum_terms3;
#else
    d_accel[iglob*3]     += sum_terms1;
    d_accel[iglob*3 + 1] += sum_terms2;
    d_accel[iglob*3 + 2] += sum_terms3;
#endif // USE_TEXTURES_FIELDS

#else // MESH_COLORING

    //mesh coloring
    if( use_mesh_coloring_gpu ){

      // no atomic operation needed, colors don't share global points between elements
#ifdef USE_TEXTURES_FIELDS
      d_accel[iglob*3]     = tex1Dfetch(d_accel_cm_tex, iglob*3) + sum_terms1;
      d_accel[iglob*3 + 1] = tex1Dfetch(d_accel_cm_tex, iglob*3 + 1) + sum_terms2;
      d_accel[iglob*3 + 2] = tex1Dfetch(d_accel_cm_tex, iglob*3 + 2) + sum_terms3;
#else
      d_accel[iglob*3]     += sum_terms1;
      d_accel[iglob*3 + 1] += sum_terms2;
      d_accel[iglob*3 + 2] += sum_terms3;
#endif // USE_TEXTURES_FIELDS

    }else{
      // no mesh coloring uses atomic updates

      atomicAdd(&d_accel[iglob*3], sum_terms1);
      atomicAdd(&d_accel[iglob*3 + 1], sum_terms2);
      atomicAdd(&d_accel[iglob*3 + 2], sum_terms3);

      // debug: for testing purposes only: w/out atomic updates
      //d_accel[iglob*3] -= (0.00000001f*tempx1l + 0.00000001f*tempx2l + 0.00000001f*tempx3l);
      //d_accel[iglob*3 + 1] -= (0.00000001f*tempy1l + 0.00000001f*tempy2l + 0.00000001f*tempy3l);
      //d_accel[iglob*3 + 2] -= (0.00000001f*tempz1l + 0.00000001f*tempz2l + 0.00000001f*tempz3l);
    }
#endif // MESH_COLORING

    // update memory variables based upon the Runge-Kutta scheme
    if( ATTENUATION && ( ! PARTIAL_PHYS_DISPERSION_ONLY ) ){

      compute_element_cm_att_memory(tx,working_element,
                                    d_muvstore,
                                    factor_common,alphaval,betaval,gammaval,
                                    R_xx,R_yy,R_xy,R_xz,R_yz,
                                    epsilondev_xx,epsilondev_yy,epsilondev_xy,
                                    epsilondev_xz,epsilondev_yz,
                                    epsilondev_xx_loc,epsilondev_yy_loc,epsilondev_xy_loc,
                                    epsilondev_xz_loc,epsilondev_yz_loc,
                                    d_c44store,ANISOTROPY,USE_3D_ATTENUATION_ARRAYS);
    }

    // save deviatoric strain for Runge-Kutta scheme
    if( COMPUTE_AND_STORE_STRAIN ){
      // fortran: epsilondev_xx(:,:,:,ispec) = epsilondev_xx_loc(:,:,:)
      epsilondev_xx[tx + working_element*NGLL3] = epsilondev_xx_loc;
      epsilondev_yy[tx + working_element*NGLL3] = epsilondev_yy_loc;
      epsilondev_xy[tx + working_element*NGLL3] = epsilondev_xy_loc;
      epsilondev_xz[tx + working_element*NGLL3] = epsilondev_xz_loc;
      epsilondev_yz[tx + working_element*NGLL3] = epsilondev_yz_loc;
    }
  } // active
}

/* ----------------------------------------------------------------------------------------------- */

void Kernel_2_crust_mantle(int nb_blocks_to_compute,Mesh* mp,
                          int d_iphase,
                          int* d_ibool,
                          int* d_ispec_is_tiso,
                          realw* d_xix,realw* d_xiy,realw* d_xiz,
                          realw* d_etax,realw* d_etay,realw* d_etaz,
                          realw* d_gammax,realw* d_gammay,realw* d_gammaz,
                          realw* d_kappavstore,realw* d_muvstore,
                          realw* d_kappahstore,realw* d_muhstore,
                          realw* d_eta_anisostore,
                          realw* d_epsilondev_xx,
                          realw* d_epsilondev_yy,
                          realw* d_epsilondev_xy,
                          realw* d_epsilondev_xz,
                          realw* d_epsilondev_yz,
                          realw* d_epsilon_trace_over_3,
                          realw* d_one_minus_sum_beta,
                          realw* d_factor_common,
                          realw* d_R_xx,
                          realw* d_R_yy,
                          realw* d_R_xy,
                          realw* d_R_xz,
                          realw* d_R_yz,
                          realw* d_c11store,realw* d_c12store,realw* d_c13store,
                          realw* d_c14store,realw* d_c15store,realw* d_c16store,
                          realw* d_c22store,realw* d_c23store,realw* d_c24store,
                          realw* d_c25store,realw* d_c26store,realw* d_c33store,
                          realw* d_c34store,realw* d_c35store,realw* d_c36store,
                          realw* d_c44store,realw* d_c45store,realw* d_c46store,
                          realw* d_c55store,realw* d_c56store,realw* d_c66store,
                          realw* d_b_epsilondev_xx,
                          realw* d_b_epsilondev_yy,
                          realw* d_b_epsilondev_xy,
                          realw* d_b_epsilondev_xz,
                          realw* d_b_epsilondev_yz,
                          realw* d_b_epsilon_trace_over_3,
                          realw* d_b_R_xx,
                          realw* d_b_R_yy,
                          realw* d_b_R_xy,
                          realw* d_b_R_xz,
                          realw* d_b_R_yz,
                          int FORWARD_OR_ADJOINT){

#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
  exit_on_cuda_error("before kernel Kernel_2_crust_mantle");
#endif

  // if the grid can handle the number of blocks, we let it be 1D
  // grid_2_x = nb_elem_color;
  // nb_elem_color is just how many blocks we are computing now

  int blocksize = NGLL3_PADDED;

  int num_blocks_x, num_blocks_y;
  get_blocks_xy(nb_blocks_to_compute,&num_blocks_x,&num_blocks_y);

  dim3 grid(num_blocks_x,num_blocks_y);
  dim3 threads(blocksize,1,1);

  // Cuda timing
  // hipEvent_t start, stop;
  // realw time;
  // hipEventCreate(&start);
  // hipEventCreate(&stop);
  // hipEventRecord( start, 0 );

  if( FORWARD_OR_ADJOINT == 1 ){
    Kernel_2_crust_mantle_impl<<<grid,threads>>>(nb_blocks_to_compute,
                                                  mp->NGLOB_CRUST_MANTLE,
                                                  d_ibool,
                                                  d_ispec_is_tiso,
                                                  mp->d_phase_ispec_inner_crust_mantle,
                                                  mp->num_phase_ispec_crust_mantle,
                                                  d_iphase,
                                                  mp->deltat,
                                                  mp->use_mesh_coloring_gpu,
                                                  mp->d_displ_crust_mantle,
                                                  mp->d_veloc_crust_mantle,
                                                  mp->d_accel_crust_mantle,
                                                  d_xix, d_xiy, d_xiz,
                                                  d_etax, d_etay, d_etaz,
                                                  d_gammax, d_gammay, d_gammaz,
                                                  mp->d_hprime_xx,
                                                  mp->d_hprimewgll_xx,
                                                  mp->d_wgllwgll_xy, mp->d_wgllwgll_xz, mp->d_wgllwgll_yz,
                                                  d_kappavstore, d_muvstore,
                                                  d_kappahstore, d_muhstore,
                                                  d_eta_anisostore,
                                                  mp->compute_and_store_strain,
                                                  d_epsilondev_xx,d_epsilondev_yy,d_epsilondev_xy,
                                                  d_epsilondev_xz,d_epsilondev_yz,
                                                  d_epsilon_trace_over_3,
                                                  mp->attenuation,
                                                  mp->partial_phys_dispersion_only,
                                                  mp->use_3d_attenuation_arrays,
                                                  d_one_minus_sum_beta,d_factor_common,
                                                  d_R_xx,d_R_yy,d_R_xy,d_R_xz,d_R_yz,
                                                  mp->d_alphaval,mp->d_betaval,mp->d_gammaval,
                                                  mp->anisotropic_3D_mantle,
                                                  d_c11store,d_c12store,d_c13store,
                                                  d_c14store,d_c15store,d_c16store,
                                                  d_c22store,d_c23store,d_c24store,
                                                  d_c25store,d_c26store,d_c33store,
                                                  d_c34store,d_c35store,d_c36store,
                                                  d_c44store,d_c45store,d_c46store,
                                                  d_c55store,d_c56store,d_c66store,
                                                  mp->gravity,
                                                  mp->d_xstore_crust_mantle,mp->d_ystore_crust_mantle,mp->d_zstore_crust_mantle,
                                                  mp->d_minus_gravity_table,
                                                  mp->d_minus_deriv_gravity_table,
                                                  mp->d_density_table,
                                                  mp->d_wgll_cube,
                                                  mp->NSPEC_CRUST_MANTLE_STRAIN_ONLY);
  }else if( FORWARD_OR_ADJOINT == 3 ){
    // debug
    DEBUG_BACKWARD_FORCES();

    Kernel_2_crust_mantle_impl<<< grid,threads>>>(nb_blocks_to_compute,
                                                   mp->NGLOB_CRUST_MANTLE,
                                                   d_ibool,
                                                   d_ispec_is_tiso,
                                                   mp->d_phase_ispec_inner_crust_mantle,
                                                   mp->num_phase_ispec_crust_mantle,
                                                   d_iphase,
                                                   mp->b_deltat,
                                                   mp->use_mesh_coloring_gpu,
                                                   mp->d_b_displ_crust_mantle,
                                                   mp->d_b_veloc_crust_mantle,
                                                   mp->d_b_accel_crust_mantle,
                                                   d_xix, d_xiy, d_xiz,
                                                   d_etax, d_etay, d_etaz,
                                                   d_gammax, d_gammay, d_gammaz,
                                                   mp->d_hprime_xx,
                                                   mp->d_hprimewgll_xx,
                                                   mp->d_wgllwgll_xy, mp->d_wgllwgll_xz, mp->d_wgllwgll_yz,
                                                   d_kappavstore, d_muvstore,
                                                   d_kappahstore, d_muhstore,
                                                   d_eta_anisostore,
                                                   mp->compute_and_store_strain,
                                                   d_b_epsilondev_xx,d_b_epsilondev_yy,d_b_epsilondev_xy,
                                                   d_b_epsilondev_xz,d_b_epsilondev_yz,
                                                   d_b_epsilon_trace_over_3,
                                                   mp->attenuation,
                                                   mp->partial_phys_dispersion_only,
                                                   mp->use_3d_attenuation_arrays,
                                                   d_one_minus_sum_beta,d_factor_common,
                                                   d_b_R_xx,d_b_R_yy,d_b_R_xy,d_b_R_xz,d_b_R_yz,
                                                   mp->d_b_alphaval,mp->d_b_betaval,mp->d_b_gammaval,
                                                   mp->anisotropic_3D_mantle,
                                                   d_c11store,d_c12store,d_c13store,
                                                   d_c14store,d_c15store,d_c16store,
                                                   d_c22store,d_c23store,d_c24store,
                                                   d_c25store,d_c26store,d_c33store,
                                                   d_c34store,d_c35store,d_c36store,
                                                   d_c44store,d_c45store,d_c46store,
                                                   d_c55store,d_c56store,d_c66store,
                                                   mp->gravity,
                                                   mp->d_xstore_crust_mantle,mp->d_ystore_crust_mantle,mp->d_zstore_crust_mantle,
                                                   mp->d_minus_gravity_table,
                                                   mp->d_minus_deriv_gravity_table,
                                                   mp->d_density_table,
                                                   mp->d_wgll_cube,
                                                   mp->NSPEC_CRUST_MANTLE_STRAIN_ONLY);
  }

  // hipEventRecord( stop, 0 );
  // hipEventSynchronize( stop );
  // hipEventElapsedTime( &time, start, stop );
  // hipEventDestroy( start );
  // hipEventDestroy( stop );
  // printf("Kernel2 Execution Time: %f ms\n",time);

  /* hipDeviceSynchronize(); */
  /* LOG("Kernel 2 finished"); */
#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
  exit_on_cuda_error("Kernel_2_crust_mantle");
#endif
}

/* ----------------------------------------------------------------------------------------------- */


extern "C"
void FC_FUNC_(compute_forces_crust_mantle_cuda,
              COMPUTE_FORCES_CRUST_MANTLE_CUDA)(long* Mesh_pointer_f,
                                                int* iphase,
                                                int* FORWARD_OR_ADJOINT_f) {

  TRACE("compute_forces_crust_mantle_cuda");

//debug time
//  printf("Running compute_forces\n");
//  double start_time = get_time();

  Mesh* mp = (Mesh*)(*Mesh_pointer_f); // get Mesh from fortran integer wrapper

  int FORWARD_OR_ADJOINT = *FORWARD_OR_ADJOINT_f;

  int num_elements;

  if( *iphase == 1 )
    num_elements = mp->nspec_outer_crust_mantle;
  else
    num_elements = mp->nspec_inner_crust_mantle;

  // checks if anything to do
  if( num_elements == 0 ) return;

  // mesh coloring
  if( mp->use_mesh_coloring_gpu ){

    // note: array offsets require sorted arrays, such that e.g. ibool starts with elastic elements
    //         and followed by acoustic ones.
    //         elastic elements also start with outer than inner element ordering

    int nb_colors,nb_blocks_to_compute;
    int istart;
    int offset,offset_nonpadded;
    int offset_nonpadded_att1,offset_nonpadded_att2,offset_nonpadded_att3;
    int offset_nonpadded_strain;
    int offset_ispec;

    // sets up color loop
    if( *iphase == 1 ){
      // outer elements
      nb_colors = mp->num_colors_outer_crust_mantle;
      istart = 0;

      // array offsets
      offset = 0;
      offset_nonpadded = 0;
      offset_nonpadded_att1 = 0;
      offset_nonpadded_att2 = 0;
      offset_nonpadded_att3 = 0;
      offset_nonpadded_strain = 0;
      offset_ispec = 0;
    }else{
      // inner elements (start after outer elements)
      nb_colors = mp->num_colors_outer_crust_mantle + mp->num_colors_inner_crust_mantle;
      istart = mp->num_colors_outer_crust_mantle;

      // array offsets
      offset = (mp->nspec_outer_crust_mantle) * NGLL3_PADDED;
      offset_nonpadded = (mp->nspec_outer_crust_mantle) * NGLL3;
      offset_nonpadded_att1 = (mp->nspec_outer_crust_mantle) * NGLL3 * N_SLS;

      // for factor_common array
      if( mp->use_3d_attenuation_arrays ){
        offset_nonpadded_att2 = (mp->nspec_outer_crust_mantle) * NGLL3;
        offset_nonpadded_att3 = (mp->nspec_outer_crust_mantle) * NGLL3 * N_SLS;
      }else{
        offset_nonpadded_att2 = (mp->nspec_outer_crust_mantle) * 1;
        offset_nonpadded_att3 = (mp->nspec_outer_crust_mantle) * 1 * N_SLS;
      }
      // for tiso models
      if( ! mp->anisotropic_3D_mantle ){
        offset_ispec = mp->nspec_outer_crust_mantle;
      }
      // for strain
      if( ! ( mp->NSPEC_CRUST_MANTLE_STRAIN_ONLY == 1 ) ){
        offset_nonpadded_strain = (mp->nspec_outer_crust_mantle) * NGLL3;
      }
    }

    // loops over colors
    for(int icolor = istart; icolor < nb_colors; icolor++){

      nb_blocks_to_compute = mp->h_num_elem_colors_crust_mantle[icolor];

#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
      // checks
      if( nb_blocks_to_compute <= 0 ){
        printf("error number of color blocks in crust_mantle: %d -- color = %d \n",
               nb_blocks_to_compute,icolor);
        exit(EXIT_FAILURE);
      }
#endif

      Kernel_2_crust_mantle(nb_blocks_to_compute,mp,
                            *iphase,
                            mp->d_ibool_crust_mantle + offset_nonpadded,
                            mp->d_ispec_is_tiso_crust_mantle + offset_ispec,
                            mp->d_xix_crust_mantle + offset,mp->d_xiy_crust_mantle + offset,mp->d_xiz_crust_mantle + offset,
                            mp->d_etax_crust_mantle + offset,mp->d_etay_crust_mantle + offset,mp->d_etaz_crust_mantle + offset,
                            mp->d_gammax_crust_mantle + offset,mp->d_gammay_crust_mantle + offset,mp->d_gammaz_crust_mantle + offset,
                            mp->d_kappavstore_crust_mantle + offset,
                            mp->d_muvstore_crust_mantle + offset,
                            mp->d_kappahstore_crust_mantle + offset,
                            mp->d_muhstore_crust_mantle + offset,
                            mp->d_eta_anisostore_crust_mantle + offset,
                            mp->d_epsilondev_xx_crust_mantle + offset_nonpadded,
                            mp->d_epsilondev_yy_crust_mantle + offset_nonpadded,
                            mp->d_epsilondev_xy_crust_mantle + offset_nonpadded,
                            mp->d_epsilondev_xz_crust_mantle + offset_nonpadded,
                            mp->d_epsilondev_yz_crust_mantle + offset_nonpadded,
                            mp->d_eps_trace_over_3_crust_mantle + offset_nonpadded_strain,
                            mp->d_one_minus_sum_beta_crust_mantle + offset_nonpadded_att2,
                            mp->d_factor_common_crust_mantle + offset_nonpadded_att3,
                            mp->d_R_xx_crust_mantle + offset_nonpadded_att1,
                            mp->d_R_yy_crust_mantle + offset_nonpadded_att1,
                            mp->d_R_xy_crust_mantle + offset_nonpadded_att1,
                            mp->d_R_xz_crust_mantle + offset_nonpadded_att1,
                            mp->d_R_yz_crust_mantle + offset_nonpadded_att1,
                            mp->d_c11store_crust_mantle + offset,mp->d_c12store_crust_mantle + offset,mp->d_c13store_crust_mantle + offset,
                            mp->d_c14store_crust_mantle + offset,mp->d_c15store_crust_mantle + offset,mp->d_c16store_crust_mantle + offset,
                            mp->d_c22store_crust_mantle + offset,mp->d_c23store_crust_mantle + offset,mp->d_c24store_crust_mantle + offset,
                            mp->d_c25store_crust_mantle + offset,mp->d_c26store_crust_mantle + offset,mp->d_c33store_crust_mantle + offset,
                            mp->d_c34store_crust_mantle + offset,mp->d_c35store_crust_mantle + offset,mp->d_c36store_crust_mantle + offset,
                            mp->d_c44store_crust_mantle + offset,mp->d_c45store_crust_mantle + offset,mp->d_c46store_crust_mantle + offset,
                            mp->d_c55store_crust_mantle + offset,mp->d_c56store_crust_mantle + offset,mp->d_c66store_crust_mantle + offset,
                            mp->d_b_epsilondev_xx_crust_mantle + offset_nonpadded,
                            mp->d_b_epsilondev_yy_crust_mantle + offset_nonpadded,
                            mp->d_b_epsilondev_xy_crust_mantle + offset_nonpadded,
                            mp->d_b_epsilondev_xz_crust_mantle + offset_nonpadded,
                            mp->d_b_epsilondev_yz_crust_mantle + offset_nonpadded,
                            mp->d_b_eps_trace_over_3_crust_mantle + offset_nonpadded,
                            mp->d_b_R_xx_crust_mantle + offset_nonpadded_att1,
                            mp->d_b_R_yy_crust_mantle + offset_nonpadded_att1,
                            mp->d_b_R_xy_crust_mantle + offset_nonpadded_att1,
                            mp->d_b_R_xz_crust_mantle + offset_nonpadded_att1,
                            mp->d_b_R_yz_crust_mantle + offset_nonpadded_att1,
                            FORWARD_OR_ADJOINT);

      // for padded and aligned arrays
      offset += nb_blocks_to_compute * NGLL3_PADDED;
      // for no-aligned arrays
      offset_nonpadded += nb_blocks_to_compute * NGLL3;
      offset_nonpadded_att1 += nb_blocks_to_compute * NGLL3 * N_SLS;
      // for factor_common array
      if( mp->use_3d_attenuation_arrays ){
        offset_nonpadded_att2 += nb_blocks_to_compute * NGLL3;
        offset_nonpadded_att3 += nb_blocks_to_compute * NGLL3 * N_SLS;
      }else{
        offset_nonpadded_att2 += nb_blocks_to_compute * 1;
        offset_nonpadded_att3 += nb_blocks_to_compute * 1 * N_SLS;
      }
      // for tiso models
      if( ! mp->anisotropic_3D_mantle ){
        offset_ispec += nb_blocks_to_compute;
      }
      // for strain
      if( ! ( mp->NSPEC_CRUST_MANTLE_STRAIN_ONLY == 1 ) ){
        offset_nonpadded_strain += nb_blocks_to_compute * NGLL3;
      }

    } // icolor

  }else{

    // no mesh coloring: uses atomic updates
    Kernel_2_crust_mantle(num_elements,mp,
                          *iphase,
                          mp->d_ibool_crust_mantle,
                          mp->d_ispec_is_tiso_crust_mantle,
                          mp->d_xix_crust_mantle,mp->d_xiy_crust_mantle,mp->d_xiz_crust_mantle,
                          mp->d_etax_crust_mantle,mp->d_etay_crust_mantle,mp->d_etaz_crust_mantle,
                          mp->d_gammax_crust_mantle,mp->d_gammay_crust_mantle,mp->d_gammaz_crust_mantle,
                          mp->d_kappavstore_crust_mantle,mp->d_muvstore_crust_mantle,
                          mp->d_kappahstore_crust_mantle,mp->d_muhstore_crust_mantle,
                          mp->d_eta_anisostore_crust_mantle,
                          mp->d_epsilondev_xx_crust_mantle,
                          mp->d_epsilondev_yy_crust_mantle,
                          mp->d_epsilondev_xy_crust_mantle,
                          mp->d_epsilondev_xz_crust_mantle,
                          mp->d_epsilondev_yz_crust_mantle,
                          mp->d_eps_trace_over_3_crust_mantle,
                          mp->d_one_minus_sum_beta_crust_mantle,
                          mp->d_factor_common_crust_mantle,
                          mp->d_R_xx_crust_mantle,
                          mp->d_R_yy_crust_mantle,
                          mp->d_R_xy_crust_mantle,
                          mp->d_R_xz_crust_mantle,
                          mp->d_R_yz_crust_mantle,
                          mp->d_c11store_crust_mantle,mp->d_c12store_crust_mantle,mp->d_c13store_crust_mantle,
                          mp->d_c14store_crust_mantle,mp->d_c15store_crust_mantle,mp->d_c16store_crust_mantle,
                          mp->d_c22store_crust_mantle,mp->d_c23store_crust_mantle,mp->d_c24store_crust_mantle,
                          mp->d_c25store_crust_mantle,mp->d_c26store_crust_mantle,mp->d_c33store_crust_mantle,
                          mp->d_c34store_crust_mantle,mp->d_c35store_crust_mantle,mp->d_c36store_crust_mantle,
                          mp->d_c44store_crust_mantle,mp->d_c45store_crust_mantle,mp->d_c46store_crust_mantle,
                          mp->d_c55store_crust_mantle,mp->d_c56store_crust_mantle,mp->d_c66store_crust_mantle,
                          mp->d_b_epsilondev_xx_crust_mantle,
                          mp->d_b_epsilondev_yy_crust_mantle,
                          mp->d_b_epsilondev_xy_crust_mantle,
                          mp->d_b_epsilondev_xz_crust_mantle,
                          mp->d_b_epsilondev_yz_crust_mantle,
                          mp->d_b_eps_trace_over_3_crust_mantle,
                          mp->d_b_R_xx_crust_mantle,
                          mp->d_b_R_yy_crust_mantle,
                          mp->d_b_R_xy_crust_mantle,
                          mp->d_b_R_xz_crust_mantle,
                          mp->d_b_R_yz_crust_mantle,
                          FORWARD_OR_ADJOINT);
  }

#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
  //double end_time = get_time();
  //printf("Elapsed time: %e\n",end_time-start_time);
  exit_on_cuda_error("compute_forces_crust_mantle_cuda");
#endif
}
