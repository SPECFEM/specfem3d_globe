
#include <hip/hip_runtime.h>
#ifndef INDEX2
#define INDEX2(isize,i,j) i + isize*j
#endif
#ifndef INDEX3
#define INDEX3(isize,jsize,i,j,k) i + isize*(j + jsize*k)
#endif
#ifndef INDEX4
#define INDEX4(isize,jsize,ksize,i,j,k,x) i + isize*(j + jsize*(k + ksize*x))
#endif
#ifndef INDEX5
#define INDEX5(isize,jsize,ksize,xsize,i,j,k,x,y) i + isize*(j + jsize*(k + ksize*(x + xsize*y)))
#endif
#ifndef NDIM
#define NDIM 3
#endif
#ifndef NGLLX
#define NGLLX 5
#endif
#ifndef NGLL2
#define NGLL2 25
#endif
#ifndef NGLL3
#define NGLL3 125
#endif
#ifndef NGLL3_PADDED
#define NGLL3_PADDED 128
#endif
#ifndef N_SLS
#define N_SLS 3
#endif
#ifndef IREGION_CRUST_MANTLE
#define IREGION_CRUST_MANTLE 1
#endif
#ifndef IREGION_INNER_CORE
#define IREGION_INNER_CORE 3
#endif
#ifndef IFLAG_IN_FICTITIOUS_CUBE
#define IFLAG_IN_FICTITIOUS_CUBE 11
#endif
#ifndef R_EARTH_KM
#define R_EARTH_KM 6371.0f
#endif
#ifndef COLORING_MIN_NSPEC_INNER_CORE
#define COLORING_MIN_NSPEC_INNER_CORE 1000
#endif
#ifndef COLORING_MIN_NSPEC_OUTER_CORE
#define COLORING_MIN_NSPEC_OUTER_CORE 1000
#endif
#ifndef BLOCKSIZE_TRANSFER
#define BLOCKSIZE_TRANSFER 256
#endif
__global__ void compute_rho_kernel(const int * ibool, const float * accel, const float * b_displ, float * rho_kl, const int NSPEC, const float deltat){
  int ispec;
  int ijk_ispec;
  int iglob;
  ispec = blockIdx.x + (blockIdx.y) * (gridDim.x);
  if(ispec < NSPEC){
    ijk_ispec = threadIdx.x + (NGLL3) * (ispec);
    iglob = ibool[ijk_ispec - 0] - (1);
    rho_kl[ijk_ispec - 0] = rho_kl[ijk_ispec - 0] + (deltat) * ((accel[0 - 0 + (iglob - (0)) * (3)]) * (b_displ[0 - 0 + (iglob - (0)) * (3)]) + (accel[1 - 0 + (iglob - (0)) * (3)]) * (b_displ[1 - 0 + (iglob - (0)) * (3)]) + (accel[2 - 0 + (iglob - (0)) * (3)]) * (b_displ[2 - 0 + (iglob - (0)) * (3)]));
  }
}
