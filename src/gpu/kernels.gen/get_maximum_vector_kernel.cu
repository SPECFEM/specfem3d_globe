
#include <hip/hip_runtime.h>
#ifndef INDEX2
#define INDEX2(isize,i,j) i + isize*j
#endif
#ifndef INDEX3
#define INDEX3(isize,jsize,i,j,k) i + isize*(j + jsize*k)
#endif
#ifndef INDEX4
#define INDEX4(isize,jsize,ksize,i,j,k,x) i + isize*(j + jsize*(k + ksize*x))
#endif
#ifndef INDEX5
#define INDEX5(isize,jsize,ksize,xsize,i,j,k,x,y) i + isize*(j + jsize*(k + ksize*(x + xsize*y)))
#endif
#ifndef NDIM
#define NDIM 3
#endif
#ifndef NGLLX
#define NGLLX 5
#endif
#ifndef NGLL2
#define NGLL2 25
#endif
#ifndef NGLL3
#define NGLL3 125
#endif
#ifndef NGLL3_PADDED
#define NGLL3_PADDED 128
#endif
#ifndef N_SLS
#define N_SLS 3
#endif
#ifndef IREGION_CRUST_MANTLE
#define IREGION_CRUST_MANTLE 1
#endif
#ifndef IREGION_INNER_CORE
#define IREGION_INNER_CORE 3
#endif
#ifndef IFLAG_IN_FICTITIOUS_CUBE
#define IFLAG_IN_FICTITIOUS_CUBE 11
#endif
#ifndef R_EARTH_KM
#define R_EARTH_KM 6371.0f
#endif
#ifndef COLORING_MIN_NSPEC_INNER_CORE
#define COLORING_MIN_NSPEC_INNER_CORE 1000
#endif
#ifndef COLORING_MIN_NSPEC_OUTER_CORE
#define COLORING_MIN_NSPEC_OUTER_CORE 1000
#endif
#ifndef BLOCKSIZE_TRANSFER
#define BLOCKSIZE_TRANSFER 256
#endif
__global__ void get_maximum_vector_kernel(const float * array, const int size, float * d_max){
  __shared__ float sdata[BLOCKSIZE_TRANSFER + 0 - (1) - (0) + 1];
  int tid;
  int bx;
  int i;
  int s;
  tid = threadIdx.x;
  bx = (blockIdx.y) * (gridDim.x) + blockIdx.x;
  i = tid + (bx) * (blockDim.x);
  sdata[tid - (0)] = (i < size ? sqrt((array[(i) * (3) + 0 - (0)]) * (array[(i) * (3) + 0 - (0)]) + (array[(i) * (3) + 1 - (0)]) * (array[(i) * (3) + 1 - (0)]) + (array[(i) * (3) + 2 - (0)]) * (array[(i) * (3) + 2 - (0)])) : 0.0f);
  __syncthreads();
  s = (blockDim.x) / (2);
  while(s > 0){
    if(tid < s){
      if(sdata[tid - (0)] < sdata[tid + s - (0)]){
        sdata[tid - (0)] = sdata[tid + s - (0)];
      }
    }
    s = s >> 1;
    __syncthreads();
  }
  if(tid == 0){
    d_max[bx - (0)] = sdata[0 - (0)];
  }
}
