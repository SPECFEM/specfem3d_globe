
#include <hip/hip_runtime.h>
#ifndef INDEX2
#define INDEX2(isize,i,j) i + isize*j
#endif
#ifndef INDEX3
#define INDEX3(isize,jsize,i,j,k) i + isize*(j + jsize*k)
#endif
#ifndef INDEX4
#define INDEX4(isize,jsize,ksize,i,j,k,x) i + isize*(j + jsize*(k + ksize*x))
#endif
#ifndef INDEX5
#define INDEX5(isize,jsize,ksize,xsize,i,j,k,x,y) i + isize*(j + jsize*(k + ksize*(x + xsize*y)))
#endif
#ifndef NDIM
#define NDIM 3
#endif
#ifndef NGLLX
#define NGLLX 5
#endif
#ifndef NGLL2
#define NGLL2 25
#endif
#ifndef NGLL3
#define NGLL3 125
#endif
#ifndef NGLL3_PADDED
#define NGLL3_PADDED 128
#endif
#ifndef N_SLS
#define N_SLS 3
#endif
#ifndef IREGION_CRUST_MANTLE
#define IREGION_CRUST_MANTLE 1
#endif
#ifndef IREGION_INNER_CORE
#define IREGION_INNER_CORE 3
#endif
#ifndef IFLAG_IN_FICTITIOUS_CUBE
#define IFLAG_IN_FICTITIOUS_CUBE 11
#endif
#ifndef R_EARTH_KM
#define R_EARTH_KM 6371.0f
#endif
#ifndef COLORING_MIN_NSPEC_INNER_CORE
#define COLORING_MIN_NSPEC_INNER_CORE 1000
#endif
#ifndef COLORING_MIN_NSPEC_OUTER_CORE
#define COLORING_MIN_NSPEC_OUTER_CORE 1000
#endif
#ifndef BLOCKSIZE_TRANSFER
#define BLOCKSIZE_TRANSFER 256
#endif
__global__ void compute_coupling_CMB_fluid_kernel(const float * displ_crust_mantle, float * accel_crust_mantle, const float * accel_outer_core, const int * ibool_crust_mantle, const int * ibelm_bottom_crust_mantle, const float * normal_top_outer_core, const float * jacobian2D_top_outer_core, const float * wgllwgll_xy, const int * ibool_outer_core, const int * ibelm_top_outer_core, const float RHO_TOP_OC, const float minus_g_cmb, int GRAVITY, const int NSPEC2D_BOTTOM_CM){
  int i;
  int j;
  int k;
  int iface;
  int k_corresp;
  int iglob_oc;
  int iglob_cm;
  float pressure;
  int ispec;
  int ispec_selected;
  float nx;
  float ny;
  float nz;
  float weight;
  i = threadIdx.x;
  j = threadIdx.y;
  iface = blockIdx.x + (gridDim.x) * (blockIdx.y);
  if(iface < NSPEC2D_BOTTOM_CM){
    ispec = ibelm_bottom_crust_mantle[iface - 0] - (1);
    ispec_selected = ibelm_top_outer_core[iface - 0] - (1);
    k = 0;
    k_corresp = NGLLX - (1);
    iglob_oc = ibool_outer_core[INDEX4(NGLLX, NGLLX, NGLLX, i, j, k_corresp, ispec_selected) - 0] - (1);
    nx = normal_top_outer_core[INDEX4(NDIM, NGLLX, NGLLX, 0, i, j, iface) - 0];
    ny = normal_top_outer_core[INDEX4(NDIM, NGLLX, NGLLX, 1, i, j, iface) - 0];
    nz = normal_top_outer_core[INDEX4(NDIM, NGLLX, NGLLX, 2, i, j, iface) - 0];
    weight = (jacobian2D_top_outer_core[INDEX3(NGLLX, NGLLX, i, j, iface) - 0]) * (wgllwgll_xy[INDEX2(NGLLX, i, j) - 0]);
    iglob_cm = ibool_crust_mantle[INDEX4(NGLLX, NGLLX, NGLLX, i, j, k, ispec) - 0] - (1);
    if(GRAVITY){
      pressure = (RHO_TOP_OC) * ((minus_g_cmb) * ((displ_crust_mantle[(iglob_cm) * (3) - 0]) * (nx) + (displ_crust_mantle[(iglob_cm) * (3) + 1 - 0]) * (ny) + (displ_crust_mantle[(iglob_cm) * (3) + 2 - 0]) * (nz)) - (accel_outer_core[iglob_oc - 0]));
    } else {
      pressure = ( -(RHO_TOP_OC)) * (accel_outer_core[iglob_oc - 0]);
    }
    atomicAdd(accel_crust_mantle + (iglob_cm) * (3) + 0, ((weight) * (nx)) * (pressure));
    atomicAdd(accel_crust_mantle + (iglob_cm) * (3) + 1, ((weight) * (ny)) * (pressure));
    atomicAdd(accel_crust_mantle + (iglob_cm) * (3) + 2, ((weight) * (nz)) * (pressure));
  }
}
