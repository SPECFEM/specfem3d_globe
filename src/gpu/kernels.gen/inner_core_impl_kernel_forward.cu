
#include <hip/hip_runtime.h>
//note: please do not modify this file manually!
//      this file has been generated automatically by BOAST version 0.9995
//      by: make boast_kernels

/*
!=====================================================================
!
!          S p e c f e m 3 D  G l o b e  V e r s i o n  6 . 0
!          --------------------------------------------------
!
!     Main historical authors: Dimitri Komatitsch and Jeroen Tromp
!                        Princeton University, USA
!                and CNRS / University of Marseille, France
!                 (there are currently many more authors!)
! (c) Princeton University and CNRS / University of Marseille, April 2014
!
! This program is free software; you can redistribute it and/or modify
! it under the terms of the GNU General Public License as published by
! the Free Software Foundation; either version 2 of the License, or
! (at your option) any later version.
!
! This program is distributed in the hope that it will be useful,
! but WITHOUT ANY WARRANTY; without even the implied warranty of
! MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
! GNU General Public License for more details.
!
! You should have received a copy of the GNU General Public License along
! with this program; if not, write to the Free Software Foundation, Inc.,
! 51 Franklin Street, Fifth Floor, Boston, MA 02110-1301 USA.
!
!=====================================================================
*/

#ifndef INDEX2
#define INDEX2(isize,i,j) i + isize*j
#endif
#ifndef INDEX3
#define INDEX3(isize,jsize,i,j,k) i + isize*(j + jsize*k)
#endif
#ifndef INDEX4
#define INDEX4(isize,jsize,ksize,i,j,k,x) i + isize*(j + jsize*(k + ksize*x))
#endif
#ifndef INDEX5
#define INDEX5(isize,jsize,ksize,xsize,i,j,k,x,y) i + isize*(j + jsize*(k + ksize*(x + xsize*y)))
#endif

#ifndef NDIM
#define NDIM 3
#endif
#ifndef NGLLX
#define NGLLX 5
#endif
#ifndef NGLL2
#define NGLL2 25
#endif
#ifndef NGLL3
#define NGLL3 125
#endif
#ifndef NGLL3_PADDED
#define NGLL3_PADDED 128
#endif
#ifndef N_SLS
#define N_SLS 3
#endif
#ifndef IREGION_CRUST_MANTLE
#define IREGION_CRUST_MANTLE 1
#endif
#ifndef IREGION_INNER_CORE
#define IREGION_INNER_CORE 3
#endif
#ifndef IFLAG_IN_FICTITIOUS_CUBE
#define IFLAG_IN_FICTITIOUS_CUBE 11
#endif
#ifndef R_EARTH_KM
#define R_EARTH_KM 6371.0f
#endif
#ifndef COLORING_MIN_NSPEC_INNER_CORE
#define COLORING_MIN_NSPEC_INNER_CORE 1000
#endif
#ifndef COLORING_MIN_NSPEC_OUTER_CORE
#define COLORING_MIN_NSPEC_OUTER_CORE 1000
#endif
#ifndef BLOCKSIZE_TRANSFER
#define BLOCKSIZE_TRANSFER 256
#endif

#ifdef USE_TEXTURES_CONSTANTS
#undef USE_TEXTURES_CONSTANTS
#endif
static __device__ void compute_element_ic_att_stress(const int tx, const int working_element, const float * R_xx, const float * R_yy, const float * R_xy, const float * R_xz, const float * R_yz, float * sigma_xx, float * sigma_yy, float * sigma_zz, float * sigma_xy, float * sigma_xz, float * sigma_yz){
  int offset;
  int i_sls;
  float R_xx_val;
  float R_yy_val;
  for (i_sls = 0; i_sls <= N_SLS - (1); i_sls += 1) {
    offset = tx + (NGLL3) * (i_sls + (N_SLS) * (working_element));
    R_xx_val = R_xx[offset - (0)];
    R_yy_val = R_yy[offset - (0)];
    sigma_xx[0 - (0)] = sigma_xx[0 - (0)] - (R_xx_val);
    sigma_yy[0 - (0)] = sigma_yy[0 - (0)] - (R_yy_val);
    sigma_zz[0 - (0)] = sigma_zz[0 - (0)] + R_xx_val + R_yy_val;
    sigma_xy[0 - (0)] = sigma_xy[0 - (0)] - (R_xy[offset - (0)]);
    sigma_xz[0 - (0)] = sigma_xz[0 - (0)] - (R_xz[offset - (0)]);
    sigma_yz[0 - (0)] = sigma_yz[0 - (0)] - (R_yz[offset - (0)]);
  }
}
static __device__ void compute_element_ic_att_memory(const int tx, const int working_element, const float * d_muv, const float * factor_common, const float * alphaval, const float * betaval, const float * gammaval, float * R_xx, float * R_yy, float * R_xy, float * R_xz, float * R_yz, const float * epsilondev_xx, const float * epsilondev_yy, const float * epsilondev_xy, const float * epsilondev_xz, const float * epsilondev_yz, const float epsilondev_xx_loc, const float epsilondev_yy_loc, const float epsilondev_xy_loc, const float epsilondev_xz_loc, const float epsilondev_yz_loc, const int USE_3D_ATTENUATION_ARRAYS){
  int offset;
  int i_sls;
  float mul;
  float alphaval_loc;
  float betaval_loc;
  float gammaval_loc;
  float factor_loc;
  float sn;
  float snp1;
  mul = d_muv[tx + (NGLL3_PADDED) * (working_element) - (0)];
  for (i_sls = 0; i_sls <= N_SLS - (1); i_sls += 1) {
    offset = tx + (NGLL3) * (i_sls + (N_SLS) * (working_element));
    if (USE_3D_ATTENUATION_ARRAYS) {
      factor_loc = (mul) * (factor_common[offset - (0)]);
    } else {
      factor_loc = (mul) * (factor_common[i_sls + (N_SLS) * (working_element) - (0)]);
    }
    alphaval_loc = alphaval[i_sls - (0)];
    betaval_loc = betaval[i_sls - (0)];
    gammaval_loc = gammaval[i_sls - (0)];
    sn = (factor_loc) * (epsilondev_xx[tx + (NGLL3) * (working_element) - (0)]);
    snp1 = (factor_loc) * (epsilondev_xx_loc);
    R_xx[offset - (0)] = (alphaval_loc) * (R_xx[offset - (0)]) + (betaval_loc) * (sn) + (gammaval_loc) * (snp1);
    sn = (factor_loc) * (epsilondev_yy[tx + (NGLL3) * (working_element) - (0)]);
    snp1 = (factor_loc) * (epsilondev_yy_loc);
    R_yy[offset - (0)] = (alphaval_loc) * (R_yy[offset - (0)]) + (betaval_loc) * (sn) + (gammaval_loc) * (snp1);
    sn = (factor_loc) * (epsilondev_xy[tx + (NGLL3) * (working_element) - (0)]);
    snp1 = (factor_loc) * (epsilondev_xy_loc);
    R_xy[offset - (0)] = (alphaval_loc) * (R_xy[offset - (0)]) + (betaval_loc) * (sn) + (gammaval_loc) * (snp1);
    sn = (factor_loc) * (epsilondev_xz[tx + (NGLL3) * (working_element) - (0)]);
    snp1 = (factor_loc) * (epsilondev_xz_loc);
    R_xz[offset - (0)] = (alphaval_loc) * (R_xz[offset - (0)]) + (betaval_loc) * (sn) + (gammaval_loc) * (snp1);
    sn = (factor_loc) * (epsilondev_yz[tx + (NGLL3) * (working_element) - (0)]);
    snp1 = (factor_loc) * (epsilondev_yz_loc);
    R_yz[offset - (0)] = (alphaval_loc) * (R_yz[offset - (0)]) + (betaval_loc) * (sn) + (gammaval_loc) * (snp1);
  }
}
static __device__ void compute_element_ic_gravity(const int tx, const int iglob, const float * __restrict__ d_xstore, const float * __restrict__ d_ystore, const float * __restrict__ d_zstore, const float * __restrict__ d_minus_gravity_table, const float * __restrict__ d_minus_deriv_gravity_table, const float * __restrict__ d_density_table, const float * __restrict__ wgll_cube, const float jacobianl, const float * s_dummyx_loc, const float * s_dummyy_loc, const float * s_dummyz_loc, float * sigma_xx, float * sigma_yy, float * sigma_zz, float * sigma_xy, float * sigma_yx, float * sigma_xz, float * sigma_zx, float * sigma_yz, float * sigma_zy, float * rho_s_H1, float * rho_s_H2, float * rho_s_H3){
  float radius;
  float theta;
  float phi;
  float cos_theta;
  float sin_theta;
  float cos_phi;
  float sin_phi;
  float cos_theta_sq;
  float sin_theta_sq;
  float cos_phi_sq;
  float sin_phi_sq;
  float minus_g;
  float minus_dg;
  float rho;
  float gxl;
  float gyl;
  float gzl;
  float minus_g_over_radius;
  float minus_dg_plus_g_over_radius;
  float Hxxl;
  float Hyyl;
  float Hzzl;
  float Hxyl;
  float Hxzl;
  float Hyzl;
  float sx_l;
  float sy_l;
  float sz_l;
  float factor;
  int int_radius;
  radius = d_xstore[iglob - (0)];
  if (radius < 1.5696123057604773e-05f) {
    radius = 1.5696123057604773e-05f;
  }
  theta = d_ystore[iglob - (0)];
  phi = d_zstore[iglob - (0)];
  sincosf(theta,  &sin_theta,  &cos_theta);
  sincosf(phi,  &sin_phi,  &cos_phi);
  int_radius = rint(((radius) * (6371.0f)) * (10.0f)) - (1);
  if (int_radius < 0) {
    int_radius = 0;
  }
  minus_g = d_minus_gravity_table[int_radius - (0)];
  minus_dg = d_minus_deriv_gravity_table[int_radius - (0)];
  rho = d_density_table[int_radius - (0)];
  gxl = ((minus_g) * (sin_theta)) * (cos_phi);
  gyl = ((minus_g) * (sin_theta)) * (sin_phi);
  gzl = (minus_g) * (cos_theta);
  minus_g_over_radius = (minus_g) / (radius);
  minus_dg_plus_g_over_radius = minus_dg - (minus_g_over_radius);
  cos_theta_sq = (cos_theta) * (cos_theta);
  sin_theta_sq = (sin_theta) * (sin_theta);
  cos_phi_sq = (cos_phi) * (cos_phi);
  sin_phi_sq = (sin_phi) * (sin_phi);
  Hxxl = (minus_g_over_radius) * ((cos_phi_sq) * (cos_theta_sq) + sin_phi_sq) + ((cos_phi_sq) * (minus_dg)) * (sin_theta_sq);
  Hyyl = (minus_g_over_radius) * (cos_phi_sq + (cos_theta_sq) * (sin_phi_sq)) + ((minus_dg) * (sin_phi_sq)) * (sin_theta_sq);
  Hzzl = (cos_theta_sq) * (minus_dg) + (minus_g_over_radius) * (sin_theta_sq);
  Hxyl = (((cos_phi) * (minus_dg_plus_g_over_radius)) * (sin_phi)) * (sin_theta_sq);
  Hxzl = (((cos_phi) * (cos_theta)) * (minus_dg_plus_g_over_radius)) * (sin_theta);
  Hyzl = (((cos_theta) * (minus_dg_plus_g_over_radius)) * (sin_phi)) * (sin_theta);
  sx_l = (rho) * (s_dummyx_loc[tx - (0)]);
  sy_l = (rho) * (s_dummyy_loc[tx - (0)]);
  sz_l = (rho) * (s_dummyz_loc[tx - (0)]);
  *(sigma_xx) = *(sigma_xx) + (sy_l) * (gyl) + (sz_l) * (gzl);
  *(sigma_yy) = *(sigma_yy) + (sx_l) * (gxl) + (sz_l) * (gzl);
  *(sigma_zz) = *(sigma_zz) + (sx_l) * (gxl) + (sy_l) * (gyl);
  *(sigma_xy) = *(sigma_xy) - ((sx_l) * (gyl));
  *(sigma_yx) = *(sigma_yx) - ((sy_l) * (gxl));
  *(sigma_xz) = *(sigma_xz) - ((sx_l) * (gzl));
  *(sigma_zx) = *(sigma_zx) - ((sz_l) * (gxl));
  *(sigma_yz) = *(sigma_yz) - ((sy_l) * (gzl));
  *(sigma_zy) = *(sigma_zy) - ((sz_l) * (gyl));
  factor = (jacobianl) * (wgll_cube[tx - (0)]);
  rho_s_H1[0 - (0)] = (factor) * ((sx_l) * (Hxxl) + (sy_l) * (Hxyl) + (sz_l) * (Hxzl));
  rho_s_H2[0 - (0)] = (factor) * ((sx_l) * (Hxyl) + (sy_l) * (Hyyl) + (sz_l) * (Hyzl));
  rho_s_H3[0 - (0)] = (factor) * ((sx_l) * (Hxzl) + (sy_l) * (Hyzl) + (sz_l) * (Hzzl));
}
__global__ 
#ifdef USE_LAUNCH_BOUNDS
__launch_bounds__(NGLL3_PADDED, LAUNCH_MIN_BLOCKS)
#endif
 void inner_core_impl_kernel_forward(const int nb_blocks_to_compute, const int * d_ibool, const int * d_idoubling, const int * d_phase_ispec_inner, const int num_phase_ispec, const int d_iphase, const float deltat, const int use_mesh_coloring_gpu, const float * __restrict__ d_displ, float * d_accel, const float * __restrict__ d_xix, const float * __restrict__ d_xiy, const float * __restrict__ d_xiz, const float * __restrict__ d_etax, const float * __restrict__ d_etay, const float * __restrict__ d_etaz, const float * __restrict__ d_gammax, const float * __restrict__ d_gammay, const float * __restrict__ d_gammaz, const float * __restrict__ d_hprime_xx, const float * __restrict__ d_hprimewgll_xx, const float * __restrict__ d_wgllwgll_xy, const float * __restrict__ d_wgllwgll_xz, const float * __restrict__ d_wgllwgll_yz, const float * __restrict__ d_kappavstore, const float * __restrict__ d_muvstore, const int COMPUTE_AND_STORE_STRAIN, float * epsilondev_xx, float * epsilondev_yy, float * epsilondev_xy, float * epsilondev_xz, float * epsilondev_yz, float * epsilon_trace_over_3, const int ATTENUATION, const int PARTIAL_PHYS_DISPERSION_ONLY, const int USE_3D_ATTENUATION_ARRAYS, const float * __restrict__ one_minus_sum_beta, const float * __restrict__ factor_common, float * R_xx, float * R_yy, float * R_xy, float * R_xz, float * R_yz, const float * __restrict__ alphaval, const float * __restrict__ betaval, const float * __restrict__ gammaval, const int ANISOTROPY, const float * __restrict__ d_c11store, const float * __restrict__ d_c12store, const float * __restrict__ d_c13store, const float * __restrict__ d_c33store, const float * __restrict__ d_c44store, const int GRAVITY, const float * __restrict__ d_xstore, const float * __restrict__ d_ystore, const float * __restrict__ d_zstore, const float * __restrict__ d_minus_gravity_table, const float * __restrict__ d_minus_deriv_gravity_table, const float * __restrict__ d_density_table, const float * __restrict__ wgll_cube, const int NSPEC_INNER_CORE_STRAIN_ONLY, const int NSPEC_INNER_CORE){
  int bx;
  int tx;
  int K;
  int J;
  int I;
#ifndef MANUALLY_UNROLLED_LOOPS
  int l;
#endif
  unsigned short active;
  int offset;
  int iglob;
  int working_element;
  float tempx1l;
  float tempx2l;
  float tempx3l;
  float tempy1l;
  float tempy2l;
  float tempy3l;
  float tempz1l;
  float tempz2l;
  float tempz3l;
  float xixl;
  float xiyl;
  float xizl;
  float etaxl;
  float etayl;
  float etazl;
  float gammaxl;
  float gammayl;
  float gammazl;
  float jacobianl;
  float duxdxl;
  float duxdyl;
  float duxdzl;
  float duydxl;
  float duydyl;
  float duydzl;
  float duzdxl;
  float duzdyl;
  float duzdzl;
  float duxdxl_plus_duydyl;
  float duxdxl_plus_duzdzl;
  float duydyl_plus_duzdzl;
  float duxdyl_plus_duydxl;
  float duzdxl_plus_duxdzl;
  float duzdyl_plus_duydzl;
  float templ;
  float fac1;
  float fac2;
  float fac3;
  float lambdal;
  float mul;
  float lambdalplus2mul;
  float kappal;
  float mul_iso;
  float mul_aniso;
  float sigma_xx;
  float sigma_xy;
  float sigma_xz;
  float sigma_yx;
  float sigma_yy;
  float sigma_yz;
  float sigma_zx;
  float sigma_zy;
  float sigma_zz;
  float epsilondev_xx_loc;
  float epsilondev_yy_loc;
  float epsilondev_xy_loc;
  float epsilondev_xz_loc;
  float epsilondev_yz_loc;
  float c11;
  float c12;
  float c13;
  float c33;
  float c44;
  float sum_terms1;
  float sum_terms2;
  float sum_terms3;
  float rho_s_H1;
  float rho_s_H2;
  float rho_s_H3;
  __shared__ float s_dummyx_loc[NGLL3 + 0 - (1) - (0) + 1];
  __shared__ float s_dummyy_loc[NGLL3 + 0 - (1) - (0) + 1];
  __shared__ float s_dummyz_loc[NGLL3 + 0 - (1) - (0) + 1];
  __shared__ float s_tempx1[NGLL3 + 0 - (1) - (0) + 1];
  __shared__ float s_tempx2[NGLL3 + 0 - (1) - (0) + 1];
  __shared__ float s_tempx3[NGLL3 + 0 - (1) - (0) + 1];
  __shared__ float s_tempy1[NGLL3 + 0 - (1) - (0) + 1];
  __shared__ float s_tempy2[NGLL3 + 0 - (1) - (0) + 1];
  __shared__ float s_tempy3[NGLL3 + 0 - (1) - (0) + 1];
  __shared__ float s_tempz1[NGLL3 + 0 - (1) - (0) + 1];
  __shared__ float s_tempz2[NGLL3 + 0 - (1) - (0) + 1];
  __shared__ float s_tempz3[NGLL3 + 0 - (1) - (0) + 1];
  __shared__ float sh_hprime_xx[NGLL2 + 0 - (1) - (0) + 1];
  __shared__ float sh_hprimewgll_xx[NGLL2 + 0 - (1) - (0) + 1];
  bx = (blockIdx.y) * (gridDim.x) + blockIdx.x;
  tx = threadIdx.x;
  K = (tx) / (NGLL2);
  J = (tx - ((K) * (NGLL2))) / (NGLLX);
  I = tx - ((K) * (NGLL2)) - ((J) * (NGLLX));
  active = (tx < NGLL3 && bx < nb_blocks_to_compute ? 1 : 0);
  if (active) {
#ifdef USE_MESH_COLORING_GPU
    working_element = bx;
#else
    if (use_mesh_coloring_gpu) {
      working_element = bx;
    } else {
      working_element = d_phase_ispec_inner[bx + (num_phase_ispec) * (d_iphase - (1)) - (0)] - (1);
    }
#endif
    if (d_idoubling[working_element - (0)] == IFLAG_IN_FICTITIOUS_CUBE) {
      active = 0;
    } else {
      iglob = d_ibool[(working_element) * (NGLL3) + tx - (0)] - (1);
#ifdef USE_TEXTURES_FIELDS
      s_dummyx_loc[tx - (0)] = tex1Dfetch(d_displ_ic_tex,(iglob) * (3) + 0);
      s_dummyy_loc[tx - (0)] = tex1Dfetch(d_displ_ic_tex,(iglob) * (3) + 1);
      s_dummyz_loc[tx - (0)] = tex1Dfetch(d_displ_ic_tex,(iglob) * (3) + 2);
#else
      s_dummyx_loc[tx - (0)] = d_displ[0 - (0) + (iglob - (0)) * (3)];
      s_dummyy_loc[tx - (0)] = d_displ[1 - (0) + (iglob - (0)) * (3)];
      s_dummyz_loc[tx - (0)] = d_displ[2 - (0) + (iglob - (0)) * (3)];
#endif
    }
  }
  if (tx < NGLL2) {
#ifdef USE_TEXTURES_CONSTANTS
    sh_hprime_xx[tx - (0)] = tex1Dfetch(d_hprime_xx_ic_tex,tx);
    sh_hprimewgll_xx[tx - (0)] = tex1Dfetch(d_hprimewgll_xx_ic_tex,tx);
#else
    sh_hprime_xx[tx - (0)] = d_hprime_xx[tx - (0)];
    sh_hprimewgll_xx[tx - (0)] = d_hprimewgll_xx[tx - (0)];
#endif
  }
  __syncthreads();
  if (active) {
    tempx1l = 0.0f;
    tempx2l = 0.0f;
    tempx3l = 0.0f;
    tempy1l = 0.0f;
    tempy2l = 0.0f;
    tempy3l = 0.0f;
    tempz1l = 0.0f;
    tempz2l = 0.0f;
    tempz3l = 0.0f;
#ifdef MANUALLY_UNROLLED_LOOPS
    fac1 = sh_hprime_xx[(0) * (NGLLX) + I - (0)];
    tempx1l = tempx1l + (s_dummyx_loc[(K) * (NGLL2) + (J) * (NGLLX) + 0 - (0)]) * (fac1);
    tempy1l = tempy1l + (s_dummyy_loc[(K) * (NGLL2) + (J) * (NGLLX) + 0 - (0)]) * (fac1);
    tempz1l = tempz1l + (s_dummyz_loc[(K) * (NGLL2) + (J) * (NGLLX) + 0 - (0)]) * (fac1);
    fac2 = sh_hprime_xx[(0) * (NGLLX) + J - (0)];
    tempx2l = tempx2l + (s_dummyx_loc[(K) * (NGLL2) + (0) * (NGLLX) + I - (0)]) * (fac2);
    tempy2l = tempy2l + (s_dummyy_loc[(K) * (NGLL2) + (0) * (NGLLX) + I - (0)]) * (fac2);
    tempz2l = tempz2l + (s_dummyz_loc[(K) * (NGLL2) + (0) * (NGLLX) + I - (0)]) * (fac2);
    fac3 = sh_hprime_xx[(0) * (NGLLX) + K - (0)];
    tempx3l = tempx3l + (s_dummyx_loc[(0) * (NGLL2) + (J) * (NGLLX) + I - (0)]) * (fac3);
    tempy3l = tempy3l + (s_dummyy_loc[(0) * (NGLL2) + (J) * (NGLLX) + I - (0)]) * (fac3);
    tempz3l = tempz3l + (s_dummyz_loc[(0) * (NGLL2) + (J) * (NGLLX) + I - (0)]) * (fac3);
    fac1 = sh_hprime_xx[(1) * (NGLLX) + I - (0)];
    tempx1l = tempx1l + (s_dummyx_loc[(K) * (NGLL2) + (J) * (NGLLX) + 1 - (0)]) * (fac1);
    tempy1l = tempy1l + (s_dummyy_loc[(K) * (NGLL2) + (J) * (NGLLX) + 1 - (0)]) * (fac1);
    tempz1l = tempz1l + (s_dummyz_loc[(K) * (NGLL2) + (J) * (NGLLX) + 1 - (0)]) * (fac1);
    fac2 = sh_hprime_xx[(1) * (NGLLX) + J - (0)];
    tempx2l = tempx2l + (s_dummyx_loc[(K) * (NGLL2) + (1) * (NGLLX) + I - (0)]) * (fac2);
    tempy2l = tempy2l + (s_dummyy_loc[(K) * (NGLL2) + (1) * (NGLLX) + I - (0)]) * (fac2);
    tempz2l = tempz2l + (s_dummyz_loc[(K) * (NGLL2) + (1) * (NGLLX) + I - (0)]) * (fac2);
    fac3 = sh_hprime_xx[(1) * (NGLLX) + K - (0)];
    tempx3l = tempx3l + (s_dummyx_loc[(1) * (NGLL2) + (J) * (NGLLX) + I - (0)]) * (fac3);
    tempy3l = tempy3l + (s_dummyy_loc[(1) * (NGLL2) + (J) * (NGLLX) + I - (0)]) * (fac3);
    tempz3l = tempz3l + (s_dummyz_loc[(1) * (NGLL2) + (J) * (NGLLX) + I - (0)]) * (fac3);
    fac1 = sh_hprime_xx[(2) * (NGLLX) + I - (0)];
    tempx1l = tempx1l + (s_dummyx_loc[(K) * (NGLL2) + (J) * (NGLLX) + 2 - (0)]) * (fac1);
    tempy1l = tempy1l + (s_dummyy_loc[(K) * (NGLL2) + (J) * (NGLLX) + 2 - (0)]) * (fac1);
    tempz1l = tempz1l + (s_dummyz_loc[(K) * (NGLL2) + (J) * (NGLLX) + 2 - (0)]) * (fac1);
    fac2 = sh_hprime_xx[(2) * (NGLLX) + J - (0)];
    tempx2l = tempx2l + (s_dummyx_loc[(K) * (NGLL2) + (2) * (NGLLX) + I - (0)]) * (fac2);
    tempy2l = tempy2l + (s_dummyy_loc[(K) * (NGLL2) + (2) * (NGLLX) + I - (0)]) * (fac2);
    tempz2l = tempz2l + (s_dummyz_loc[(K) * (NGLL2) + (2) * (NGLLX) + I - (0)]) * (fac2);
    fac3 = sh_hprime_xx[(2) * (NGLLX) + K - (0)];
    tempx3l = tempx3l + (s_dummyx_loc[(2) * (NGLL2) + (J) * (NGLLX) + I - (0)]) * (fac3);
    tempy3l = tempy3l + (s_dummyy_loc[(2) * (NGLL2) + (J) * (NGLLX) + I - (0)]) * (fac3);
    tempz3l = tempz3l + (s_dummyz_loc[(2) * (NGLL2) + (J) * (NGLLX) + I - (0)]) * (fac3);
    fac1 = sh_hprime_xx[(3) * (NGLLX) + I - (0)];
    tempx1l = tempx1l + (s_dummyx_loc[(K) * (NGLL2) + (J) * (NGLLX) + 3 - (0)]) * (fac1);
    tempy1l = tempy1l + (s_dummyy_loc[(K) * (NGLL2) + (J) * (NGLLX) + 3 - (0)]) * (fac1);
    tempz1l = tempz1l + (s_dummyz_loc[(K) * (NGLL2) + (J) * (NGLLX) + 3 - (0)]) * (fac1);
    fac2 = sh_hprime_xx[(3) * (NGLLX) + J - (0)];
    tempx2l = tempx2l + (s_dummyx_loc[(K) * (NGLL2) + (3) * (NGLLX) + I - (0)]) * (fac2);
    tempy2l = tempy2l + (s_dummyy_loc[(K) * (NGLL2) + (3) * (NGLLX) + I - (0)]) * (fac2);
    tempz2l = tempz2l + (s_dummyz_loc[(K) * (NGLL2) + (3) * (NGLLX) + I - (0)]) * (fac2);
    fac3 = sh_hprime_xx[(3) * (NGLLX) + K - (0)];
    tempx3l = tempx3l + (s_dummyx_loc[(3) * (NGLL2) + (J) * (NGLLX) + I - (0)]) * (fac3);
    tempy3l = tempy3l + (s_dummyy_loc[(3) * (NGLL2) + (J) * (NGLLX) + I - (0)]) * (fac3);
    tempz3l = tempz3l + (s_dummyz_loc[(3) * (NGLL2) + (J) * (NGLLX) + I - (0)]) * (fac3);
    fac1 = sh_hprime_xx[(4) * (NGLLX) + I - (0)];
    tempx1l = tempx1l + (s_dummyx_loc[(K) * (NGLL2) + (J) * (NGLLX) + 4 - (0)]) * (fac1);
    tempy1l = tempy1l + (s_dummyy_loc[(K) * (NGLL2) + (J) * (NGLLX) + 4 - (0)]) * (fac1);
    tempz1l = tempz1l + (s_dummyz_loc[(K) * (NGLL2) + (J) * (NGLLX) + 4 - (0)]) * (fac1);
    fac2 = sh_hprime_xx[(4) * (NGLLX) + J - (0)];
    tempx2l = tempx2l + (s_dummyx_loc[(K) * (NGLL2) + (4) * (NGLLX) + I - (0)]) * (fac2);
    tempy2l = tempy2l + (s_dummyy_loc[(K) * (NGLL2) + (4) * (NGLLX) + I - (0)]) * (fac2);
    tempz2l = tempz2l + (s_dummyz_loc[(K) * (NGLL2) + (4) * (NGLLX) + I - (0)]) * (fac2);
    fac3 = sh_hprime_xx[(4) * (NGLLX) + K - (0)];
    tempx3l = tempx3l + (s_dummyx_loc[(4) * (NGLL2) + (J) * (NGLLX) + I - (0)]) * (fac3);
    tempy3l = tempy3l + (s_dummyy_loc[(4) * (NGLL2) + (J) * (NGLLX) + I - (0)]) * (fac3);
    tempz3l = tempz3l + (s_dummyz_loc[(4) * (NGLL2) + (J) * (NGLLX) + I - (0)]) * (fac3);
#else
    for (l = 0; l <= NGLLX - (1); l += 1) {
      fac1 = sh_hprime_xx[(l) * (NGLLX) + I - (0)];
      tempx1l = tempx1l + (s_dummyx_loc[(K) * (NGLL2) + (J) * (NGLLX) + l - (0)]) * (fac1);
      tempy1l = tempy1l + (s_dummyy_loc[(K) * (NGLL2) + (J) * (NGLLX) + l - (0)]) * (fac1);
      tempz1l = tempz1l + (s_dummyz_loc[(K) * (NGLL2) + (J) * (NGLLX) + l - (0)]) * (fac1);
      fac2 = sh_hprime_xx[(l) * (NGLLX) + J - (0)];
      tempx2l = tempx2l + (s_dummyx_loc[(K) * (NGLL2) + (l) * (NGLLX) + I - (0)]) * (fac2);
      tempy2l = tempy2l + (s_dummyy_loc[(K) * (NGLL2) + (l) * (NGLLX) + I - (0)]) * (fac2);
      tempz2l = tempz2l + (s_dummyz_loc[(K) * (NGLL2) + (l) * (NGLLX) + I - (0)]) * (fac2);
      fac3 = sh_hprime_xx[(l) * (NGLLX) + K - (0)];
      tempx3l = tempx3l + (s_dummyx_loc[(l) * (NGLL2) + (J) * (NGLLX) + I - (0)]) * (fac3);
      tempy3l = tempy3l + (s_dummyy_loc[(l) * (NGLL2) + (J) * (NGLLX) + I - (0)]) * (fac3);
      tempz3l = tempz3l + (s_dummyz_loc[(l) * (NGLL2) + (J) * (NGLLX) + I - (0)]) * (fac3);
    }
#endif
    offset = (working_element) * (NGLL3_PADDED) + tx;
    xixl = d_xix[offset - (0)];
    etaxl = d_etax[offset - (0)];
    gammaxl = d_gammax[offset - (0)];
    xiyl = d_xiy[offset - (0)];
    etayl = d_etay[offset - (0)];
    gammayl = d_gammay[offset - (0)];
    xizl = d_xiz[offset - (0)];
    etazl = d_etaz[offset - (0)];
    gammazl = d_gammaz[offset - (0)];
    duxdxl = (xixl) * (tempx1l) + (etaxl) * (tempx2l) + (gammaxl) * (tempx3l);
    duxdyl = (xiyl) * (tempx1l) + (etayl) * (tempx2l) + (gammayl) * (tempx3l);
    duxdzl = (xizl) * (tempx1l) + (etazl) * (tempx2l) + (gammazl) * (tempx3l);
    duydxl = (xixl) * (tempy1l) + (etaxl) * (tempy2l) + (gammaxl) * (tempy3l);
    duydyl = (xiyl) * (tempy1l) + (etayl) * (tempy2l) + (gammayl) * (tempy3l);
    duydzl = (xizl) * (tempy1l) + (etazl) * (tempy2l) + (gammazl) * (tempy3l);
    duzdxl = (xixl) * (tempz1l) + (etaxl) * (tempz2l) + (gammaxl) * (tempz3l);
    duzdyl = (xiyl) * (tempz1l) + (etayl) * (tempz2l) + (gammayl) * (tempz3l);
    duzdzl = (xizl) * (tempz1l) + (etazl) * (tempz2l) + (gammazl) * (tempz3l);
    duxdxl_plus_duydyl = duxdxl + duydyl;
    duxdxl_plus_duzdzl = duxdxl + duzdzl;
    duydyl_plus_duzdzl = duydyl + duzdzl;
    duxdyl_plus_duydxl = duxdyl + duydxl;
    duzdxl_plus_duxdzl = duzdxl + duxdzl;
    duzdyl_plus_duydzl = duzdyl + duydzl;
    if (COMPUTE_AND_STORE_STRAIN) {
      templ = (duxdxl + duydyl + duzdzl) * (0.3333333333333333f);
      epsilondev_xx_loc = duxdxl - (templ);
      epsilondev_yy_loc = duydyl - (templ);
      epsilondev_xy_loc = (duxdyl_plus_duydxl) * (0.5f);
      epsilondev_xz_loc = (duzdxl_plus_duxdzl) * (0.5f);
      epsilondev_yz_loc = (duzdyl_plus_duydzl) * (0.5f);
      if (NSPEC_INNER_CORE_STRAIN_ONLY == 1) {
        epsilon_trace_over_3[tx - (0)] = templ;
      } else {
        epsilon_trace_over_3[tx + (working_element) * (NGLL3) - (0)] = templ;
      }
    }
    kappal = d_kappavstore[offset - (0)];
    mul = d_muvstore[offset - (0)];
    if (ATTENUATION) {
      if (USE_3D_ATTENUATION_ARRAYS) {
        mul_iso = (mul) * (one_minus_sum_beta[tx + (working_element) * (NGLL3) - (0)]);
        mul_aniso = (mul) * (one_minus_sum_beta[tx + (working_element) * (NGLL3) - (0)] - (1.0f));
      } else {
        mul_iso = (mul) * (one_minus_sum_beta[working_element - (0)]);
        mul_aniso = (mul) * (one_minus_sum_beta[working_element - (0)] - (1.0f));
      }
    } else {
      mul_iso = mul;
    }
    if (ANISOTROPY) {
      c11 = d_c11store[offset - (0)];
      c12 = d_c12store[offset - (0)];
      c13 = d_c13store[offset - (0)];
      c33 = d_c33store[offset - (0)];
      c44 = d_c44store[offset - (0)];
      if (ATTENUATION) {
        c11 = c11 + (mul_aniso) * (1.3333333333333333f);
        c12 = c12 - ((mul_aniso) * (0.6666666666666666f));
        c13 = c13 - ((mul_aniso) * (0.6666666666666666f));
        c33 = c33 + (mul_aniso) * (1.3333333333333333f);
        c44 = c44 + mul_aniso;
      }
      sigma_xx = (c11) * (duxdxl) + (c12) * (duydyl) + (c13) * (duzdzl);
      sigma_yy = (c12) * (duxdxl) + (c11) * (duydyl) + (c13) * (duzdzl);
      sigma_zz = (c13) * (duxdxl) + (c13) * (duydyl) + (c33) * (duzdzl);
      sigma_xy = ((c11 - (c12)) * (duxdyl_plus_duydxl)) * (0.5f);
      sigma_xz = (c44) * (duzdxl_plus_duxdzl);
      sigma_yz = (c44) * (duzdyl_plus_duydzl);
    } else {
      lambdalplus2mul = kappal + (mul_iso) * (1.3333333333333333f);
      lambdal = lambdalplus2mul - ((mul_iso) * (2.0f));
      sigma_xx = (lambdalplus2mul) * (duxdxl) + (lambdal) * (duydyl_plus_duzdzl);
      sigma_yy = (lambdalplus2mul) * (duydyl) + (lambdal) * (duxdxl_plus_duzdzl);
      sigma_zz = (lambdalplus2mul) * (duzdzl) + (lambdal) * (duxdxl_plus_duydyl);
      sigma_xy = (mul) * (duxdyl_plus_duydxl);
      sigma_xz = (mul) * (duzdxl_plus_duxdzl);
      sigma_yz = (mul) * (duzdyl_plus_duydzl);
    }
    if (ATTENUATION &&  ! PARTIAL_PHYS_DISPERSION_ONLY) {
      compute_element_ic_att_stress(tx, working_element, R_xx, R_yy, R_xy, R_xz, R_yz,  &sigma_xx,  &sigma_yy,  &sigma_zz,  &sigma_xy,  &sigma_xz,  &sigma_yz);
    }
    sigma_yx = sigma_xy;
    sigma_zx = sigma_xz;
    sigma_zy = sigma_yz;
    jacobianl = (1.0f) / ((xixl) * ((etayl) * (gammazl) - ((etazl) * (gammayl))) - ((xiyl) * ((etaxl) * (gammazl) - ((etazl) * (gammaxl)))) + (xizl) * ((etaxl) * (gammayl) - ((etayl) * (gammaxl))));
    if (GRAVITY) {
      compute_element_ic_gravity(tx, iglob, d_xstore, d_ystore, d_zstore, d_minus_gravity_table, d_minus_deriv_gravity_table, d_density_table, wgll_cube, jacobianl, s_dummyx_loc, s_dummyy_loc, s_dummyz_loc,  &sigma_xx,  &sigma_yy,  &sigma_zz,  &sigma_xy,  &sigma_yx,  &sigma_xz,  &sigma_zx,  &sigma_yz,  &sigma_zy,  &rho_s_H1,  &rho_s_H2,  &rho_s_H3);
    }
    s_tempx1[tx - (0)] = (jacobianl) * ((sigma_xx) * (xixl) + (sigma_yx) * (xiyl) + (sigma_zx) * (xizl));
    s_tempy1[tx - (0)] = (jacobianl) * ((sigma_xy) * (xixl) + (sigma_yy) * (xiyl) + (sigma_zy) * (xizl));
    s_tempz1[tx - (0)] = (jacobianl) * ((sigma_xz) * (xixl) + (sigma_yz) * (xiyl) + (sigma_zz) * (xizl));
    s_tempx2[tx - (0)] = (jacobianl) * ((sigma_xx) * (etaxl) + (sigma_yx) * (etayl) + (sigma_zx) * (etazl));
    s_tempy2[tx - (0)] = (jacobianl) * ((sigma_xy) * (etaxl) + (sigma_yy) * (etayl) + (sigma_zy) * (etazl));
    s_tempz2[tx - (0)] = (jacobianl) * ((sigma_xz) * (etaxl) + (sigma_yz) * (etayl) + (sigma_zz) * (etazl));
    s_tempx3[tx - (0)] = (jacobianl) * ((sigma_xx) * (gammaxl) + (sigma_yx) * (gammayl) + (sigma_zx) * (gammazl));
    s_tempy3[tx - (0)] = (jacobianl) * ((sigma_xy) * (gammaxl) + (sigma_yy) * (gammayl) + (sigma_zy) * (gammazl));
    s_tempz3[tx - (0)] = (jacobianl) * ((sigma_xz) * (gammaxl) + (sigma_yz) * (gammayl) + (sigma_zz) * (gammazl));
  }
  __syncthreads();
  if (active) {
    tempx1l = 0.0f;
    tempx2l = 0.0f;
    tempx3l = 0.0f;
    tempy1l = 0.0f;
    tempy2l = 0.0f;
    tempy3l = 0.0f;
    tempz1l = 0.0f;
    tempz2l = 0.0f;
    tempz3l = 0.0f;
#ifdef MANUALLY_UNROLLED_LOOPS
    fac1 = sh_hprimewgll_xx[(I) * (NGLLX) + 0 - (0)];
    offset = (K) * (NGLL2) + (J) * (NGLLX) + 0;
    tempx1l = tempx1l + (s_tempx1[offset - (0)]) * (fac1);
    tempy1l = tempy1l + (s_tempy1[offset - (0)]) * (fac1);
    tempz1l = tempz1l + (s_tempz1[offset - (0)]) * (fac1);
    fac2 = sh_hprimewgll_xx[(J) * (NGLLX) + 0 - (0)];
    offset = (K) * (NGLL2) + (0) * (NGLLX) + I;
    tempx2l = tempx2l + (s_tempx2[offset - (0)]) * (fac2);
    tempy2l = tempy2l + (s_tempy2[offset - (0)]) * (fac2);
    tempz2l = tempz2l + (s_tempz2[offset - (0)]) * (fac2);
    fac3 = sh_hprimewgll_xx[(K) * (NGLLX) + 0 - (0)];
    offset = (0) * (NGLL2) + (J) * (NGLLX) + I;
    tempx3l = tempx3l + (s_tempx3[offset - (0)]) * (fac3);
    tempy3l = tempy3l + (s_tempy3[offset - (0)]) * (fac3);
    tempz3l = tempz3l + (s_tempz3[offset - (0)]) * (fac3);
    fac1 = sh_hprimewgll_xx[(I) * (NGLLX) + 1 - (0)];
    offset = (K) * (NGLL2) + (J) * (NGLLX) + 1;
    tempx1l = tempx1l + (s_tempx1[offset - (0)]) * (fac1);
    tempy1l = tempy1l + (s_tempy1[offset - (0)]) * (fac1);
    tempz1l = tempz1l + (s_tempz1[offset - (0)]) * (fac1);
    fac2 = sh_hprimewgll_xx[(J) * (NGLLX) + 1 - (0)];
    offset = (K) * (NGLL2) + (1) * (NGLLX) + I;
    tempx2l = tempx2l + (s_tempx2[offset - (0)]) * (fac2);
    tempy2l = tempy2l + (s_tempy2[offset - (0)]) * (fac2);
    tempz2l = tempz2l + (s_tempz2[offset - (0)]) * (fac2);
    fac3 = sh_hprimewgll_xx[(K) * (NGLLX) + 1 - (0)];
    offset = (1) * (NGLL2) + (J) * (NGLLX) + I;
    tempx3l = tempx3l + (s_tempx3[offset - (0)]) * (fac3);
    tempy3l = tempy3l + (s_tempy3[offset - (0)]) * (fac3);
    tempz3l = tempz3l + (s_tempz3[offset - (0)]) * (fac3);
    fac1 = sh_hprimewgll_xx[(I) * (NGLLX) + 2 - (0)];
    offset = (K) * (NGLL2) + (J) * (NGLLX) + 2;
    tempx1l = tempx1l + (s_tempx1[offset - (0)]) * (fac1);
    tempy1l = tempy1l + (s_tempy1[offset - (0)]) * (fac1);
    tempz1l = tempz1l + (s_tempz1[offset - (0)]) * (fac1);
    fac2 = sh_hprimewgll_xx[(J) * (NGLLX) + 2 - (0)];
    offset = (K) * (NGLL2) + (2) * (NGLLX) + I;
    tempx2l = tempx2l + (s_tempx2[offset - (0)]) * (fac2);
    tempy2l = tempy2l + (s_tempy2[offset - (0)]) * (fac2);
    tempz2l = tempz2l + (s_tempz2[offset - (0)]) * (fac2);
    fac3 = sh_hprimewgll_xx[(K) * (NGLLX) + 2 - (0)];
    offset = (2) * (NGLL2) + (J) * (NGLLX) + I;
    tempx3l = tempx3l + (s_tempx3[offset - (0)]) * (fac3);
    tempy3l = tempy3l + (s_tempy3[offset - (0)]) * (fac3);
    tempz3l = tempz3l + (s_tempz3[offset - (0)]) * (fac3);
    fac1 = sh_hprimewgll_xx[(I) * (NGLLX) + 3 - (0)];
    offset = (K) * (NGLL2) + (J) * (NGLLX) + 3;
    tempx1l = tempx1l + (s_tempx1[offset - (0)]) * (fac1);
    tempy1l = tempy1l + (s_tempy1[offset - (0)]) * (fac1);
    tempz1l = tempz1l + (s_tempz1[offset - (0)]) * (fac1);
    fac2 = sh_hprimewgll_xx[(J) * (NGLLX) + 3 - (0)];
    offset = (K) * (NGLL2) + (3) * (NGLLX) + I;
    tempx2l = tempx2l + (s_tempx2[offset - (0)]) * (fac2);
    tempy2l = tempy2l + (s_tempy2[offset - (0)]) * (fac2);
    tempz2l = tempz2l + (s_tempz2[offset - (0)]) * (fac2);
    fac3 = sh_hprimewgll_xx[(K) * (NGLLX) + 3 - (0)];
    offset = (3) * (NGLL2) + (J) * (NGLLX) + I;
    tempx3l = tempx3l + (s_tempx3[offset - (0)]) * (fac3);
    tempy3l = tempy3l + (s_tempy3[offset - (0)]) * (fac3);
    tempz3l = tempz3l + (s_tempz3[offset - (0)]) * (fac3);
    fac1 = sh_hprimewgll_xx[(I) * (NGLLX) + 4 - (0)];
    offset = (K) * (NGLL2) + (J) * (NGLLX) + 4;
    tempx1l = tempx1l + (s_tempx1[offset - (0)]) * (fac1);
    tempy1l = tempy1l + (s_tempy1[offset - (0)]) * (fac1);
    tempz1l = tempz1l + (s_tempz1[offset - (0)]) * (fac1);
    fac2 = sh_hprimewgll_xx[(J) * (NGLLX) + 4 - (0)];
    offset = (K) * (NGLL2) + (4) * (NGLLX) + I;
    tempx2l = tempx2l + (s_tempx2[offset - (0)]) * (fac2);
    tempy2l = tempy2l + (s_tempy2[offset - (0)]) * (fac2);
    tempz2l = tempz2l + (s_tempz2[offset - (0)]) * (fac2);
    fac3 = sh_hprimewgll_xx[(K) * (NGLLX) + 4 - (0)];
    offset = (4) * (NGLL2) + (J) * (NGLLX) + I;
    tempx3l = tempx3l + (s_tempx3[offset - (0)]) * (fac3);
    tempy3l = tempy3l + (s_tempy3[offset - (0)]) * (fac3);
    tempz3l = tempz3l + (s_tempz3[offset - (0)]) * (fac3);
#else
    for (l = 0; l <= NGLLX - (1); l += 1) {
      fac1 = sh_hprimewgll_xx[(I) * (NGLLX) + l - (0)];
      offset = (K) * (NGLL2) + (J) * (NGLLX) + l;
      tempx1l = tempx1l + (s_tempx1[offset - (0)]) * (fac1);
      tempy1l = tempy1l + (s_tempy1[offset - (0)]) * (fac1);
      tempz1l = tempz1l + (s_tempz1[offset - (0)]) * (fac1);
      fac2 = sh_hprimewgll_xx[(J) * (NGLLX) + l - (0)];
      offset = (K) * (NGLL2) + (l) * (NGLLX) + I;
      tempx2l = tempx2l + (s_tempx2[offset - (0)]) * (fac2);
      tempy2l = tempy2l + (s_tempy2[offset - (0)]) * (fac2);
      tempz2l = tempz2l + (s_tempz2[offset - (0)]) * (fac2);
      fac3 = sh_hprimewgll_xx[(K) * (NGLLX) + l - (0)];
      offset = (l) * (NGLL2) + (J) * (NGLLX) + I;
      tempx3l = tempx3l + (s_tempx3[offset - (0)]) * (fac3);
      tempy3l = tempy3l + (s_tempy3[offset - (0)]) * (fac3);
      tempz3l = tempz3l + (s_tempz3[offset - (0)]) * (fac3);
    }
#endif
    fac1 = d_wgllwgll_yz[(K) * (NGLLX) + J - (0)];
    fac2 = d_wgllwgll_xz[(K) * (NGLLX) + I - (0)];
    fac3 = d_wgllwgll_xy[(J) * (NGLLX) + I - (0)];
    sum_terms1 =  -((fac1) * (tempx1l) + (fac2) * (tempx2l) + (fac3) * (tempx3l));
    sum_terms2 =  -((fac1) * (tempy1l) + (fac2) * (tempy2l) + (fac3) * (tempy3l));
    sum_terms3 =  -((fac1) * (tempz1l) + (fac2) * (tempz2l) + (fac3) * (tempz3l));
    if (GRAVITY) {
      sum_terms1 = sum_terms1 + rho_s_H1;
      sum_terms2 = sum_terms2 + rho_s_H2;
      sum_terms3 = sum_terms3 + rho_s_H3;
    }
#ifdef USE_MESH_COLORING_GPU
#ifdef USE_TEXTURES_FIELDS
    d_accel[0 - (0) + (iglob - (0)) * (3)] = tex1Dfetch(d_accel_ic_tex,(iglob) * (3) + 0) + sum_terms1;
    d_accel[1 - (0) + (iglob - (0)) * (3)] = tex1Dfetch(d_accel_ic_tex,(iglob) * (3) + 1) + sum_terms2;
    d_accel[2 - (0) + (iglob - (0)) * (3)] = tex1Dfetch(d_accel_ic_tex,(iglob) * (3) + 2) + sum_terms3;
#else
    d_accel[0 - (0) + (iglob - (0)) * (3)] = d_accel[0 - (0) + (iglob - (0)) * (3)] + sum_terms1;
    d_accel[1 - (0) + (iglob - (0)) * (3)] = d_accel[1 - (0) + (iglob - (0)) * (3)] + sum_terms2;
    d_accel[2 - (0) + (iglob - (0)) * (3)] = d_accel[2 - (0) + (iglob - (0)) * (3)] + sum_terms3;
#endif
#else
    if (use_mesh_coloring_gpu) {
      if (NSPEC_INNER_CORE > 1000) {
#ifdef USE_TEXTURES_FIELDS
        d_accel[0 - (0) + (iglob - (0)) * (3)] = tex1Dfetch(d_accel_ic_tex,(iglob) * (3) + 0) + sum_terms1;
        d_accel[1 - (0) + (iglob - (0)) * (3)] = tex1Dfetch(d_accel_ic_tex,(iglob) * (3) + 1) + sum_terms2;
        d_accel[2 - (0) + (iglob - (0)) * (3)] = tex1Dfetch(d_accel_ic_tex,(iglob) * (3) + 2) + sum_terms3;
#else
        d_accel[0 - (0) + (iglob - (0)) * (3)] = d_accel[0 - (0) + (iglob - (0)) * (3)] + sum_terms1;
        d_accel[1 - (0) + (iglob - (0)) * (3)] = d_accel[1 - (0) + (iglob - (0)) * (3)] + sum_terms2;
        d_accel[2 - (0) + (iglob - (0)) * (3)] = d_accel[2 - (0) + (iglob - (0)) * (3)] + sum_terms3;
#endif
      } else {
        atomicAdd(d_accel + (iglob) * (3) + 0, sum_terms1);
        atomicAdd(d_accel + (iglob) * (3) + 1, sum_terms2);
        atomicAdd(d_accel + (iglob) * (3) + 2, sum_terms3);
      }
    } else {
      atomicAdd(d_accel + (iglob) * (3) + 0, sum_terms1);
      atomicAdd(d_accel + (iglob) * (3) + 1, sum_terms2);
      atomicAdd(d_accel + (iglob) * (3) + 2, sum_terms3);
    }
#endif
    if (ATTENUATION &&  ! PARTIAL_PHYS_DISPERSION_ONLY) {
      compute_element_ic_att_memory(tx, working_element, d_muvstore, factor_common, alphaval, betaval, gammaval, R_xx, R_yy, R_xy, R_xz, R_yz, epsilondev_xx, epsilondev_yy, epsilondev_xy, epsilondev_xz, epsilondev_yz, epsilondev_xx_loc, epsilondev_yy_loc, epsilondev_xy_loc, epsilondev_xz_loc, epsilondev_yz_loc, USE_3D_ATTENUATION_ARRAYS);
    }
    if (COMPUTE_AND_STORE_STRAIN) {
      epsilondev_xx[tx + (working_element) * (NGLL3) - (0)] = epsilondev_xx_loc;
      epsilondev_yy[tx + (working_element) * (NGLL3) - (0)] = epsilondev_yy_loc;
      epsilondev_xy[tx + (working_element) * (NGLL3) - (0)] = epsilondev_xy_loc;
      epsilondev_xz[tx + (working_element) * (NGLL3) - (0)] = epsilondev_xz_loc;
      epsilondev_yz[tx + (working_element) * (NGLL3) - (0)] = epsilondev_yz_loc;
    }
  }
}
