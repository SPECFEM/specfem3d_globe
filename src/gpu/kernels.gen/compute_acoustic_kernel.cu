
#include <hip/hip_runtime.h>
#ifndef INDEX2
#define INDEX2(isize,i,j) i + isize*j
#endif
#ifndef INDEX3
#define INDEX3(isize,jsize,i,j,k) i + isize*(j + jsize*k)
#endif
#ifndef INDEX4
#define INDEX4(isize,jsize,ksize,i,j,k,x) i + isize*(j + jsize*(k + ksize*x))
#endif
#ifndef INDEX5
#define INDEX5(isize,jsize,ksize,xsize,i,j,k,x,y) i + isize*(j + jsize*(k + ksize*(x + xsize*y)))
#endif
#ifndef NDIM
#define NDIM 3
#endif
#ifndef NGLLX
#define NGLLX 5
#endif
#ifndef NGLL2
#define NGLL2 25
#endif
#ifndef NGLL3
#define NGLL3 125
#endif
#ifndef NGLL3_PADDED
#define NGLL3_PADDED 128
#endif
#ifndef N_SLS
#define N_SLS 3
#endif
#ifndef IREGION_CRUST_MANTLE
#define IREGION_CRUST_MANTLE 1
#endif
#ifndef IREGION_INNER_CORE
#define IREGION_INNER_CORE 3
#endif
#ifndef IFLAG_IN_FICTITIOUS_CUBE
#define IFLAG_IN_FICTITIOUS_CUBE 11
#endif
#ifndef R_EARTH_KM
#define R_EARTH_KM 6371.0f
#endif
#ifndef COLORING_MIN_NSPEC_INNER_CORE
#define COLORING_MIN_NSPEC_INNER_CORE 1000
#endif
#ifndef COLORING_MIN_NSPEC_OUTER_CORE
#define COLORING_MIN_NSPEC_OUTER_CORE 1000
#endif
#ifndef BLOCKSIZE_TRANSFER
#define BLOCKSIZE_TRANSFER 256
#endif
static __device__ void compute_gradient_kernel(const int ijk, const int ispec, const float * scalar_field, float * vector_field_element, const float * hprime_xx, const float * d_xix, const float * d_xiy, const float * d_xiz, const float * d_etax, const float * d_etay, const float * d_etaz, const float * d_gammax, const float * d_gammay, const float * d_gammaz){
  float temp1l;
  float temp2l;
  float temp3l;
  float hp1;
  float hp2;
  float hp3;
  float xixl;
  float xiyl;
  float xizl;
  float etaxl;
  float etayl;
  float etazl;
  float gammaxl;
  float gammayl;
  float gammazl;
  int l;
  int offset;
  int offset1;
  int offset2;
  int offset3;
  int I;
  int J;
  int K;
  K = (ijk) / (NGLL2);
  J = (ijk - ((K) * (NGLL2))) / (NGLLX);
  I = ijk - ((K) * (NGLL2)) - ((J) * (NGLLX));
  temp1l = 0.0f;
  temp2l = 0.0f;
  temp3l = 0.0f;
  for(l=0; l<=NGLLX - (1); l+=1){
    hp1 = hprime_xx[(l) * (NGLLX) + I - (0)];
    hp2 = hprime_xx[(l) * (NGLLX) + J - (0)];
    hp3 = hprime_xx[(l) * (NGLLX) + K - (0)];
    offset1 = (K) * (NGLL2) + (J) * (NGLLX) + l;
    offset2 = (K) * (NGLL2) + (l) * (NGLLX) + I;
    offset3 = (l) * (NGLL2) + (J) * (NGLLX) + I;
    temp1l = temp1l + (scalar_field[offset1 - (0)]) * (hp1);
    temp2l = temp2l + (scalar_field[offset2 - (0)]) * (hp2);
    temp3l = temp3l + (scalar_field[offset3 - (0)]) * (hp3);
  }
  offset = (ispec) * (NGLL3_PADDED) + ijk;
  xixl = d_xix[offset - (0)];
  xiyl = d_xiy[offset - (0)];
  xizl = d_xiz[offset - (0)];
  etaxl = d_etax[offset - (0)];
  etayl = d_etay[offset - (0)];
  etazl = d_etaz[offset - (0)];
  gammaxl = d_gammax[offset - (0)];
  gammayl = d_gammay[offset - (0)];
  gammazl = d_gammaz[offset - (0)];
  vector_field_element[0 - (0)] = (temp1l) * (xixl) + (temp2l) * (etaxl) + (temp3l) * (gammaxl);
  vector_field_element[1 - (0)] = (temp1l) * (xiyl) + (temp2l) * (etayl) + (temp3l) * (gammayl);
  vector_field_element[2 - (0)] = (temp1l) * (xizl) + (temp2l) * (etazl) + (temp3l) * (gammazl);
}
__global__ void compute_acoustic_kernel(const int * ibool, const float * rhostore, const float * kappastore, const float * hprime_xx, const float * d_xix, const float * d_xiy, const float * d_xiz, const float * d_etax, const float * d_etay, const float * d_etaz, const float * d_gammax, const float * d_gammay, const float * d_gammaz, const float * potential_dot_dot_acoustic, const float * b_potential_acoustic, const float * b_potential_dot_dot_acoustic, float * rho_ac_kl, float * kappa_ac_kl, const float deltat, const int NSPEC){
  int ispec;
  int ijk;
  int ijk_ispec;
  int ijk_ispec_padded;
  int iglob;
  float accel_elm[3];
  float b_displ_elm[3];
  float rhol;
  float kappal;
  float div_displ;
  float b_div_displ;
  __shared__ float scalar_field_displ[NGLL3 + 0 - (1) - (0) + 1];
  __shared__ float scalar_field_accel[NGLL3 + 0 - (1) - (0) + 1];
  ispec = blockIdx.x + (blockIdx.y) * (gridDim.x);
  if(ispec < NSPEC){
    ijk = threadIdx.x;
    ijk_ispec = ijk + (NGLL3) * (ispec);
    ijk_ispec_padded = ijk + (NGLL3_PADDED) * (ispec);
    iglob = ibool[ijk_ispec - (0)] - (1);
    scalar_field_displ[ijk - (0)] = b_potential_acoustic[iglob - (0)];
    scalar_field_accel[ijk - (0)] = potential_dot_dot_acoustic[iglob - (0)];
    __syncthreads();
    compute_gradient_kernel(ijk, ispec, scalar_field_displ, b_displ_elm, hprime_xx, d_xix, d_xiy, d_xiz, d_etax, d_etay, d_etaz, d_gammax, d_gammay, d_gammaz);
    compute_gradient_kernel(ijk, ispec, scalar_field_accel, accel_elm, hprime_xx, d_xix, d_xiy, d_xiz, d_etax, d_etay, d_etaz, d_gammax, d_gammay, d_gammaz);
    rhol = rhostore[ijk_ispec_padded - (0)];
    rho_ac_kl[ijk_ispec - (0)] = rho_ac_kl[ijk_ispec - (0)] + ((deltat) * (rhol)) * ((accel_elm[0 - (0)]) * (b_displ_elm[0 - (0)]) + (accel_elm[1 - (0)]) * (b_displ_elm[1 - (0)]) + (accel_elm[2 - (0)]) * (b_displ_elm[2 - (0)]));
    kappal = (rhol) / (kappastore[ijk_ispec_padded - (0)]);
    div_displ = (kappal) * (potential_dot_dot_acoustic[iglob - (0)]);
    b_div_displ = (kappal) * (b_potential_dot_dot_acoustic[iglob - (0)]);
    kappa_ac_kl[ijk_ispec - (0)] = kappa_ac_kl[ijk_ispec - (0)] + ((deltat) * (div_displ)) * (b_div_displ);
  }
}
