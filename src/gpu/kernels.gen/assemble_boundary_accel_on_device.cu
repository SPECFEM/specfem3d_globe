
#include <hip/hip_runtime.h>
#ifndef INDEX2
#define INDEX2(isize,i,j) i + isize*j
#endif
#ifndef INDEX3
#define INDEX3(isize,jsize,i,j,k) i + isize*(j + jsize*k)
#endif
#ifndef INDEX4
#define INDEX4(isize,jsize,ksize,i,j,k,x) i + isize*(j + jsize*(k + ksize*x))
#endif
#ifndef INDEX5
#define INDEX5(isize,jsize,ksize,xsize,i,j,k,x,y) i + isize*(j + jsize*(k + ksize*(x + xsize*y)))
#endif
#ifndef NDIM
#define NDIM 3
#endif
#ifndef NGLLX
#define NGLLX 5
#endif
#ifndef NGLL2
#define NGLL2 25
#endif
#ifndef NGLL3
#define NGLL3 125
#endif
#ifndef NGLL3_PADDED
#define NGLL3_PADDED 128
#endif
#ifndef N_SLS
#define N_SLS 3
#endif
#ifndef IREGION_CRUST_MANTLE
#define IREGION_CRUST_MANTLE 1
#endif
#ifndef IREGION_INNER_CORE
#define IREGION_INNER_CORE 3
#endif
#ifndef IFLAG_IN_FICTITIOUS_CUBE
#define IFLAG_IN_FICTITIOUS_CUBE 11
#endif
#ifndef R_EARTH_KM
#define R_EARTH_KM 6371.0f
#endif
#ifndef COLORING_MIN_NSPEC_INNER_CORE
#define COLORING_MIN_NSPEC_INNER_CORE 1000
#endif
#ifndef COLORING_MIN_NSPEC_OUTER_CORE
#define COLORING_MIN_NSPEC_OUTER_CORE 1000
#endif
#ifndef BLOCKSIZE_TRANSFER
#define BLOCKSIZE_TRANSFER 256
#endif
__global__ void assemble_boundary_accel_on_device(float * d_accel, const float * d_send_accel_buffer, const int num_interfaces, const int max_nibool_interfaces, const int * d_nibool_interfaces, const int * d_ibool_interfaces){
  int id;
  int iglob;
  int iloc;
  int iinterface;
  id = threadIdx.x + (blockIdx.x) * (blockDim.x) + ((gridDim.x) * (blockDim.x)) * (threadIdx.y + (blockIdx.y) * (blockDim.y));
  for(iinterface=0; iinterface<=num_interfaces - (1); iinterface+=1){
    if(id < d_nibool_interfaces[iinterface - 0]){
      iloc = id + (max_nibool_interfaces) * (iinterface);
      iglob = d_ibool_interfaces[iloc - 0] - (1);
      atomicAdd(d_accel + (iglob) * (3) + 0, d_send_accel_buffer[(iloc) * (3) + 0 - 0]);
      atomicAdd(d_accel + (iglob) * (3) + 1, d_send_accel_buffer[(iloc) * (3) + 1 - 0]);
      atomicAdd(d_accel + (iglob) * (3) + 2, d_send_accel_buffer[(iloc) * (3) + 2 - 0]);
    }
  }
}
