
#include <hip/hip_runtime.h>
//note: please do not modify this file manually!
//      this file has been generated automatically by BOAST version 2.1.0
//      by: make boast_kernels

/*
!=====================================================================
!
!          S p e c f e m 3 D  G l o b e  V e r s i o n  7 . 0
!          --------------------------------------------------
!
!     Main historical authors: Dimitri Komatitsch and Jeroen Tromp
!                        Princeton University, USA
!                and CNRS / University of Marseille, France
!                 (there are currently many more authors!)
! (c) Princeton University and CNRS / University of Marseille, April 2014
!
! This program is free software; you can redistribute it and/or modify
! it under the terms of the GNU General Public License as published by
! the Free Software Foundation; either version 3 of the License, or
! (at your option) any later version.
!
! This program is distributed in the hope that it will be useful,
! but WITHOUT ANY WARRANTY; without even the implied warranty of
! MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
! GNU General Public License for more details.
!
! You should have received a copy of the GNU General Public License along
! with this program; if not, write to the Free Software Foundation, Inc.,
! 51 Franklin Street, Fifth Floor, Boston, MA 02110-1301 USA.
!
!=====================================================================
*/

#ifndef INDEX2
#define INDEX2(isize,i,j) i + isize*j
#endif
#ifndef INDEX3
#define INDEX3(isize,jsize,i,j,k) i + isize*(j + jsize*k)
#endif
#ifndef INDEX4
#define INDEX4(isize,jsize,ksize,i,j,k,x) i + isize*(j + jsize*(k + ksize*x))
#endif
#ifndef INDEX5
#define INDEX5(isize,jsize,ksize,xsize,i,j,k,x,y) i + isize*(j + jsize*(k + ksize*(x + xsize*y)))
#endif

#ifndef NDIM
#define NDIM 3
#endif
#ifndef NGLLX
#define NGLLX 5
#endif
#ifndef NGLL2
#define NGLL2 25
#endif
#ifndef NGLL3
#define NGLL3 125
#endif
#ifndef NGLL3_PADDED
#define NGLL3_PADDED 128
#endif
#ifndef N_SLS
#define N_SLS 3
#endif
#ifndef IREGION_CRUST_MANTLE
#define IREGION_CRUST_MANTLE 1
#endif
#ifndef IREGION_INNER_CORE
#define IREGION_INNER_CORE 3
#endif
#ifndef IFLAG_IN_FICTITIOUS_CUBE
#define IFLAG_IN_FICTITIOUS_CUBE 11
#endif
#ifndef R_EARTH_KM
#define R_EARTH_KM 6371.0f
#endif
#ifndef COLORING_MIN_NSPEC_INNER_CORE
#define COLORING_MIN_NSPEC_INNER_CORE 1000
#endif
#ifndef COLORING_MIN_NSPEC_OUTER_CORE
#define COLORING_MIN_NSPEC_OUTER_CORE 1000
#endif
#ifndef BLOCKSIZE_TRANSFER
#define BLOCKSIZE_TRANSFER 256
#endif

#ifdef USE_TEXTURES_CONSTANTS
#undef USE_TEXTURES_CONSTANTS
#endif

static __device__ void compute_element_ic_att_stress(const int tx, const int working_element, const float * R_xx, const float * R_yy, const float * R_xy, const float * R_xz, const float * R_yz, float * sigma_xx, float * sigma_yy, float * sigma_zz, float * sigma_xy, float * sigma_xz, float * sigma_yz){
  int offset;
  int i_sls;
  float R_xx_val;
  float R_yy_val;
  for (i_sls = 0; i_sls <= N_SLS - (1); i_sls += 1) {
    offset = tx + (NGLL3) * (i_sls + (N_SLS) * (working_element));
    R_xx_val = R_xx[offset];
    R_yy_val = R_yy[offset];
    sigma_xx[0] = sigma_xx[0] - (R_xx_val);
    sigma_yy[0] = sigma_yy[0] - (R_yy_val);
    sigma_zz[0] = sigma_zz[0] + R_xx_val + R_yy_val;
    sigma_xy[0] = sigma_xy[0] - (R_xy[offset]);
    sigma_xz[0] = sigma_xz[0] - (R_xz[offset]);
    sigma_yz[0] = sigma_yz[0] - (R_yz[offset]);
  }
}

static __device__ void compute_element_ic_att_memory(const int tx, const int working_element, const float * d_muv, const float * factor_common, const float * alphaval, const float * betaval, const float * gammaval, float * R_xx, float * R_yy, float * R_xy, float * R_xz, float * R_yz, const float * epsilondev_xx, const float * epsilondev_yy, const float * epsilondev_xy, const float * epsilondev_xz, const float * epsilondev_yz, const float epsilondev_xx_loc, const float epsilondev_yy_loc, const float epsilondev_xy_loc, const float epsilondev_xz_loc, const float epsilondev_yz_loc, const int USE_3D_ATTENUATION_ARRAYS){
  int offset;
  int i_sls;
  float mul;
  float alphaval_loc;
  float betaval_loc;
  float gammaval_loc;
  float factor_loc;
  float sn;
  float snp1;
  mul = d_muv[tx + (NGLL3_PADDED) * (working_element)];
  for (i_sls = 0; i_sls <= N_SLS - (1); i_sls += 1) {
    offset = tx + (NGLL3) * (i_sls + (N_SLS) * (working_element));
    if (USE_3D_ATTENUATION_ARRAYS) {
      factor_loc = (mul) * (factor_common[offset]);
    } else {
      factor_loc = (mul) * (factor_common[i_sls + (N_SLS) * (working_element)]);
    }
    alphaval_loc = alphaval[i_sls];
    betaval_loc = betaval[i_sls];
    gammaval_loc = gammaval[i_sls];
    sn = (factor_loc) * (epsilondev_xx[tx + (NGLL3) * (working_element)]);
    snp1 = (factor_loc) * (epsilondev_xx_loc);
    R_xx[offset] = (alphaval_loc) * (R_xx[offset]) + (betaval_loc) * (sn) + (gammaval_loc) * (snp1);
    sn = (factor_loc) * (epsilondev_yy[tx + (NGLL3) * (working_element)]);
    snp1 = (factor_loc) * (epsilondev_yy_loc);
    R_yy[offset] = (alphaval_loc) * (R_yy[offset]) + (betaval_loc) * (sn) + (gammaval_loc) * (snp1);
    sn = (factor_loc) * (epsilondev_xy[tx + (NGLL3) * (working_element)]);
    snp1 = (factor_loc) * (epsilondev_xy_loc);
    R_xy[offset] = (alphaval_loc) * (R_xy[offset]) + (betaval_loc) * (sn) + (gammaval_loc) * (snp1);
    sn = (factor_loc) * (epsilondev_xz[tx + (NGLL3) * (working_element)]);
    snp1 = (factor_loc) * (epsilondev_xz_loc);
    R_xz[offset] = (alphaval_loc) * (R_xz[offset]) + (betaval_loc) * (sn) + (gammaval_loc) * (snp1);
    sn = (factor_loc) * (epsilondev_yz[tx + (NGLL3) * (working_element)]);
    snp1 = (factor_loc) * (epsilondev_yz_loc);
    R_yz[offset] = (alphaval_loc) * (R_yz[offset]) + (betaval_loc) * (sn) + (gammaval_loc) * (snp1);
  }
}

static __device__ void compute_element_ic_gravity(const int tx, const int iglob, const float * __restrict__ d_rstore, const float * __restrict__ d_minus_gravity_table, const float * __restrict__ d_minus_deriv_gravity_table, const float * __restrict__ d_density_table, const float * __restrict__ wgll_cube, const float jacobianl, const float * s_dummyx_loc, const float * s_dummyy_loc, const float * s_dummyz_loc, float * sigma_xx, float * sigma_yy, float * sigma_zz, float * sigma_xy, float * sigma_yx, float * sigma_xz, float * sigma_zx, float * sigma_yz, float * sigma_zy, float * rho_s_H1, float * rho_s_H2, float * rho_s_H3){
  float radius;
  float theta;
  float phi;
  float cos_theta;
  float sin_theta;
  float cos_phi;
  float sin_phi;
  float cos_theta_sq;
  float sin_theta_sq;
  float cos_phi_sq;
  float sin_phi_sq;
  float minus_g;
  float minus_dg;
  float rho;
  float gxl;
  float gyl;
  float gzl;
  float minus_g_over_radius;
  float minus_dg_plus_g_over_radius;
  float Hxxl;
  float Hyyl;
  float Hzzl;
  float Hxyl;
  float Hxzl;
  float Hyzl;
  float sx_l;
  float sy_l;
  float sz_l;
  float factor;
  int int_radius;
  radius = d_rstore[0 + (3) * (iglob)];
  theta = d_rstore[1 + (3) * (iglob)];
  phi = d_rstore[2 + (3) * (iglob)];
  if (radius < 1.5696123057604773e-05f) {
    radius = 1.5696123057604773e-05f;
  }
  sincosf(theta,  &sin_theta,  &cos_theta);
  sincosf(phi,  &sin_phi,  &cos_phi);
  int_radius = rint(((radius) * (6371.0f)) * (10.0f)) - (1);
  if (int_radius < 0) {
    int_radius = 0;
  }
  minus_g = d_minus_gravity_table[int_radius];
  minus_dg = d_minus_deriv_gravity_table[int_radius];
  rho = d_density_table[int_radius];
  gxl = ((minus_g) * (sin_theta)) * (cos_phi);
  gyl = ((minus_g) * (sin_theta)) * (sin_phi);
  gzl = (minus_g) * (cos_theta);
  minus_g_over_radius = (minus_g) / (radius);
  minus_dg_plus_g_over_radius = minus_dg - (minus_g_over_radius);
  cos_theta_sq = (cos_theta) * (cos_theta);
  sin_theta_sq = (sin_theta) * (sin_theta);
  cos_phi_sq = (cos_phi) * (cos_phi);
  sin_phi_sq = (sin_phi) * (sin_phi);
  Hxxl = (minus_g_over_radius) * ((cos_phi_sq) * (cos_theta_sq) + sin_phi_sq) + ((cos_phi_sq) * (minus_dg)) * (sin_theta_sq);
  Hyyl = (minus_g_over_radius) * (cos_phi_sq + (cos_theta_sq) * (sin_phi_sq)) + ((minus_dg) * (sin_phi_sq)) * (sin_theta_sq);
  Hzzl = (cos_theta_sq) * (minus_dg) + (minus_g_over_radius) * (sin_theta_sq);
  Hxyl = (((cos_phi) * (minus_dg_plus_g_over_radius)) * (sin_phi)) * (sin_theta_sq);
  Hxzl = (((cos_phi) * (cos_theta)) * (minus_dg_plus_g_over_radius)) * (sin_theta);
  Hyzl = (((cos_theta) * (minus_dg_plus_g_over_radius)) * (sin_phi)) * (sin_theta);
  sx_l = (rho) * (s_dummyx_loc[tx]);
  sy_l = (rho) * (s_dummyy_loc[tx]);
  sz_l = (rho) * (s_dummyz_loc[tx]);
  *(sigma_xx) = *(sigma_xx) + (sy_l) * (gyl) + (sz_l) * (gzl);
  *(sigma_yy) = *(sigma_yy) + (sx_l) * (gxl) + (sz_l) * (gzl);
  *(sigma_zz) = *(sigma_zz) + (sx_l) * (gxl) + (sy_l) * (gyl);
  *(sigma_xy) = *(sigma_xy) - ((sx_l) * (gyl));
  *(sigma_yx) = *(sigma_yx) - ((sy_l) * (gxl));
  *(sigma_xz) = *(sigma_xz) - ((sx_l) * (gzl));
  *(sigma_zx) = *(sigma_zx) - ((sz_l) * (gxl));
  *(sigma_yz) = *(sigma_yz) - ((sy_l) * (gzl));
  *(sigma_zy) = *(sigma_zy) - ((sz_l) * (gyl));
  factor = (jacobianl) * (wgll_cube[tx]);
  rho_s_H1[0] = (factor) * ((sx_l) * (Hxxl) + (sy_l) * (Hxyl) + (sz_l) * (Hxzl));
  rho_s_H2[0] = (factor) * ((sx_l) * (Hxyl) + (sy_l) * (Hyyl) + (sz_l) * (Hyzl));
  rho_s_H3[0] = (factor) * ((sx_l) * (Hxzl) + (sy_l) * (Hyzl) + (sz_l) * (Hzzl));
}


/*----------------------------------------------*/
// main function
/*----------------------------------------------*/

__global__
#ifdef USE_LAUNCH_BOUNDS
__launch_bounds__(NGLL3_PADDED, LAUNCH_MIN_BLOCKS)
#endif
 void inner_core_impl_kernel_adjoint(const int nb_blocks_to_compute, const int * d_ibool, const int * d_idoubling, const int * d_phase_ispec_inner, const int num_phase_ispec, const int d_iphase, const float deltat, const int use_mesh_coloring_gpu, const float * __restrict__ d_displ, float * d_accel, const float * __restrict__ d_xix, const float * __restrict__ d_xiy, const float * __restrict__ d_xiz, const float * __restrict__ d_etax, const float * __restrict__ d_etay, const float * __restrict__ d_etaz, const float * __restrict__ d_gammax, const float * __restrict__ d_gammay, const float * __restrict__ d_gammaz, const float * __restrict__ d_hprime_xx, const float * __restrict__ d_hprimewgll_xx, const float * __restrict__ d_wgllwgll_xy, const float * __restrict__ d_wgllwgll_xz, const float * __restrict__ d_wgllwgll_yz, const float * __restrict__ d_kappavstore, const float * __restrict__ d_muvstore, const int COMPUTE_AND_STORE_STRAIN, float * epsilondev_xx, float * epsilondev_yy, float * epsilondev_xy, float * epsilondev_xz, float * epsilondev_yz, float * epsilon_trace_over_3, const int ATTENUATION, const int PARTIAL_PHYS_DISPERSION_ONLY, const int USE_3D_ATTENUATION_ARRAYS, const float * __restrict__ one_minus_sum_beta, const float * __restrict__ factor_common, float * R_xx, float * R_yy, float * R_xy, float * R_xz, float * R_yz, const float * __restrict__ alphaval, const float * __restrict__ betaval, const float * __restrict__ gammaval, const int ANISOTROPY, const float * __restrict__ d_c11store, const float * __restrict__ d_c12store, const float * __restrict__ d_c13store, const float * __restrict__ d_c33store, const float * __restrict__ d_c44store, const int GRAVITY, const float * __restrict__ d_rstore, const float * __restrict__ d_minus_gravity_table, const float * __restrict__ d_minus_deriv_gravity_table, const float * __restrict__ d_density_table, const float * __restrict__ wgll_cube, const int NSPEC_INNER_CORE_STRAIN_ONLY, const int NSPEC_INNER_CORE){
  int bx;
  int tx;
  int K;
  int J;
  int I;
#ifndef MANUALLY_UNROLLED_LOOPS
  int l;
#endif
  unsigned short active_1;
  int offset;
  int iglob_1;
  int working_element;
  float tempx1l;
  float tempx2l;
  float tempx3l;
  float tempy1l;
  float tempy2l;
  float tempy3l;
  float tempz1l;
  float tempz2l;
  float tempz3l;
  float xixl;
  float xiyl;
  float xizl;
  float etaxl;
  float etayl;
  float etazl;
  float gammaxl;
  float gammayl;
  float gammazl;
  float jacobianl;
  float duxdxl;
  float duxdyl;
  float duxdzl;
  float duydxl;
  float duydyl;
  float duydzl;
  float duzdxl;
  float duzdyl;
  float duzdzl;
  float duxdxl_plus_duydyl;
  float duxdxl_plus_duzdzl;
  float duydyl_plus_duzdzl;
  float duxdyl_plus_duydxl;
  float duzdxl_plus_duxdzl;
  float duzdyl_plus_duydzl;
  float templ;
  float fac1;
  float fac2;
  float fac3;
  float lambdal;
  float mul;
  float lambdalplus2mul;
  float kappal;
  float mul_iso;
  float mul_aniso;
  float sigma_xx;
  float sigma_xy;
  float sigma_xz;
  float sigma_yx;
  float sigma_yy;
  float sigma_yz;
  float sigma_zx;
  float sigma_zy;
  float sigma_zz;
  float epsilondev_xx_loc_1;
  float epsilondev_yy_loc_1;
  float epsilondev_xy_loc_1;
  float epsilondev_xz_loc_1;
  float epsilondev_yz_loc_1;
  float c11;
  float c12;
  float c13;
  float c33;
  float c44;
  float sum_terms1;
  float sum_terms2;
  float sum_terms3;
  float rho_s_H_1_1;
  float rho_s_H_1_2;
  float rho_s_H_1_3;
  __shared__ float s_dummyx_loc[(NGLL3)];
  __shared__ float s_dummyy_loc[(NGLL3)];
  __shared__ float s_dummyz_loc[(NGLL3)];
  __shared__ float s_tempx1[(NGLL3)];
  __shared__ float s_tempx2[(NGLL3)];
  __shared__ float s_tempx3[(NGLL3)];
  __shared__ float s_tempy1[(NGLL3)];
  __shared__ float s_tempy2[(NGLL3)];
  __shared__ float s_tempy3[(NGLL3)];
  __shared__ float s_tempz1[(NGLL3)];
  __shared__ float s_tempz2[(NGLL3)];
  __shared__ float s_tempz3[(NGLL3)];
  __shared__ float sh_hprime_xx[(NGLL2)];
  __shared__ float sh_hprimewgll_xx[(NGLL2)];

  bx = (blockIdx.y) * (gridDim.x) + blockIdx.x;
  if (bx >= nb_blocks_to_compute) {
     return ;
  }

  tx = threadIdx.x + ((NGLL3_PADDED) * (0)) / (1);
  active_1 = (tx < NGLL3 ? 1 : 0);

  if (active_1) {
#ifdef USE_MESH_COLORING_GPU
    working_element = bx;
#else
    if (use_mesh_coloring_gpu) {
      working_element = bx;
    } else {
      working_element = d_phase_ispec_inner[bx + (num_phase_ispec) * (d_iphase - (1))] - (1);
    }
#endif
    if (d_idoubling[working_element] == IFLAG_IN_FICTITIOUS_CUBE) {
      active_1 = 0;
    } else {
      iglob_1 = d_ibool[(working_element) * (NGLL3) + tx] - (1);
#ifdef USE_TEXTURES_FIELDS
      s_dummyx_loc[tx] = tex1Dfetch(d_b_displ_ic_tex,(iglob_1) * (3) + 0);
      s_dummyy_loc[tx] = tex1Dfetch(d_b_displ_ic_tex,(iglob_1) * (3) + 1);
      s_dummyz_loc[tx] = tex1Dfetch(d_b_displ_ic_tex,(iglob_1) * (3) + 2);
#else
      s_dummyx_loc[tx] = d_displ[0 + (3) * (iglob_1)];
      s_dummyy_loc[tx] = d_displ[1 + (3) * (iglob_1)];
      s_dummyz_loc[tx] = d_displ[2 + (3) * (iglob_1)];
#endif
    }
  }

  if (tx < NGLL2) {
#ifdef USE_TEXTURES_CONSTANTS
    sh_hprime_xx[tx] = tex1Dfetch(d_hprime_xx_ic_tex,tx);
    sh_hprimewgll_xx[tx] = tex1Dfetch(d_hprimewgll_xx_ic_tex,tx);
#else
    sh_hprime_xx[tx] = d_hprime_xx[tx];
    sh_hprimewgll_xx[tx] = d_hprimewgll_xx[tx];
#endif
  }
  __syncthreads();

  K = (tx) / (NGLL2);
  J = (tx - ((K) * (NGLL2))) / (NGLLX);
  I = tx - ((K) * (NGLL2)) - ((J) * (NGLLX));

  if (active_1) {
    tempx1l = 0.0f;
    tempx2l = 0.0f;
    tempx3l = 0.0f;
    tempy1l = 0.0f;
    tempy2l = 0.0f;
    tempy3l = 0.0f;
    tempz1l = 0.0f;
    tempz2l = 0.0f;
    tempz3l = 0.0f;
#ifdef MANUALLY_UNROLLED_LOOPS
    fac1 = sh_hprime_xx[(0) * (NGLLX) + I];
    tempx1l = tempx1l + (s_dummyx_loc[(K) * (NGLL2) + (J) * (NGLLX) + 0]) * (fac1);
    tempy1l = tempy1l + (s_dummyy_loc[(K) * (NGLL2) + (J) * (NGLLX) + 0]) * (fac1);
    tempz1l = tempz1l + (s_dummyz_loc[(K) * (NGLL2) + (J) * (NGLLX) + 0]) * (fac1);
    fac2 = sh_hprime_xx[(0) * (NGLLX) + J];
    tempx2l = tempx2l + (s_dummyx_loc[(K) * (NGLL2) + (0) * (NGLLX) + I]) * (fac2);
    tempy2l = tempy2l + (s_dummyy_loc[(K) * (NGLL2) + (0) * (NGLLX) + I]) * (fac2);
    tempz2l = tempz2l + (s_dummyz_loc[(K) * (NGLL2) + (0) * (NGLLX) + I]) * (fac2);
    fac3 = sh_hprime_xx[(0) * (NGLLX) + K];
    tempx3l = tempx3l + (s_dummyx_loc[(0) * (NGLL2) + (J) * (NGLLX) + I]) * (fac3);
    tempy3l = tempy3l + (s_dummyy_loc[(0) * (NGLL2) + (J) * (NGLLX) + I]) * (fac3);
    tempz3l = tempz3l + (s_dummyz_loc[(0) * (NGLL2) + (J) * (NGLLX) + I]) * (fac3);
    fac1 = sh_hprime_xx[(1) * (NGLLX) + I];
    tempx1l = tempx1l + (s_dummyx_loc[(K) * (NGLL2) + (J) * (NGLLX) + 1]) * (fac1);
    tempy1l = tempy1l + (s_dummyy_loc[(K) * (NGLL2) + (J) * (NGLLX) + 1]) * (fac1);
    tempz1l = tempz1l + (s_dummyz_loc[(K) * (NGLL2) + (J) * (NGLLX) + 1]) * (fac1);
    fac2 = sh_hprime_xx[(1) * (NGLLX) + J];
    tempx2l = tempx2l + (s_dummyx_loc[(K) * (NGLL2) + (1) * (NGLLX) + I]) * (fac2);
    tempy2l = tempy2l + (s_dummyy_loc[(K) * (NGLL2) + (1) * (NGLLX) + I]) * (fac2);
    tempz2l = tempz2l + (s_dummyz_loc[(K) * (NGLL2) + (1) * (NGLLX) + I]) * (fac2);
    fac3 = sh_hprime_xx[(1) * (NGLLX) + K];
    tempx3l = tempx3l + (s_dummyx_loc[(1) * (NGLL2) + (J) * (NGLLX) + I]) * (fac3);
    tempy3l = tempy3l + (s_dummyy_loc[(1) * (NGLL2) + (J) * (NGLLX) + I]) * (fac3);
    tempz3l = tempz3l + (s_dummyz_loc[(1) * (NGLL2) + (J) * (NGLLX) + I]) * (fac3);
    fac1 = sh_hprime_xx[(2) * (NGLLX) + I];
    tempx1l = tempx1l + (s_dummyx_loc[(K) * (NGLL2) + (J) * (NGLLX) + 2]) * (fac1);
    tempy1l = tempy1l + (s_dummyy_loc[(K) * (NGLL2) + (J) * (NGLLX) + 2]) * (fac1);
    tempz1l = tempz1l + (s_dummyz_loc[(K) * (NGLL2) + (J) * (NGLLX) + 2]) * (fac1);
    fac2 = sh_hprime_xx[(2) * (NGLLX) + J];
    tempx2l = tempx2l + (s_dummyx_loc[(K) * (NGLL2) + (2) * (NGLLX) + I]) * (fac2);
    tempy2l = tempy2l + (s_dummyy_loc[(K) * (NGLL2) + (2) * (NGLLX) + I]) * (fac2);
    tempz2l = tempz2l + (s_dummyz_loc[(K) * (NGLL2) + (2) * (NGLLX) + I]) * (fac2);
    fac3 = sh_hprime_xx[(2) * (NGLLX) + K];
    tempx3l = tempx3l + (s_dummyx_loc[(2) * (NGLL2) + (J) * (NGLLX) + I]) * (fac3);
    tempy3l = tempy3l + (s_dummyy_loc[(2) * (NGLL2) + (J) * (NGLLX) + I]) * (fac3);
    tempz3l = tempz3l + (s_dummyz_loc[(2) * (NGLL2) + (J) * (NGLLX) + I]) * (fac3);
    fac1 = sh_hprime_xx[(3) * (NGLLX) + I];
    tempx1l = tempx1l + (s_dummyx_loc[(K) * (NGLL2) + (J) * (NGLLX) + 3]) * (fac1);
    tempy1l = tempy1l + (s_dummyy_loc[(K) * (NGLL2) + (J) * (NGLLX) + 3]) * (fac1);
    tempz1l = tempz1l + (s_dummyz_loc[(K) * (NGLL2) + (J) * (NGLLX) + 3]) * (fac1);
    fac2 = sh_hprime_xx[(3) * (NGLLX) + J];
    tempx2l = tempx2l + (s_dummyx_loc[(K) * (NGLL2) + (3) * (NGLLX) + I]) * (fac2);
    tempy2l = tempy2l + (s_dummyy_loc[(K) * (NGLL2) + (3) * (NGLLX) + I]) * (fac2);
    tempz2l = tempz2l + (s_dummyz_loc[(K) * (NGLL2) + (3) * (NGLLX) + I]) * (fac2);
    fac3 = sh_hprime_xx[(3) * (NGLLX) + K];
    tempx3l = tempx3l + (s_dummyx_loc[(3) * (NGLL2) + (J) * (NGLLX) + I]) * (fac3);
    tempy3l = tempy3l + (s_dummyy_loc[(3) * (NGLL2) + (J) * (NGLLX) + I]) * (fac3);
    tempz3l = tempz3l + (s_dummyz_loc[(3) * (NGLL2) + (J) * (NGLLX) + I]) * (fac3);
    fac1 = sh_hprime_xx[(4) * (NGLLX) + I];
    tempx1l = tempx1l + (s_dummyx_loc[(K) * (NGLL2) + (J) * (NGLLX) + 4]) * (fac1);
    tempy1l = tempy1l + (s_dummyy_loc[(K) * (NGLL2) + (J) * (NGLLX) + 4]) * (fac1);
    tempz1l = tempz1l + (s_dummyz_loc[(K) * (NGLL2) + (J) * (NGLLX) + 4]) * (fac1);
    fac2 = sh_hprime_xx[(4) * (NGLLX) + J];
    tempx2l = tempx2l + (s_dummyx_loc[(K) * (NGLL2) + (4) * (NGLLX) + I]) * (fac2);
    tempy2l = tempy2l + (s_dummyy_loc[(K) * (NGLL2) + (4) * (NGLLX) + I]) * (fac2);
    tempz2l = tempz2l + (s_dummyz_loc[(K) * (NGLL2) + (4) * (NGLLX) + I]) * (fac2);
    fac3 = sh_hprime_xx[(4) * (NGLLX) + K];
    tempx3l = tempx3l + (s_dummyx_loc[(4) * (NGLL2) + (J) * (NGLLX) + I]) * (fac3);
    tempy3l = tempy3l + (s_dummyy_loc[(4) * (NGLL2) + (J) * (NGLLX) + I]) * (fac3);
    tempz3l = tempz3l + (s_dummyz_loc[(4) * (NGLL2) + (J) * (NGLLX) + I]) * (fac3);
#else
    for (l = 0; l <= NGLLX - (1); l += 1) {
      fac1 = sh_hprime_xx[(l) * (NGLLX) + I];
      tempx1l = tempx1l + (s_dummyx_loc[(K) * (NGLL2) + (J) * (NGLLX) + l]) * (fac1);
      tempy1l = tempy1l + (s_dummyy_loc[(K) * (NGLL2) + (J) * (NGLLX) + l]) * (fac1);
      tempz1l = tempz1l + (s_dummyz_loc[(K) * (NGLL2) + (J) * (NGLLX) + l]) * (fac1);
      fac2 = sh_hprime_xx[(l) * (NGLLX) + J];
      tempx2l = tempx2l + (s_dummyx_loc[(K) * (NGLL2) + (l) * (NGLLX) + I]) * (fac2);
      tempy2l = tempy2l + (s_dummyy_loc[(K) * (NGLL2) + (l) * (NGLLX) + I]) * (fac2);
      tempz2l = tempz2l + (s_dummyz_loc[(K) * (NGLL2) + (l) * (NGLLX) + I]) * (fac2);
      fac3 = sh_hprime_xx[(l) * (NGLLX) + K];
      tempx3l = tempx3l + (s_dummyx_loc[(l) * (NGLL2) + (J) * (NGLLX) + I]) * (fac3);
      tempy3l = tempy3l + (s_dummyy_loc[(l) * (NGLL2) + (J) * (NGLLX) + I]) * (fac3);
      tempz3l = tempz3l + (s_dummyz_loc[(l) * (NGLL2) + (J) * (NGLLX) + I]) * (fac3);
    }
#endif
    offset = (working_element) * (NGLL3_PADDED) + tx;
    xixl = d_xix[offset];
    etaxl = d_etax[offset];
    gammaxl = d_gammax[offset];
    xiyl = d_xiy[offset];
    etayl = d_etay[offset];
    gammayl = d_gammay[offset];
    xizl = d_xiz[offset];
    etazl = d_etaz[offset];
    gammazl = d_gammaz[offset];
    duxdxl = (xixl) * (tempx1l) + (etaxl) * (tempx2l) + (gammaxl) * (tempx3l);
    duxdyl = (xiyl) * (tempx1l) + (etayl) * (tempx2l) + (gammayl) * (tempx3l);
    duxdzl = (xizl) * (tempx1l) + (etazl) * (tempx2l) + (gammazl) * (tempx3l);
    duydxl = (xixl) * (tempy1l) + (etaxl) * (tempy2l) + (gammaxl) * (tempy3l);
    duydyl = (xiyl) * (tempy1l) + (etayl) * (tempy2l) + (gammayl) * (tempy3l);
    duydzl = (xizl) * (tempy1l) + (etazl) * (tempy2l) + (gammazl) * (tempy3l);
    duzdxl = (xixl) * (tempz1l) + (etaxl) * (tempz2l) + (gammaxl) * (tempz3l);
    duzdyl = (xiyl) * (tempz1l) + (etayl) * (tempz2l) + (gammayl) * (tempz3l);
    duzdzl = (xizl) * (tempz1l) + (etazl) * (tempz2l) + (gammazl) * (tempz3l);
    duxdxl_plus_duydyl = duxdxl + duydyl;
    duxdxl_plus_duzdzl = duxdxl + duzdzl;
    duydyl_plus_duzdzl = duydyl + duzdzl;
    duxdyl_plus_duydxl = duxdyl + duydxl;
    duzdxl_plus_duxdzl = duzdxl + duxdzl;
    duzdyl_plus_duydzl = duzdyl + duydzl;

    if (COMPUTE_AND_STORE_STRAIN) {
      templ = (duxdxl + duydyl + duzdzl) * (0.3333333333333333f);
      epsilondev_xx_loc_1 = duxdxl - (templ);
      epsilondev_yy_loc_1 = duydyl - (templ);
      epsilondev_xy_loc_1 = (duxdyl_plus_duydxl) * (0.5f);
      epsilondev_xz_loc_1 = (duzdxl_plus_duxdzl) * (0.5f);
      epsilondev_yz_loc_1 = (duzdyl_plus_duydzl) * (0.5f);
      if (NSPEC_INNER_CORE_STRAIN_ONLY == 1) {
        epsilon_trace_over_3[tx] = templ;
      } else {
        epsilon_trace_over_3[tx + (working_element) * (NGLL3)] = templ;
      }
    }

    kappal = d_kappavstore[offset];
    mul = d_muvstore[offset];

    if (ATTENUATION) {
      if (USE_3D_ATTENUATION_ARRAYS) {
        mul_iso = (mul) * (one_minus_sum_beta[tx + (working_element) * (NGLL3)]);
        mul_aniso = (mul) * (one_minus_sum_beta[tx + (working_element) * (NGLL3)] - (1.0f));
      } else {
        mul_iso = (mul) * (one_minus_sum_beta[working_element]);
        mul_aniso = (mul) * (one_minus_sum_beta[working_element] - (1.0f));
      }
    } else {
      mul_iso = mul;
    }

    if (ANISOTROPY) {
      c11 = d_c11store[offset];
      c12 = d_c12store[offset];
      c13 = d_c13store[offset];
      c33 = d_c33store[offset];
      c44 = d_c44store[offset];
      if (ATTENUATION) {
        c11 = c11 + (mul_aniso) * (1.3333333333333333f);
        c12 = c12 - ((mul_aniso) * (0.6666666666666666f));
        c13 = c13 - ((mul_aniso) * (0.6666666666666666f));
        c33 = c33 + (mul_aniso) * (1.3333333333333333f);
        c44 = c44 + mul_aniso;
      }
      sigma_xx = (c11) * (duxdxl) + (c12) * (duydyl) + (c13) * (duzdzl);
      sigma_yy = (c12) * (duxdxl) + (c11) * (duydyl) + (c13) * (duzdzl);
      sigma_zz = (c13) * (duxdxl) + (c13) * (duydyl) + (c33) * (duzdzl);
      sigma_xy = ((c11 - (c12)) * (duxdyl_plus_duydxl)) * (0.5f);
      sigma_xz = (c44) * (duzdxl_plus_duxdzl);
      sigma_yz = (c44) * (duzdyl_plus_duydzl);
    } else {
      lambdalplus2mul = kappal + (mul_iso) * (1.3333333333333333f);
      lambdal = lambdalplus2mul - ((mul_iso) * (2.0f));
      sigma_xx = (lambdalplus2mul) * (duxdxl) + (lambdal) * (duydyl_plus_duzdzl);
      sigma_yy = (lambdalplus2mul) * (duydyl) + (lambdal) * (duxdxl_plus_duzdzl);
      sigma_zz = (lambdalplus2mul) * (duzdzl) + (lambdal) * (duxdxl_plus_duydyl);
      sigma_xy = (mul) * (duxdyl_plus_duydxl);
      sigma_xz = (mul) * (duzdxl_plus_duxdzl);
      sigma_yz = (mul) * (duzdyl_plus_duydzl);
    }

    if (ATTENUATION &&  !(PARTIAL_PHYS_DISPERSION_ONLY)) {
      compute_element_ic_att_stress(tx, working_element, R_xx, R_yy, R_xy, R_xz, R_yz,  &sigma_xx,  &sigma_yy,  &sigma_zz,  &sigma_xy,  &sigma_xz,  &sigma_yz);
    }

    sigma_yx = sigma_xy;
    sigma_zx = sigma_xz;
    sigma_zy = sigma_yz;
    jacobianl = (1.0f) / ((xixl) * ((etayl) * (gammazl) - ((etazl) * (gammayl))) - ((xiyl) * ((etaxl) * (gammazl) - ((etazl) * (gammaxl)))) + (xizl) * ((etaxl) * (gammayl) - ((etayl) * (gammaxl))));

    if (GRAVITY) {
      compute_element_ic_gravity(tx, iglob_1, d_rstore, d_minus_gravity_table, d_minus_deriv_gravity_table, d_density_table, wgll_cube, jacobianl, s_dummyx_loc, s_dummyy_loc, s_dummyz_loc,  &sigma_xx,  &sigma_yy,  &sigma_zz,  &sigma_xy,  &sigma_yx,  &sigma_xz,  &sigma_zx,  &sigma_yz,  &sigma_zy,  &rho_s_H_1_1,  &rho_s_H_1_2,  &rho_s_H_1_3);
    }

    s_tempx1[tx] = (jacobianl) * ((sigma_xx) * (xixl) + (sigma_yx) * (xiyl) + (sigma_zx) * (xizl));
    s_tempy1[tx] = (jacobianl) * ((sigma_xy) * (xixl) + (sigma_yy) * (xiyl) + (sigma_zy) * (xizl));
    s_tempz1[tx] = (jacobianl) * ((sigma_xz) * (xixl) + (sigma_yz) * (xiyl) + (sigma_zz) * (xizl));
    s_tempx2[tx] = (jacobianl) * ((sigma_xx) * (etaxl) + (sigma_yx) * (etayl) + (sigma_zx) * (etazl));
    s_tempy2[tx] = (jacobianl) * ((sigma_xy) * (etaxl) + (sigma_yy) * (etayl) + (sigma_zy) * (etazl));
    s_tempz2[tx] = (jacobianl) * ((sigma_xz) * (etaxl) + (sigma_yz) * (etayl) + (sigma_zz) * (etazl));
    s_tempx3[tx] = (jacobianl) * ((sigma_xx) * (gammaxl) + (sigma_yx) * (gammayl) + (sigma_zx) * (gammazl));
    s_tempy3[tx] = (jacobianl) * ((sigma_xy) * (gammaxl) + (sigma_yy) * (gammayl) + (sigma_zy) * (gammazl));
    s_tempz3[tx] = (jacobianl) * ((sigma_xz) * (gammaxl) + (sigma_yz) * (gammayl) + (sigma_zz) * (gammazl));
  }
  __syncthreads();

  if (active_1) {
    tempx1l = 0.0f;
    tempx2l = 0.0f;
    tempx3l = 0.0f;
    tempy1l = 0.0f;
    tempy2l = 0.0f;
    tempy3l = 0.0f;
    tempz1l = 0.0f;
    tempz2l = 0.0f;
    tempz3l = 0.0f;
#ifdef MANUALLY_UNROLLED_LOOPS
    fac1 = sh_hprimewgll_xx[(I) * (NGLLX) + 0];
    offset = (K) * (NGLL2) + (J) * (NGLLX) + 0;
    tempx1l = tempx1l + (s_tempx1[offset]) * (fac1);
    tempy1l = tempy1l + (s_tempy1[offset]) * (fac1);
    tempz1l = tempz1l + (s_tempz1[offset]) * (fac1);
    fac2 = sh_hprimewgll_xx[(J) * (NGLLX) + 0];
    offset = (K) * (NGLL2) + (0) * (NGLLX) + I;
    tempx2l = tempx2l + (s_tempx2[offset]) * (fac2);
    tempy2l = tempy2l + (s_tempy2[offset]) * (fac2);
    tempz2l = tempz2l + (s_tempz2[offset]) * (fac2);
    fac3 = sh_hprimewgll_xx[(K) * (NGLLX) + 0];
    offset = (0) * (NGLL2) + (J) * (NGLLX) + I;
    tempx3l = tempx3l + (s_tempx3[offset]) * (fac3);
    tempy3l = tempy3l + (s_tempy3[offset]) * (fac3);
    tempz3l = tempz3l + (s_tempz3[offset]) * (fac3);
    fac1 = sh_hprimewgll_xx[(I) * (NGLLX) + 1];
    offset = (K) * (NGLL2) + (J) * (NGLLX) + 1;
    tempx1l = tempx1l + (s_tempx1[offset]) * (fac1);
    tempy1l = tempy1l + (s_tempy1[offset]) * (fac1);
    tempz1l = tempz1l + (s_tempz1[offset]) * (fac1);
    fac2 = sh_hprimewgll_xx[(J) * (NGLLX) + 1];
    offset = (K) * (NGLL2) + (1) * (NGLLX) + I;
    tempx2l = tempx2l + (s_tempx2[offset]) * (fac2);
    tempy2l = tempy2l + (s_tempy2[offset]) * (fac2);
    tempz2l = tempz2l + (s_tempz2[offset]) * (fac2);
    fac3 = sh_hprimewgll_xx[(K) * (NGLLX) + 1];
    offset = (1) * (NGLL2) + (J) * (NGLLX) + I;
    tempx3l = tempx3l + (s_tempx3[offset]) * (fac3);
    tempy3l = tempy3l + (s_tempy3[offset]) * (fac3);
    tempz3l = tempz3l + (s_tempz3[offset]) * (fac3);
    fac1 = sh_hprimewgll_xx[(I) * (NGLLX) + 2];
    offset = (K) * (NGLL2) + (J) * (NGLLX) + 2;
    tempx1l = tempx1l + (s_tempx1[offset]) * (fac1);
    tempy1l = tempy1l + (s_tempy1[offset]) * (fac1);
    tempz1l = tempz1l + (s_tempz1[offset]) * (fac1);
    fac2 = sh_hprimewgll_xx[(J) * (NGLLX) + 2];
    offset = (K) * (NGLL2) + (2) * (NGLLX) + I;
    tempx2l = tempx2l + (s_tempx2[offset]) * (fac2);
    tempy2l = tempy2l + (s_tempy2[offset]) * (fac2);
    tempz2l = tempz2l + (s_tempz2[offset]) * (fac2);
    fac3 = sh_hprimewgll_xx[(K) * (NGLLX) + 2];
    offset = (2) * (NGLL2) + (J) * (NGLLX) + I;
    tempx3l = tempx3l + (s_tempx3[offset]) * (fac3);
    tempy3l = tempy3l + (s_tempy3[offset]) * (fac3);
    tempz3l = tempz3l + (s_tempz3[offset]) * (fac3);
    fac1 = sh_hprimewgll_xx[(I) * (NGLLX) + 3];
    offset = (K) * (NGLL2) + (J) * (NGLLX) + 3;
    tempx1l = tempx1l + (s_tempx1[offset]) * (fac1);
    tempy1l = tempy1l + (s_tempy1[offset]) * (fac1);
    tempz1l = tempz1l + (s_tempz1[offset]) * (fac1);
    fac2 = sh_hprimewgll_xx[(J) * (NGLLX) + 3];
    offset = (K) * (NGLL2) + (3) * (NGLLX) + I;
    tempx2l = tempx2l + (s_tempx2[offset]) * (fac2);
    tempy2l = tempy2l + (s_tempy2[offset]) * (fac2);
    tempz2l = tempz2l + (s_tempz2[offset]) * (fac2);
    fac3 = sh_hprimewgll_xx[(K) * (NGLLX) + 3];
    offset = (3) * (NGLL2) + (J) * (NGLLX) + I;
    tempx3l = tempx3l + (s_tempx3[offset]) * (fac3);
    tempy3l = tempy3l + (s_tempy3[offset]) * (fac3);
    tempz3l = tempz3l + (s_tempz3[offset]) * (fac3);
    fac1 = sh_hprimewgll_xx[(I) * (NGLLX) + 4];
    offset = (K) * (NGLL2) + (J) * (NGLLX) + 4;
    tempx1l = tempx1l + (s_tempx1[offset]) * (fac1);
    tempy1l = tempy1l + (s_tempy1[offset]) * (fac1);
    tempz1l = tempz1l + (s_tempz1[offset]) * (fac1);
    fac2 = sh_hprimewgll_xx[(J) * (NGLLX) + 4];
    offset = (K) * (NGLL2) + (4) * (NGLLX) + I;
    tempx2l = tempx2l + (s_tempx2[offset]) * (fac2);
    tempy2l = tempy2l + (s_tempy2[offset]) * (fac2);
    tempz2l = tempz2l + (s_tempz2[offset]) * (fac2);
    fac3 = sh_hprimewgll_xx[(K) * (NGLLX) + 4];
    offset = (4) * (NGLL2) + (J) * (NGLLX) + I;
    tempx3l = tempx3l + (s_tempx3[offset]) * (fac3);
    tempy3l = tempy3l + (s_tempy3[offset]) * (fac3);
    tempz3l = tempz3l + (s_tempz3[offset]) * (fac3);
#else
    for (l = 0; l <= NGLLX - (1); l += 1) {
      fac1 = sh_hprimewgll_xx[(I) * (NGLLX) + l];
      offset = (K) * (NGLL2) + (J) * (NGLLX) + l;
      tempx1l = tempx1l + (s_tempx1[offset]) * (fac1);
      tempy1l = tempy1l + (s_tempy1[offset]) * (fac1);
      tempz1l = tempz1l + (s_tempz1[offset]) * (fac1);
      fac2 = sh_hprimewgll_xx[(J) * (NGLLX) + l];
      offset = (K) * (NGLL2) + (l) * (NGLLX) + I;
      tempx2l = tempx2l + (s_tempx2[offset]) * (fac2);
      tempy2l = tempy2l + (s_tempy2[offset]) * (fac2);
      tempz2l = tempz2l + (s_tempz2[offset]) * (fac2);
      fac3 = sh_hprimewgll_xx[(K) * (NGLLX) + l];
      offset = (l) * (NGLL2) + (J) * (NGLLX) + I;
      tempx3l = tempx3l + (s_tempx3[offset]) * (fac3);
      tempy3l = tempy3l + (s_tempy3[offset]) * (fac3);
      tempz3l = tempz3l + (s_tempz3[offset]) * (fac3);
    }
#endif
    fac1 = d_wgllwgll_yz[(K) * (NGLLX) + J];
    fac2 = d_wgllwgll_xz[(K) * (NGLLX) + I];
    fac3 = d_wgllwgll_xy[(J) * (NGLLX) + I];
    sum_terms1 =  -((fac1) * (tempx1l) + (fac2) * (tempx2l) + (fac3) * (tempx3l));
    sum_terms2 =  -((fac1) * (tempy1l) + (fac2) * (tempy2l) + (fac3) * (tempy3l));
    sum_terms3 =  -((fac1) * (tempz1l) + (fac2) * (tempz2l) + (fac3) * (tempz3l));

    if (GRAVITY) {
      sum_terms1 = sum_terms1 + rho_s_H_1_1;
      sum_terms2 = sum_terms2 + rho_s_H_1_2;
      sum_terms3 = sum_terms3 + rho_s_H_1_3;
    }

#ifdef USE_MESH_COLORING_GPU
#ifdef USE_TEXTURES_FIELDS
    d_accel[0 + (3) * (iglob_1)] = tex1Dfetch(d_b_accel_ic_tex,(iglob_1) * (3) + 0) + sum_terms1;
    d_accel[1 + (3) * (iglob_1)] = tex1Dfetch(d_b_accel_ic_tex,(iglob_1) * (3) + 1) + sum_terms2;
    d_accel[2 + (3) * (iglob_1)] = tex1Dfetch(d_b_accel_ic_tex,(iglob_1) * (3) + 2) + sum_terms3;
#else
    d_accel[0 + (3) * (iglob_1)] = d_accel[0 + (3) * (iglob_1)] + sum_terms1;
    d_accel[1 + (3) * (iglob_1)] = d_accel[1 + (3) * (iglob_1)] + sum_terms2;
    d_accel[2 + (3) * (iglob_1)] = d_accel[2 + (3) * (iglob_1)] + sum_terms3;
#endif
#else
    if (use_mesh_coloring_gpu) {
      if (NSPEC_INNER_CORE > 1000) {
#ifdef USE_TEXTURES_FIELDS
        d_accel[0 + (3) * (iglob_1)] = tex1Dfetch(d_b_accel_ic_tex,(iglob_1) * (3) + 0) + sum_terms1;
        d_accel[1 + (3) * (iglob_1)] = tex1Dfetch(d_b_accel_ic_tex,(iglob_1) * (3) + 1) + sum_terms2;
        d_accel[2 + (3) * (iglob_1)] = tex1Dfetch(d_b_accel_ic_tex,(iglob_1) * (3) + 2) + sum_terms3;
#else
        d_accel[0 + (3) * (iglob_1)] = d_accel[0 + (3) * (iglob_1)] + sum_terms1;
        d_accel[1 + (3) * (iglob_1)] = d_accel[1 + (3) * (iglob_1)] + sum_terms2;
        d_accel[2 + (3) * (iglob_1)] = d_accel[2 + (3) * (iglob_1)] + sum_terms3;
#endif
      } else {
        atomicAdd(d_accel + (iglob_1) * (3) + 0, sum_terms1);
        atomicAdd(d_accel + (iglob_1) * (3) + 1, sum_terms2);
        atomicAdd(d_accel + (iglob_1) * (3) + 2, sum_terms3);
      }
    } else {
      atomicAdd(d_accel + (iglob_1) * (3) + 0, sum_terms1);
      atomicAdd(d_accel + (iglob_1) * (3) + 1, sum_terms2);
      atomicAdd(d_accel + (iglob_1) * (3) + 2, sum_terms3);
    }
#endif

    if (ATTENUATION &&  !(PARTIAL_PHYS_DISPERSION_ONLY)) {
      compute_element_ic_att_memory(tx, working_element, d_muvstore, factor_common, alphaval, betaval, gammaval, R_xx, R_yy, R_xy, R_xz, R_yz, epsilondev_xx, epsilondev_yy, epsilondev_xy, epsilondev_xz, epsilondev_yz, epsilondev_xx_loc_1, epsilondev_yy_loc_1, epsilondev_xy_loc_1, epsilondev_xz_loc_1, epsilondev_yz_loc_1, USE_3D_ATTENUATION_ARRAYS);
    }

    if (COMPUTE_AND_STORE_STRAIN) {
      epsilondev_xx[tx + (working_element) * (NGLL3)] = epsilondev_xx_loc_1;
      epsilondev_yy[tx + (working_element) * (NGLL3)] = epsilondev_yy_loc_1;
      epsilondev_xy[tx + (working_element) * (NGLL3)] = epsilondev_xy_loc_1;
      epsilondev_xz[tx + (working_element) * (NGLL3)] = epsilondev_xz_loc_1;
      epsilondev_yz[tx + (working_element) * (NGLL3)] = epsilondev_yz_loc_1;
    }
  }
}
