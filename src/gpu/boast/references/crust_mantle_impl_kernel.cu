
#include <hip/hip_runtime.h>
// from compute_forces_crust_mantle_cuda.cu

#define NDIM 3
#define NGLLX 5
#define NGLL2 25
#define NGLL3 125
#define NGLL3_PADDED 128
#define N_SLS 3
#define R_EARTH_KM 6371.0f

typedef float realw;
typedef float * realw_p;
typedef const float* __restrict__ realw_const_p;

#ifdef USE_TEXTURES_FIELDS
//forward
realw_texture d_displ_cm_tex;
realw_texture d_accel_cm_tex;
//backward/reconstructed
realw_texture d_b_displ_cm_tex;
realw_texture d_b_accel_cm_tex;

//note: texture variables are implicitly static, and cannot be passed as arguments to cuda kernels;
//      thus, 1) we thus use if-statements (FORWARD_OR_ADJOINT) to determine from which texture to fetch from
//            2) we use templates
//      since if-statements are a bit slower as the variable is only known at runtime, we use option 2)

// templates definitions
template<int FORWARD_OR_ADJOINT> __device__ float texfetch_displ_cm(int x);
template<int FORWARD_OR_ADJOINT> __device__ float texfetch_accel_cm(int x);

// templates for texture fetching
// FORWARD_OR_ADJOINT == 1 <- forward arrays
template<> __device__ float texfetch_displ_cm<1>(int x) { return tex1Dfetch(d_displ_cm_tex, x); }
template<> __device__ float texfetch_accel_cm<1>(int x) { return tex1Dfetch(d_accel_cm_tex, x); }
// FORWARD_OR_ADJOINT == 3 <- backward/reconstructed arrays
template<> __device__ float texfetch_displ_cm<3>(int x) { return tex1Dfetch(d_b_displ_cm_tex, x); }
template<> __device__ float texfetch_accel_cm<3>(int x) { return tex1Dfetch(d_b_accel_cm_tex, x); }
#endif

#ifdef USE_TEXTURES_CONSTANTS
realw_texture d_hprime_xx_tex;
__constant__ size_t d_hprime_xx_tex_offset;
// weighted
realw_texture d_hprimewgll_xx_tex;
__constant__ size_t d_hprimewgll_xx_tex_offset;
#endif


/* ----------------------------------------------------------------------------------------------- */

// elemental routines

/* ----------------------------------------------------------------------------------------------- */

// updates stress

__device__ void compute_element_cm_att_stress(int tx,int working_element,
                                              realw_p R_xx,
                                              realw_p R_yy,
                                              realw_p R_xy,
                                              realw_p R_xz,
                                              realw_p R_yz,
                                              realw* sigma_xx,
                                              realw* sigma_yy,
                                              realw* sigma_zz,
                                              realw* sigma_xy,
                                              realw* sigma_xz,
                                              realw* sigma_yz) {

  realw R_xx_val,R_yy_val;
  int offset_sls;

  for(int i_sls = 0; i_sls < N_SLS; i_sls++){
    // index
    // note: index for R_xx,.. here is (i,j,k,i_sls,ispec) and not (i,j,k,ispec,i_sls) as in local version
    //       see local version: offset_sls = tx + NGLL3*(working_element + NSPEC*i_sls);
    // indexing examples:
    //   (i,j,k,ispec,i_sls) -> offset_sls = tx + NGLL3*(working_element + NSPEC*i_sls)
    //   (i_sls,i,j,k,ispec) -> offset_sls = i_sls + N_SLS*(tx + NGLL3*working_element)
    //   (i,j,k,i_sls,ispec) -> offset_sls = tx + NGLL3*(i_sls + N_SLS*working_element)
    offset_sls = tx + NGLL3*(i_sls + N_SLS*working_element);

    R_xx_val = R_xx[offset_sls];
    R_yy_val = R_yy[offset_sls];

    *sigma_xx = *sigma_xx - R_xx_val;
    *sigma_yy = *sigma_yy - R_yy_val;
    *sigma_zz = *sigma_zz + R_xx_val + R_yy_val;
    *sigma_xy = *sigma_xy - R_xy[offset_sls];
    *sigma_xz = *sigma_xz - R_xz[offset_sls];
    *sigma_yz = *sigma_yz - R_yz[offset_sls];
  }
}


/* ----------------------------------------------------------------------------------------------- */

// updates R_memory

__device__ void compute_element_cm_att_memory(int tx,int working_element,
                                              realw_const_p d_muvstore,
                                              realw_const_p factor_common,
                                              realw_const_p alphaval,realw_const_p betaval,realw_const_p gammaval,
                                              realw_p R_xx,realw_p R_yy,realw_p R_xy,realw_p R_xz,realw_p R_yz,
                                              realw_p epsilondev_xx,realw_p epsilondev_yy,realw_p epsilondev_xy,
                                              realw_p epsilondev_xz,realw_p epsilondev_yz,
                                              realw epsilondev_xx_loc,realw epsilondev_yy_loc,realw epsilondev_xy_loc,
                                              realw epsilondev_xz_loc,realw epsilondev_yz_loc,
                                              realw_const_p d_c44store,
                                              const int ANISOTROPY,
                                              const int USE_3D_ATTENUATION_ARRAYS) {

  realw fac;
  realw factor_loc;
  realw alphaval_loc,betaval_loc,gammaval_loc;
  realw Sn,Snp1;
  int offset_sls;

  // shear moduli for common factor (only Q_mu attenuation)
  if (ANISOTROPY){
    fac = d_c44store[tx + NGLL3_PADDED * working_element];
  }else{
    fac = d_muvstore[tx + NGLL3_PADDED * working_element];
  }

  // use Runge-Kutta scheme to march in time
  for(int i_sls = 0; i_sls < N_SLS; i_sls++){
    // indices
    // note: index for R_xx,... here is (i,j,k,i_sls,ispec) and not (i,j,k,ispec,i_sls) as in local version
    //
    // index:
    // (i,j,k,i_sls,ispec) -> offset_sls = tx + NGLL3*(i_sls + N_SLS*working_element)
    offset_sls = tx + NGLL3*(i_sls + N_SLS*working_element);

    // either mustore(i,j,k,ispec) * factor_common(i,j,k,i_sls,ispec)
    // or       factor_common(i_sls,:,:,:,ispec) * c44store(:,:,:,ispec)
    if (USE_3D_ATTENUATION_ARRAYS){
      // array dimension: factor_common(N_SLS,NGLLX,NGLLY,NGLLZ,NSPEC)
      factor_loc = fac * factor_common[offset_sls];
    }else{
      // array dimension: factor_common(1,1,1,N_SLS,NSPEC)
      factor_loc = fac * factor_common[i_sls + N_SLS*working_element];
    }

    alphaval_loc = alphaval[i_sls]; // (i_sls)
    betaval_loc = betaval[i_sls];
    gammaval_loc = gammaval[i_sls];


    // term in xx
    Sn   = factor_loc * epsilondev_xx[tx + NGLL3 * working_element]; //(i,j,k,ispec)
    Snp1   = factor_loc * epsilondev_xx_loc; //(i,j,k)
    R_xx[offset_sls] = alphaval_loc * R_xx[offset_sls] + betaval_loc * Sn + gammaval_loc * Snp1;

    // term in yy
    Sn   = factor_loc * epsilondev_yy[tx + NGLL3 * working_element];
    Snp1   = factor_loc * epsilondev_yy_loc;
    R_yy[offset_sls] = alphaval_loc * R_yy[offset_sls] + betaval_loc * Sn + gammaval_loc * Snp1;

    // term in zz not computed since zero trace

    // term in xy
    Sn   = factor_loc * epsilondev_xy[tx + NGLL3 * working_element];
    Snp1   = factor_loc * epsilondev_xy_loc;
    R_xy[offset_sls] = alphaval_loc * R_xy[offset_sls] + betaval_loc * Sn + gammaval_loc * Snp1;

    // term in xz
    Sn   = factor_loc * epsilondev_xz[tx + NGLL3 * working_element];
    Snp1   = factor_loc * epsilondev_xz_loc;
    R_xz[offset_sls] = alphaval_loc * R_xz[offset_sls] + betaval_loc * Sn + gammaval_loc * Snp1;

    // term in yz
    Sn   = factor_loc * epsilondev_yz[tx + NGLL3 * working_element];
    Snp1   = factor_loc * epsilondev_yz_loc;
    R_yz[offset_sls] = alphaval_loc * R_yz[offset_sls] + betaval_loc * Sn + gammaval_loc * Snp1;
  }
}

/* ----------------------------------------------------------------------------------------------- */

// pre-computes gravity term

__device__ void compute_element_cm_gravity(int tx,
                                          const int iglob,
                                          realw_const_p d_xstore,realw_const_p d_ystore,realw_const_p d_zstore,
                                          realw_const_p d_minus_gravity_table,
                                          realw_const_p d_minus_deriv_gravity_table,
                                          realw_const_p d_density_table,
                                          realw_const_p wgll_cube,
                                          realw jacobianl,
                                          realw* s_dummyx_loc,
                                          realw* s_dummyy_loc,
                                          realw* s_dummyz_loc,
                                          realw* sigma_xx,
                                          realw* sigma_yy,
                                          realw* sigma_zz,
                                          realw* sigma_xy,
                                          realw* sigma_yx,
                                          realw* sigma_xz,
                                          realw* sigma_zx,
                                          realw* sigma_yz,
                                          realw* sigma_zy,
                                          realw* rho_s_H1,
                                          realw* rho_s_H2,
                                          realw* rho_s_H3){

  realw radius,theta,phi;
  realw cos_theta,sin_theta,cos_phi,sin_phi;
  realw minus_g,minus_dg;
  realw rho;
  realw gxl,gyl,gzl;
  realw minus_g_over_radius,minus_dg_plus_g_over_radius;
  realw cos_theta_sq,sin_theta_sq,cos_phi_sq,sin_phi_sq;
  realw Hxxl,Hyyl,Hzzl,Hxyl,Hxzl,Hyzl;
  realw sx_l,sy_l,sz_l;
  realw factor;
  int int_radius;

  // R_EARTH_KM is the radius of the bottom of the oceans (radius of Earth in km)
  //const realw R_EARTH_KM = 6371.0f;
  // uncomment line below for PREM with oceans
  //const realw R_EARTH_KM = 6368.0f;

  // compute non-symmetric terms for gravity

  // use mesh coordinates to get theta and phi
  // x y z contain r theta phi
  radius = d_xstore[iglob];
  theta = d_ystore[iglob];
  phi = d_zstore[iglob];

  if (sizeof( realw ) == sizeof( float )){
    // float operations
    // sincos function return sinus and cosine for given value
    sincosf(theta, &sin_theta, &cos_theta);
    sincosf(phi, &sin_phi, &cos_phi);
  }else{
    cos_theta = cos(theta);
    sin_theta = sin(theta);
    cos_phi = cos(phi);
    sin_phi = sin(phi);
  }

  // for efficiency replace with lookup table every 100 m in radial direction
  // note: radius in crust mantle should never be zero,
  //          and arrays in C start from 0, thus we need to subtract -1
  int_radius = rint(radius * R_EARTH_KM * 10.0f ) - 1;

  // get g, rho and dg/dr=dg
  // spherical components of the gravitational acceleration
  // for efficiency replace with lookup table every 100 m in radial direction
  minus_g = d_minus_gravity_table[int_radius];
  minus_dg = d_minus_deriv_gravity_table[int_radius];
  rho = d_density_table[int_radius];

  // Cartesian components of the gravitational acceleration
  gxl = minus_g*sin_theta*cos_phi;
  gyl = minus_g*sin_theta*sin_phi;
  gzl = minus_g*cos_theta;

  // Cartesian components of gradient of gravitational acceleration
  // obtained from spherical components

  minus_g_over_radius = minus_g / radius;
  minus_dg_plus_g_over_radius = minus_dg - minus_g_over_radius;

  cos_theta_sq = cos_theta*cos_theta;
  sin_theta_sq = sin_theta*sin_theta;
  cos_phi_sq = cos_phi*cos_phi;
  sin_phi_sq = sin_phi*sin_phi;

  Hxxl = minus_g_over_radius*(cos_phi_sq*cos_theta_sq + sin_phi_sq) + cos_phi_sq*minus_dg*sin_theta_sq;
  Hyyl = minus_g_over_radius*(cos_phi_sq + cos_theta_sq*sin_phi_sq) + minus_dg*sin_phi_sq*sin_theta_sq;
  Hzzl = cos_theta_sq*minus_dg + minus_g_over_radius*sin_theta_sq;
  Hxyl = cos_phi*minus_dg_plus_g_over_radius*sin_phi*sin_theta_sq;
  Hxzl = cos_phi*cos_theta*minus_dg_plus_g_over_radius*sin_theta;
  Hyzl = cos_theta*minus_dg_plus_g_over_radius*sin_phi*sin_theta;

  // get displacement and multiply by density to compute G tensor
  sx_l = rho * s_dummyx_loc[tx];
  sy_l = rho * s_dummyy_loc[tx];
  sz_l = rho * s_dummyz_loc[tx];

  // compute G tensor from s . g and add to sigma (not symmetric)
  *sigma_xx = *sigma_xx + sy_l*gyl + sz_l*gzl;
  *sigma_yy = *sigma_yy + sx_l*gxl + sz_l*gzl;
  *sigma_zz = *sigma_zz + sx_l*gxl + sy_l*gyl;

  *sigma_xy = *sigma_xy - sx_l * gyl;
  *sigma_yx = *sigma_yx - sy_l * gxl;

  *sigma_xz = *sigma_xz - sx_l * gzl;
  *sigma_zx = *sigma_zx - sz_l * gxl;

  *sigma_yz = *sigma_yz - sy_l * gzl;
  *sigma_zy = *sigma_zy - sz_l * gyl;

  // precompute vector
  factor = jacobianl * wgll_cube[tx];
  *rho_s_H1 = factor * (sx_l * Hxxl + sy_l * Hxyl + sz_l * Hxzl);
  *rho_s_H2 = factor * (sx_l * Hxyl + sy_l * Hyyl + sz_l * Hyzl);
  *rho_s_H3 = factor * (sx_l * Hxzl + sy_l * Hyzl + sz_l * Hzzl);
}

/* ----------------------------------------------------------------------------------------------- */

// computes stresses for anisotropic element

__device__ void compute_element_cm_aniso(int offset,
                                         realw_const_p d_c11store,realw_const_p d_c12store,realw_const_p d_c13store,
                                         realw_const_p d_c14store,realw_const_p d_c15store,realw_const_p d_c16store,
                                         realw_const_p d_c22store,realw_const_p d_c23store,realw_const_p d_c24store,
                                         realw_const_p d_c25store,realw_const_p d_c26store,realw_const_p d_c33store,
                                         realw_const_p d_c34store,realw_const_p d_c35store,realw_const_p d_c36store,
                                         realw_const_p d_c44store,realw_const_p d_c45store,realw_const_p d_c46store,
                                         realw_const_p d_c55store,realw_const_p d_c56store,realw_const_p d_c66store,
                                         const int ATTENUATION,
                                         realw one_minus_sum_beta_use,
                                         realw duxdxl,realw duxdyl,realw duxdzl,
                                         realw duydxl,realw duydyl,realw duydzl,
                                         realw duzdxl,realw duzdyl,realw duzdzl,
                                         realw duxdyl_plus_duydxl,realw duzdxl_plus_duxdzl,realw duzdyl_plus_duydzl,
                                         realw* sigma_xx,realw* sigma_yy,realw* sigma_zz,
                                         realw* sigma_xy,realw* sigma_xz,realw* sigma_yz
                                         ){

  realw c11,c12,c13,c14,c15,c16,c22,c23,c24,c25,c26,c33,c34,c35,c36,c44,c45,c46,c55,c56,c66;
  realw mul,minus_sum_beta;

  c11 = d_c11store[offset];
  c12 = d_c12store[offset];
  c13 = d_c13store[offset];
  c14 = d_c14store[offset];
  c15 = d_c15store[offset];
  c16 = d_c16store[offset];
  c22 = d_c22store[offset];
  c23 = d_c23store[offset];
  c24 = d_c24store[offset];
  c25 = d_c25store[offset];
  c26 = d_c26store[offset];
  c33 = d_c33store[offset];
  c34 = d_c34store[offset];
  c35 = d_c35store[offset];
  c36 = d_c36store[offset];
  c44 = d_c44store[offset];
  c45 = d_c45store[offset];
  c46 = d_c46store[offset];
  c55 = d_c55store[offset];
  c56 = d_c56store[offset];
  c66 = d_c66store[offset];

  // use unrelaxed parameters if attenuation
  if (ATTENUATION){
    minus_sum_beta = one_minus_sum_beta_use - 1.0f;
    mul = c44;

    c11 = c11 + 1.33333333333333333333f * minus_sum_beta * mul;
    c12 = c12 - 0.66666666666666666666f * minus_sum_beta * mul;
    c13 = c13 - 0.66666666666666666666f * minus_sum_beta * mul;
    c22 = c22 + 1.33333333333333333333f * minus_sum_beta * mul;
    c23 = c23 - 0.66666666666666666666f * minus_sum_beta * mul;
    c33 = c33 + 1.33333333333333333333f * minus_sum_beta * mul;
    c44 = c44 + minus_sum_beta * mul;
    c55 = c55 + minus_sum_beta * mul;
    c66 = c66 + minus_sum_beta * mul;
  }

  *sigma_xx = c11*duxdxl + c16*duxdyl_plus_duydxl + c12*duydyl +
             c15*duzdxl_plus_duxdzl + c14*duzdyl_plus_duydzl + c13*duzdzl;
  *sigma_yy = c12*duxdxl + c26*duxdyl_plus_duydxl + c22*duydyl +
             c25*duzdxl_plus_duxdzl + c24*duzdyl_plus_duydzl + c23*duzdzl;
  *sigma_zz = c13*duxdxl + c36*duxdyl_plus_duydxl + c23*duydyl +
             c35*duzdxl_plus_duxdzl + c34*duzdyl_plus_duydzl + c33*duzdzl;
  *sigma_xy = c16*duxdxl + c66*duxdyl_plus_duydxl + c26*duydyl +
             c56*duzdxl_plus_duxdzl + c46*duzdyl_plus_duydzl + c36*duzdzl;
  *sigma_xz = c15*duxdxl + c56*duxdyl_plus_duydxl + c25*duydyl +
             c55*duzdxl_plus_duxdzl + c45*duzdyl_plus_duydzl + c35*duzdzl;
  *sigma_yz = c14*duxdxl + c46*duxdyl_plus_duydxl + c24*duydyl +
             c45*duzdxl_plus_duxdzl + c44*duzdyl_plus_duydzl + c34*duzdzl;
}

/* ----------------------------------------------------------------------------------------------- */

// computes stresses for isotropic element

__device__ void compute_element_cm_iso(int offset,
                                       realw_const_p d_kappavstore,realw_const_p d_muvstore,
                                       const int ATTENUATION,
                                       realw one_minus_sum_beta_use,
                                       realw duxdxl,realw duydyl,realw duzdzl,
                                       realw duxdxl_plus_duydyl,realw duxdxl_plus_duzdzl,realw duydyl_plus_duzdzl,
                                       realw duxdyl_plus_duydxl,realw duzdxl_plus_duxdzl,realw duzdyl_plus_duydzl,
                                       realw* sigma_xx,realw* sigma_yy,realw* sigma_zz,
                                       realw* sigma_xy,realw* sigma_xz,realw* sigma_yz){

  realw lambdal,mul,lambdalplus2mul,kappal;

  // compute elements with an elastic isotropic rheology
  kappal = d_kappavstore[offset];
  mul = d_muvstore[offset];

  // use unrelaxed parameters if attenuation
  if (ATTENUATION ){
    mul = mul * one_minus_sum_beta_use;
  }

  lambdalplus2mul = kappal + 1.33333333333333333333f * mul;  // 4./3. = 1.3333333
  lambdal = lambdalplus2mul - 2.0f * mul;

  // compute the six components of the stress tensor sigma
  *sigma_xx = lambdalplus2mul*duxdxl + lambdal*duydyl_plus_duzdzl;
  *sigma_yy = lambdalplus2mul*duydyl + lambdal*duxdxl_plus_duzdzl;
  *sigma_zz = lambdalplus2mul*duzdzl + lambdal*duxdxl_plus_duydyl;

  *sigma_xy = mul*duxdyl_plus_duydxl;
  *sigma_xz = mul*duzdxl_plus_duxdzl;
  *sigma_yz = mul*duzdyl_plus_duydzl;

}

/* ----------------------------------------------------------------------------------------------- */

// computes stresses for transversely isotropic element

__device__ void compute_element_cm_tiso(int offset,
                                        realw_const_p d_kappavstore,realw_const_p d_muvstore,
                                        realw_const_p d_kappahstore,realw_const_p d_muhstore,realw_const_p d_eta_anisostore,
                                        const int ATTENUATION,
                                        realw one_minus_sum_beta_use,
                                        realw duxdxl,realw duxdyl,realw duxdzl,
                                        realw duydxl,realw duydyl,realw duydzl,
                                        realw duzdxl,realw duzdyl,realw duzdzl,
                                        realw duxdyl_plus_duydxl,realw duzdxl_plus_duxdzl,realw duzdyl_plus_duydzl,
                                        int iglob,
                                        realw_const_p d_ystore, realw_const_p d_zstore,
                                        realw* sigma_xx,realw* sigma_yy,realw* sigma_zz,
                                        realw* sigma_xy,realw* sigma_xz,realw* sigma_yz){

  realw kappavl,muvl,kappahl,muhl;
  realw rhovpvsq,rhovphsq,rhovsvsq,rhovshsq,eta_aniso;
  realw costheta,sintheta,cosphi,sinphi;
  realw costhetasq,sinthetasq,cosphisq,sinphisq,costhetafour,sinthetafour,cosphifour,sinphifour;
  realw costwotheta,sintwotheta,costwophi,sintwophi,cosfourtheta,cosfourphi;
  realw costwothetasq,costwophisq,sintwophisq;
  realw etaminone,twoetaminone;
  realw two_eta_aniso,four_eta_aniso,six_eta_aniso;
  realw two_rhovsvsq,two_rhovshsq;
  realw four_rhovsvsq,four_rhovshsq;
  realw c11,c12,c13,c14,c15,c16,c22,c23,c24,c25,c26,c33,c34,c35,c36,c44,c45,c46,c55,c56,c66;
  // cosine and sine function in CUDA only supported for float
  realw theta,phi;

  // use Kappa and mu from transversely isotropic model
  kappavl = d_kappavstore[offset];
  muvl = d_muvstore[offset];

  kappahl = d_kappahstore[offset];
  muhl = d_muhstore[offset];

  // use unrelaxed parameters if attenuation
  // eta does not need to be shifted since it is a ratio
  if (ATTENUATION ){
    muvl = muvl * one_minus_sum_beta_use;
    muhl = muhl * one_minus_sum_beta_use;
  }

  rhovpvsq = kappavl + 1.33333333333333333333f * muvl ; //!!! that is C
  rhovphsq = kappahl + 1.33333333333333333333f * muhl ; //!!! that is A

  rhovsvsq = muvl; // !!! that is L
  rhovshsq = muhl; //!!! that is N

  eta_aniso = d_eta_anisostore[offset]; // !!! that is  F / (A - 2 L)

  // use mesh coordinates to get theta and phi
  //ystore and zstore contain theta and phi
  theta = d_ystore[iglob];
  phi = d_zstore[iglob];

  if (sizeof( realw ) == sizeof( float )){
    // float operations

    // sincos function return sinus and cosine for given value
    // example:
    //   sincosf(theta, &sintheta, &costheta);
    // or with loss of accuracy:  __sincosf(theta, &sintheta, &costheta);
    // or compile with: -use_fast_math

    //costheta = cosf(theta);
    //sintheta = sinf(theta);
    sincosf(theta, &sintheta, &costheta);

    //cosphi = cosf(phi);
    //sinphi = sinf(phi);
    sincosf(phi, &sinphi, &cosphi);

    //costwotheta = cosf(2.0f * theta);
    //sintwotheta = sinf(2.0f * theta);
    sincosf(2.0f * theta, &sintwotheta, &costwotheta);

    //costwophi = cosf(2.0f * phi);
    //sintwophi = sinf(2.0f * phi);
    sincosf(2.0f * phi, &sintwophi, &costwophi);

    cosfourtheta = cosf(4.0f * theta);
    cosfourphi = cosf(4.0f * phi);

  }else{
    // double operations
    costheta = cos(theta);
    sintheta = sin(theta);

    cosphi = cos(phi);
    sinphi = sin(phi);

    costwotheta = cos(2.0f * theta);
    sintwotheta = sin(2.0f * theta);
    costwophi = cos(2.0f * phi);
    sintwophi = sin(2.0f * phi);

    cosfourtheta = cos(4.0f * theta);
    cosfourphi = cos(4.0f * phi);
  }

  costhetasq = costheta * costheta;
  sinthetasq = sintheta * sintheta;
  cosphisq = cosphi * cosphi;
  sinphisq = sinphi * sinphi;

  costhetafour = costhetasq * costhetasq;
  sinthetafour = sinthetasq * sinthetasq;
  cosphifour = cosphisq * cosphisq;
  sinphifour = sinphisq * sinphisq;

  costwothetasq = costwotheta * costwotheta;

  costwophisq = costwophi * costwophi;
  sintwophisq = sintwophi * sintwophi;

  etaminone = eta_aniso - 1.0f;
  twoetaminone = 2.0f * eta_aniso - 1.0f;

  // precompute some products to reduce the CPU time
  two_eta_aniso = 2.0f * eta_aniso;
  four_eta_aniso = 4.0f * eta_aniso;
  six_eta_aniso = 6.0f * eta_aniso;

  two_rhovsvsq = 2.0f * rhovsvsq;
  two_rhovshsq = 2.0f * rhovshsq;

  four_rhovsvsq = 4.0f * rhovsvsq;
  four_rhovshsq = 4.0f * rhovshsq;

  // the 21 anisotropic coefficients computed using Mathematica
  c11 = rhovphsq*sinphifour + 2.0f*cosphisq*sinphisq*
        (rhovphsq*costhetasq + (eta_aniso*rhovphsq + two_rhovsvsq - two_eta_aniso*rhovsvsq)*
        sinthetasq) + cosphifour*
        (rhovphsq*costhetafour + 2.0f*(eta_aniso*rhovphsq + two_rhovsvsq - two_eta_aniso*rhovsvsq)*
        costhetasq*sinthetasq + rhovpvsq*sinthetafour);

  c12 = ((rhovphsq - two_rhovshsq)*(3.0f + cosfourphi)*costhetasq)*0.25f -
        four_rhovshsq*cosphisq*costhetasq*sinphisq +
        (rhovphsq*(11.0f + 4.0f*costwotheta + cosfourtheta)*sintwophisq)*0.03125f +
        eta_aniso*(rhovphsq - two_rhovsvsq)*(cosphifour +
        2.0f*cosphisq*costhetasq*sinphisq + sinphifour)*sinthetasq +
        rhovpvsq*cosphisq*sinphisq*sinthetafour -
        rhovsvsq*sintwophisq*sinthetafour;

  c13 = (cosphisq*(rhovphsq + six_eta_aniso*rhovphsq + rhovpvsq - four_rhovsvsq -
        12.0f*eta_aniso*rhovsvsq + (twoetaminone*rhovphsq - rhovpvsq + four_rhovsvsq -
        four_eta_aniso*rhovsvsq)*cosfourtheta))*0.125f +
        sinphisq*(eta_aniso*(rhovphsq - two_rhovsvsq)*costhetasq +
        (rhovphsq - two_rhovshsq)*sinthetasq);

  c14 = costheta*sinphi*((cosphisq*
        (-rhovphsq + rhovpvsq + four_rhovshsq - four_rhovsvsq +
        (-rhovphsq + two_eta_aniso*rhovphsq - rhovpvsq + four_rhovsvsq -
        four_eta_aniso*rhovsvsq)*costwotheta))*0.5f +
        (etaminone*rhovphsq + 2.0f*(rhovshsq - eta_aniso*rhovsvsq))*sinphisq)* sintheta;

  c15 = cosphi*costheta*((cosphisq* (-rhovphsq + rhovpvsq +
        (twoetaminone*rhovphsq - rhovpvsq + four_rhovsvsq - four_eta_aniso*rhovsvsq)*
        costwotheta))*0.5f + etaminone*(rhovphsq - two_rhovsvsq)*sinphisq)*sintheta;

  c16 = (cosphi*sinphi*(cosphisq* (-rhovphsq + rhovpvsq +
        (-rhovphsq + two_eta_aniso*rhovphsq - rhovpvsq + four_rhovsvsq -
        four_eta_aniso*rhovsvsq)*costwotheta) +
        2.0f*etaminone*(rhovphsq - two_rhovsvsq)*sinphisq)*sinthetasq)*0.5f;

  c22 = rhovphsq*cosphifour + 2.0f*cosphisq*sinphisq*
        (rhovphsq*costhetasq + (eta_aniso*rhovphsq + two_rhovsvsq - two_eta_aniso*rhovsvsq)*
        sinthetasq) + sinphifour*
        (rhovphsq*costhetafour + 2.0f*(eta_aniso*rhovphsq + two_rhovsvsq - two_eta_aniso*rhovsvsq)*
        costhetasq*sinthetasq + rhovpvsq*sinthetafour);

  c23 = ((rhovphsq + six_eta_aniso*rhovphsq + rhovpvsq - four_rhovsvsq - 12.0f*eta_aniso*rhovsvsq +
        (twoetaminone*rhovphsq - rhovpvsq + four_rhovsvsq - four_eta_aniso*rhovsvsq)*
        cosfourtheta)*sinphisq)*0.125f +
        cosphisq*(eta_aniso*(rhovphsq - two_rhovsvsq)*costhetasq +
        (rhovphsq - two_rhovshsq)*sinthetasq);

  c24 = costheta*sinphi*(etaminone*(rhovphsq - two_rhovsvsq)*cosphisq +
        ((-rhovphsq + rhovpvsq + (twoetaminone*rhovphsq - rhovpvsq +
        four_rhovsvsq - four_eta_aniso*rhovsvsq)*costwotheta)*sinphisq)*0.5f)*sintheta;

  c25 = cosphi*costheta*((etaminone*rhovphsq + 2.0f*(rhovshsq - eta_aniso*rhovsvsq))*
        cosphisq + ((-rhovphsq + rhovpvsq + four_rhovshsq - four_rhovsvsq +
        (-rhovphsq + two_eta_aniso*rhovphsq - rhovpvsq + four_rhovsvsq -
        four_eta_aniso*rhovsvsq)*costwotheta)*sinphisq)*0.5f)*sintheta;

  c26 = (cosphi*sinphi*(2.0f*etaminone*(rhovphsq - two_rhovsvsq)*cosphisq +
        (-rhovphsq + rhovpvsq + (-rhovphsq + two_eta_aniso*rhovphsq - rhovpvsq + four_rhovsvsq -
        four_eta_aniso*rhovsvsq)*costwotheta)*sinphisq)*sinthetasq)*0.5f;

  c33 = rhovpvsq*costhetafour + 2.0f*(eta_aniso*(rhovphsq - two_rhovsvsq) + two_rhovsvsq)*
        costhetasq*sinthetasq + rhovphsq*sinthetafour;

  c34 = -((rhovphsq - rhovpvsq + (twoetaminone*rhovphsq - rhovpvsq + four_rhovsvsq
        - four_eta_aniso*rhovsvsq)*costwotheta)*sinphi*sintwotheta)*0.25f;

  c35 = -(cosphi*(rhovphsq - rhovpvsq +
        (twoetaminone*rhovphsq - rhovpvsq + four_rhovsvsq - four_eta_aniso*rhovsvsq)*
        costwotheta)*sintwotheta)*0.25f;

  c36 = -((rhovphsq - rhovpvsq - four_rhovshsq + four_rhovsvsq +
        (twoetaminone*rhovphsq - rhovpvsq + four_rhovsvsq - four_eta_aniso*rhovsvsq)*
        costwotheta)*sintwophi*sinthetasq)*0.25f;

  c44 = cosphisq*(rhovsvsq*costhetasq + rhovshsq*sinthetasq) +
        sinphisq*(rhovsvsq*costwothetasq +
        (rhovphsq - two_eta_aniso*rhovphsq + rhovpvsq + four_eta_aniso*rhovsvsq)*costhetasq* sinthetasq);

  c45 = ((rhovphsq - two_eta_aniso*rhovphsq + rhovpvsq - two_rhovshsq - two_rhovsvsq +
        four_eta_aniso*rhovsvsq + (rhovphsq - two_eta_aniso*rhovphsq + rhovpvsq +
        4.0f*etaminone*rhovsvsq)*costwotheta)*sintwophi*sinthetasq)*0.25f;

  c46 = -(cosphi*costheta*((rhovshsq - rhovsvsq)*cosphisq -
        ((rhovphsq - two_eta_aniso*rhovphsq + rhovpvsq - two_rhovshsq - two_rhovsvsq +
        four_eta_aniso*rhovsvsq + (-rhovphsq + two_eta_aniso*rhovphsq - rhovpvsq +
        four_rhovsvsq - four_eta_aniso*rhovsvsq)*costwotheta)*sinphisq)*0.5f)* sintheta);

  c55 = sinphisq*(rhovsvsq*costhetasq + rhovshsq*sinthetasq) +
        cosphisq*(rhovsvsq*costwothetasq +
        (rhovphsq - two_eta_aniso*rhovphsq + rhovpvsq + four_eta_aniso*rhovsvsq)*costhetasq* sinthetasq);

  c56 = costheta*sinphi*((cosphisq*
        (rhovphsq - two_eta_aniso*rhovphsq + rhovpvsq - two_rhovshsq - two_rhovsvsq +
        four_eta_aniso*rhovsvsq + (-rhovphsq + two_eta_aniso*rhovphsq - rhovpvsq +
        four_rhovsvsq - four_eta_aniso*rhovsvsq)*costwotheta))*0.5f +
        (-rhovshsq + rhovsvsq)*sinphisq)*sintheta;

  c66 = rhovshsq*costwophisq*costhetasq -
        2.0f*(rhovphsq - two_rhovshsq)*cosphisq*costhetasq*sinphisq +
        (rhovphsq*(11.0f + 4.0f*costwotheta + cosfourtheta)*sintwophisq)*0.03125f -
        (rhovsvsq*(-6.0f - 2.0f*cosfourphi + cos(4.0f*phi - 2.0f*theta) - 2.0f*costwotheta +
        cos(2.0f*(2.0f*phi + theta)))*sinthetasq)*0.125f +
        rhovpvsq*cosphisq*sinphisq*sinthetafour -
        (eta_aniso*(rhovphsq - two_rhovsvsq)*sintwophisq*sinthetafour)*0.5f;

  // general expression of stress tensor for full Cijkl with 21 coefficients

  *sigma_xx = c11*duxdxl + c16*duxdyl_plus_duydxl + c12*duydyl +
              c15*duzdxl_plus_duxdzl + c14*duzdyl_plus_duydzl + c13*duzdzl;

  *sigma_yy = c12*duxdxl + c26*duxdyl_plus_duydxl + c22*duydyl +
              c25*duzdxl_plus_duxdzl + c24*duzdyl_plus_duydzl + c23*duzdzl;

  *sigma_zz = c13*duxdxl + c36*duxdyl_plus_duydxl + c23*duydyl +
              c35*duzdxl_plus_duxdzl + c34*duzdyl_plus_duydzl + c33*duzdzl;

  *sigma_xy = c16*duxdxl + c66*duxdyl_plus_duydxl + c26*duydyl +
              c56*duzdxl_plus_duxdzl + c46*duzdyl_plus_duydzl + c36*duzdzl;

  *sigma_xz = c15*duxdxl + c56*duxdyl_plus_duydxl + c25*duydyl +
              c55*duzdxl_plus_duxdzl + c45*duzdyl_plus_duydzl + c35*duzdzl;

  *sigma_yz = c14*duxdxl + c46*duxdyl_plus_duydxl + c24*duydyl +
              c45*duzdxl_plus_duxdzl + c44*duzdyl_plus_duydzl + c34*duzdzl;
}

/* ----------------------------------------------------------------------------------------------- */


// loads displacement into shared memory for element

template<int FORWARD_OR_ADJOINT>
__device__ void load_shared_memory_cm(const int* tx, const int* iglob,
                                      realw_const_p d_displ,
                                      realw* s_dummyx_loc,
                                      realw* s_dummyy_loc,
                                      realw* s_dummyz_loc){

  // copy from global memory to shared memory
  // each thread writes one of the NGLL^3 = 125 data points
#ifdef USE_TEXTURES_FIELDS
  s_dummyx_loc[(*tx)] = texfetch_displ_cm<FORWARD_OR_ADJOINT>((*iglob)*3);
  s_dummyy_loc[(*tx)] = texfetch_displ_cm<FORWARD_OR_ADJOINT>((*iglob)*3 + 1);
  s_dummyz_loc[(*tx)] = texfetch_displ_cm<FORWARD_OR_ADJOINT>((*iglob)*3 + 2);
#else
  // changing iglob indexing to match fortran row changes fast style
  s_dummyx_loc[(*tx)] = d_displ[(*iglob)*3];
  s_dummyy_loc[(*tx)] = d_displ[(*iglob)*3 + 1];
  s_dummyz_loc[(*tx)] = d_displ[(*iglob)*3 + 2];
#endif
}

/* ----------------------------------------------------------------------------------------------- */

// loads hprime into shared memory for element

__device__ void load_shared_memory_hprime(const int* tx,
                                          realw_const_p d_hprime_xx,
                                          realw* sh_hprime_xx){

  // each thread reads its corresponding value
  // (might be faster sometimes...)
#ifdef USE_TEXTURES_CONSTANTS
  // hprime
  sh_hprime_xx[(*tx)] = tex1Dfetch(d_hprime_xx_tex,tx + d_hprime_xx_tex_offset);
#else
  // hprime
  sh_hprime_xx[(*tx)] = d_hprime_xx[(*tx)];
#endif
}


/* ----------------------------------------------------------------------------------------------- */

// loads hprimewgll into shared memory for element

__device__ void load_shared_memory_hprimewgll(const int* tx,
                                              realw_const_p d_hprimewgll_xx,
                                              realw* sh_hprimewgll_xx ){

  // each thread reads its corresponding value
#ifdef USE_TEXTURES_CONSTANTS
  // weighted hprime
  sh_hprimewgll_xx[(*tx)] = tex1Dfetch(d_hprimewgll_xx_tex,tx + d_hprimewgll_xx_tex_offset);
#else
  // weighted hprime
  sh_hprimewgll_xx[(*tx)] = d_hprimewgll_xx[(*tx)];
#endif
}

/* ----------------------------------------------------------------------------------------------- */

// KERNEL 2
//
// for crust_mantle

/* ----------------------------------------------------------------------------------------------- */

template<int FORWARD_OR_ADJOINT> __global__ void
#ifdef USE_LAUNCH_BOUNDS
// adds compiler specification
__launch_bounds__(NGLL3_PADDED,LAUNCH_MIN_BLOCKS)
#endif
crust_mantle_impl_kernel( int nb_blocks_to_compute,
                          const int* d_ibool,
                          const int* d_ispec_is_tiso,
                          const int* d_phase_ispec_inner,
                          int num_phase_ispec,
                          const int d_iphase,
                          realw deltat,
                          const int use_mesh_coloring_gpu,
                          realw_const_p d_displ,
                          realw_p d_accel,
                          realw_const_p d_xix, realw_const_p d_xiy, realw_const_p d_xiz,
                          realw_const_p d_etax, realw_const_p d_etay, realw_const_p d_etaz,
                          realw_const_p d_gammax, realw_const_p d_gammay, realw_const_p d_gammaz,
                          realw_const_p d_hprime_xx,
                          realw_const_p d_hprimewgll_xx,
                          realw_const_p d_wgllwgll_xy,
                          realw_const_p d_wgllwgll_xz,
                          realw_const_p d_wgllwgll_yz,
                          realw_const_p d_kappavstore,
                          realw_const_p d_muvstore,
                          realw_const_p d_kappahstore,
                          realw_const_p d_muhstore,
                          realw_const_p d_eta_anisostore,
                          const int COMPUTE_AND_STORE_STRAIN,
                          realw_p epsilondev_xx,
                          realw_p epsilondev_yy,
                          realw_p epsilondev_xy,
                          realw_p epsilondev_xz,
                          realw_p epsilondev_yz,
                          realw_p epsilon_trace_over_3,
                          const int ATTENUATION,
                          const int PARTIAL_PHYS_DISPERSION_ONLY,
                          const int USE_3D_ATTENUATION_ARRAYS,
                          realw_const_p one_minus_sum_beta,
                          realw_const_p factor_common,
                          realw_p R_xx, realw_p R_yy, realw_p R_xy, realw_p R_xz, realw_p R_yz,
                          realw_const_p alphaval,
                          realw_const_p betaval,
                          realw_const_p gammaval,
                          const int ANISOTROPY,
                          realw_const_p d_c11store,
                          realw_const_p d_c12store,
                          realw_const_p d_c13store,
                          realw_const_p d_c14store,
                          realw_const_p d_c15store,
                          realw_const_p d_c16store,
                          realw_const_p d_c22store,
                          realw_const_p d_c23store,
                          realw_const_p d_c24store,
                          realw_const_p d_c25store,
                          realw_const_p d_c26store,
                          realw_const_p d_c33store,
                          realw_const_p d_c34store,
                          realw_const_p d_c35store,
                          realw_const_p d_c36store,
                          realw_const_p d_c44store,
                          realw_const_p d_c45store,
                          realw_const_p d_c46store,
                          realw_const_p d_c55store,
                          realw_const_p d_c56store,
                          realw_const_p d_c66store,
                          const int GRAVITY,
                          realw_const_p d_xstore,
                          realw_const_p d_ystore,
                          realw_const_p d_zstore,
                          realw_const_p d_minus_gravity_table,
                          realw_const_p d_minus_deriv_gravity_table,
                          realw_const_p d_density_table,
                          realw_const_p wgll_cube,
                          const int NSPEC_CRUST_MANTLE_STRAIN_ONLY ){

  // block id == spectral-element id
  int bx = blockIdx.y*gridDim.x+blockIdx.x;
  // thread id == GLL point id
  int tx = threadIdx.x;

  int K = (tx/NGLL2);
  int J = ((tx-K*NGLL2)/NGLLX);
  int I = (tx-K*NGLL2-J*NGLLX);

  unsigned short int active;
  int iglob,offset;
  int working_element;

  realw tempx1l,tempx2l,tempx3l,tempy1l,tempy2l,tempy3l,tempz1l,tempz2l,tempz3l;
  realw xixl,xiyl,xizl,etaxl,etayl,etazl,gammaxl,gammayl,gammazl,jacobianl;
  realw duxdxl,duxdyl,duxdzl,duydxl,duydyl,duydzl,duzdxl,duzdyl,duzdzl;
  realw duxdxl_plus_duydyl,duxdxl_plus_duzdzl,duydyl_plus_duzdzl;
  realw duxdyl_plus_duydxl,duzdxl_plus_duxdzl,duzdyl_plus_duydzl;
  realw templ;

  realw fac1,fac2,fac3;
  realw one_minus_sum_beta_use;

  realw sigma_xx,sigma_yy,sigma_zz,sigma_xy,sigma_xz,sigma_yz;
  realw epsilondev_xx_loc,epsilondev_yy_loc,epsilondev_xy_loc,epsilondev_xz_loc,epsilondev_yz_loc;
  realw sum_terms1,sum_terms2,sum_terms3;

  // gravity variables
  realw sigma_yx,sigma_zx,sigma_zy;
  realw rho_s_H1,rho_s_H2,rho_s_H3;

#ifndef MANUALLY_UNROLLED_LOOPS
  int l;
#endif

  // shared memory arrays
  __shared__ realw s_dummyx_loc[NGLL3];
  __shared__ realw s_dummyy_loc[NGLL3];
  __shared__ realw s_dummyz_loc[NGLL3];

  __shared__ realw s_tempx1[NGLL3];
  __shared__ realw s_tempx2[NGLL3];
  __shared__ realw s_tempx3[NGLL3];

  __shared__ realw s_tempy1[NGLL3];
  __shared__ realw s_tempy2[NGLL3];
  __shared__ realw s_tempy3[NGLL3];

  __shared__ realw s_tempz1[NGLL3];
  __shared__ realw s_tempz2[NGLL3];
  __shared__ realw s_tempz3[NGLL3];

  // note: using shared memory for hprime's improves performance
  //       (but could tradeoff with occupancy)
  __shared__ realw sh_hprime_xx[NGLL2];
  __shared__ realw sh_hprimewgll_xx[NGLL2];

  // use only NGLL^3 = 125 active threads, plus 3 inactive/ghost threads,
  // because we used memory padding from NGLL^3 = 125 to 128 to get coalescent memory accesses
  active = (tx < NGLL3 && bx < nb_blocks_to_compute) ? 1:0;

  // determines spectral element to work on
  if (active) {
#ifdef USE_MESH_COLORING_GPU
    working_element = bx;
#else
    //mesh coloring
    if (use_mesh_coloring_gpu){
      working_element = bx;
    }else{
      // iphase-1 and working_element-1 for Fortran->C array conventions
      working_element = d_phase_ispec_inner[bx + num_phase_ispec*(d_iphase-1)]-1;
    }
#endif
    // local padded index
    offset = working_element*NGLL3_PADDED + tx;

    // global index
    iglob = d_ibool[working_element*NGLL3 + tx]-1;

    // copy displacement from global memory to shared memory
    load_shared_memory_cm<FORWARD_OR_ADJOINT>(&tx,&iglob,d_displ,s_dummyx_loc,s_dummyy_loc,s_dummyz_loc);
  } // active

  // loads hprime's into shared memory
  if (tx < NGLL2) {
    // copy hprime from global memory to shared memory
    load_shared_memory_hprime(&tx,d_hprime_xx,sh_hprime_xx);
    // copy hprime from global memory to shared memory
    load_shared_memory_hprimewgll(&tx,d_hprimewgll_xx,sh_hprimewgll_xx);
  }

// synchronize all the threads (one thread for each of the NGLL grid points of the
// current spectral element) because we need the whole element to be ready in order
// to be able to compute the matrix products along cut planes of the 3D element below
  __syncthreads();

  if (active) {

#ifdef MANUALLY_UNROLLED_LOOPS
    tempx1l = s_dummyx_loc[K*NGLL2+J*NGLLX]*sh_hprime_xx[I]
            + s_dummyx_loc[K*NGLL2+J*NGLLX+1]*sh_hprime_xx[NGLLX+I]
            + s_dummyx_loc[K*NGLL2+J*NGLLX+2]*sh_hprime_xx[2*NGLLX+I]
            + s_dummyx_loc[K*NGLL2+J*NGLLX+3]*sh_hprime_xx[3*NGLLX+I]
            + s_dummyx_loc[K*NGLL2+J*NGLLX+4]*sh_hprime_xx[4*NGLLX+I];

    tempy1l = s_dummyy_loc[K*NGLL2+J*NGLLX]*sh_hprime_xx[I]
            + s_dummyy_loc[K*NGLL2+J*NGLLX+1]*sh_hprime_xx[NGLLX+I]
            + s_dummyy_loc[K*NGLL2+J*NGLLX+2]*sh_hprime_xx[2*NGLLX+I]
            + s_dummyy_loc[K*NGLL2+J*NGLLX+3]*sh_hprime_xx[3*NGLLX+I]
            + s_dummyy_loc[K*NGLL2+J*NGLLX+4]*sh_hprime_xx[4*NGLLX+I];

    tempz1l = s_dummyz_loc[K*NGLL2+J*NGLLX]*sh_hprime_xx[I]
            + s_dummyz_loc[K*NGLL2+J*NGLLX+1]*sh_hprime_xx[NGLLX+I]
            + s_dummyz_loc[K*NGLL2+J*NGLLX+2]*sh_hprime_xx[2*NGLLX+I]
            + s_dummyz_loc[K*NGLL2+J*NGLLX+3]*sh_hprime_xx[3*NGLLX+I]
            + s_dummyz_loc[K*NGLL2+J*NGLLX+4]*sh_hprime_xx[4*NGLLX+I];

    tempx2l = s_dummyx_loc[K*NGLL2+I]*sh_hprime_xx[J]
            + s_dummyx_loc[K*NGLL2+NGLLX+I]*sh_hprime_xx[NGLLX+J]
            + s_dummyx_loc[K*NGLL2+2*NGLLX+I]*sh_hprime_xx[2*NGLLX+J]
            + s_dummyx_loc[K*NGLL2+3*NGLLX+I]*sh_hprime_xx[3*NGLLX+J]
            + s_dummyx_loc[K*NGLL2+4*NGLLX+I]*sh_hprime_xx[4*NGLLX+J];

    tempy2l = s_dummyy_loc[K*NGLL2+I]*sh_hprime_xx[J]
            + s_dummyy_loc[K*NGLL2+NGLLX+I]*sh_hprime_xx[NGLLX+J]
            + s_dummyy_loc[K*NGLL2+2*NGLLX+I]*sh_hprime_xx[2*NGLLX+J]
            + s_dummyy_loc[K*NGLL2+3*NGLLX+I]*sh_hprime_xx[3*NGLLX+J]
            + s_dummyy_loc[K*NGLL2+4*NGLLX+I]*sh_hprime_xx[4*NGLLX+J];

    tempz2l = s_dummyz_loc[K*NGLL2+I]*sh_hprime_xx[J]
            + s_dummyz_loc[K*NGLL2+NGLLX+I]*sh_hprime_xx[NGLLX+J]
            + s_dummyz_loc[K*NGLL2+2*NGLLX+I]*sh_hprime_xx[2*NGLLX+J]
            + s_dummyz_loc[K*NGLL2+3*NGLLX+I]*sh_hprime_xx[3*NGLLX+J]
            + s_dummyz_loc[K*NGLL2+4*NGLLX+I]*sh_hprime_xx[4*NGLLX+J];

    tempx3l = s_dummyx_loc[J*NGLLX+I]*sh_hprime_xx[K]
            + s_dummyx_loc[NGLL2+J*NGLLX+I]*sh_hprime_xx[NGLLX+K]
            + s_dummyx_loc[2*NGLL2+J*NGLLX+I]*sh_hprime_xx[2*NGLLX+K]
            + s_dummyx_loc[3*NGLL2+J*NGLLX+I]*sh_hprime_xx[3*NGLLX+K]
            + s_dummyx_loc[4*NGLL2+J*NGLLX+I]*sh_hprime_xx[4*NGLLX+K];

    tempy3l = s_dummyy_loc[J*NGLLX+I]*sh_hprime_xx[K]
            + s_dummyy_loc[NGLL2+J*NGLLX+I]*sh_hprime_xx[NGLLX+K]
            + s_dummyy_loc[2*NGLL2+J*NGLLX+I]*sh_hprime_xx[2*NGLLX+K]
            + s_dummyy_loc[3*NGLL2+J*NGLLX+I]*sh_hprime_xx[3*NGLLX+K]
            + s_dummyy_loc[4*NGLL2+J*NGLLX+I]*sh_hprime_xx[4*NGLLX+K];

    tempz3l = s_dummyz_loc[J*NGLLX+I]*sh_hprime_xx[K]
            + s_dummyz_loc[NGLL2+J*NGLLX+I]*sh_hprime_xx[NGLLX+K]
            + s_dummyz_loc[2*NGLL2+J*NGLLX+I]*sh_hprime_xx[2*NGLLX+K]
            + s_dummyz_loc[3*NGLL2+J*NGLLX+I]*sh_hprime_xx[3*NGLLX+K]
            + s_dummyz_loc[4*NGLL2+J*NGLLX+I]*sh_hprime_xx[4*NGLLX+K];
#else
    tempx1l = 0.f;
    tempx2l = 0.f;
    tempx3l = 0.f;

    tempy1l = 0.f;
    tempy2l = 0.f;
    tempy3l = 0.f;

    tempz1l = 0.f;
    tempz2l = 0.f;
    tempz3l = 0.f;

    for (l=0;l<NGLLX;l++) {
        fac1 = sh_hprime_xx[l*NGLLX+I];
        tempx1l += s_dummyx_loc[K*NGLL2+J*NGLLX+l]*fac1;
        tempy1l += s_dummyy_loc[K*NGLL2+J*NGLLX+l]*fac1;
        tempz1l += s_dummyz_loc[K*NGLL2+J*NGLLX+l]*fac1;

        fac2 = sh_hprime_xx[l*NGLLX+J];
        tempx2l += s_dummyx_loc[K*NGLL2+l*NGLLX+I]*fac2;
        tempy2l += s_dummyy_loc[K*NGLL2+l*NGLLX+I]*fac2;
        tempz2l += s_dummyz_loc[K*NGLL2+l*NGLLX+I]*fac2;

        fac3 = sh_hprime_xx[l*NGLLX+K];
        tempx3l += s_dummyx_loc[l*NGLL2+J*NGLLX+I]*fac3;
        tempy3l += s_dummyy_loc[l*NGLL2+J*NGLLX+I]*fac3;
        tempz3l += s_dummyz_loc[l*NGLL2+J*NGLLX+I]*fac3;
    }
#endif

    // compute derivatives of ux, uy and uz with respect to x, y and z
    xixl = d_xix[offset];
    xiyl = d_xiy[offset];
    xizl = d_xiz[offset];
    etaxl = d_etax[offset];
    etayl = d_etay[offset];
    etazl = d_etaz[offset];
    gammaxl = d_gammax[offset];
    gammayl = d_gammay[offset];
    gammazl = d_gammaz[offset];

    duxdxl = xixl*tempx1l + etaxl*tempx2l + gammaxl*tempx3l;
    duxdyl = xiyl*tempx1l + etayl*tempx2l + gammayl*tempx3l;
    duxdzl = xizl*tempx1l + etazl*tempx2l + gammazl*tempx3l;

    duydxl = xixl*tempy1l + etaxl*tempy2l + gammaxl*tempy3l;
    duydyl = xiyl*tempy1l + etayl*tempy2l + gammayl*tempy3l;
    duydzl = xizl*tempy1l + etazl*tempy2l + gammazl*tempy3l;

    duzdxl = xixl*tempz1l + etaxl*tempz2l + gammaxl*tempz3l;
    duzdyl = xiyl*tempz1l + etayl*tempz2l + gammayl*tempz3l;
    duzdzl = xizl*tempz1l + etazl*tempz2l + gammazl*tempz3l;

    // precompute some sums to save CPU time
    duxdxl_plus_duydyl = duxdxl + duydyl;
    duxdxl_plus_duzdzl = duxdxl + duzdzl;
    duydyl_plus_duzdzl = duydyl + duzdzl;
    duxdyl_plus_duydxl = duxdyl + duydxl;
    duzdxl_plus_duxdzl = duzdxl + duxdzl;
    duzdyl_plus_duydzl = duzdyl + duydzl;

    // computes deviatoric strain attenuation and/or for kernel calculations
    if(COMPUTE_AND_STORE_STRAIN) {
      templ = 0.33333333333333333333f * (duxdxl + duydyl + duzdzl); // 1./3. = 0.33333

      // local storage: stresses at this current time step
      epsilondev_xx_loc = duxdxl - templ;
      epsilondev_yy_loc = duydyl - templ;
      epsilondev_xy_loc = 0.5f * duxdyl_plus_duydxl;
      epsilondev_xz_loc = 0.5f * duzdxl_plus_duxdzl;
      epsilondev_yz_loc = 0.5f * duzdyl_plus_duydzl;

      if(NSPEC_CRUST_MANTLE_STRAIN_ONLY == 1) {
        epsilon_trace_over_3[tx] = templ;
      }else{
        epsilon_trace_over_3[tx + working_element*NGLL3] = templ;
      }
    }

    // attenuation
    if(ATTENUATION){
      // use unrelaxed parameters if attenuation
      if (USE_3D_ATTENUATION_ARRAYS){
        one_minus_sum_beta_use = one_minus_sum_beta[tx+working_element*NGLL3]; // (i,j,k,ispec)
      }else{
        one_minus_sum_beta_use = one_minus_sum_beta[working_element]; // (1,1,1,ispec)
      }
    }

    // computes stresses
    if(ANISOTROPY){
      // full anisotropic case, stress calculations
      compute_element_cm_aniso(offset,
                            d_c11store,d_c12store,d_c13store,d_c14store,d_c15store,d_c16store,d_c22store,
                            d_c23store,d_c24store,d_c25store,d_c26store,d_c33store,d_c34store,d_c35store,
                            d_c36store,d_c44store,d_c45store,d_c46store,d_c55store,d_c56store,d_c66store,
                            ATTENUATION,
                            one_minus_sum_beta_use,
                            duxdxl,duxdyl,duxdzl,duydxl,duydyl,duydzl,duzdxl,duzdyl,duzdzl,
                            duxdyl_plus_duydxl,duzdxl_plus_duxdzl,duzdyl_plus_duydzl,
                            &sigma_xx,&sigma_yy,&sigma_zz,
                            &sigma_xy,&sigma_xz,&sigma_yz);

    }else{
      if (! d_ispec_is_tiso[working_element]){
        // isotropic case
        compute_element_cm_iso(offset,
                            d_kappavstore,d_muvstore,
                            ATTENUATION,
                            one_minus_sum_beta_use,
                            duxdxl,duydyl,duzdzl,
                            duxdxl_plus_duydyl,duxdxl_plus_duzdzl,duydyl_plus_duzdzl,
                            duxdyl_plus_duydxl,duzdxl_plus_duxdzl,duzdyl_plus_duydzl,
                            &sigma_xx,&sigma_yy,&sigma_zz,
                            &sigma_xy,&sigma_xz,&sigma_yz);
      }else{
        // transverse isotropy
        compute_element_cm_tiso(offset,
                              d_kappavstore,d_muvstore,
                              d_kappahstore,d_muhstore,d_eta_anisostore,
                              ATTENUATION,
                              one_minus_sum_beta_use,
                              duxdxl,duxdyl,duxdzl,
                              duydxl,duydyl,duydzl,
                              duzdxl,duzdyl,duzdzl,
                              duxdyl_plus_duydxl,duzdxl_plus_duxdzl,duzdyl_plus_duydzl,
                              iglob,
                              d_ystore,d_zstore,
                              &sigma_xx,&sigma_yy,&sigma_zz,
                              &sigma_xy,&sigma_xz,&sigma_yz);
      }
    } // ! end of test whether isotropic or anisotropic element

    if(ATTENUATION && (! PARTIAL_PHYS_DISPERSION_ONLY ) ){
      // subtracts memory variables if attenuation
      compute_element_cm_att_stress(tx,working_element,
                                    R_xx,R_yy,R_xy,R_xz,R_yz,
                                    &sigma_xx,&sigma_yy,&sigma_zz,&sigma_xy,&sigma_xz,&sigma_yz);
    }

    // define symmetric components (needed for non-symmetric dot product and sigma for gravity)
    sigma_yx = sigma_xy;
    sigma_zx = sigma_xz;
    sigma_zy = sigma_yz;

    // jacobian
    jacobianl = 1.0f / (xixl*(etayl*gammazl-etazl*gammayl)
                      - xiyl*(etaxl*gammazl-etazl*gammaxl)
                      + xizl*(etaxl*gammayl-etayl*gammaxl));

    if (GRAVITY){
      //  computes non-symmetric terms for gravity
      compute_element_cm_gravity(tx,iglob,
                                 d_xstore,d_ystore,d_zstore,
                                 d_minus_gravity_table,d_minus_deriv_gravity_table,d_density_table,
                                 wgll_cube,jacobianl,
                                 s_dummyx_loc,s_dummyy_loc,s_dummyz_loc,
                                 &sigma_xx,&sigma_yy,&sigma_zz,&sigma_xy,&sigma_yx,
                                 &sigma_xz,&sigma_zx,&sigma_yz,&sigma_zy,
                                 &rho_s_H1,&rho_s_H2,&rho_s_H3);
    }

    // form dot product with test vector, non-symmetric form
    s_tempx1[tx] = jacobianl * (sigma_xx*xixl + sigma_yx*xiyl + sigma_zx*xizl);
    s_tempy1[tx] = jacobianl * (sigma_xy*xixl + sigma_yy*xiyl + sigma_zy*xizl);
    s_tempz1[tx] = jacobianl * (sigma_xz*xixl + sigma_yz*xiyl + sigma_zz*xizl);

    s_tempx2[tx] = jacobianl * (sigma_xx*etaxl + sigma_yx*etayl + sigma_zx*etazl);
    s_tempy2[tx] = jacobianl * (sigma_xy*etaxl + sigma_yy*etayl + sigma_zy*etazl);
    s_tempz2[tx] = jacobianl * (sigma_xz*etaxl + sigma_yz*etayl + sigma_zz*etazl);

    s_tempx3[tx] = jacobianl * (sigma_xx*gammaxl + sigma_yx*gammayl + sigma_zx*gammazl);
    s_tempy3[tx] = jacobianl * (sigma_xy*gammaxl + sigma_yy*gammayl + sigma_zy*gammazl);
    s_tempz3[tx] = jacobianl * (sigma_xz*gammaxl + sigma_yz*gammayl + sigma_zz*gammazl);
  }

// synchronize all the threads (one thread for each of the NGLL grid points of the
// current spectral element) because we need the whole element to be ready in order
// to be able to compute the matrix products along cut planes of the 3D element below
  __syncthreads();

  if (active) {

#ifdef MANUALLY_UNROLLED_LOOPS
    tempx1l = s_tempx1[K*NGLL2+J*NGLLX]*sh_hprimewgll_xx[I*NGLLX]
            + s_tempx1[K*NGLL2+J*NGLLX+1]*sh_hprimewgll_xx[I*NGLLX+1]
            + s_tempx1[K*NGLL2+J*NGLLX+2]*sh_hprimewgll_xx[I*NGLLX+2]
            + s_tempx1[K*NGLL2+J*NGLLX+3]*sh_hprimewgll_xx[I*NGLLX+3]
            + s_tempx1[K*NGLL2+J*NGLLX+4]*sh_hprimewgll_xx[I*NGLLX+4];

    tempy1l = s_tempy1[K*NGLL2+J*NGLLX]*sh_hprimewgll_xx[I*NGLLX]
            + s_tempy1[K*NGLL2+J*NGLLX+1]*sh_hprimewgll_xx[I*NGLLX+1]
            + s_tempy1[K*NGLL2+J*NGLLX+2]*sh_hprimewgll_xx[I*NGLLX+2]
            + s_tempy1[K*NGLL2+J*NGLLX+3]*sh_hprimewgll_xx[I*NGLLX+3]
            + s_tempy1[K*NGLL2+J*NGLLX+4]*sh_hprimewgll_xx[I*NGLLX+4];

    tempz1l = s_tempz1[K*NGLL2+J*NGLLX]*sh_hprimewgll_xx[I*NGLLX]
            + s_tempz1[K*NGLL2+J*NGLLX+1]*sh_hprimewgll_xx[I*NGLLX+1]
            + s_tempz1[K*NGLL2+J*NGLLX+2]*sh_hprimewgll_xx[I*NGLLX+2]
            + s_tempz1[K*NGLL2+J*NGLLX+3]*sh_hprimewgll_xx[I*NGLLX+3]
            + s_tempz1[K*NGLL2+J*NGLLX+4]*sh_hprimewgll_xx[I*NGLLX+4];

    tempx2l = s_tempx2[K*NGLL2+I]*sh_hprimewgll_xx[J*NGLLX]
            + s_tempx2[K*NGLL2+NGLLX+I]*sh_hprimewgll_xx[J*NGLLX+1]
            + s_tempx2[K*NGLL2+2*NGLLX+I]*sh_hprimewgll_xx[J*NGLLX+2]
            + s_tempx2[K*NGLL2+3*NGLLX+I]*sh_hprimewgll_xx[J*NGLLX+3]
            + s_tempx2[K*NGLL2+4*NGLLX+I]*sh_hprimewgll_xx[J*NGLLX+4];

    tempy2l = s_tempy2[K*NGLL2+I]*sh_hprimewgll_xx[J*NGLLX]
            + s_tempy2[K*NGLL2+NGLLX+I]*sh_hprimewgll_xx[J*NGLLX+1]
            + s_tempy2[K*NGLL2+2*NGLLX+I]*sh_hprimewgll_xx[J*NGLLX+2]
            + s_tempy2[K*NGLL2+3*NGLLX+I]*sh_hprimewgll_xx[J*NGLLX+3]
            + s_tempy2[K*NGLL2+4*NGLLX+I]*sh_hprimewgll_xx[J*NGLLX+4];

    tempz2l = s_tempz2[K*NGLL2+I]*sh_hprimewgll_xx[J*NGLLX]
            + s_tempz2[K*NGLL2+NGLLX+I]*sh_hprimewgll_xx[J*NGLLX+1]
            + s_tempz2[K*NGLL2+2*NGLLX+I]*sh_hprimewgll_xx[J*NGLLX+2]
            + s_tempz2[K*NGLL2+3*NGLLX+I]*sh_hprimewgll_xx[J*NGLLX+3]
            + s_tempz2[K*NGLL2+4*NGLLX+I]*sh_hprimewgll_xx[J*NGLLX+4];

    tempx3l = s_tempx3[J*NGLLX+I]*sh_hprimewgll_xx[K*NGLLX]
            + s_tempx3[NGLL2+J*NGLLX+I]*sh_hprimewgll_xx[K*NGLLX+1]
            + s_tempx3[2*NGLL2+J*NGLLX+I]*sh_hprimewgll_xx[K*NGLLX+2]
            + s_tempx3[3*NGLL2+J*NGLLX+I]*sh_hprimewgll_xx[K*NGLLX+3]
            + s_tempx3[4*NGLL2+J*NGLLX+I]*sh_hprimewgll_xx[K*NGLLX+4];

    tempy3l = s_tempy3[J*NGLLX+I]*sh_hprimewgll_xx[K*NGLLX]
            + s_tempy3[NGLL2+J*NGLLX+I]*sh_hprimewgll_xx[K*NGLLX+1]
            + s_tempy3[2*NGLL2+J*NGLLX+I]*sh_hprimewgll_xx[K*NGLLX+2]
            + s_tempy3[3*NGLL2+J*NGLLX+I]*sh_hprimewgll_xx[K*NGLLX+3]
            + s_tempy3[4*NGLL2+J*NGLLX+I]*sh_hprimewgll_xx[K*NGLLX+4];

    tempz3l = s_tempz3[J*NGLLX+I]*sh_hprimewgll_xx[K*NGLLX]
            + s_tempz3[NGLL2+J*NGLLX+I]*sh_hprimewgll_xx[K*NGLLX+1]
            + s_tempz3[2*NGLL2+J*NGLLX+I]*sh_hprimewgll_xx[K*NGLLX+2]
            + s_tempz3[3*NGLL2+J*NGLLX+I]*sh_hprimewgll_xx[K*NGLLX+3]
            + s_tempz3[4*NGLL2+J*NGLLX+I]*sh_hprimewgll_xx[K*NGLLX+4];
#else
    tempx1l = 0.f;
    tempy1l = 0.f;
    tempz1l = 0.f;

    tempx2l = 0.f;
    tempy2l = 0.f;
    tempz2l = 0.f;

    tempx3l = 0.f;
    tempy3l = 0.f;
    tempz3l = 0.f;

    for (l=0;l<NGLLX;l++) {
      fac1 = sh_hprimewgll_xx[I*NGLLX+l];
      tempx1l += s_tempx1[K*NGLL2+J*NGLLX+l]*fac1;
      tempy1l += s_tempy1[K*NGLL2+J*NGLLX+l]*fac1;
      tempz1l += s_tempz1[K*NGLL2+J*NGLLX+l]*fac1;

      // assume hprimewgll_xx == hprimewgll_yy == hprimewgll_zz
      fac2 = sh_hprimewgll_xx[J*NGLLX+l];
      tempx2l += s_tempx2[K*NGLL2+l*NGLLX+I]*fac2;
      tempy2l += s_tempy2[K*NGLL2+l*NGLLX+I]*fac2;
      tempz2l += s_tempz2[K*NGLL2+l*NGLLX+I]*fac2;

      fac3 = sh_hprimewgll_xx[K*NGLLX+l];
      tempx3l += s_tempx3[l*NGLL2+J*NGLLX+I]*fac3;
      tempy3l += s_tempy3[l*NGLL2+J*NGLLX+I]*fac3;
      tempz3l += s_tempz3[l*NGLL2+J*NGLLX+I]*fac3;
    }
#endif

    fac1 = d_wgllwgll_yz[K*NGLLX+J];
    fac2 = d_wgllwgll_xz[K*NGLLX+I];
    fac3 = d_wgllwgll_xy[J*NGLLX+I];

    sum_terms1 = - (fac1*tempx1l + fac2*tempx2l + fac3*tempx3l);
    sum_terms2 = - (fac1*tempy1l + fac2*tempy2l + fac3*tempy3l);
    sum_terms3 = - (fac1*tempz1l + fac2*tempz2l + fac3*tempz3l);

    // adds gravity term
    if (GRAVITY){
      sum_terms1 += rho_s_H1;
      sum_terms2 += rho_s_H2;
      sum_terms3 += rho_s_H3;
    }

#ifdef USE_MESH_COLORING_GPU
    // no atomic operation needed, colors don't share global points between elements

#ifdef USE_TEXTURES_FIELDS
    d_accel[iglob*3]     = texfetch_accel_cm<FORWARD_OR_ADJOINT>(iglob*3) + sum_terms1;
    d_accel[iglob*3 + 1] = texfetch_accel_cm<FORWARD_OR_ADJOINT>(iglob*3 + 1) + sum_terms2;
    d_accel[iglob*3 + 2] = texfetch_accel_cm<FORWARD_OR_ADJOINT>(iglob*3 + 2) + sum_terms3;
#else
    d_accel[iglob*3]     += sum_terms1;
    d_accel[iglob*3 + 1] += sum_terms2;
    d_accel[iglob*3 + 2] += sum_terms3;
#endif // USE_TEXTURES_FIELDS

#else // MESH_COLORING

    //mesh coloring
    if (use_mesh_coloring_gpu){

      // no atomic operation needed, colors don't share global points between elements
#ifdef USE_TEXTURES_FIELDS
      d_accel[iglob*3]     = texfetch_accel_cm<FORWARD_OR_ADJOINT>(iglob*3) + sum_terms1;
      d_accel[iglob*3 + 1] = texfetch_accel_cm<FORWARD_OR_ADJOINT>(iglob*3 + 1) + sum_terms2;
      d_accel[iglob*3 + 2] = texfetch_accel_cm<FORWARD_OR_ADJOINT>(iglob*3 + 2) + sum_terms3;
#else
      d_accel[iglob*3]     += sum_terms1;
      d_accel[iglob*3 + 1] += sum_terms2;
      d_accel[iglob*3 + 2] += sum_terms3;
#endif // USE_TEXTURES_FIELDS

    }else{
      // no mesh coloring uses atomic updates

      atomicAdd(&d_accel[iglob*3], sum_terms1);
      atomicAdd(&d_accel[iglob*3 + 1], sum_terms2);
      atomicAdd(&d_accel[iglob*3 + 2], sum_terms3);

      // debug: for testing purposes only: w/out atomic updates
      //d_accel[iglob*3] -= (0.00000001f*tempx1l + 0.00000001f*tempx2l + 0.00000001f*tempx3l);
      //d_accel[iglob*3 + 1] -= (0.00000001f*tempy1l + 0.00000001f*tempy2l + 0.00000001f*tempy3l);
      //d_accel[iglob*3 + 2] -= (0.00000001f*tempz1l + 0.00000001f*tempz2l + 0.00000001f*tempz3l);
    }
#endif // MESH_COLORING

    // update memory variables based upon the Runge-Kutta scheme
    if (ATTENUATION && ( ! PARTIAL_PHYS_DISPERSION_ONLY ) ){
      compute_element_cm_att_memory(tx,working_element,
                                    d_muvstore,
                                    factor_common,alphaval,betaval,gammaval,
                                    R_xx,R_yy,R_xy,R_xz,R_yz,
                                    epsilondev_xx,epsilondev_yy,epsilondev_xy,
                                    epsilondev_xz,epsilondev_yz,
                                    epsilondev_xx_loc,epsilondev_yy_loc,epsilondev_xy_loc,
                                    epsilondev_xz_loc,epsilondev_yz_loc,
                                    d_c44store,ANISOTROPY,USE_3D_ATTENUATION_ARRAYS);
    }

    // save deviatoric strain for Runge-Kutta scheme
    if (COMPUTE_AND_STORE_STRAIN){
      // fortran: epsilondev_xx(:,:,:,ispec) = epsilondev_xx_loc(:,:,:)
      epsilondev_xx[tx + working_element*NGLL3] = epsilondev_xx_loc;
      epsilondev_yy[tx + working_element*NGLL3] = epsilondev_yy_loc;
      epsilondev_xy[tx + working_element*NGLL3] = epsilondev_xy_loc;
      epsilondev_xz[tx + working_element*NGLL3] = epsilondev_xz_loc;
      epsilondev_yz[tx + working_element*NGLL3] = epsilondev_yz_loc;
    }
  } // active
}
