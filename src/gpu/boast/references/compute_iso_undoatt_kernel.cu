
#include <hip/hip_runtime.h>
// from compute_kernels_cuda.cu
#define NGLLX 5
#define NGLL2 25
#define NGLL3 125
#define NGLL3_PADDED 128

typedef float realw;

__device__ void compute_element_strain_undoatt(int ispec,int ijk_ispec,
                                               int* d_ibool,
                                               realw* s_dummyx_loc,
                                               realw* s_dummyy_loc,
                                               realw* s_dummyz_loc,
                                               realw* d_xix,realw* d_xiy,realw* d_xiz,
                                               realw* d_etax,realw* d_etay,realw* d_etaz,
                                               realw* d_gammax,realw* d_gammay,realw* d_gammaz,
                                               realw* sh_hprime_xx,
                                               realw* epsilondev_loc,
                                               realw* epsilon_trace_over_3) {


  // thread id == GLL point id
  int tx = threadIdx.x;
  int K = (tx/NGLL2);
  int J = ((tx-K*NGLL2)/NGLLX);
  int I = (tx-K*NGLL2-J*NGLLX);

  int offset;

  realw tempx1l,tempx2l,tempx3l,tempy1l,tempy2l,tempy3l,tempz1l,tempz2l,tempz3l;
  realw xixl,xiyl,xizl,etaxl,etayl,etazl,gammaxl,gammayl,gammazl;
  realw duxdxl,duxdyl,duxdzl,duydxl,duydyl,duydzl,duzdxl,duzdyl,duzdzl;
  realw templ;
  realw fac1,fac2,fac3;

  int l;

// copy from global memory to shared memory
// each thread writes one of the NGLL^3 = 125 data points

  tempx1l = 0.f;
  tempx2l = 0.f;
  tempx3l = 0.f;

  tempy1l = 0.f;
  tempy2l = 0.f;
  tempy3l = 0.f;

  tempz1l = 0.f;
  tempz2l = 0.f;
  tempz3l = 0.f;

  for (l=0;l<NGLLX;l++) {
      fac1 = sh_hprime_xx[l*NGLLX+I];
      tempx1l += s_dummyx_loc[K*NGLL2+J*NGLLX+l]*fac1;
      tempy1l += s_dummyy_loc[K*NGLL2+J*NGLLX+l]*fac1;
      tempz1l += s_dummyz_loc[K*NGLL2+J*NGLLX+l]*fac1;

      fac2 = sh_hprime_xx[l*NGLLX+J];
      tempx2l += s_dummyx_loc[K*NGLL2+l*NGLLX+I]*fac2;
      tempy2l += s_dummyy_loc[K*NGLL2+l*NGLLX+I]*fac2;
      tempz2l += s_dummyz_loc[K*NGLL2+l*NGLLX+I]*fac2;

      fac3 = sh_hprime_xx[l*NGLLX+K];
      tempx3l += s_dummyx_loc[l*NGLL2+J*NGLLX+I]*fac3;
      tempy3l += s_dummyy_loc[l*NGLL2+J*NGLLX+I]*fac3;
      tempz3l += s_dummyz_loc[l*NGLL2+J*NGLLX+I]*fac3;
  }

  // compute derivatives of ux, uy and uz with respect to x, y and z
  offset = ispec*NGLL3_PADDED + tx;

  xixl = d_xix[offset];
  xiyl = d_xiy[offset];
  xizl = d_xiz[offset];
  etaxl = d_etax[offset];
  etayl = d_etay[offset];
  etazl = d_etaz[offset];
  gammaxl = d_gammax[offset];
  gammayl = d_gammay[offset];
  gammazl = d_gammaz[offset];

  duxdxl = xixl*tempx1l + etaxl*tempx2l + gammaxl*tempx3l;
  duxdyl = xiyl*tempx1l + etayl*tempx2l + gammayl*tempx3l;
  duxdzl = xizl*tempx1l + etazl*tempx2l + gammazl*tempx3l;

  duydxl = xixl*tempy1l + etaxl*tempy2l + gammaxl*tempy3l;
  duydyl = xiyl*tempy1l + etayl*tempy2l + gammayl*tempy3l;
  duydzl = xizl*tempy1l + etazl*tempy2l + gammazl*tempy3l;

  duzdxl = xixl*tempz1l + etaxl*tempz2l + gammaxl*tempz3l;
  duzdyl = xiyl*tempz1l + etayl*tempz2l + gammayl*tempz3l;
  duzdzl = xizl*tempz1l + etazl*tempz2l + gammazl*tempz3l;

  // computes deviatoric strain attenuation and/or for kernel calculations
  templ = 0.33333333333333333333f * (duxdxl + duydyl + duzdzl); // 1./3. = 0.33333

  // local storage: stresses at this current time step
  epsilondev_loc[0] = duxdxl - templ;   // xx
  epsilondev_loc[1] = duydyl - templ;   // yy
  epsilondev_loc[2] = 0.5f * ( duxdyl + duydxl ); // xy
  epsilondev_loc[3] = 0.5f * ( duzdxl + duxdzl ); // xz
  epsilondev_loc[4] = 0.5f * ( duzdyl + duydzl ); // yz
  *epsilon_trace_over_3 = templ;
}

/* ----------------------------------------------------------------------------------------------- */

__global__ void compute_iso_undoatt_kernel(realw* epsilondev_xx,
                                           realw* epsilondev_yy,
                                           realw* epsilondev_xy,
                                           realw* epsilondev_xz,
                                           realw* epsilondev_yz,
                                           realw* epsilon_trace_over_3,
                                           realw* mu_kl,
                                           realw* kappa_kl,
                                           int NSPEC,
                                           realw deltat,
                                           int* d_ibool,
                                           realw* d_b_displ,
                                           realw* d_xix,
                                           realw* d_xiy,
                                           realw* d_xiz,
                                           realw* d_etax,
                                           realw* d_etay,
                                           realw* d_etaz,
                                           realw* d_gammax,
                                           realw* d_gammay,
                                           realw* d_gammaz,
                                           realw* d_hprime_xx) {

  int ispec = blockIdx.x + blockIdx.y*gridDim.x;
  int ijk_ispec = threadIdx.x + NGLL3*ispec;

  int tx = threadIdx.x;
  int iglob;

  __shared__ realw s_dummyx_loc[NGLL3];
  __shared__ realw s_dummyy_loc[NGLL3];
  __shared__ realw s_dummyz_loc[NGLL3];

  __shared__ realw sh_hprime_xx[NGLL2];

  // loads element displacements
  // all threads load their displacement into shared memory
  if (ispec < NSPEC){
    iglob = d_ibool[ijk_ispec]-1;
    // changing iglob indexing to match fortran row changes fast style
    s_dummyx_loc[tx] = d_b_displ[iglob*3];
    s_dummyy_loc[tx] = d_b_displ[iglob*3 + 1];
    s_dummyz_loc[tx] = d_b_displ[iglob*3 + 2];

    // main thread loads hprime
    if (threadIdx.x == 0){
      for(int m=0; m < NGLL2; m++){
        // hprime
        sh_hprime_xx[m] = d_hprime_xx[m];
      }
    }
  }

  // synchronizes threads
  __syncthreads();

  // handles case when there is 1 extra block (due to rectangular grid)
  if(ispec < NSPEC) {

    realw eps_trace_over_3,b_eps_trace_over_3;
    realw epsdev[5];
    realw b_epsdev[5];

    // strain from adjoint wavefield
    epsdev[0] = epsilondev_xx[ijk_ispec];
    epsdev[1] = epsilondev_yy[ijk_ispec];
    epsdev[2] = epsilondev_xy[ijk_ispec];
    epsdev[3] = epsilondev_xz[ijk_ispec];
    epsdev[4] = epsilondev_yz[ijk_ispec];
    eps_trace_over_3 = epsilon_trace_over_3[ijk_ispec];

    // strain from backward/reconstructed forward wavefield
    compute_element_strain_undoatt(ispec,ijk_ispec,
                                   d_ibool,
                                   s_dummyx_loc,s_dummyy_loc,s_dummyz_loc,
                                   d_xix,d_xiy,d_xiz,d_etax,d_etay,d_etaz,d_gammax,d_gammay,d_gammaz,
                                   sh_hprime_xx,
                                   b_epsdev,&b_eps_trace_over_3);

    // isotropic kernel contributions
    // shear modulus kernel
    mu_kl[ijk_ispec] += deltat * ( epsdev[0]*b_epsdev[0] + epsdev[1]*b_epsdev[1]
                                   + (epsdev[0]+epsdev[1])*(b_epsdev[0]+b_epsdev[1])
                                   + 2*( epsdev[2]*b_epsdev[2] + epsdev[3]*b_epsdev[3] + epsdev[4]*b_epsdev[4]) );

    // bulk modulus kernel
    kappa_kl[ijk_ispec] += deltat * ( 9 * eps_trace_over_3 * b_eps_trace_over_3);
  }
}
