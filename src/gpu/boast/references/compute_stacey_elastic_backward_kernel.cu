
#include <hip/hip_runtime.h>
// from compute_stacey_elastic_cuda.cu
#define NDIM 3
#define NGLLX 5
#define NGLL2 25
#define INDEX2(xsize,x,y) x + (y)*xsize
#define INDEX3(xsize,ysize,x,y,z) x + xsize*(y + ysize*z)
#define INDEX4(xsize,ysize,zsize,x,y,z,i) x + xsize*(y + ysize*(z + zsize*i))

typedef float realw;

__global__ void compute_stacey_elastic_backward_kernel(realw* b_accel,
                                                       realw* b_absorb_field,
                                                       int interface_type,
                                                       int num_abs_boundary_faces,
                                                       int* abs_boundary_ispec,
                                                       int* nkmin_xi, int* nkmin_eta,
                                                       int* njmin, int* njmax,
                                                       int* nimin, int* nimax,
                                                       int* ibool) {

  int igll = threadIdx.x; // tx
  int iface = blockIdx.x + gridDim.x*blockIdx.y; // bx

  int i,j,k,iglob,ispec;

  // don't compute surface faces outside of range
  // and don't compute points outside NGLLSQUARE==NGLL2==25
  //if(igll < NGLL2 && iface < num_abs_boundary_faces) {

  // way 2: only check face, no further check needed since blocksize = 25
  if (iface < num_abs_boundary_faces){

    // "-1" from index values to convert from Fortran-> C indexing
    ispec = abs_boundary_ispec[iface]-1;

    // determines indices i,j,k depending on absorbing boundary type
    switch( interface_type ){
      case 0:
        // xmin
        if (nkmin_xi[INDEX2(2,0,iface)] == 0 || njmin[INDEX2(2,0,iface)] == 0) return;

        i = 0; // index -1
        k = (igll/NGLLX);
        j = (igll-k*NGLLX);

        if (k < nkmin_xi[INDEX2(2,0,iface)]-1 || k > NGLLX-1) return;
        if (j < njmin[INDEX2(2,0,iface)]-1 || j > NGLLX-1) return;

        break;

      case 1:
        // xmax
        if (nkmin_xi[INDEX2(2,1,iface)] == 0 || njmin[INDEX2(2,1,iface)] == 0) return;

        i = NGLLX-1;
        k = (igll/NGLLX);
        j = (igll-k*NGLLX);

        if (k < nkmin_xi[INDEX2(2,1,iface)]-1 || k > NGLLX-1) return;
        if (j < njmin[INDEX2(2,1,iface)]-1 || j > njmax[INDEX2(2,1,iface)]-1) return;

        break;

      case 2:
        // ymin
        if (nkmin_eta[INDEX2(2,0,iface)] == 0 || nimin[INDEX2(2,0,iface)] == 0) return;

        j = 0;
        k = (igll/NGLLX);
        i = (igll-k*NGLLX);

        if (k < nkmin_eta[INDEX2(2,0,iface)]-1 || k > NGLLX-1) return;
        if (i < nimin[INDEX2(2,0,iface)]-1 || i > nimax[INDEX2(2,0,iface)]-1) return;

        break;

      case 3:
        // ymax
        if (nkmin_eta[INDEX2(2,1,iface)] == 0 || nimin[INDEX2(2,1,iface)] == 0) return;

        j = NGLLX-1;
        k = (igll/NGLLX);
        i = (igll-k*NGLLX);

        if (k < nkmin_eta[INDEX2(2,1,iface)]-1 || k > NGLLX-1) return;
        if (i < nimin[INDEX2(2,1,iface)]-1 || i > nimax[INDEX2(2,1,iface)]-1) return;

        break;
    }

    iglob = ibool[INDEX4(NGLLX,NGLLX,NGLLX,i,j,k,ispec)]-1;

    atomicAdd(&b_accel[iglob*3  ],-b_absorb_field[INDEX3(NDIM,NGLL2,0,igll,iface)]);
    atomicAdd(&b_accel[iglob*3+1],-b_absorb_field[INDEX3(NDIM,NGLL2,1,igll,iface)]);
    atomicAdd(&b_accel[iglob*3+2],-b_absorb_field[INDEX3(NDIM,NGLL2,2,igll,iface)]);

  } // num_abs_boundary_faces
}

