
#include <hip/hip_runtime.h>
// from compute_kernels_cuda.cu
#define NGLL3 125

typedef float realw;

__global__ void compute_hess_kernel(int* ibool,
                                    realw* accel,
                                    realw* b_accel,
                                    realw* hess_kl,
                                    realw deltat,
                                    int NSPEC_AB,
                                    int USE_SOURCE_RECEIVER_HESSIAN) {

  int ispec = blockIdx.x + blockIdx.y*gridDim.x;

  // handles case when there is 1 extra block (due to rectangular grid)
  if(ispec < NSPEC_AB) {

    int ijk = threadIdx.x;
    int ijk_ispec = ijk + NGLL3*ispec;
    int iglob = ibool[ijk_ispec] - 1 ;

    // approximate hessian
    if (USE_SOURCE_RECEIVER_HESSIAN) {
      hess_kl[ijk_ispec] += deltat * (accel[3*iglob]*b_accel[3*iglob] +
                                      accel[3*iglob+1]*b_accel[3*iglob+1] +
                                      accel[3*iglob+2]*b_accel[3*iglob+2]);
    } else {
      hess_kl[ijk_ispec] += deltat * (accel[3*iglob]*accel[3*iglob] +
                                      accel[3*iglob+1]*accel[3*iglob+1] +
                                      accel[3*iglob+2]*accel[3*iglob+2]);
    }
  }
}
