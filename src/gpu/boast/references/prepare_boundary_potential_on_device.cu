
#include <hip/hip_runtime.h>
// from assemble_MPI_scalar_cuda.cu
typedef float realw;

__global__ void prepare_boundary_potential_on_device(realw* d_potential_dot_dot_acoustic,
                                                     realw* d_send_potential_dot_dot_buffer,
                                                     int num_interfaces,
                                                     int max_nibool_interfaces,
                                                     int* d_nibool_interfaces,
                                                     int* d_ibool_interfaces) {

  int id = threadIdx.x + blockIdx.x*blockDim.x + blockIdx.y*gridDim.x*blockDim.x;
  int iglob,iloc;

  for( int iinterface=0; iinterface < num_interfaces; iinterface++) {
    if(id<d_nibool_interfaces[iinterface]) {

      iloc = id + max_nibool_interfaces*iinterface;
      iglob = d_ibool_interfaces[iloc] - 1;

      // fills buffer
      d_send_potential_dot_dot_buffer[iloc] = d_potential_dot_dot_acoustic[iglob];
    }
  }

}
