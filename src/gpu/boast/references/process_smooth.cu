#include "hip/hip_runtime.h"
// original implementation

#ifndef NGLL3
#define NGLL3 125
#endif

__global__ void process_smooth(realw_const_p xstore_me,
                               realw_const_p ystore_me,
                               realw_const_p zstore_me,
                               realw_const_p xstore_other,
                               realw_const_p ystore_other,
                               realw_const_p zstore_other,
                               realw_const_p data_other,
                               const realw sigma_h2_inv,
                               const realw sigma_v2_inv,
                               const int iker,
                               const int nspec_me,
                               const int nspec_other,
                               const realw v_criterion,
                               const realw h_criterion,
                               realw_const_p integ_factor,
                               realw_p data_smooth,
                               realw_p normalisation){

  int ispec = blockIdx.x + gridDim.x*blockIdx.y;
  int igll = threadIdx.x;

  int gll_other;
  realw x_me,y_me,z_me;
  realw x_other,y_other,z_other;
  realw center_x,center_y,center_z;
  realw alpha,ratio,theta;
  realw r0_squared,r1_squared;
  realw dist_h,dist_v;
  realw val,val_gaussian;
  realw coef, normalisation_slice;
  realw dat;

  // PI squared
  const realw PI2 = 9.869604401089358f;

  __shared__ int sh_test[NGLL3];
  __shared__ realw sh_x_other[NGLL3];
  __shared__ realw sh_y_other[NGLL3];
  __shared__ realw sh_z_other[NGLL3];
  __shared__ realw sh_integ_factor[NGLL3];
  __shared__ realw sh_data[NGLL3];

  // for each reference GLL point, we can check a block of 125 neighbor elements
  // by convenience, the block size is set to the number of threads 125 of this kernel
  int n_loop = nspec_other/NGLL3 + 1;

  // reference GLL point position
  x_me = xstore_me[NGLL3*ispec + igll];
  y_me = ystore_me[NGLL3*ispec + igll];
  z_me = zstore_me[NGLL3*ispec + igll];

  __syncthreads();

  dat = 0.f;
  normalisation_slice = 0.f;

  // We test 125 spectral elements at a time
  for (int i=0; i < n_loop; i++){
    __syncthreads();

    // each thread helps to test a different element in the other slice (using the center position)
    // number of threads == NGLL3 == 125
    // for i==0: element range [0,124]
    // for i==1: element range [125,(125+124)]
    // ..
    // for i==n_loop-1: element range [NGLL3*(nloop-1),NGLL3*(nloop-1)+124]
    //                  where NGLL3*(nloop-1)+124 is equal to nspec_other (or slightly greater)
    int ispec_other = NGLL3*i + igll;

    if (ispec_other < nspec_other){
      // center position
      center_x = (xstore_other[ispec_other * NGLL3] + xstore_other[ispec_other * NGLL3 + (NGLL3 - 1)]) * 0.5f;
      center_y = (ystore_other[ispec_other * NGLL3] + ystore_other[ispec_other * NGLL3 + (NGLL3 - 1)]) * 0.5f;
      center_z = (zstore_other[ispec_other * NGLL3] + zstore_other[ispec_other * NGLL3 + (NGLL3 - 1)]) * 0.5f;

      // note: instead of distance we use distance squared to avoid too many sqrt() operations

      // Cartesian case
      // distance horizontal = (x-x0)**2 + (y-y0)**2, and vertical = (z-z0)**2
      //dist_h = (x_me - center_x)*(x_me - center_x) + (y_me - center_y)*(y_me - center_y);
      //dist_v = (z_me - center_z)*(z_me - center_z);

      // Spherical case
      // vertical distance
      r0_squared = x_me*x_me + y_me*y_me + z_me*z_me;
      r1_squared = center_x*center_x + center_y*center_y + center_z*center_z;

      // vertical distance (squared)
      // dist_v = (r1 - r0)*(r1 - r0)
      //        = r1**2 + r0**2 - 2 * alpha
      //          with alpha = sqrt( r0**2 * r1**2 ) = r0 * r1
      // this avoids using sqrt() function too often which is costly
      alpha = sqrt( r0_squared * r1_squared );
      dist_v = r1_squared + r0_squared - 2.0f * alpha;

      // epicentral distance
      // (accounting for spherical curvature)
      // calculates distance of circular segment
      // angle between r0 and r1 in radian
      // given by dot-product of two vectors
      if (alpha > 0.0f){
        ratio = (x_me*center_x + y_me*center_y + z_me*center_z) / alpha;
      } else {
        ratio = 1.0f;
      }

      // checks boundaries of ratio (due to numerical inaccuracies)
      if (ratio >= 1.0f){
        // ratio = 1.0_CUSTOM_REAL
        // -> acos(1) = 0
        // -> dist_h = 0
        dist_h = 0.0f;
      } else if (ratio <= -1.0f) {
        // ratio = -1.0_CUSTOM_REAL
        // -> acos(-1) = PI
        // -> dist_h = r1**2 * PI**2
        dist_h = r1_squared * PI2;
      } else {
        theta = acos( ratio );
        // segment length at heigth of r1 (squared)
        dist_h = r1_squared * (theta*theta);
      }
    } else {
      // artificial high values
      // (h_criterion and v_criterion are normalized in global version)
      dist_v = 99999999.f;
      dist_h = 99999999.f;
    }

    // tests if element is too far away
    sh_test[igll] = ( ispec_other >= nspec_other
                    || dist_h > h_criterion
                    || dist_v > v_criterion ) ? 1 : 0 ;


    __syncthreads();

    // loops over each spectral element tested
    for (int k=0; k < NGLL3; k++){
      __syncthreads();

      // skips element if test was true (too far away)
      if (sh_test[k]) continue ;

      // loads data from other slice to shared memory
      int ispec_test = i*NGLL3 + k;
      sh_x_other[igll] = xstore_other[ispec_test*NGLL3 + igll];
      sh_y_other[igll] = ystore_other[ispec_test*NGLL3 + igll];
      sh_z_other[igll] = zstore_other[ispec_test*NGLL3 + igll];

      sh_data[igll] = data_other[ispec_test*NGLL3 + igll];
      sh_integ_factor[igll] = integ_factor[ispec_test*NGLL3 + igll];

      __syncthreads();

      // loops over gll points
      for (int j=0; j < NGLL3; j++){
        gll_other = (igll + j) % NGLL3;

        x_other = sh_x_other[gll_other];
        y_other = sh_y_other[gll_other];
        z_other = sh_z_other[gll_other];

        // Cartesian case
        // distance horizontal = (x-x0)**2 + (y-y0)**2, and vertical = (z-z0)**2
        //dist_h = (x_me - x_other)*(x_me - x_other) + (y_me - y_other)*(y_me - y_other);
        //dist_v = (z_me - z_other)*(z_me - z_other);
        //coef = expf(- sigma_h2_inv * dist_h - sigma_v2_inv * dist_v) * sh_integ_factor[gll_other];

        // Spherical case
        // vertical distance
        r0_squared = x_me*x_me + y_me*y_me + z_me*z_me;
        r1_squared = x_other*x_other + y_other*y_other + z_other*z_other;

        // vertical distance (squared)
        // dist_v = (r1 - r0)*(r1 - r0)
        //        = r1**2 + r0**2 - 2 * alpha
        //          with alpha = sqrt( r0**2 * r1**2 ) = r0 * r1
        // this avoids using sqrt() function too often which is costly
        alpha = sqrt( r0_squared * r1_squared );
        dist_v = r1_squared + r0_squared - 2.0f * alpha;

        // epicentral distance
        // (accounting for spherical curvature)
        // calculates distance of circular segment
        // angle between r0 and r1 in radian
        // given by dot-product of two vectors
        if (alpha > 0.0f){
          ratio = (x_me*x_other + y_me*y_other + z_me*z_other) / alpha;
        } else {
          ratio = 1.0f;
        }

        // checks boundaries of ratio (due to numerical inaccuracies)
        if (ratio >= 1.0f){
          // ratio = 1.0_CUSTOM_REAL
          // -> acos(1) = 0
          // -> dist_h = 0
          dist_h = 0.0f;
        } else if (ratio <= -1.0f) {
          // ratio = -1.0_CUSTOM_REAL
          // -> acos(-1) = PI
          // -> dist_h = r1**2 * PI**2
          dist_h = r1_squared * PI2;
        } else {
          theta = acos( ratio );
          // segment length at heigth of r1 (squared)
          dist_h = r1_squared * (theta*theta);
        }

        // Gaussian function
        val = - dist_h*sigma_h2_inv - dist_v*sigma_v2_inv;

        // limits to single precision
        if (val < - 86.0f){
          // smaller than numerical precision: exp(-86) < 1.e-37
          val_gaussian = 0.0f;
        } else {
          val_gaussian = expf(val);
        }

        coef = val_gaussian * sh_integ_factor[gll_other];

        normalisation_slice = normalisation_slice + coef;
        dat += sh_data[gll_other] * coef;
      } //loop on each gll_other
    } //loop on each spec_other tested
  } //loop on each serie of 125 spec_other

  data_smooth[NGLL3*nspec_me*iker + NGLL3*ispec + igll] += dat;

  // note: normalization coefficient is added nker times
  normalisation[NGLL3*ispec + igll] += normalisation_slice;
}

