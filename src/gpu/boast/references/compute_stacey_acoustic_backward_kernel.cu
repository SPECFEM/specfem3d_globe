
#include <hip/hip_runtime.h>
// from compute_stacey_acoustic_cuda.cu
#define NGLLX 5
#define NGLL2 25
#define INDEX2(xsize,x,y) x + (y)*xsize
#define INDEX4(xsize,ysize,zsize,x,y,z,i) x + xsize*(y + ysize*(z + zsize*i))

typedef float realw;

__global__ void compute_stacey_acoustic_backward_kernel(realw* b_potential_dot_dot_acoustic,
                                                        realw* b_absorb_potential,
                                                        int interface_type,
                                                        int num_abs_boundary_faces,
                                                        int* abs_boundary_ispec,
                                                        int* nkmin_xi, int* nkmin_eta,
                                                        int* njmin, int* njmax,
                                                        int* nimin, int* nimax,
                                                        int* ibool) {

  int igll = threadIdx.x;
  int iface = blockIdx.x + gridDim.x*blockIdx.y;

  int i,j,k,iglob,ispec;

  // don't compute points outside NGLLSQUARE==NGLL2==25
  // way 2: no further check needed since blocksize = 25
  if (iface < num_abs_boundary_faces){

  //  if(igll<NGLL2 && iface < num_abs_boundary_faces) {

    // "-1" from index values to convert from Fortran-> C indexing
    ispec = abs_boundary_ispec[iface]-1;

    // determines indices i,j,k depending on absorbing boundary type
    switch( interface_type ){
      case 4:
        // xmin
        if (nkmin_xi[INDEX2(2,0,iface)] == 0 || njmin[INDEX2(2,0,iface)] == 0) return;

        i = 0; // index -1
        k = (igll/NGLLX);
        j = (igll-k*NGLLX);

        if (k < nkmin_xi[INDEX2(2,0,iface)]-1 || k > NGLLX-1) return;
        if (j < njmin[INDEX2(2,0,iface)]-1 || j > njmax[INDEX2(2,0,iface)]-1) return;

        break;

      case 5:
        // xmax
        if (nkmin_xi[INDEX2(2,1,iface)] == 0 || njmin[INDEX2(2,1,iface)] == 0) return;

        i = NGLLX-1;
        k = (igll/NGLLX);
        j = (igll-k*NGLLX);

        if (k < nkmin_xi[INDEX2(2,1,iface)]-1 || k > NGLLX-1) return;
        if (j < njmin[INDEX2(2,1,iface)]-1 || j > njmax[INDEX2(2,1,iface)]-1) return;

        break;

      case 6:
        // ymin
        if (nkmin_eta[INDEX2(2,0,iface)] == 0 || nimin[INDEX2(2,0,iface)] == 0) return;

        j = 0;
        k = (igll/NGLLX);
        i = (igll-k*NGLLX);

        if (k < nkmin_eta[INDEX2(2,0,iface)]-1 || k > NGLLX-1) return;
        if (i < nimin[INDEX2(2,0,iface)]-1 || i > nimax[INDEX2(2,0,iface)]-1) return;

        break;

      case 7:
        // ymax
        if (nkmin_eta[INDEX2(2,1,iface)] == 0 || nimin[INDEX2(2,1,iface)] == 0) return;

        j = NGLLX-1;
        k = (igll/NGLLX);
        i = (igll-k*NGLLX);

        if (k < nkmin_eta[INDEX2(2,1,iface)]-1 || k > NGLLX-1) return;
        if (i < nimin[INDEX2(2,1,iface)]-1 || i > nimax[INDEX2(2,1,iface)]-1) return;

        break;

      case 8:
        // zmin
        k = 0;
        j = (igll/NGLLX);
        i = (igll-j*NGLLX);

        if (j < 0 || j > NGLLX-1) return;
        if (i < 0 || i > NGLLX-1) return;

        break;

    }

    iglob = ibool[INDEX4(5,5,5,i,j,k,ispec)]-1;

    // Sommerfeld condition
    atomicAdd(&b_potential_dot_dot_acoustic[iglob],-b_absorb_potential[INDEX2(NGLL2,igll,iface)]);
  }
}
